#include "BasicMaterials.cuh"

SceneError ConstantMat::InitializeGroup(const NodeListing& materialNodes, double time,
                                        const std::string& scenePath)
{
    constexpr const char* ALBEDO = "albedo";

    std::vector<Vector3> albedoCPU;
    for(const auto& sceneNode : materialNodes)
    {
        std::vector<Vector3> albedos = sceneNode->AccessVector3(ALBEDO);
        albedoCPU.insert(albedoCPU.end(), albedos.begin(), albedos.end());
    }

    // Generate Id List
    SceneError e = SceneError::OK;
    if((e = GenerateInnerIds(materialNodes)) != SceneError::OK)
        return e;

    // Alloc etc
    size_t dAlbedoSize = albedoCPU.size() * sizeof(Vector3);
    memory = std::move(DeviceMemory(dAlbedoSize));
    Vector3f* dAlbedo = static_cast<Vector3f*>(memory);
    CUDA_CHECK(hipMemcpy(dAlbedo, albedoCPU.data(), dAlbedoSize,
                          hipMemcpyHostToDevice));

    dData = AlbedoMatData{dAlbedo};
    return SceneError::OK;
}

SceneError ConstantMat::ChangeTime(const NodeListing& materialNodes, double time,
                                   const std::string& scenePath)
{
    // TODO: Implement
    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}