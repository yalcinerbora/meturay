#include "SampleMaterials.cuh"

SceneError EmissiveMat::InitializeGroup(const NodeListing& materialNodes, double time,
                                        const std::string& scenePath)
{
    constexpr const char* IRRADIANCE = "radiance";

    std::vector<Vector3> irradianceCPU;
    uint32_t i = 0;
    for(const auto& sceneNode : materialNodes)
    {
        std::vector<Vector3> irradiances = sceneNode->AccessVector3(IRRADIANCE);
        irradianceCPU.insert(irradianceCPU.end(), irradiances.begin(), irradiances.end());

        const auto& ids = sceneNode->Ids();
        for(IdPair id : ids)
        {
            innerIds.emplace(std::make_pair(id.first, i));
            i++;
        }
    }

    // Alloc etc
    size_t dIrradianceSize = irradianceCPU.size() * sizeof(Vector3);
    memory = std::move(DeviceMemory(dIrradianceSize));
    Vector3f* dIrradiance = static_cast<Vector3f*>(memory);
    CUDA_CHECK(hipMemcpy(dIrradiance, irradianceCPU.data(), dIrradianceSize,
               hipMemcpyHostToDevice));

    dData = EmissiveMatData{dIrradiance};
    return SceneError::OK;
}

SceneError EmissiveMat::ChangeTime(const NodeListing& materialNodes, double time,
                                   const std::string& scenePath)
{
    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}

int EmissiveMat::InnerId(uint32_t materialId) const
{
    return innerIds.at(materialId);
}

// -------------
SceneError LambertMat::InitializeGroup(const NodeListing& materialNodes, double time,
                                       const std::string& scenePath)
{
    constexpr const char* ALBEDO = "albedo";

    std::vector<Vector3> albedoCPU;
    uint32_t i = 0;
    for(const auto& sceneNode : materialNodes)
    {
        std::vector<Vector3> albedos = sceneNode->AccessVector3(ALBEDO);
        albedoCPU.insert(albedoCPU.end(), albedos.begin(), albedos.end());

        const auto& ids = sceneNode->Ids();
        for(IdPair id : ids)
        {
            innerIds.emplace(std::make_pair(id.first, i));
            i++;
        }
    }

    // Alloc etc
    size_t dAlbedoSize = albedoCPU.size() * sizeof(Vector3);
    memory = std::move(DeviceMemory(dAlbedoSize));
    Vector3f* dAlbedo = static_cast<Vector3f*>(memory);
    CUDA_CHECK(hipMemcpy(dAlbedo, albedoCPU.data(), dAlbedoSize,
                          hipMemcpyHostToDevice));

    dData = AlbedoMatData{dAlbedo};
    return SceneError::OK;
}

SceneError LambertMat::ChangeTime(const NodeListing& materialNodes, double time,
                                  const std::string& scenePath)
{
    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}

int LambertMat::InnerId(uint32_t materialId) const
{
    return innerIds.at(materialId);
}

// -------------
SceneError ReflectMat::InitializeGroup(const NodeListing& materialNodes, double time,
                                       const std::string& scenePath)
{
    constexpr const char* ALBEDO = "albedo";
    constexpr const char* ROUGHNESS = "roughness";

    std::vector<Vector4> matDataCPU;
    uint32_t i = 0;
    for(const auto& sceneNode : materialNodes)
    {
        std::vector<Vector3> albedos = sceneNode->AccessVector3(ALBEDO);
        std::vector<float> rougnessList = sceneNode->AccessFloat(ROUGHNESS);

        const auto& ids = sceneNode->Ids();
        for(IdPair id : ids)
        {
            Vector4 data = Vector4(albedos[i], rougnessList[i]);
            matDataCPU.push_back(data);

            innerIds.emplace(std::make_pair(id.first, i));
            i++;
        }
    }

    // Alloc etc
    size_t dMatDataSize = matDataCPU.size() * sizeof(Vector4);
    memory = std::move(DeviceMemory(dMatDataSize));
    Vector4f* dMemory = static_cast<Vector4f*>(memory);
    CUDA_CHECK(hipMemcpy(dMemory, matDataCPU.data(), dMatDataSize,
                          hipMemcpyHostToDevice));

    dData = ReflectMatData{dMemory};
    return SceneError::OK;
}

SceneError ReflectMat::ChangeTime(const NodeListing& materialNodes, double time,
                                  const std::string& scenePath)
{
    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}

int ReflectMat::InnerId(uint32_t materialId) const
{
    return innerIds.at(materialId);
}

// -------------
SceneError RefractMat::InitializeGroup(const NodeListing& materialNodes, double time,
                                       const std::string& scenePath)
{
    //constexpr const char* ALBEDO = "albedo";
    //constexpr const char* ROUGHNESS = "roughness";

    //std::vector<Vector4> matDataCPU;
    //uint32_t i = 0;
    //for(const auto& sceneNode : materialNodes)
    //{
    //    std::vector<Vector3> albedos = sceneNode->AccessVector3(ALBEDO);
    //    std::vector<float> rougnessList = sceneNode->AccessFloat(ROUGHNESS);

    //    const auto& ids = sceneNode->Ids();
    //    for(IdPair id : ids)
    //    {
    //        Vector4 data = Vector4(albedos[i], rougnessList[i]);
    //        matDataCPU.push_back(data);

    //        innerIds.emplace(std::make_pair(id.first, i));
    //        i++;
    //    }
    //}

    //// Alloc etc
    //size_t dMatDataSize = matDataCPU.size() * sizeof(Vector4);
    //memory = std::move(DeviceMemory(dMatDataSize));
    //Vector4f* dMemory = static_cast<Vector4f*>(memory);
    //CUDA_CHECK(hipMemcpy(dMemory, matDataCPU.data(), dMatDataSize,
    //                      hipMemcpyHostToDevice));

    //dData = ReflectMatData{dMemory};
    //return SceneError::OK;
}

SceneError RefractMat::ChangeTime(const NodeListing& materialNodes, double time,
                                  const std::string& scenePath)
{
    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}

int RefractMat::InnerId(uint32_t materialId) const
{
    return innerIds.at(materialId);
}