#include "SampleMaterials.cuh"

SceneError EmissiveMat::InitializeGroup(const NodeListing& materialNodes, double time,
                                        const std::string& scenePath)
{
    constexpr const char* IRRADIANCE = "radiance";

    std::vector<Vector3> irradianceCPU;
    uint32_t i = 0;
    for(const auto& sceneNode : materialNodes)
    {
        std::vector<Vector3> irradiances = sceneNode->AccessVector3(IRRADIANCE);
        irradianceCPU.insert(irradianceCPU.end(), irradiances.begin(), irradiances.end());

        const auto& ids = sceneNode->Ids();
        for(IdPair id : ids)
        {
            innerIds.emplace(std::make_pair(id.first, i));
            i++;
        }
    }

    // Generate Id List
    SceneError e = SceneError::OK;
    if((e = GenerateInnerIds(materialNodes)) != SceneError::OK)
        return e;

    // Alloc etc
    size_t dIrradianceSize = irradianceCPU.size() * sizeof(Vector3);
    memory = std::move(DeviceMemory(dIrradianceSize));
    Vector3f* dIrradiance = static_cast<Vector3f*>(memory);
    CUDA_CHECK(hipMemcpy(dIrradiance, irradianceCPU.data(), dIrradianceSize,
               hipMemcpyHostToDevice));

    dData = EmissiveMatData{dIrradiance};
    return SceneError::OK;
}

SceneError EmissiveMat::ChangeTime(const NodeListing& materialNodes, double time,
                                   const std::string& scenePath)
{
    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}

// -------------
SceneError LambertMat::InitializeGroup(const NodeListing& materialNodes, double time,
                                       const std::string& scenePath)
{
    constexpr const char* ALBEDO = "albedo";

    std::vector<Vector3> albedoCPU;
    for(const auto& sceneNode : materialNodes)
    {
        std::vector<Vector3> albedos = sceneNode->AccessVector3(ALBEDO);
        albedoCPU.insert(albedoCPU.end(), albedos.begin(), albedos.end());
    }

    // Generate Id List
    SceneError e = SceneError::OK;
    if((e = GenerateInnerIds(materialNodes)) != SceneError::OK)
        return e;

    // Alloc etc
    size_t dAlbedoSize = albedoCPU.size() * sizeof(Vector3);
    memory = std::move(DeviceMemory(dAlbedoSize));
    Vector3f* dAlbedo = static_cast<Vector3f*>(memory);
    CUDA_CHECK(hipMemcpy(dAlbedo, albedoCPU.data(), dAlbedoSize,
                          hipMemcpyHostToDevice));

    dData = AlbedoMatData{dAlbedo};
    return SceneError::OK;
}

SceneError LambertMat::ChangeTime(const NodeListing& materialNodes, double time,
                                  const std::string& scenePath)
{
    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}

// -------------
SceneError ReflectMat::InitializeGroup(const NodeListing& materialNodes, double time,
                                       const std::string& scenePath)
{
    constexpr const char* ALBEDO = "albedo";
    constexpr const char* ROUGHNESS = "roughness";

    std::vector<Vector4> matDataCPU;
    uint32_t i = 0;
    for(const auto& sceneNode : materialNodes)
    {
        std::vector<Vector3> albedos = sceneNode->AccessVector3(ALBEDO);
        std::vector<float> rougnessList = sceneNode->AccessFloat(ROUGHNESS);

        const auto& ids = sceneNode->Ids();
        for(IdPair id : ids)
        {
            Vector4 data = Vector4(albedos[i], rougnessList[i]);
            matDataCPU.push_back(data);
            
            innerIds.emplace(std::make_pair(id.first, i));
            i++;
        }
    }

    // Alloc etc
    size_t dMatDataSize = matDataCPU.size() * sizeof(Vector4);
    memory = std::move(DeviceMemory(dMatDataSize));
    Vector4f* dMemory = static_cast<Vector4f*>(memory);
    CUDA_CHECK(hipMemcpy(dMemory, matDataCPU.data(), dMatDataSize,
                          hipMemcpyHostToDevice));

    dData = ReflectMatData{dMemory};
    return SceneError::OK;
}

SceneError ReflectMat::ChangeTime(const NodeListing& materialNodes, double time,
                                  const std::string& scenePath)
{
    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}

// -------------
SceneError RefractMat::InitializeGroup(const NodeListing& materialNodes, double time,
                                       const std::string& scenePath)
{
    constexpr const char* ALBEDO = "albedo";
    constexpr const char* INDEX = "index";

    std::vector<Vector4> matDataCPU;
    uint32_t i = 0;
    for(const auto& sceneNode : materialNodes)
    {
        std::vector<Vector3> albedos = sceneNode->AccessVector3(ALBEDO);
        std::vector<float> indices = sceneNode->AccessFloat(INDEX);

        const auto& ids = sceneNode->Ids();
        for(IdPair id : ids)
        {
            Vector4 data = Vector4(albedos[i], indices[i]);
            matDataCPU.push_back(data);

            innerIds.emplace(std::make_pair(id.first, i));
            i++;
        }
    }

    // Alloc etc
    size_t dMatDataSize = matDataCPU.size() * sizeof(Vector4);
    memory = std::move(DeviceMemory(dMatDataSize));
    Vector4f* dMemory = static_cast<Vector4f*>(memory);
    CUDA_CHECK(hipMemcpy(dMemory, matDataCPU.data(), dMatDataSize,
                          hipMemcpyHostToDevice));

    //dData = RefractMatData{dMemory, ..., ...};
    return SceneError::OK;
}

SceneError RefractMat::ChangeTime(const NodeListing& materialNodes, double time,
                                  const std::string& scenePath)
{
    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}