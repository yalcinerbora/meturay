#include "RayCasterCUDA.h"
#include "CudaSystem.h"

#include "RayLib/GPUSceneI.h"
#include "GPUAcceleratorI.h"
#include "RayLib/TracerStructs.h"

// Constructors & Destructor
RayCasterCUDA::RayCasterCUDA(const GPUSceneI& gpuScene,
                             const CudaSystem& system)
    : rayMemory(system.BestGPU())
    , maxAccelBits(DetermineMaxBitFromId(gpuScene.MaxAccelIds()))
    , maxWorkBits(DetermineMaxBitFromId(gpuScene.MaxMatIds()))
    , maxHitSize(gpuScene.HitStructUnionSize())
    , boundaryTransformIndex(gpuScene.BoundaryTransformIndex())
    , cudaSystem(system)
    , baseAccelerator(*gpuScene.BaseAccelerator())
    , accelBatches(gpuScene.AcceleratorBatchMappings())
    , currentRayCount(0)
{}

TracerError RayCasterCUDA::ConstructAccelerators(const GPUTransformI** dTransforms,
                                                 uint32_t identityTransformIndex)
{
    TracerError e = TracerError::OK;

    // Attach Transform gpu pointer to the Accelerator Batches
    for(const auto& acc : accelBatches)
        acc.second->AttachGlobalTransformArray(dTransforms, identityTransformIndex);

    // Construct Accelerators
    SurfaceAABBList allSurfaceAABBs;
    for(const auto& accBatch : accelBatches)
    {
        GPUAcceleratorGroupI* acc = accBatch.second;
        if((e = acc->ConstructAccelerators(cudaSystem)) != TracerError::OK)
            return e;
        // Acquire surface aabb listings for base accelerator construction
        allSurfaceAABBs.insert(acc->AcceleratorAABBs().cbegin(),
                               acc->AcceleratorAABBs().cend());
    }
    // Construct Base accelerator using aabb list
    if((e = baseAccelerator.Construct(cudaSystem, allSurfaceAABBs)) != TracerError::OK)
        return e;

    return e;
}

RayPartitions<uint32_t> RayCasterCUDA::HitAndPartitionRays()
{
    // Sort and Partition happens on the leader device
    CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice().DeviceId()));

    // Tracer Logic interface
    const Vector2i& accBitCounts = maxAccelBits;
    const AcceleratorBatchMap& subAccelerators = accelBatches;
    // Reset Hit Memory for hit loop
    rayMemory.ResetHitMemory(boundaryTransformIndex, currentRayCount, maxHitSize);
    // Make Base Accelerator to get ready for hitting
    baseAccelerator.GetReady(cudaSystem, currentRayCount);
    // Ray Memory Pointers
    RayGMem* dRays = rayMemory.Rays();
    HitKey* dWorkKeys = rayMemory.WorkKeys();
    TransformId* dTransfomIds = rayMemory.TransformIds();
    PrimitiveId* dPrimitiveIds = rayMemory.PrimitiveIds();
    HitStructPtr dHitStructs = rayMemory.HitStructs();
    // These are sorted etc.
    HitKey* dCurrentKeys = rayMemory.CurrentKeys();
    RayId*  dCurrentRayIds = rayMemory.CurrentIds();

    //CUDA_CHECK(hipMemset(dTransfomIds, 0xFF, currentRayCount * sizeof(TransformId)));
    //Debug::DumpMemToFile("dTransforms", dTransfomIds, currentRayCount);

    // Try to hit rays until no ray is left
    // (these rays will be assigned with a material)
    // outside rays are also assigned with a material (which is special)
    uint32_t rayCount = currentRayCount;
    // At start all rays are valid
    uint32_t validRayOffset = 0;
    while(rayCount > 0)
    {
        // Traverse accelerator
        // Base accelerator provides potential hits
        // Cannot provide an absolute hit (its not its job)
        baseAccelerator.Hit(cudaSystem,
                            dCurrentKeys + validRayOffset,
                            dRays,
                            dCurrentRayIds + validRayOffset,
                            rayCount);
        // Wait all GPUs to finish...
        cudaSystem.SyncAllGPUsMainStreamOnly();

        // Base accelerator traverses the data partially
        // Updates current key (which represents inner accelerator batch and id)

        // After that, system sorts rays according to the keys
        // and partitions the array according to batches
        // Sort and Partition happens on the leader device
        CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice().DeviceId()));

        // Sort initial results (in order to partition and launch kernels accordingly)
        // Sort is radix sort.
        // We sort inner indices in addition to batches results for better data locality
        // We only sort up-to a certain bit (radix sort) which is tied to
        // accelerator count
        // Move offset to skip null batches
        rayMemory.SortKeys(dCurrentRayIds, dCurrentKeys, rayCount, accBitCounts);
        // Partition to sub accelerators
        //
        // There may be invalid rays sprinkled along the array.
        // Holes occur in the structure since in previous iteration,
        // a material may required to write N rays for its output (which is defined
        // by the material) but it wrote < N rays.
        //
        // One of the main examples for such behavior can be transparent objects
        // where ray may be only reflected (instead of refracting and reflecting) because
        // of the total internal reflection phenomena.
        auto portions = rayMemory.Partition(rayCount);

        // TODO: Reorder partitions for efficient calls
        // (group partitions into gpus and order for better async access)
        const int totalGPU = static_cast<int>(cudaSystem.SystemGPUs().size());
        const auto& gpus = cudaSystem.SystemGPUs();
        auto currentGPU = gpus.begin();
        // For each partition
        for(const auto& p : portions)
        {
            // Find Accelerator
            // Since there is no batch for invalid keys
            // that partition will be automatically be skipped
            auto loc = subAccelerators.find(p.portionId);
            if(loc == subAccelerators.end()) continue;

            RayId* dRayIdStart = dCurrentRayIds + validRayOffset + p.offset;
            HitKey* dCurrentKeyStart = dCurrentKeys + validRayOffset + p.offset;

            // Run local hit kernels
            // Local hit kernels returns a material key
            // and primitive inner id.
            // Since materials are batched for both material and
            loc->second->Hit(*currentGPU,
                             // O
                             dWorkKeys,
                             dTransfomIds,
                             dPrimitiveIds,
                             dHitStructs,
                             // I-O
                             dRays,
                             // Input
                             dRayIdStart,
                             dCurrentKeyStart,
                             static_cast<uint32_t>(p.count));

            // Split to GPUs
            currentGPU++;
            if(currentGPU == gpus.end()) currentGPU = gpus.begin();

            // Hit function updates material key,
            // primitive id and struct if this hit is accepted
        }
        // Update new ray count
        // On partition array check first partition
        // it may contain invalid key meaning
        // those rays are totally processed
        // change the offset so that
        // we skip those rays
        auto nullPortion = portions.begin();
        if(nullPortion->portionId == HitKey::NullBatch)
            rayCount = static_cast<uint32_t>(nullPortion->offset);

        // Iteration is done
        // We cant continue loop until these kernels are finished
        // on gpu(s)
        //
        // Tracer logic mostly utilizes multiple GPUs so we need to
        // wait all GPUs to finish
        cudaSystem.SyncAllGPUs();
    }
    // At the end of iteration, all rays found a material, a primitive
    // and an interpolation weights (which should be on hitStruct)

    // Partition rays for work kernel calls
    // Copy materialKeys to currentKeys
    // to make it ready for sorting
    rayMemory.FillMatIdsForSort(currentRayCount);
    // Sort with respect to the materials keys
    rayMemory.SortKeys(dCurrentRayIds, dCurrentKeys, currentRayCount, maxWorkBits);
    // Partition w.r.t. material batch
    RayPartitions<uint32_t> workPartition;
    workPartition.clear();
    workPartition = rayMemory.Partition(currentRayCount);

    return workPartition;
}

void RayCasterCUDA::WorkRays(const WorkBatchMap& workMap,
                             const RayPartitionsMulti<uint32_t>& outPortions,
                             const RayPartitions<uint32_t>& inPartitions,
                             RNGeneratorCPUI& rngCPU,
                             uint32_t totalRayOut,
                             HitKey baseBoundMatKey)
{
    // Sort and Partition happens on leader device
    CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice().DeviceId()));

    // Ray Memory Pointers
    const RayGMem* dRays = rayMemory.Rays();
    const HitStructPtr dHitStructs = rayMemory.HitStructs();
    const PrimitiveId* dPrimitiveIds = rayMemory.PrimitiveIds();
    const TransformId* dTransformIds = rayMemory.TransformIds();
    // These are sorted etc.
    HitKey* dCurrentKeys = rayMemory.CurrentKeys();
    RayId* dCurrentRayIds = rayMemory.CurrentIds();

    // Allocate output ray memory
    rayMemory.ResizeRayOut(totalRayOut, baseBoundMatKey);
    RayGMem* dRaysOut = rayMemory.RaysOut();
    HitKey* dBoundKeyOut = rayMemory.WorkKeys();
    // Wait that "ResizeRayOut" is completed on the leader device
    rayMemory.LeaderDevice().WaitMainStream();

    // For each partition
    for(const auto& p : inPartitions)
    {
        // Skip if null batch or not found material
        if(p.portionId == HitKey::NullBatch) continue;
        auto loc = workMap.find(p.portionId);
        if(loc == workMap.end()) continue;

        // TODO: change this loop to combine iterator instead of find
        //const auto& pIn = *(workPartition.find<uint32_t>(p.portionId));
        const auto& pOut = *(outPortions.find(MultiArrayPortion<uint32_t>{p.portionId}));

        // Relativize input & output pointers
        const RayId* dRayIdStart = dCurrentRayIds + p.offset;
        const HitKey* dKeyStart = dCurrentKeys + p.offset;

        assert(pOut.counts.size() == pOut.offsets.size());
        assert(pOut.counts.size() == loc->second.size());

        // Actual Shade Calls
        int i = 0;
        for(auto& workBatch : loc->second)
        {
            // Output
            RayGMem* dRayOutStart = dRaysOut + pOut.offsets[i];
            HitKey* dBoundKeyStart = dBoundKeyOut + pOut.offsets[i];

            workBatch->Work(dBoundKeyStart,
                            dRayOutStart,
                            //  Input
                            dRays,
                            dPrimitiveIds,
                            dTransformIds,
                            dHitStructs,
                            // Ids
                            dKeyStart,
                            dRayIdStart,
                            //
                            static_cast<uint32_t>(p.count),
                            rngCPU);

            i++;
        }
    }
    currentRayCount = totalRayOut;
    // Again wait all of the GPU's since
    // CUDA functions will be on multiple-gpus
    cudaSystem.SyncAllGPUs();
    // Shading complete
    // Now make "RayOut" -> "RayIn"
    // and continue
    rayMemory.SwapRays();
}

// Memory Usage
size_t RayCasterCUDA::UsedGPUMemory() const
{
    size_t mem = 0;
    for(const auto& accel : accelBatches)
        mem += accel.second->UsedGPUMemory();
    mem += baseAccelerator.UsedGPUMemory();
    mem += rayMemory.UsedGPUMemory();
    return mem;
}