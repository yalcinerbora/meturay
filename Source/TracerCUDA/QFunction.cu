#include "QFunction.cuh"
#include "CudaSystem.h"
#include "CudaSystem.hpp"
#include "ParallelMemset.cuh"

#include <execution>

#include "TracerDebug.h"

void QFunctionCPU::RecalculateDistributions(const CudaSystem& system)
{
    //Debug::DumpBatchedMemToFile("qFunc", qFuncGPU.gQFunction,
    //                            qFuncGPU.dataPerNode.Multiply(),
    //                            qFuncGPU.dataPerNode.Multiply() * qFuncGPU.nodeCount);

    distributions.UpdateDistributions(qFuncGPU.gQFunction, true,
                                      system, hipMemcpyDeviceToDevice);
}

TracerError QFunctionCPU::Initialize(const CudaSystem& system)
{
    // Initially set all values to uniform
    const CudaGPU& gpu = system.BestGPU();

    uint32_t dataCount = spatialCount * qFuncGPU.dataPerNode.Multiply();
    gpu.GridStrideKC_X(0, (hipStream_t)0, dataCount,
                       //
                       KCMemset<float>,
                       //
                       qFuncGPU.gQFunction,
                       1.0f,
                       dataCount);


    // Generate Distributions over this
    distributions = PWCDistStaticCPU2D(qFuncGPU.gQFunction,
                                       spatialCount,
                                       qFuncGPU.dataPerNode,
                                       true,
                                       system);

    qFuncGPU.gDistributions = distributions.DistributionGPU();
    return TracerError::OK;
}

void QFunctionCPU::DumpFunctionAsBinary(std::vector<Byte>& dataOut) const
{
    size_t size = qFuncGPU.dataPerNode.Multiply() * qFuncGPU.nodeCount * sizeof(float);
    dataOut.resize(size);

    CUDA_CHECK(hipMemcpy(dataOut.data(), qFuncGPU.gQFunction,
                          size, hipMemcpyDeviceToHost));
}