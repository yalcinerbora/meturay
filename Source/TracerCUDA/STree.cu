﻿#include "STree.cuh"
#include "PathNode.cuh"
#include "CudaSystem.h"
#include "CudaSystem.hpp"
#include "ParallelPartition.cuh"
#include "STreeKC.cuh"

#include "RayLib/MemoryAlignment.h"

#include <hipcub/hipcub.hpp>

static constexpr size_t AlignedOffsetSTreeGPU = Memory::AlignSize(sizeof(STreeGPU));

struct FetchTreeIdFunctor
{
    __device__ __host__ __forceinline__
    uint32_t operator()(const PathGuidingNode& node) const
    {
        return node.nearestDTreeIndex;
    }
};

struct IsSplittedLeafFunctor
{
    __device__ __host__ __forceinline__
    bool operator()(const uint32_t& index) const
    {
        return (index != INVALID_NODE);
    }
};

DeviceMemory STree::LinearizeDTreeGPUPtrs(bool readTree)
{
    std::vector<DTreeGPU*> hTreePtrs(dTrees.size());
    uint32_t i = 0;
    for(DTree& tree : dTrees)
    {
        hTreePtrs[i] = tree.TreeGPU(readTree);
        i++;
    }
    DeviceMemory treePtrs(dTrees.size() * sizeof(DTreeGPU*));
    CUDA_CHECK(hipMemcpy(static_cast<DTreeGPU**>(treePtrs),
                          hTreePtrs.data(),
                          dTrees.size() * sizeof(DTreeGPU*),
                          hipMemcpyHostToDevice));
    return std::move(treePtrs);
}

void STree::ExpandTree(size_t newNodeCount)
{
    // If its already large do not do stuff
    size_t currentCapacity = 0;
    if(memory.Size() > AlignedOffsetSTreeGPU)
        currentCapacity = (memory.Size() - AlignedOffsetSTreeGPU) / sizeof(STreeGPU);    
    if(currentCapacity >= newNodeCount) return;
    
    DeviceMemory newMem(AlignedOffsetSTreeGPU + newNodeCount * sizeof(STreeNode));
    // Copy the old stuff
    if(memory.Size() > 0)
        CUDA_CHECK(hipMemcpy(static_cast<Byte*>(newMem),
                              static_cast<const Byte*>(memory),
                              AlignedOffsetSTreeGPU + nodeCount * sizeof(STreeNode),
                              hipMemcpyHostToDevice));

    // Set new STree
    dSTree = static_cast<STreeGPU*>(newMem);
    // Copy the new node pointer
    Byte* nodeStart = static_cast<Byte*>(newMem) + AlignedOffsetSTreeGPU;
    Byte* nodePtrLoc = static_cast<Byte*>(newMem) + offsetof(STreeGPU, gRoot);
    CUDA_CHECK(hipMemcpy(nodePtrLoc, &nodeStart, sizeof(STreeNode*),
                          hipMemcpyHostToDevice));
    
    memory = std::move(newMem);    
}

STree::STree(const AABB3f& sceneExtents)
    : nodeCount(0)
    , dSTree(nullptr)
{
    ExpandTree(INITIAL_NODE_CAPACITY);

    // There should be at least one node
    STreeNode node;
    node.isLeaf = true;
    node.splitAxis = STreeNode::AxisType::X;
    node.index = 0; // This shows tree index since isLeaf is true
    Byte* nodeStart = static_cast<Byte*>(memory) + AlignedOffsetSTreeGPU;
    CUDA_CHECK(hipMemcpy(nodeStart, &node, sizeof(STreeNode),
                          hipMemcpyHostToDevice));

    // Update total node count aswell
    nodeCount = 1;
    Byte* nodeCountLocPtr = static_cast<Byte*>(memory) + offsetof(STreeGPU, nodeCount);
    CUDA_CHECK(hipMemcpy(nodeCountLocPtr, &nodeCount, sizeof(uint32_t),
               hipMemcpyHostToDevice));
    // Create a default single tree
    dTrees.reserve(INITIAL_TREE_RESERVE_COUNT);
    dTrees.emplace_back();
}

void STree::SplitLeaves(uint32_t maxSamplesPerNode,
                        const CudaSystem& system)
{
    const CudaGPU& gpu = system.BestGPU();
    CUDA_CHECK(hipSetDevice(gpu.DeviceId()));
    // Check the split cretaria on the leaf and respond
    DeviceMemory splitMarks(nodeCount * sizeof(uint32_t));
    // Use Read DTrees for determination
    // Allocate temp writeTree Buffer   
    DeviceMemory writeDTreeGPUBuffer = std::move(LinearizeDTreeGPUPtrs(false));
    DTreeGPU** dWriteDTrees = static_cast<DTreeGPU**>(writeDTreeGPUBuffer);
    
    // Mark Leafs
    gpu.GridStrideKC_X(0, 0, nodeCount,
                       //
                       KCMarkSTreeSplitLeaf,
                       //
                       static_cast<uint32_t*>(splitMarks),
                       *dSTree,
                       dWriteDTrees,
                       maxSamplesPerNode,
                       static_cast<uint32_t>(nodeCount));
    
    // Make dense leaf indices from sparse mark indices
    size_t tempMemSize;
    hipcub::DeviceSelect::If(nullptr, tempMemSize,
                          static_cast<uint32_t*>(splitMarks),
                          static_cast<uint32_t*>(splitMarks),
                          static_cast<uint32_t*>(splitMarks),
                          static_cast<int>(nodeCount),
                          IsSplittedLeafFunctor());
    // Output array and Temp
    DeviceMemory tempMemory(tempMemSize);
    DeviceMemory selectedIndices((nodeCount + 1) * sizeof(uint32_t));
    uint32_t* dDenseIndexCount = static_cast<uint32_t*>(selectedIndices);
    uint32_t* dDenseIndices = static_cast<uint32_t*>(selectedIndices) + 1;
    hipcub::DeviceSelect::If(static_cast<void*>(tempMemory), tempMemSize,
                          static_cast<uint32_t*>(splitMarks),
                          dDenseIndices, dDenseIndexCount,
                          static_cast<int>(nodeCount),
                          IsSplittedLeafFunctor());
    // Clear Mems
    tempMemory = std::move(DeviceMemory());
    splitMarks = std::move(DeviceMemory());

    // Check how many new trees we need to create
    // then allocate these trees
    uint32_t hSplitLeafCount = 0;
    CUDA_CHECK(hipMemcpy(&hSplitLeafCount, dDenseIndexCount,
                          sizeof(uint32_t),
                          hipMemcpyDeviceToHost));

    // No need to continue since there are no leaves to split
    if(hSplitLeafCount == 0) return;   
    // Each individual node will create two childs
    uint32_t extraChildCount = hSplitLeafCount * 2;
    // And we need one extra tree
    uint32_t extraTreeCount = hSplitLeafCount;

    // Old Tree count will be the next "allocation"
    uint32_t oldTreeCount = static_cast<uint32_t>(dTrees.size());    
    // Expand nodes
    uint32_t oldNodeCount = static_cast<uint32_t>(nodeCount);
    ExpandTree(nodeCount + extraChildCount);
    nodeCount += extraChildCount;

    DeviceMemory oldTreeIds(hSplitLeafCount * sizeof(uint32_t));
    gpu.GridStrideKC_X(0, 0, hSplitLeafCount,
                       //
                       KCSplitSTree,
                       //
                       static_cast<uint32_t*>(oldTreeIds),
                       *dSTree,
                       //
                       dDenseIndices,
                       oldNodeCount,
                       oldTreeCount,
                       hSplitLeafCount);
    // Copy old indices to the CPU
    std::vector<uint32_t> hOldTreeIds(hSplitLeafCount);
    CUDA_CHECK(hipMemcpy(hOldTreeIds.data(),
                          static_cast<uint32_t*>(oldTreeIds),
                          hSplitLeafCount * sizeof(uint32_t),
                          hipMemcpyDeviceToHost));

    // Create the tree copies
    for(uint32_t i = 0; i < extraTreeCount; i++)
    {
        // Copy the old tree to the new
        DTree& oldTree = dTrees[hOldTreeIds[i]];
        dTrees.push_back(oldTree);
    }        
}

void STree::AccumulateRaidances(const PathGuidingNode* dPGNodes,
                                uint32_t totalNodeCount,
                                uint32_t maxPathNodePerRay,                                
                                const CudaSystem& system)
{   
    const CudaGPU& bestGPU = system.BestGPU();

    std::set<ArrayPortion<uint32_t>> partitions;
    DeviceMemory sortedIndices;

    CUDA_CHECK(hipSetDevice(bestGPU.DeviceId()));    
    PartitionGPU(partitions, sortedIndices,
                 dPGNodes, totalNodeCount,
                 FetchTreeIdFunctor(),
                 static_cast<uint32_t>(dTrees.size()));

    const GPUList& gpuList = system.GPUList();
    auto currentGPU = gpuList.cbegin();
    // Call kernels
    for(const auto& partition : partitions)
    {        
        uint32_t treeIndex = partition.portionId;
        // Skip if these nodes are invalid
        if(treeIndex == InvalidDTreeIndex) continue;

        dTrees[treeIndex].AddRadiancesFromPaths(static_cast<uint32_t*>(sortedIndices),
                                                dPGNodes, partition,
                                                maxPathNodePerRay,
                                                *currentGPU);
        // Get a next GPU if exausted all gpus
        // rool back to start
        currentGPU++;
        if(currentGPU == gpuList.cend()) currentGPU = gpuList.cbegin();
    }
    
    // Wait all gpus to finish
    system.SyncAllGPUs();
}

void STree::SwapTrees(float fluxRatio, uint32_t depthLimit, const CudaSystem& system)
{
    const GPUList& gpuList = system.GPUList();
    auto currentGPU = gpuList.cbegin();
    for(DTree& dTree : dTrees)
    {
        // Call swap function for a kernel
        dTree.SwapTrees(fluxRatio, depthLimit, *currentGPU);
        // Get a next GPU if exausted all gpus
        // rool back to start
        currentGPU++;
        if(currentGPU == gpuList.cend()) currentGPU = gpuList.cbegin();
    }
    CUDA_CHECK(hipDeviceSynchronize());
}

void STree::SplitAndSwapTrees(uint32_t sTreeMaxSamplePerLeaf,
                              float dTreeFluxRatio, 
                              uint32_t dTreeDepthLimit,
                              const CudaSystem& system)
{   
    SplitLeaves(sTreeMaxSamplePerLeaf, system);
    SwapTrees(dTreeFluxRatio, dTreeDepthLimit, system);

    // Adjust DTree pointers for Tracer Kernels
    readDTreeGPUBuffer = std::move(LinearizeDTreeGPUPtrs(true));
    dReadDTrees = static_cast<const DTreeGPU**>(readDTreeGPUBuffer);
}

void STree::GetTreeToCPU(STreeGPU& treeCPU, std::vector<STreeNode>& nodesCPU) const
{
    CUDA_CHECK(hipMemcpy(&treeCPU, dSTree, sizeof(STreeGPU),
                          hipMemcpyDeviceToHost));
    nodesCPU.resize(nodeCount);
    const STreeNode* dSTreeNodes = treeCPU.gRoot;
    CUDA_CHECK(hipMemcpy(nodesCPU.data(), dSTreeNodes, 
                          nodeCount * sizeof(STreeNode),
                          hipMemcpyDeviceToHost));
}