﻿#include "STree.cuh"
#include "PathNode.cuh"
#include "CudaSystem.h"
#include "CudaSystem.hpp"
#include "STreeKC.cuh"

#include "TracerDebug.h"

#include "RayLib/MemoryAlignment.h"

#include <hipcub/hipcub.hpp>

static constexpr size_t AlignedOffsetSTreeGPU = Memory::AlignSize(sizeof(STreeGPU));

struct IsSplittedLeafFunctor
{
    __device__ __host__ __forceinline__
    bool operator()(const uint32_t& index) const
    {
        return (index != INVALID_NODE);
    }
};

void STree::ExpandTree(size_t newNodeCount)
{
    // If its already large do not do stuff
    size_t currentCapacity = 0;
    if(memory.Size() > AlignedOffsetSTreeGPU)
        currentCapacity = (memory.Size() - AlignedOffsetSTreeGPU) / sizeof(STreeGPU);
    if(currentCapacity >= newNodeCount) return;

    DeviceMemory newMem(AlignedOffsetSTreeGPU + newNodeCount * sizeof(STreeNode));
    // Copy the old stuff
    if(memory.Size() > 0)
        CUDA_CHECK(hipMemcpy(static_cast<Byte*>(newMem),
                              static_cast<const Byte*>(memory),
                              AlignedOffsetSTreeGPU + nodeCount * sizeof(STreeNode),
                              hipMemcpyHostToDevice));

    // Set new STree
    dSTree = static_cast<STreeGPU*>(newMem);
    // Copy the new node pointer
    Byte* nodeStart = static_cast<Byte*>(newMem) + AlignedOffsetSTreeGPU;
    Byte* nodePtrLoc = static_cast<Byte*>(newMem) + offsetof(STreeGPU, gRoot);
    CUDA_CHECK(hipMemcpy(nodePtrLoc, &nodeStart, sizeof(STreeNode*),
                          hipMemcpyHostToDevice));
    memory = std::move(newMem);
}

STree::STree(const AABB3f& sceneExtents,
             const CudaSystem& system)
    : nodeCount(0)
    , dSTree(nullptr)
{
    ExpandTree(INITIAL_NODE_CAPACITY);

    // There should be at least one node
    STreeNode node;
    node.isLeaf = true;
    node.splitAxis = STreeNode::AxisType::X;
    node.index = 0; // This shows tree index since isLeaf is true
    Byte* nodeStart = static_cast<Byte*>(memory) + AlignedOffsetSTreeGPU;
    CUDA_CHECK(hipMemcpy(nodeStart, &node, sizeof(STreeNode),
                          hipMemcpyHostToDevice));

    // Update total node count aswell
    nodeCount = 1;
    Byte* nodeCountLocPtr = static_cast<Byte*>(memory) + offsetof(STreeGPU, nodeCount);
    CUDA_CHECK(hipMemcpy(nodeCountLocPtr, &nodeCount, sizeof(uint32_t),
               hipMemcpyHostToDevice));
    // Copy AABB aswell
    // Copy slightly larger AABB to prevent numerical unstabilities
    AABB3f sceneAABB = AABB3f(sceneExtents.Min() - MathConstants::Epsilon,
                              sceneExtents.Max() + MathConstants::Epsilon);
    Byte* nodeAABBLoc = static_cast<Byte*>(memory) + offsetof(STreeGPU, extents);
    CUDA_CHECK(hipMemcpy(nodeAABBLoc, &sceneAABB, sizeof(AABB3f),
                          hipMemcpyHostToDevice));

    dTrees.AllocateDefaultTrees(1, system);
}

void STree::SplitLeaves(uint32_t maxSamplesPerNode,
                        const CudaSystem& system)
{
    const CudaGPU& gpu = system.BestGPU();
    CUDA_CHECK(hipSetDevice(gpu.DeviceId()));
    // Temporary memories
    size_t deviceIfTempMemSize;
    DeviceMemory writeDTreeGPUBuffer;
    DeviceMemory oldTreeIds;
    DeviceMemory tempMemory;
    DeviceMemory selectedIndices;
    DeviceMemory splitMarks;

    // Loop untill no subdivision is left
    uint32_t offset = 0;
    uint32_t processedNodeCount = static_cast<uint32_t>(nodeCount);
    while(processedNodeCount > 0)
    {
        // Resize if buffer if required
        hipcub::DeviceSelect::If(nullptr, deviceIfTempMemSize,
                              static_cast<uint32_t*>(splitMarks),
                              static_cast<uint32_t*>(splitMarks),
                              static_cast<uint32_t*>(splitMarks),
                              static_cast<int>(processedNodeCount),
                              IsSplittedLeafFunctor());

        DeviceMemory::EnlargeBuffer(tempMemory, deviceIfTempMemSize);
        DeviceMemory::EnlargeBuffer(splitMarks, processedNodeCount * sizeof(uint32_t));
        DeviceMemory::EnlargeBuffer(selectedIndices, (processedNodeCount + 1) *sizeof(uint32_t));

        // Mark Leafs
        gpu.GridStrideKC_X(0, 0, nodeCount,
                           //
                           KCMarkSTreeSplitLeaf,
                           //
                           static_cast<uint32_t*>(splitMarks),
                           *dSTree,
                           dTrees.WriteTrees(),
                           maxSamplesPerNode,
                           offset,
                           static_cast<uint32_t>(processedNodeCount));

        // Make dense leaf indices from sparse mark indices
        uint32_t* dDenseIndexCount = static_cast<uint32_t*>(selectedIndices);
        uint32_t* dDenseIndices = static_cast<uint32_t*>(selectedIndices) + 1;
        hipcub::DeviceSelect::If(static_cast<void*>(tempMemory), deviceIfTempMemSize,
                              static_cast<uint32_t*>(splitMarks),
                              dDenseIndices, dDenseIndexCount,
                              static_cast<int>(processedNodeCount),
                              IsSplittedLeafFunctor());

        // Check how many new trees we need to create
        // then allocate these trees
        uint32_t hSubdivisionCount;
        CUDA_CHECK(hipMemcpy(&hSubdivisionCount, dDenseIndexCount,
                              sizeof(uint32_t),
                              hipMemcpyDeviceToHost));

        // No need to continue since there are no leaves to split
        if(hSubdivisionCount == 0) break;
        // Each individual node will create two childs
        uint32_t extraChildCount = hSubdivisionCount * 2;
        // And we need one extra tree
        //uint32_t extraTreeCount = hSubdivisionCount;

        // Old Tree count will be the next "allocation"
        uint32_t oldTreeCount = dTrees.TreeCount();
        // Expand nodes
        uint32_t oldNodeCount = static_cast<uint32_t>(nodeCount);
        ExpandTree(nodeCount + extraChildCount);
        nodeCount += extraChildCount;

        DeviceMemory::EnlargeBuffer(oldTreeIds, hSubdivisionCount * sizeof(uint32_t));
        gpu.GridStrideKC_X(0, 0, hSubdivisionCount,
                           //
                           KCSplitSTree,
                           //
                           static_cast<uint32_t*>(oldTreeIds),
                           *dSTree,
                           //
                           dDenseIndices,
                           //
                           offset,
                           oldNodeCount,
                           oldTreeCount,
                           hSubdivisionCount);
        // Copy old indices to the CPU
        std::vector<uint32_t> hOldTreeIds(hSubdivisionCount);
        CUDA_CHECK(hipMemcpy(hOldTreeIds.data(),
                              static_cast<uint32_t*>(oldTreeIds),
                              hSubdivisionCount * sizeof(uint32_t),
                              hipMemcpyDeviceToHost));

        // Create the tree copies
        dTrees.AllocateExtra(hOldTreeIds, system);

        // Now get redy for next iteration
        offset = oldNodeCount;
        processedNodeCount = extraChildCount;

    }
    // Finally copy the new node count to the GPU
    Byte* nodeCountLocPtr = static_cast<Byte*>(memory) + offsetof(STreeGPU, nodeCount);
    CUDA_CHECK(hipMemcpy(nodeCountLocPtr, &nodeCount, sizeof(uint32_t),
                          hipMemcpyHostToDevice));

    // Subdivided recursively untill all leaf nodes
    // have sample count less than "maxSamplesPerNode"
    // All done!
}

void STree::AccumulateRaidances(const PathGuidingNode* dPGNodes,
                                uint32_t totalNodeCount,
                                uint32_t maxPathNodePerRay,
                                const CudaSystem& system)
{
    if(totalNodeCount == 0) return;

    dTrees.AddRadiancesFromPaths(dPGNodes,
                                 totalNodeCount,
                                 maxPathNodePerRay,
                                 system);
}

void STree::SwapTrees(float fluxRatio, uint32_t depthLimit, const CudaSystem& system)
{
    dTrees.SwapTrees(fluxRatio, depthLimit, system);
    CUDA_CHECK(hipDeviceSynchronize());
}

void STree::SplitAndSwapTrees(uint32_t sTreeMaxSamplePerLeaf,
                              float dTreeFluxRatio,
                              uint32_t dTreeDepthLimit,
                              const CudaSystem& system)
{
    SplitLeaves(sTreeMaxSamplePerLeaf, system);
    SwapTrees(dTreeFluxRatio, dTreeDepthLimit, system);
}

void STree::GetTreeToCPU(STreeGPU& treeCPU, std::vector<STreeNode>& nodesCPU) const
{
    CUDA_CHECK(hipMemcpy(&treeCPU, dSTree, sizeof(STreeGPU),
                          hipMemcpyDeviceToHost));
    nodesCPU.resize(nodeCount);
    const STreeNode* dSTreeNodes = treeCPU.gRoot;
    CUDA_CHECK(hipMemcpy(nodesCPU.data(), dSTreeNodes,
                          nodeCount * sizeof(STreeNode),
                          hipMemcpyDeviceToHost));
}

const DTreeGroup& STree::DTrees() const
{
    return dTrees;
}

void STree::GetAllDTreesToCPU(std::vector<DTreeGPU>& dTreeStructs,
                              std::vector<std::vector<DTreeNode>>& dTreeNodes,
                              bool fetchReadTree) const
{
    dTreeStructs.reserve(dTrees.TreeCount());
    dTreeNodes.reserve(dTrees.TreeCount());
    for(uint32_t i = 0 ; i < dTrees.TreeCount(); i++)
    {
        DTreeGPU currentStruct;
        std::vector<DTreeNode> currentNodes;
        if(fetchReadTree)
        {
            dTrees.GetReadTreeToCPU(currentStruct, currentNodes, i);
        }
        else
        {
            dTrees.GetWriteTreeToCPU(currentStruct, currentNodes, i);
        }
        dTreeStructs.push_back(std::move(currentStruct));
        dTreeNodes.push_back(std::move(currentNodes));
    }
}

void STree::DumpSDTreeAsBinary(std::vector<Byte>& data,
                               bool fetchReadTree) const
{
    std::vector<Byte> sTree;
    std::vector<std::vector<Byte>> dTreeBinary(dTrees.TreeCount());
    std::vector<Vector2ul> countOffsetPairs(dTrees.TreeCount());

    uint64_t sTreeStartOffset = (sizeof(uint64_t) +
                                 sizeof(uint64_t) +
                                 sizeof(uint64_t) +
                                 sizeof(Vector2ul) * dTrees.TreeCount());


    size_t offset = (sTreeStartOffset +
                     sizeof(AABB3f) +
                     nodeCount * sizeof(STreeNode));
    for(uint32_t i = 0; i < dTrees.TreeCount(); i++)
    {
        dTrees.DumpTreeAsBinary(dTreeBinary[i], i, fetchReadTree);
        countOffsetPairs[i] = Vector2ul(static_cast<uint64_t>(offset),
                                        static_cast<uint64_t>(dTrees.NodeCount(i, fetchReadTree)));
        offset += dTreeBinary[i].size();
        i++;
    }
    data.reserve(offset);

    // Write STree Start Offset
    data.insert(data.end(),
                reinterpret_cast<Byte*>(&sTreeStartOffset),
                reinterpret_cast<Byte*>(&sTreeStartOffset) + sizeof(uint64_t));
    // Write STree Node Count
    uint64_t sTreeNodeCount = static_cast<uint64_t>(nodeCount);
    data.insert(data.end(),
                reinterpret_cast<Byte*>(&sTreeNodeCount),
                reinterpret_cast<Byte*>(&sTreeNodeCount) + sizeof(uint64_t));
    // Write DTree Count
    uint64_t dTreeCount = static_cast<uint64_t>(dTrees.TreeCount());
    assert(countOffsetPairs.size() == dTreeCount);
    data.insert(data.end(),
                reinterpret_cast<Byte*>(&dTreeCount),
                reinterpret_cast<Byte*>(&dTreeCount) + sizeof(uint64_t));
    // Write DTree Offset/Count Pairs
    data.insert(data.end(),
                reinterpret_cast<Byte*>(countOffsetPairs.data()),
                (reinterpret_cast<Byte*>(countOffsetPairs.data()) +
                 sizeof(Vector2ul) * countOffsetPairs.size()));
    // Write STree
    STreeGPU sTreeBase;
    std::vector<STreeNode> sTreeNodes;
    GetTreeToCPU(sTreeBase, sTreeNodes);
    data.insert(data.end(),
                reinterpret_cast<Byte*>(&sTreeBase.extents),
                reinterpret_cast<Byte*>(&sTreeBase.extents) + sizeof(AABB3f));
    data.insert(data.end(),
                reinterpret_cast<Byte*>(sTreeNodes.data()),
                (reinterpret_cast<Byte*>(sTreeNodes.data()) +
                 sizeof(STreeNode) * sTreeNodes.size()));

    // Write DTrees in order
    for(const std::vector<Byte>& dTree : dTreeBinary)
    {
        data.insert(data.end(), dTree.cbegin(), dTree.cend());
    }
}