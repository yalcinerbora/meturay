﻿#include "DirectTracerWorks.cuh"
// ===================================================
// Direct Tracer Boundary Work Batches
template class DirectTracerBoundaryWork<CPULightGroupNull>;
template class DirectTracerBoundaryWork<CPULightGroupSkySphere>;
// ===================================================
// Direct Tracer Work Batches
template class DirectTracerFurnaceWork<BarycentricMat, GPUPrimitiveTriangle>;
template class DirectTracerFurnaceWork<SphericalMat, GPUPrimitiveSphere>;

template class DirectTracerFurnaceWork<NormalRenderMat, GPUPrimitiveTriangle>;
template class DirectTracerFurnaceWork<NormalRenderMat, GPUPrimitiveSphere>;

template class DirectTracerFurnaceWork<LambertCMat, GPUPrimitiveTriangle>;
template class DirectTracerFurnaceWork<LambertCMat, GPUPrimitiveSphere>;

template class DirectTracerFurnaceWork<LambertMat, GPUPrimitiveTriangle>;
template class DirectTracerFurnaceWork<LambertMat, GPUPrimitiveSphere>;

template class DirectTracerFurnaceWork<UnrealMat, GPUPrimitiveTriangle>;
template class DirectTracerFurnaceWork<UnrealMat, GPUPrimitiveSphere>;
// ===================================================
template class DirectTracerNormalWork<GPUPrimitiveEmpty>;
template class DirectTracerNormalWork<GPUPrimitiveTriangle>;
template class DirectTracerNormalWork<GPUPrimitiveSphere>;