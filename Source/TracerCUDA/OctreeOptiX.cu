#include "hip/hip_runtime.h"
#include "OctreeOptiX.h"
#include "OptixCheck.h"

#include "RayLib/CPUTimer.h"

#include <optix_stack_size.h>

template<class T>
__global__
void KCGenAABBAndMortonCode(// Ouptuts
                            AABB3f* gAABBs,
                            T* gMortonCodes,
                            // Inputs
                            AnisoSVOctreeGPU svo,
                            uint32_t level,
                            uint32_t nodeCount)
{
    float levelVoxSize = svo.LevelVoxelSize(level);
    uint32_t levelOffset = svo.LevelNodeStart(level);
    bool isLeaf = (level == svo.LeafDepth());

    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < nodeCount;
        threadId += (blockDim.x * gridDim.x))
    {
        uint32_t nodeId = levelOffset + threadId;
        uint32_t depth;
        Vector3ui voxelId = svo.NodeVoxelId(depth, nodeId, isLeaf);

        // Gen AABB using VoxelId
        Vector3f voxIdF = Vector3f(voxelId);
        Vector3f voxAABBMin = svo.OctreeAABB().Min() + voxIdF * levelVoxSize;
        Vector3f voxAABBMax = voxAABBMin + levelVoxSize;

        // Write
        gMortonCodes[threadId] = MortonCode::Compose3D<T>(voxelId);
        gAABBs[threadId] = AABB3f(voxAABBMin, voxAABBMax);
    }
}

SVOOptixConeCaster::SVOOptixConeCaster(const OptiXSystem& optixSystem)
    : optixSystem(optixSystem)
    , paramsMemory(&optixSystem.OptixCapableDevices()[0].first,
                   sizeof(OpitXBaseAccelParams))
    , sbtMemory(&optixSystem.OptixCapableDevices()[0].first)
    , mortonMemory(&optixSystem.OptixCapableDevices()[0].first)
{
    const auto& [gpu, optixContext] = optixSystem.OptixCapableDevices()[0];

    // Now do OptiX boilerplate
    // =============================== //
    //       MODULE GENERATION         //
    // =============================== //
    OptixModuleCompileOptions moduleCompileOpts = {};
    OptixPipelineCompileOptions pipelineCompileOpts = {};
    moduleCompileOpts.maxRegisterCount = OPTIX_COMPILE_DEFAULT_MAX_REGISTER_COUNT;
    if constexpr(METU_DEBUG_BOOL)
    {
        moduleCompileOpts.optLevel = OPTIX_COMPILE_OPTIMIZATION_LEVEL_0;
        moduleCompileOpts.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_FULL;
        pipelineCompileOpts.exceptionFlags = (OPTIX_EXCEPTION_FLAG_DEBUG |
                                              OPTIX_EXCEPTION_FLAG_TRACE_DEPTH |
                                              OPTIX_EXCEPTION_FLAG_STACK_OVERFLOW);
    }
    else
    {
        moduleCompileOpts.optLevel = OPTIX_COMPILE_OPTIMIZATION_DEFAULT;

        #if OPTIX_VERSION > 70300
        moduleCompileOpts.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_MINIMAL;
        #else
        moduleCompileOpts.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_NONE;
        #endif
        pipelineCompileOpts.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
    }
    pipelineCompileOpts.usesMotionBlur = false;
    pipelineCompileOpts.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
    pipelineCompileOpts.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
    pipelineCompileOpts.numPayloadValues = 3;
    pipelineCompileOpts.numAttributeValues = 0;
    pipelineCompileOpts.pipelineLaunchParamsVariableName = "params";

    TracerError err = TracerError::OK;
    std::vector<Byte> ptxSource;
    if((err = OptiXSystem::LoadPTXFile(ptxSource, gpu, MODULE_BASE_NAME)) != TracerError::OK)
        throw TracerException(err);

    OPTIX_CHECK(optixModuleCreateFromPTX(optixContext,
                                         &moduleCompileOpts, &pipelineCompileOpts,
                                         reinterpret_cast<const char*>(ptxSource.data()),
                                         ptxSource.size(),
                                         nullptr,
                                         nullptr,
                                         &mdl));

    OptixProgramGroupOptions pgOpts = {};
    OptixModule gpuModule = mdl;

    // RADIANCE GEN RAYGEN NAME
    programGroups.emplace_back();
    OptixProgramGroupDesc rgRadGenProgramDesc = {};
    rgRadGenProgramDesc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
    rgRadGenProgramDesc.raygen.module = gpuModule;
    rgRadGenProgramDesc.raygen.entryFunctionName = RAYGEN_RAD_FUNC_NAME;
    OPTIX_CHECK(optixProgramGroupCreate(optixContext,
                                        &rgRadGenProgramDesc, 1,
                                        &pgOpts,
                                        nullptr, 0,
                                        &programGroups.back()));
    assert(programGroups.size() == (RAD_RAYGEN_PG_INDEX + 1));

    // CAMERA GEN RAYGEN NAME
    programGroups.emplace_back();
    OptixProgramGroupDesc rgCamGenProgramDesc = {};
    rgCamGenProgramDesc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
    rgCamGenProgramDesc.raygen.module = gpuModule;
    rgCamGenProgramDesc.raygen.entryFunctionName = RAYGEN_CAM_FUNC_NAME;
    OPTIX_CHECK(optixProgramGroupCreate(optixContext,
                                        &rgCamGenProgramDesc, 1,
                                        &pgOpts,
                                        nullptr, 0,
                                        &programGroups.back()));
    assert(programGroups.size() == (CAM_RAYGEN_PG_INDEX + 1));

    // EMPTY MISS NAME
    programGroups.emplace_back();
    OptixProgramGroupDesc missProgramDesc = {};
    missProgramDesc.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;
    missProgramDesc.miss.module = gpuModule;
    missProgramDesc.miss.entryFunctionName = MISS_FUNC_NAME;
    OPTIX_CHECK(optixProgramGroupCreate(optixContext,
                                        &missProgramDesc, 1,
                                        &pgOpts,
                                        nullptr, 0,
                                        &programGroups.back()));
    assert(programGroups.size() == (MISS_PG_INDEX + 1));

    // HIT GROUP NAME (MORTON 32)
    programGroups.emplace_back();
    OptixProgramGroupDesc h32ProgramDesc = {};
    h32ProgramDesc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
    h32ProgramDesc.hitgroup.moduleCH = gpuModule;
    h32ProgramDesc.hitgroup.entryFunctionNameCH = CHIT_FUNC_NAME;
    h32ProgramDesc.hitgroup.moduleAH = nullptr;
    h32ProgramDesc.hitgroup.entryFunctionNameAH = nullptr;
    h32ProgramDesc.hitgroup.moduleIS = gpuModule;
    h32ProgramDesc.hitgroup.entryFunctionNameIS = INTERSECT32_FUNC_NAME;
    OPTIX_CHECK(optixProgramGroupCreate(optixContext,
                                        &h32ProgramDesc, 1,
                                        &pgOpts,
                                        nullptr, 0,
                                        &programGroups.back()));
    assert(programGroups.size() == (MORTON32_HIT_PG_INDEX + 1));

    // HIT GROUP NAME (MORTON 64)
    programGroups.emplace_back();
    OptixProgramGroupDesc h64ProgramDesc = {};
    h64ProgramDesc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
    h64ProgramDesc.hitgroup.moduleCH = gpuModule;
    h64ProgramDesc.hitgroup.entryFunctionNameCH = CHIT_FUNC_NAME;
    h64ProgramDesc.hitgroup.moduleAH = nullptr;
    h64ProgramDesc.hitgroup.entryFunctionNameAH = nullptr;
    h64ProgramDesc.hitgroup.moduleIS = gpuModule;
    h64ProgramDesc.hitgroup.entryFunctionNameIS = INTERSECT64_FUNC_NAME;
    OPTIX_CHECK(optixProgramGroupCreate(optixContext,
                                        &h64ProgramDesc, 1,
                                        &pgOpts,
                                        nullptr, 0,
                                        &programGroups.back()));
    assert(programGroups.size() == (MORTON64_HIT_PG_INDEX + 1));

    // =============================== //
    //      PIPELINE GENERATION        //
    // =============================== //
    OptixPipelineLinkOptions pipelineLinkOpts = {};
    pipelineLinkOpts.maxTraceDepth = 1;
    pipelineLinkOpts.debugLevel = moduleCompileOpts.debugLevel;

    const std::vector<OptixProgramGroup>& pgs = programGroups;

    OPTIX_CHECK(optixPipelineCreate(optixContext,
                                    &pipelineCompileOpts, &pipelineLinkOpts,
                                    pgs.data(),
                                    static_cast<uint32_t>(pgs.size()),
                                    nullptr, nullptr,
                                    &pipeline));

    // We need to specify the max traversal depth.  Calculate the stack sizes, so we can specify all
    // parameters to optixPipelineSetStackSize.
    OptixStackSizes stack_sizes = {};
    for(const auto& pg : pgs)
        OPTIX_CHECK(optixUtilAccumulateStackSizes(pg, &stack_sizes));

    uint32_t dcStackSizeTraverse;
    uint32_t dcStackSizeState;
    uint32_t contStackSize;
    OPTIX_CHECK(optixUtilComputeStackSizes(&stack_sizes,
                                           1,   // max trace depth
                                           0, 0,
                                           &dcStackSizeTraverse,
                                           &dcStackSizeState,
                                           &contStackSize));

    const uint32_t maxTraversalDepth = 1; // Single GAS
    OPTIX_CHECK(optixPipelineSetStackSize(pipeline,
                                          dcStackSizeTraverse,
                                          dcStackSizeState,
                                          contStackSize,
                                          maxTraversalDepth));
}

SVOOptixConeCaster::~SVOOptixConeCaster()
{
    OPTIX_CHECK(optixPipelineDestroy(pipeline));
    for(auto& pg : programGroups)
        OPTIX_CHECK(optixProgramGroupDestroy(pg));
    OPTIX_CHECK(optixModuleDestroy(mdl));
}

void SVOOptixConeCaster::GenerateSVOTraversable(const AnisoSVOctreeCPU& svoCPU)
{
    Utility::CPUTimer t;
    t.Start();

    const auto& [gpu, optixContext] = optixSystem.OptixCapableDevices()[0];

    std::vector<uint32_t> levelNodeOffsets = svoCPU.LevelNodeOffsets();
    AnisoSVOctreeGPU svo = svoCPU.TreeGPU();
    hSVOOptixLaunchParams.svo = svo;

    // Allocate mortonCodeMemory
    // For records
    // Slightly improve memory here use 32-bit for 1024 levels
    static constexpr uint32_t VOXEL_MORTON3D_FIT_THRESHOLD = 10;
    static constexpr std::array<size_t, 2> MORTON_SIZE = {sizeof(uint32_t), sizeof(uint64_t)};
    std::vector<size_t> mortonOffsets;
    mortonOffsets.reserve(svo.LeafDepth() + 1);

    // Record type for the svo level
    std::vector<uint32_t> levelRecordTypeIndex;

    size_t offset = 0;
    // Determine Size & Record Type (32-bit or 64-bit)
    for(uint32_t i = 1; i <= svo.LeafDepth(); i++)
    {
        uint32_t localPrimCount = levelNodeOffsets[i + 1] - levelNodeOffsets[i];
        bool is32BitRecord = (i <= VOXEL_MORTON3D_FIT_THRESHOLD);

        size_t mortonSize = (is32BitRecord) ? MORTON_SIZE[0] : MORTON_SIZE[1];

        levelRecordTypeIndex.push_back((is32BitRecord) ? LEVEL_32_BIT : LEVEL_64_BIT);
        size_t localSize = localPrimCount * mortonSize;
        mortonOffsets.push_back(offset);
        offset += localSize;
    }
    mortonOffsets.push_back(offset);
    mortonMemory = DeviceLocalMemory(&gpu, mortonOffsets.back());

    std::vector<uint32_t*> dMortonPtrs32(svo.LeafDepth() + 1, nullptr);
    std::vector<uint64_t*> dMortonPtrs64(svo.LeafDepth() + 1, nullptr);
    for(uint32_t i = 1; i <= svo.LeafDepth(); i++)
    {
        uint32_t localPrimCount = levelNodeOffsets[i + 1] - levelNodeOffsets[i];
        Byte* dMortonStart = static_cast<Byte*>(mortonMemory) + mortonOffsets[i];
        if(i <= VOXEL_MORTON3D_FIT_THRESHOLD)
            dMortonPtrs32[i] = reinterpret_cast<uint32_t*>(dMortonStart);
        else
            dMortonPtrs64[i] = reinterpret_cast<uint64_t*>(dMortonStart);
    }

    DeviceLocalMemory tempAABB(&gpu, svo.LeafCount() * sizeof(AABB3f));
    AABB3f* dTempAABBs = static_cast<AABB3f*>(tempAABB);

    // Call a kernel for each level to generate AABB
    for(uint32_t i = 1; i <= svo.LeafDepth(); i++)
    {
        uint32_t localPrimCount = levelNodeOffsets[i + 1] - levelNodeOffsets[i];

        // Generate AABB
        if(dMortonPtrs32[i] != nullptr)
        {
            gpu.GridStrideKC_X(0, (hipStream_t)0, localPrimCount,
                               //
                               KCGenAABBAndMortonCode<uint32_t>,
                               // Outputs
                               dTempAABBs,
                               dMortonPtrs32[i],
                               // Inputs
                               svo,
                               i,
                               localPrimCount);
        }
        else
        {
            gpu.GridStrideKC_X(0, (hipStream_t)0, localPrimCount,
                               //
                               KCGenAABBAndMortonCode<uint64_t>,
                               // Outputs
                               dTempAABBs,
                               dMortonPtrs64[i],
                               // Inputs
                               svo,
                               i,
                               localPrimCount);
        }
        //
        uint32_t flags = OPTIX_GEOMETRY_FLAG_DISABLE_ANYHIT;
        hipDeviceptr_t aabbBuffer = AsOptixPtr(dTempAABBs);

        OptixBuildInput buildInput = {};
        buildInput.type = OPTIX_BUILD_INPUT_TYPE_CUSTOM_PRIMITIVES;
        // AABB
        buildInput.customPrimitiveArray.aabbBuffers = &aabbBuffer;
        buildInput.customPrimitiveArray.numPrimitives = static_cast<uint32_t>(localPrimCount);
        buildInput.customPrimitiveArray.strideInBytes = sizeof(AABB3f);
        buildInput.customPrimitiveArray.primitiveIndexOffset = 0;
        // SBT
        buildInput.customPrimitiveArray.flags = &flags;
        buildInput.customPrimitiveArray.numSbtRecords = 1;
        buildInput.customPrimitiveArray.sbtIndexOffsetBuffer = 0;
        buildInput.customPrimitiveArray.sbtIndexOffsetSizeInBytes = sizeof(uint32_t);
        buildInput.customPrimitiveArray.sbtIndexOffsetStrideInBytes = sizeof(uint32_t);

        OptixAccelBuildOptions accelOptions = {};
        accelOptions.buildFlags = OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
        accelOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

        OptixAccelBufferSizes accelMemorySizes;
        OPTIX_CHECK(optixAccelComputeMemoryUsage
        (
            optixContext,
            &accelOptions, &buildInput,
            1, &accelMemorySizes
        ));

        // Allocate Temp Buffer for Build
        DeviceLocalMemory buildBuffer(&gpu, accelMemorySizes.outputSizeInBytes);
        Byte* dTempBuild = static_cast<Byte*>(buildBuffer);
        Byte* dTemp;
        uint64_t* dCompactedSize;
        DeviceLocalMemory tempMemory(&gpu);
        GPUMemFuncs::AllocateMultiData(std::tie(dTemp, dCompactedSize), tempMemory,
                                       {accelMemorySizes.tempSizeInBytes, 1}, 128);

        // While building fetch compacted output size
        OptixAccelEmitDesc emitProperty = {};
        emitProperty.type = OPTIX_PROPERTY_TYPE_COMPACTED_SIZE;
        emitProperty.result = AsOptixPtr(dCompactedSize);

        OptixTraversableHandle traversable;
        OPTIX_CHECK(optixAccelBuild(optixContext, (hipStream_t)0,
                                    &accelOptions,
                                    // Build Inputs
                                    &buildInput, 1,
                                    // Temp Memory
                                    AsOptixPtr(dTemp), accelMemorySizes.tempSizeInBytes,
                                    // Output Memory
                                    AsOptixPtr(dTempBuild), accelMemorySizes.outputSizeInBytes,
                                    &traversable, &emitProperty, 1));
        CUDA_KERNEL_CHECK();

        // Get compacted size to CPU
        uint64_t hCompactAccelSize;
        CUDA_CHECK(hipMemcpy(&hCompactAccelSize, dCompactedSize,
                              sizeof(uint64_t), hipMemcpyDeviceToHost));

        if(hCompactAccelSize < buildBuffer.Size())
        {
            DeviceLocalMemory compactedMemory(&gpu, hCompactAccelSize);

            // use handle as input and output
            OPTIX_CHECK(optixAccelCompact(optixContext, (hipStream_t)0,
                                          traversable,
                                          AsOptixPtr(compactedMemory),
                                          hCompactAccelSize,
                                          &traversable));
            CUDA_KERNEL_CHECK();

            svoLevelAcceleratorMemory.emplace_back(std::move(compactedMemory));
        }
        else
            svoLevelAcceleratorMemory.emplace_back(std::move(buildBuffer));

        svoLevelAccelerators.emplace_back(traversable);
    }

    t.Split();
    METU_LOG("OptiX SVO GAS hierarchy generated in {:f} ms.",
             t.Elapsed<CPUTimeMillis>());


    // =============================== //
    //     SHADER BINDING TABLES       //
    // =============================== //
    CUDA_CHECK(hipSetDevice(gpu.DeviceId()));
    // Calculate the HR Amount
    SVOEmptyRecord* dRadRaygenRecord;
    SVOEmptyRecord* dCamRaygenRecord;
    SVOEmptyRecord* dMissRecord;
    SVOHitRecord<uint64_t>* dHitRecords;
    // Sanity Check
    static_assert(sizeof(SVOHitRecord<uint64_t>) == sizeof(SVOHitRecord<uint32_t>));
    static constexpr uint32_t HIT_RECORD_STRIDE = static_cast<uint32_t>(std::max(sizeof(SVOHitRecord<uint64_t>),
                                                                                 sizeof(SVOHitRecord<uint32_t>)));

    const uint32_t recordCount = static_cast<uint32_t>(svo.LeafDepth());

    GPUMemFuncs::AllocateMultiData(std::tie(dCamRaygenRecord,
                                            dRadRaygenRecord,
                                            dMissRecord,
                                            dHitRecords),
                                   sbtMemory,
                                   {1, 1, 1, (svo.LeafDepth() + 1)},
                                   OPTIX_SBT_RECORD_ALIGNMENT);

    SVOEmptyRecord hRadRGRecord = SVOEmptyRecord{};
    SVOEmptyRecord hCamRGRecord = SVOEmptyRecord{};
    SVOEmptyRecord hMissRecord = SVOEmptyRecord{};
    SVOHitRecord<uint32_t> hHitRecord32 = SVOHitRecord<uint32_t>{};
    SVOHitRecord<uint64_t> hHitRecord64 = SVOHitRecord<uint64_t>{};

    // Set Raygen Record
    OPTIX_CHECK(optixSbtRecordPackHeader(programGroups[RAD_RAYGEN_PG_INDEX],
                                         &hRadRGRecord));
    CUDA_CHECK(hipMemcpy(dRadRaygenRecord, &hRadRGRecord, sizeof(SVOEmptyRecord),
                          hipMemcpyHostToDevice));
    OPTIX_CHECK(optixSbtRecordPackHeader(programGroups[CAM_RAYGEN_PG_INDEX],
                                         &hCamRGRecord));
    CUDA_CHECK(hipMemcpy(dCamRaygenRecord, &hCamRGRecord, sizeof(SVOEmptyRecord),
                          hipMemcpyHostToDevice));
    // Set Miss Record
    OPTIX_CHECK(optixSbtRecordPackHeader(programGroups[MISS_PG_INDEX], &hMissRecord));
    CUDA_CHECK(hipMemcpy(dMissRecord, &hMissRecord, sizeof(SVOEmptyRecord),
                          hipMemcpyHostToDevice));
    // Set Hit Record
    // Preset the headers (According to docs it is opaque but copyable if each loc uses same
    // program)
    OPTIX_CHECK(optixSbtRecordPackHeader(programGroups[MORTON32_HIT_PG_INDEX], &hHitRecord32));
    OPTIX_CHECK(optixSbtRecordPackHeader(programGroups[MORTON64_HIT_PG_INDEX], &hHitRecord64));

    for(uint32_t i = 0; i < svo.LeafDepth(); i++)
    {
        if(levelRecordTypeIndex[i] == LEVEL_32_BIT)
        {
            hHitRecord32.dMortonCode = dMortonPtrs32[i];
            CUDA_CHECK(hipMemcpy(dHitRecords + i,
                                  reinterpret_cast<SVOHitRecord<uint64_t>*>(&hHitRecord32),
                                  sizeof(SVOHitRecord<uint32_t>),
                                  hipMemcpyHostToDevice));
        }
        else
        {
            hHitRecord64.dMortonCode = dMortonPtrs64[i];
            CUDA_CHECK(hipMemcpy(dHitRecords + i,
                                  &hHitRecord64, sizeof(SVOHitRecord<uint64_t>),
                                  hipMemcpyHostToDevice));
        }
    }

    // SBT CAM GEN
    // Although we do not use the miss shader
    // Optix mandates these to be set
    sbtRadGen.raygenRecord = AsOptixPtr(dRadRaygenRecord);
    //
    sbtRadGen.missRecordBase = AsOptixPtr(dMissRecord);
    sbtRadGen.missRecordCount = 1;
    sbtRadGen.missRecordStrideInBytes = sizeof(EmptyRecord);
    //
    sbtRadGen.hitgroupRecordBase = AsOptixPtr(dHitRecords);
    sbtRadGen.hitgroupRecordStrideInBytes = sizeof(HIT_RECORD_STRIDE);
    sbtRadGen.hitgroupRecordCount = recordCount;
    // SBT RAD GEN
    sbtCamGen.raygenRecord = AsOptixPtr(dRadRaygenRecord);
    //
    sbtCamGen.missRecordBase = AsOptixPtr(dMissRecord);
    sbtCamGen.missRecordCount = 1;
    sbtCamGen.missRecordStrideInBytes = sizeof(EmptyRecord);
    //
    sbtCamGen.hitgroupRecordBase = AsOptixPtr(dHitRecords);
    sbtCamGen.hitgroupRecordStrideInBytes = sizeof(HIT_RECORD_STRIDE);
    sbtCamGen.hitgroupRecordCount = recordCount;

}

void SVOOptixConeCaster::ConeTraceFromCamera(// Output
                                             CamSampleGMem<Vector3f> gSamples,
                                             // Input
                                             const GPUCameraI* gCamera,
                                             WFPGRenderMode mode,
                                             uint32_t maxQueryLevelOffset,
                                             const Vector2i& totalPixelCount)
{

    // TODO:
    hSVOOptixLaunchParams = {};

    CUDA_CHECK(hipMemcpyAsync(dSVOOptixLaunchParams,
                               &hSVOOptixLaunchParams,
                               sizeof(OctreeAccelParams),
                               hipMemcpyHostToDevice,
                               (hipStream_t)0));
    OPTIX_CHECK(optixLaunch(pipeline, (hipStream_t)0,
                            AsOptixPtr(dSVOOptixLaunchParams),
                            sizeof(OctreeAccelParams),
                            &sbtCamGen,
                            totalPixelCount[0],
                            totalPixelCount[1],
                            1));
}