#include "hip/hip_runtime.h"
﻿#include "GPUCameraSpherical.cuh"
#include "CudaSystem.hpp"
#include "RayLib/MemoryAlignment.h"

__global__
void KCConstructGPUCameraSpherical(GPUCameraSpherical* gCameraLocations,
                                   //
                                   const CPUCameraGroupSpherical::Data* gData,
                                   //
                                   const uint16_t* gMediumIndices,
                                   const HitKey* gWorkKeys,
                                   const TransformId* gTransformIds,
                                   //
                                   const GPUTransformI** gTransforms,
                                   uint32_t camCaount)
{
    for(uint32_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
        globalId < camCaount;
        globalId += blockDim.x * gridDim.x)
    {
        CPUCameraGroupSpherical::Data data = gData[globalId];
        new (gCameraLocations + globalId) GPUCameraSpherical(data.pixelRatio,
                                                             data.position,
                                                             data.direction,
                                                             data.up,
                                                             data.nearFar,
                                                             // Base class
                                                             gMediumIndices[globalId],
                                                             gWorkKeys[globalId],
                                                             *gTransforms[gTransformIds[globalId]]);
    }
}

SceneError CPUCameraGroupSpherical::InitializeGroup(const EndpointGroupDataList& cameraNodes,
                                                    const TextureNodeMap& textures,
                                                    const std::map<uint32_t, uint32_t>& mediumIdIndexPairs,
                                                    const std::map<uint32_t, uint32_t>& transformIdIndexPairs,
                                                    uint32_t batchId, double time,
                                                    const std::string& scenePath)
{
    SceneError e = SceneError::OK;

    if((e = InitializeCommon(cameraNodes, textures,
                             mediumIdIndexPairs,
                             transformIdIndexPairs,
                             batchId, time,
                             scenePath)) != SceneError::OK)
        return e;


    hCameraData.reserve(cameraCount);
    for(const auto& node : cameraNodes)
    {
        const auto position = node.node->CommonVector3(POSITION_NAME);
        const auto up = node.node->CommonVector3(UP_NAME);
        const auto direction = node.node->CommonVector3(DIR_NAME);
        const auto nearFar = node.node->CommonVector2(PLANES_NAME);
        const auto pixRatio = node.node->CommonFloat(PIX_RATIO_NAME);

        Data data = {};
        data.position = position;
        data.up = up;
        data.direction = direction;
        data.nearFar = nearFar;
        data.pixelRatio = pixRatio;

        hCameraData.push_back(data);
    }

    return SceneError::OK;
}

SceneError CPUCameraGroupSpherical::ChangeTime(const NodeListing& lightNodes, double time,
                                               const std::string& scenePath)
{
    return SceneError::CAMERA_TYPE_INTERNAL_ERROR;
}

TracerError CPUCameraGroupSpherical::ConstructEndpoints(const GPUTransformI** dGlobalTransformArray,
                                                        const CudaSystem& system)
{
    // Gen Temporary Memory
    DeviceMemory tempMemory;

    const uint16_t* dMediumIndices;
    const TransformId* dTransformIds;
    const HitKey* dWorkKeys;
    const Data* dData;
    DeviceMemory::AllocateMultiData(std::tie(dMediumIndices, dTransformIds,
                                             dWorkKeys, dData),
                                    tempMemory,
                                    {cameraCount, cameraCount,
                                    cameraCount, cameraCount});

    // Set a GPU
    const CudaGPU& gpu = system.BestGPU();
    CUDA_CHECK(hipSetDevice(gpu.DeviceId()));
    // Load Data to Temp Memory
    CUDA_CHECK(hipMemcpy(const_cast<uint16_t*>(dMediumIndices),
                          hMediumIds.data(),
                          sizeof(uint16_t) * cameraCount,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(const_cast<TransformId*>(dTransformIds),
                          hTransformIds.data(),
                          sizeof(TransformId) * cameraCount,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(const_cast<HitKey*>(dWorkKeys),
                          hWorkKeys.data(),
                          sizeof(HitKey) * cameraCount,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(const_cast<Data*>(dData),
                          hCameraData.data(),
                          sizeof(Data) * cameraCount,
                          hipMemcpyHostToDevice));


    // Call allocation kernel
    gpu.GridStrideKC_X(0, 0,
                       cameraCount,
                       //
                       KCConstructGPUCameraSpherical,
                       //
                       const_cast<GPUCameraSpherical*>(dGPUCameras),
                       //
                       dData,
                       //
                       dMediumIndices,
                       dWorkKeys,
                       dTransformIds,
                       //
                       dGlobalTransformArray,
                       cameraCount);

    gpu.WaitMainStream();

    SetCameraLists();

    return TracerError::OK;
}