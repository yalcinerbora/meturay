#include "hip/hip_runtime.h"
#include "GPULightDisk.cuh"
#include "TypeTraits.h"
#include "RayLib/MemoryAlignment.h"
#include "CudaConstants.hpp"

__global__ void KCConstructGPULightDisk(GPULightDisk* gLightLocations,
                                        //
                                        const Vector3f* gCenters,
                                        const Vector3f* gNormals,
                                        const float* gRadius,
                                        //
                                        const TransformId* gTransformIds,
                                        const uint16_t* gMediumIndices,
                                        const HitKey* gLightMaterialIds,
                                        //
                                        const GPUTransformI** gTransforms,
                                        uint32_t lightCount)
{
    for(uint32_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
        globalId < lightCount;
        globalId += blockDim.x * gridDim.x)
    {
        new (gLightLocations + globalId) GPULightDisk(gCenters[globalId],
                                                      gNormals[globalId],
                                                      gRadius[globalId],
                                                      *gTransforms[gTransformIds[globalId]],
                                                      //
                                                      gLightMaterialIds[globalId],
                                                      gMediumIndices[globalId]);
    }
}

SceneError CPULightGroupDisk::InitializeGroup(const ConstructionDataList& lightNodes,
                                              const std::map<uint32_t, uint32_t>& mediumIdIndexPairs,
                                              const std::map<uint32_t, uint32_t>& transformIdIndexPairs,
                                              const MaterialKeyListing& allMaterialKeys,
                                              double time,
                                              const std::string& scenePath)
{
    lightCount = static_cast<uint32_t>(lightNodes.size());
    hHitKeys.reserve(lightCount);
    hMediumIds.reserve(lightCount);
    hTransformIds.reserve(lightCount);

    hCenters.reserve(lightCount);
    hNormals.reserve(lightCount);
    hRadius.reserve(lightCount);

    for(const auto& node : lightNodes)
    {
        // Convert Ids to inner index
        uint32_t mediumIndex = mediumIdIndexPairs.at(node.mediumId);
        uint32_t transformIndex = transformIdIndexPairs.at(node.transformId);
        HitKey materialKey = allMaterialKeys.at(std::make_pair(BaseConstants::EMPTY_PRIMITIVE_NAME,
                                                               node.materialId));

        const auto centers = node.node->AccessVector3(NAME_POSITION);
        const auto normals = node.node->AccessVector3(NAME_NORMAL);
        const auto radius = node.node->AccessFloat(NAME_RADIUS);

        // Load to host memory
        hHitKeys.push_back(materialKey);
        hMediumIds.push_back(mediumIndex);
        hTransformIds.push_back(transformIndex);
        hCenters.insert(hCenters.end(), centers.begin(), centers.end());
        hNormals.insert(hNormals.end(), normals.begin(), normals.end());
        hRadius.insert(hRadius.end(), radius.begin(), radius.end());
    }

    // Allocate for GPULight classes
    size_t totalClassSize = sizeof(GPULightDisk) * lightCount;
    totalClassSize = Memory::AlignSize(totalClassSize);

    DeviceMemory::EnlargeBuffer(memory, totalClassSize);

    size_t offset = 0;
    std::uint8_t* dBasePtr = static_cast<uint8_t*>(memory);
    dGPULights = reinterpret_cast<const GPULightDisk*>(dBasePtr + offset);
    offset += totalClassSize;
    assert(totalClassSize == offset);

    return SceneError::OK;
}

SceneError CPULightGroupDisk::ChangeTime(const NodeListing& lightNodes, double time,
                                         const std::string& scenePath)
{
    // TODO: Implement
    return SceneError::LIGHT_TYPE_INTERNAL_ERRROR;
}

TracerError CPULightGroupDisk::ConstructLights(const CudaSystem& system,
                                               const GPUTransformI** dGlobalTransformArray)
{
    // Gen Temporary Memory
    DeviceMemory tempMemory;
    // Allocate for GPULight classes
    size_t matKeySize = sizeof(HitKey) * lightCount;
    matKeySize = Memory::AlignSize(matKeySize);
    size_t mediumSize = sizeof(uint16_t) * lightCount;
    mediumSize = Memory::AlignSize(mediumSize);
    size_t transformIdSize = sizeof(TransformId) * lightCount;
    transformIdSize = Memory::AlignSize(transformIdSize);
    size_t centerSize = sizeof(Vector3f) * lightCount;
    centerSize = Memory::AlignSize(centerSize);
    size_t normalSize = sizeof(Vector3f) * lightCount;
    normalSize = Memory::AlignSize(normalSize);
    size_t radiusSize = sizeof(float) * lightCount;
    radiusSize = Memory::AlignSize(radiusSize);

    size_t totalSize = (matKeySize +
                        mediumSize +
                        transformIdSize +
                        centerSize +
                        normalSize +
                        radiusSize);
    DeviceMemory::EnlargeBuffer(tempMemory, totalSize);

    size_t offset = 0;
    std::uint8_t* dBasePtr = static_cast<uint8_t*>(tempMemory);
    const HitKey* dLightMaterialIds = reinterpret_cast<const HitKey*>(dBasePtr + offset);
    offset += matKeySize;
    const uint16_t* dMediumIndices = reinterpret_cast<const uint16_t*>(dBasePtr + offset);
    offset += mediumSize;
    const TransformId* dTransformIds = reinterpret_cast<const TransformId*>(dBasePtr + offset);
    offset += transformIdSize;
    const Vector3f* dCenters = reinterpret_cast<const Vector3f*>(dBasePtr + offset);
    offset += centerSize;
    const Vector3f* dNormals = reinterpret_cast<const Vector3f*>(dBasePtr + offset);
    offset += normalSize;
    const float* dRadius = reinterpret_cast<const float*>(dBasePtr + offset);
    offset += radiusSize;
    assert(totalSize == offset);

    // Set a GPU
    const CudaGPU& gpu = system.BestGPU();
    CUDA_CHECK(hipSetDevice(gpu.DeviceId()));
    // Load Data to Temp Memory
    CUDA_CHECK(hipMemcpy(const_cast<HitKey*>(dLightMaterialIds),
                          hHitKeys.data(),
                          sizeof(HitKey) * lightCount,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(const_cast<uint16_t*>(dMediumIndices),
                          hMediumIds.data(),
                          sizeof(uint16_t) * lightCount,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(const_cast<TransformId*>(dTransformIds),
                          hTransformIds.data(),
                          sizeof(TransformId) * lightCount,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(const_cast<Vector3*>(dCenters),
                          hCenters.data(),
                          sizeof(Vector3) * lightCount,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(const_cast<Vector3*>(dNormals),
                          hNormals.data(),
                          sizeof(Vector3) * lightCount,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(const_cast<float*>(dRadius),
                          hRadius.data(),
                          sizeof(float) * lightCount,
                          hipMemcpyHostToDevice));

    // Call allocation kernel
    gpu.GridStrideKC_X(0, 0,
                       LightCount(),
                       //
                       KCConstructGPULightDisk,
                       //
                       const_cast<GPULightDisk*>(dGPULights),
                       //
                       dCenters,
                       dNormals,
                       dRadius,
                       //
                       dTransformIds,
                       dMediumIndices,
                       dLightMaterialIds,
                       //
                       dGlobalTransformArray,
                       LightCount());

    gpu.WaitAllStreams();

    // Generate transform list
    for(uint32_t i = 0; i < LightCount(); i++)
    {
        const auto* ptr = static_cast<const GPULightI*>(dGPULights + i);
        gpuLightList.push_back(ptr);
    }
    return TracerError::OK;
}