#include "hip/hip_runtime.h"
#include "TracerDebug.h"
#include "ImageMemory.h"
#include "DefaultLeaf.h"

#include "DTreeKC.cuh"
#include "STreeKC.cuh"
#include "PathNode.cuh"
#include "LinearBVH.cuh"

#include "ImageIO/EntryPoint.h"

namespace Debug
{
namespace Detail
{
    void OutputHitPairs(std::ostream& s, const RayId* ids, const HitKey* keys, size_t count);
}
}

void Debug::Detail::OutputHitPairs(std::ostream& s, const RayId* ids, const HitKey* keys, size_t count)
{
    // Do Sync this makes memory to be accessible from Host
    for(size_t i = 0; i < count; i++)
    {
        s << "{" << std::hex << std::setw(8) << std::setfill('0') << keys[i] << ", "
                 << std::dec << std::setw(0) << std::setfill(' ') << ids[i] << "}" << " ";
    }
}

void Debug::DumpImage(const std::string& fName,
                      const ImageMemory& iMem)
{
    CUDA_CHECK(hipDeviceSynchronize());
    const ImageIOI& io = *ImageIOInstance();
    Vector2ui size(iMem.SegmentSize()[0],
                   iMem.SegmentSize()[1]);
    auto image = iMem.GMem<Vector4f>();

    ImageIOError e = ImageIOError::OK;
    if((e = io.WriteImage(reinterpret_cast<const Byte*>(image.gPixels),
                          size,
                          iMem.Format(), ImageType::PNG,
                          fName)) != ImageIOError::OK)
        METU_ERROR_LOG(static_cast<std::string>(e));
}

void Debug::DumpImage(const std::string& fName,
                      const Vector4* iMem,
                      const Vector2ui& resolution)
{
    const ImageIOI& io = *ImageIOInstance();
    ImageIOError e = ImageIOError::OK;
    if((e = io.WriteImage(reinterpret_cast<const Byte*>(iMem),
                          resolution,
                          PixelFormat::RGBA_FLOAT, ImageType::PNG,
                          fName)) != ImageIOError::OK)
        METU_ERROR_LOG(static_cast<std::string>(e));
}

void Debug::DumpTextureMip(const std::string& fName,
                           const Texture<2, Vector4f>& texture,
                           uint32_t mipLevel)
{
    CUDA_CHECK(hipDeviceSynchronize());

    const ImageIOI& io = *ImageIOInstance();

    std::vector<Byte> hPixels;
    texture.GetRawPixelData(hPixels, mipLevel);

    Vector2ui mipDim = texture.Dimensions();
    mipDim[0] = std::max(1u, mipDim[0] >> mipLevel);
    mipDim[1] = std::max(1u, mipDim[1] >> mipLevel);

    std::string fNameWithExt = fName + ".exr";

    ImageIOError e = ImageIOError::OK;
    if((e = io.WriteImage(hPixels.data(),
                          mipDim,
                          PixelFormat::RGBA_FLOAT, ImageType::EXR,
                          fNameWithExt)) != ImageIOError::OK)
        METU_ERROR_LOG(static_cast<std::string>(e));
}

void Debug::Dump2DDataToImage(const std::string& fName,
                              const float* images,
                              Vector2i texSize,
                              uint32_t imageCount)
{
    CUDA_CHECK(hipDeviceSynchronize());

    const ImageIOI& io = *ImageIOInstance();

    // Square out the image
    // Find the padding
    uint32_t columnCount = static_cast<uint32_t>(std::sqrt(static_cast<float>(imageCount)));
    uint32_t rowCount = (imageCount + columnCount - 1) / columnCount;
    assert((columnCount * rowCount) >= imageCount);
    uint32_t rowPixels = rowCount * texSize[1];
    uint32_t columnPixels = columnCount * texSize[0];
    // Allocate memory
    std::vector<float> hPixels(rowPixels * columnPixels);
    CUDA_CHECK(hipMemcpy(hPixels.data(), images,
                          texSize.Multiply() * imageCount * sizeof(float),
                          hipMemcpyDeviceToHost));

    // Convert to Row-major
    std::vector<float> hPixelsRM(rowPixels * columnPixels, 0.0f);
    uint32_t readLinearIndex = 0;
    for(uint32_t y = 0; y < rowCount; y++)
    for(uint32_t x = 0; x < columnCount; x++)
    {
        for(int32_t j = 0; j < texSize[1]; j++)
        for(int32_t i = 0; i < texSize[0]; i++)
        {
            uint32_t writeRow = y * texSize[1] + j;
            uint32_t writeColumn = x * texSize[0] + i;
            uint32_t writeLinear = writeRow * columnPixels + writeColumn;
            assert(readLinearIndex < hPixels.size());
            hPixelsRM[writeLinear] = hPixels[readLinearIndex];
            readLinearIndex++;
        }
    }

    Vector2ui totalPixels = Vector2ui(columnPixels, rowPixels);
    std::string fNameWithExt = fName + ".exr";
    ImageIOError e = ImageIOError::OK;
    if((e = io.WriteImage(reinterpret_cast<Byte*>(hPixelsRM.data()),
                          totalPixels,
                          PixelFormat::R_FLOAT, ImageType::EXR,
                          fNameWithExt)) != ImageIOError::OK)
        METU_ERROR_LOG(static_cast<std::string>(e));
}

void Debug::DumpBitmap(const std::string& fName,
                       const Byte* bits,
                       const Vector2ui& resolution)
{
    const ImageIOI& io = *ImageIOInstance();
    ImageIOError e = ImageIOError::OK;
    if((e = io.WriteBitmap(bits, resolution, ImageType::PNG, fName)) != ImageIOError::OK)
        METU_ERROR_LOG(static_cast<std::string>(e));
}

void Debug::PrintHitPairs(const RayId* ids, const HitKey* keys, size_t count)
{
    std::stringstream s;
    Detail::OutputHitPairs(s, ids, keys, count);
    METU_LOG(s.str());
}

void Debug::WriteHitPairs(const RayId* ids, const HitKey* keys, size_t count, const std::string& file)
{
    std::ofstream f(file);
    Detail::OutputHitPairs(f, ids, keys, count);
}

std::ostream& operator<<(std::ostream& stream, const Vector2ui& v)
{
    stream << std::setw(0)
        << v[0] << ", "
        << v[1];
    return stream;
}

std::ostream& operator<<(std::ostream& stream, const Vector2ul& v)
{
    stream << std::setw(0)
        << v[0] << ", "
        << v[1];
    return stream;
}

std::ostream& operator<<(std::ostream& stream, const Vector2f& v)
{
    stream << std::setw(0)
           << v[0] << ", "
           << v[1];
    return stream;
}

std::ostream& operator<<(std::ostream& stream, const Vector3f& v)
{
    stream << std::setw(0)
        << v[0] << ", "
        << v[1] << ", "
        << v[2];
    return stream;
}

std::ostream& operator<<(std::ostream& stream, const Vector4f& v)
{
    stream << std::setw(0)
        << v[0] << ", "
        << v[1] << ", "
        << v[2] << ", "
        << v[3];
    return stream;
}

std::ostream& operator<<(std::ostream& stream, const AABB3f& aabb)
{
    stream << std::setw(0)
        << "{("
        << aabb.Min() << "), ("
        << aabb.Max()
        << ")}";
    return stream;
}

std::ostream& operator<<(std::ostream& stream, const RayGMem& r)
{
    stream << std::setw(0)
           << "{" << r.pos[0] << ", " << r.pos[1] << ", " << r.pos[2] << "} "
           << "{" << r.dir[0] << ", " << r.dir[1] << ", " << r.dir[2] << "} "
           << "{" << r.tMin << ", " << r.tMax << "}";
    return stream;
}

std::ostream& operator<<(std::ostream& stream, const HitKey& key)
{
    stream << std::hex << std::setfill('0')
           << std::setw(HitKey::BatchBits / 4) << HitKey::FetchBatchPortion(key)
           << ":"
           << std::setw(HitKey::IdBits / 4) << HitKey::FetchIdPortion(key);
    stream << std::dec;
    return stream;
}

std::ostream& operator<<(std::ostream& stream, const DefaultLeaf& l)
{
    stream << std::setw(0)
           << "{ mat: " << l.matId << ", primId: " << l.primitiveId << "} ";
    return stream;
}

std::ostream& operator<<(std::ostream& s, const STreeNode& n)
{
    s << "C{";
    if(n.isLeaf)
    {
        s << "-, -} ";
        s << "T{" << n.index << "} ";
    }
    else
    {
        s << n.index << ", " << (n.index + 1) << "} ";
        s << "T{-} ";
    }

    //static constexpr const char* XYZ = "XYZ";
    s << "Axis {";
    s << "XYZ"[static_cast<int>(n.splitAxis)];
    s << "}";
    return s;
}

std::ostream& operator<<(std::ostream& s, const STreeGPU& n)
{
    s << "NodeCount  : " << n.nodeCount << std::endl;
    s << "Extents    : {{"
        << n.extents.Min()[0] << ", " << n.extents.Min()[1] << ", " << n.extents.Min()[2] << "}, {"
        << n.extents.Max()[0] << ", " << n.extents.Max()[1] << ", " << n.extents.Max()[2];
    s << "}}" << std::endl;
    return s;
}

std::ostream& operator<<(std::ostream& s, const PPGPathNode& n)
{
    s << "W: { "
      << n.worldPosition[0] << ", "
      << n.worldPosition[1] << ", "
      << n.worldPosition[2] << "} "
      << "PN: {"
      << static_cast<uint32_t>(n.prevNext[0]) << ", "
      << static_cast<uint32_t>(n.prevNext[1]) << "} "
      << "R: {"
      << n.totalRadiance[0] << ", "
      << n.totalRadiance[1] << ", "
      << n.totalRadiance[2] << "} "
      << "DT: "
      << n.dataStructIndex;
    return s;
}

std::ostream& operator<<(std::ostream& s, const PathGuidingNode& n)
{
    s << "W: { "
        << n.worldPosition[0] << ", "
        << n.worldPosition[1] << ", "
        << n.worldPosition[2] << "} "
        << "PN: {"
        << static_cast<uint32_t>(n.prevNext[0]) << ", "
        << static_cast<uint32_t>(n.prevNext[1]) << "} "
        << "R: {"
        << n.totalRadiance[0] << ", "
        << n.totalRadiance[1] << ", "
        << n.totalRadiance[2] << "} "
        << "TP: {"
        << n.radFactor[0] << ", "
        << n.radFactor[1] << ", "
        << n.radFactor[2] << "} ";
    return s;
}

std::ostream& operator<<(std::ostream& s, const DTreeNode& n)
{
    constexpr uint32_t UINT32_T_MAX = std::numeric_limits<uint32_t>::max();

    s << "P{";
    if(n.parentIndex == UINT32_T_MAX) s << "-";
    else s << n.parentIndex;
    s << "} ";
    s << "C{";
    if(n.childIndices[0] == UINT32_T_MAX) s << "-";
    else s << n.childIndices[0];
    s << ", ";
    if(n.childIndices[1] == UINT32_T_MAX) s << "-";
    else s << n.childIndices[1];
    s << ", ";
    if(n.childIndices[2] == UINT32_T_MAX) s << "-";
    else s << n.childIndices[2];
    s << ", ";
    if(n.childIndices[3] == UINT32_T_MAX) s << "-";
    else s << n.childIndices[3];
    s << "} ";
    s << "I{"
      << n.irradianceEstimates[0] << ", "
      << n.irradianceEstimates[1] << ", "
      << n.irradianceEstimates[2] << ", "
      << n.irradianceEstimates[3] << "}";
    return s;
}

std::ostream& operator<<(std::ostream& s, const DTreeGPU& n)
{
    s << "Irradiance  : " << n.irradiance << std::endl;
    s << "NodeCount  : " << n.nodeCount << std::endl;
    s << "SampleCount: " << n.totalSamples << std::endl;
    return s;
}