#include "hip/hip_runtime.h"
#include "RNGSobol.cuh"
#include "CudaSystem.h"
#include "CudaSystem.hpp"

#include <random>
#include <execution>

__global__ void KCInitRNGStatesSobol(RNGSobolGPU* dGenerators,
                                     RNGeneratorGPUI** dGenPtrs,
                                     hiprandDirectionVectors32_t* gDirectionVectors,
                                     const uint32_t* gOffsets,
                                     const uint32_t* gScrambleConsts,
                                     uint32_t totalCount)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < totalCount;
        threadId += (blockDim.x * gridDim.x))
    {
        new (dGenerators + threadId) RNGSobolGPU(gDirectionVectors[threadId],
                                                 gOffsets[threadId],
                                                 gScrambleConsts[threadId]);
        dGenPtrs[threadId] = dGenerators + threadId;
    }
}

RNGSobolCPU::RNGSobolCPU(uint32_t seed,
                         const CudaSystem& system)
{
    // RNG for seeding each thread in the gpu(s)
    std::mt19937 rng;
    rng.seed(seed);

    static constexpr uint32_t VECTOR_PER_THREAD = 32;
    // Determine GPU Sizes and Offsets
    size_t totalCount = 0;
    std::vector<Vector2ul> offsetAndCounts;
    for(const auto& gpu : system.SystemGPUs())
    {
        offsetAndCounts.push_back(Vector2ul(0));
        offsetAndCounts.back()[0] = totalCount;
        offsetAndCounts.back()[1] = gpu.MaxActiveBlockPerSM() * gpu.SMCount() * StaticThreadPerBlock1D;
        totalCount += offsetAndCounts.back()[1];
    }

    // Get Directions Vectors & Scramble Constants from
    // CPU API
    uint32_t* hScrambleConstants;
    hiprandDirectionVectors32_t* hDirectionVectors;
    hiprandStatus_t s = hiprandGetDirectionVectors32(&hDirectionVectors,
                                                   HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6);
    if(s != HIPRAND_STATUS_SUCCESS) assert(false);
    s = hiprandGetScrambleConstants32(&hScrambleConstants);
    if(s != HIPRAND_STATUS_SUCCESS) assert(false);

    // Copy to temp memory
    uint32_t* dOffsets;
    uint32_t* dScrambleConstants;
    hiprandDirectionVectors32_t* dDirectionVectors;
    DeviceMemory tempMemory;
    GPUMemFuncs::AllocateMultiData(std::tie(dOffsets, dScrambleConstants,
                                            dDirectionVectors),
                                   tempMemory,
                                   {totalCount, totalCount,
                                   VECTOR_PER_THREAD * totalCount});

    // Before touching gpu mem from cpu do a sync
    // since other initialization probably launched a kernel
    system.SyncAllGPUs();
    std::for_each(dOffsets, dOffsets + totalCount,
                  [&](uint32_t& t) { t = rng(); });

    // Rest is copied from host
    CUDA_CHECK(hipMemcpy(dScrambleConstants, hScrambleConstants,
                          totalCount * sizeof(uint32_t),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dDirectionVectors, hDirectionVectors,
                          VECTOR_PER_THREAD * totalCount * sizeof(uint32_t),
                          hipMemcpyHostToDevice));

    // Allocate Actual State Data
    RNGSobolGPU* dGenerators;
    RNGeneratorGPUI** dGenPtrs;
    GPUMemFuncs::AllocateMultiData(std::tie(dGenerators, dGenPtrs),
                                   memRandom,
                                   {totalCount, totalCount});

    size_t totalOffset = 0;
    for(const auto& gpu : system.SystemGPUs())
    {
        uint32_t gpuRNGStateCount = gpu.MaxActiveBlockPerSM() * gpu.SMCount() * StaticThreadPerBlock1D;
        deviceGenerators.emplace(&gpu, dGenPtrs + totalOffset);
        totalOffset += gpuRNGStateCount;
    }
    assert(totalCount == totalOffset);

    // Make all GPU do its own initialization
    int i = 0;
    for(const auto& gpu : system.SystemGPUs())
    {
        uint32_t localCount = static_cast<uint32_t>(offsetAndCounts[i][1]);

        gpu.GridStrideKC_X(0, 0, localCount,
                           //
                           KCInitRNGStatesSobol,
                           //
                           dGenerators + offsetAndCounts[i][0],
                           dGenPtrs + offsetAndCounts[i][0],
                           dDirectionVectors + offsetAndCounts[i][0],
                           dOffsets + offsetAndCounts[i][0],
                           dScrambleConstants + offsetAndCounts[i][0],
                           localCount);
        i++;
    }
    // All Done!
}

RNGSobolCPU::RNGSobolCPU(uint32_t seed,
                         const CudaGPU& gpu)
{
    CUDA_CHECK(hipSetDevice(gpu.DeviceId()));
    static constexpr uint32_t VECTOR_PER_THREAD = 32;

    // CPU Mersenne Twister
    std::mt19937 rng;
    rng.seed(seed);
    // Determine GPU
    size_t offset = 0;
    uint32_t count = gpu.MaxActiveBlockPerSM() * gpu.SMCount() * StaticThreadPerBlock1D;
    // Get Directions Vectors & Scramble Constants from
    // CPU API
    uint32_t* hScrambleConstants;
    hiprandDirectionVectors32_t* hDirectionVectors;
    hiprandStatus_t s = hiprandGetDirectionVectors32(&hDirectionVectors,
                                                   HIPRAND_SCRAMBLED_DIRECTION_VECTORS_64_JOEKUO6);
    if(s != HIPRAND_STATUS_SUCCESS) assert(false);
    s = hiprandGetScrambleConstants32(&hScrambleConstants);
    if(s != HIPRAND_STATUS_SUCCESS) assert(false);
    // Copy to temp memory
    uint32_t* dOffsets;
    uint32_t* dScrambleConstants;
    hiprandDirectionVectors32_t* dDirectionVectors;
    DeviceMemory tempMemory;
    GPUMemFuncs::AllocateMultiData(std::tie(dOffsets, dScrambleConstants,
                                            dDirectionVectors),
                                   tempMemory,
                                   {count, count,
                                   VECTOR_PER_THREAD * count});
    // Before touching gpu mem from cpu do a sync
    // since other initialization probably launched a kernel
    CUDA_CHECK(hipDeviceSynchronize());
    std::for_each(dOffsets, dOffsets + count,
                  [&](uint32_t& t) { t = rng(); });
    // Rest is copied from host
    CUDA_CHECK(hipMemcpy(dScrambleConstants, hScrambleConstants,
                          count * sizeof(uint32_t),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dDirectionVectors, hDirectionVectors,
                          VECTOR_PER_THREAD * count * sizeof(uint32_t),
                          hipMemcpyHostToDevice));
    // Actual Allocation
    RNGSobolGPU* dGenerators;
    RNGeneratorGPUI** dGenPtrs;
    GPUMemFuncs::AllocateMultiData(std::tie(dGenerators, dGenPtrs),
                                   memRandom,
                                   {count, count});

    size_t totalOffset = 0;
    uint32_t gpuRNGStateCount = gpu.MaxActiveBlockPerSM() * gpu.SMCount() * StaticThreadPerBlock1D;
    deviceGenerators.emplace(&gpu, dGenPtrs + totalOffset);
    totalOffset += gpuRNGStateCount;
    assert(count == static_cast<uint32_t>(totalOffset));

    // Initialize the States
    gpu.GridStrideKC_X(0, 0, count,
                       //
                       KCInitRNGStatesSobol,
                       //
                       dGenerators + offset,
                       dGenPtrs + offset,
                       dDirectionVectors + offset,
                       dOffsets + offset,
                       dScrambleConstants + offset,
                       count);
}

RNGeneratorGPUI** RNGSobolCPU::GetGPUGenerators(const CudaGPU& gpu)
{
    return deviceGenerators.at(&gpu);
}