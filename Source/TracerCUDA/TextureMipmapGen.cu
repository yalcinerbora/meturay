#include "hip/hip_runtime.h"
#include "TextureMipmapGen.cuh"

#include "CudaSystem.h"
#include "CudaSystem.hpp"
#include "TextureMipmapGen.cuh"
#include "TextureReference.cuh"
#include "GPUReconFilterMitchell.h"
#include "GPUReconFilterGaussian.h"


#include <hipcub/hipcub.hpp>

//#include "TracerDebug.h"

template <uint32_t TPB, class WriteType, class FilterFunc>
__global__
static void KCGenMipmap(// Outputs
                        hipSurfaceObject_t sObj,
                        // Inputs
                        hipTextureObject_t texObj,
                        // Constants
                        Vector2ui mipTexSize,
                        Vector2ui texSize,
                        uint32_t mipLevel,
                        uint32_t sampleXY,
                        float filterRadius,
                        FilterFunc filter)
{
    static constexpr uint32_t WARP_PER_BLOCK = TPB / WARP_SIZE;
    using ReadType = TexFloatType_t<3>;
    using WarpValueReduce = hipcub::WarpReduce<ReadType>;
    using WarpFloatReduce = hipcub::WarpReduce<float>;

    struct SharedMemory
    {
        union
        {
            typename WarpValueReduce::TempStorage valReduceMem;
            typename WarpFloatReduce::TempStorage floatReduceMem;
        } warp[WARP_PER_BLOCK];
    };

    __shared__ SharedMemory shMem;

    // TODO: Put this on a library
    // https://link.springer.com/content/pdf/10.1007/978-1-4842-4427-2_16.pdf
    auto CocentricDiskSample = [=](const Vector2f& uv) -> Vector2f
    {
        const float R = filterRadius;
        float a = 2 * uv[0] - 1;
        float b = 2 * uv[1] - 1;
        float r;
        float phi;
        if(a * a > b * b)
        {
            r = R * a;
            phi = (MathConstants::Pi * 0.25f) * (b / a);
        }
        else
        {
            r = R * b;
            phi = ((MathConstants::Pi * 0.5) -
                   (MathConstants::Pi * 0.25f) * (a / b));
        }
        // Prevent nan here a/b == 0.0 or b/a depending on the branch.
        // Also prevent inf since cos(inf) is undefined as well
        // Both happens when sampler samples very close to the origin
        // so you can safely assume phi is zero
        phi = (isnan(phi) || isinf(phi)) ? 0.0f : phi;

        // Convert to relative pixel index
        return Vector2f(r * cos(phi),
                        r * sin(phi));
    };

    // Wrap the texture to device class for better readability
    // TODO: we dont use the mip count and dim here zeroed it out. Change to proper
    // texture system here.
    const TextureRef<2, ReadType> texture(texObj, Vector2ui(0), 0);

    // Each warp is responsible for single pixel on the leaf
    const uint32_t kernelWarpCount = (blockDim.x * gridDim.x) / WARP_SIZE;
    const uint32_t pixelCount = mipTexSize.Multiply();

    uint32_t globalThreadId = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t globalWarpId = globalThreadId / WARP_SIZE;
    uint32_t localWarpId = threadIdx.x / WARP_SIZE;
    uint32_t laneId = globalThreadId % WARP_SIZE;

    uint32_t samplePerPixel = sampleXY * sampleXY;
    uint32_t warpLoopCount = (samplePerPixel + WARP_SIZE - 1) / WARP_SIZE;

    // Warp stride Loop
    for(uint32_t pixelId = globalWarpId; pixelId < pixelCount;
        pixelId += kernelWarpCount)
    {
        Vector2ui pixIndex2D = Vector2ui(pixelId % mipTexSize[0],
                                         pixelId / mipTexSize[0]);
        Vector2f pixCenterFloat = Vector2f(pixIndex2D) + 0.5f;
        Vector2f mipZeroUV = pixCenterFloat / Vector2f(mipTexSize);


        ReadType leaderTotal = ReadType{0};
        float leaderWeightTotal = 0.0f;

        // Do not loop over using laneId and warpSize
        // reduction operation may block because of inactive threads
        for(uint32_t i = 0; i < warpLoopCount; i++)
        {
            // Generate fake random variables
            // stratify 2D [0,1) space wrt. total sample count
            uint32_t localSampleId = i * WARP_SIZE + laneId;
            bool isValid = (localSampleId < samplePerPixel);
            Vector2ui laneStratId(localSampleId % sampleXY,
                                  localSampleId / sampleXY);

            Vector2f sampleXi = (Vector2f(laneStratId) + Vector2f(0.5f)) / Vector2f(sampleXY);

            Vector2f sampleOffset = CocentricDiskSample(sampleXi);

            // Current filter functions requires filter center coordinates
            // and the filtering location image space coordinate
            float weight = filter(pixCenterFloat,
                                  pixCenterFloat + sampleOffset);

            // UV of the miplevel0 is required scale offset accordingly
            sampleOffset *= static_cast<float>(1 << mipLevel);
            Vector2f texReadUV = mipZeroUV + sampleOffset / Vector2f(texSize);
            ReadType texVal = texture(texReadUV, static_cast<float>(mipLevel - 1));
            ReadType weightedVal = texVal * weight;

            weightedVal = (isValid) ? weightedVal : ReadType{0};
            weight = (isValid) ? weight : 0.0f;

            leaderTotal += WarpValueReduce(shMem.warp[localWarpId].valReduceMem).Sum(weightedVal);
            leaderWeightTotal += WarpFloatReduce(shMem.warp[localWarpId].floatReduceMem).Sum(weight);
        }

        // Finally leader writes to the surface object
        if(laneId == 0)
        {
            // Sanity check
            static_assert(sizeof(Vector4f) == sizeof(float4));
            ReadType total = leaderTotal / leaderWeightTotal;
            float4 writeVal = {total[0], total[1], total[2], 1.0f};
            surf2Dwrite(writeVal, sObj,
                        static_cast<int>(pixIndex2D[0] * sizeof(Vector4f)),
                        static_cast<int>(pixIndex2D[1]));
        }
    }

}

__host__
Texture<2, Vector4f> GenerateMipmaps(const Texture<2, Vector4f>& texture, uint32_t upToMip)
{
    const CudaGPU& textureGPU = *texture.Device();
    std::vector<CudaSurfaceRAII> surfaces;
    CUDA_CHECK(hipSetDevice(texture.Device()->DeviceId()));

    // Allocate new mipmapped texture
    Texture<2, Vector4f> newTexture = texture.EmptyMipmappedTexture(upToMip);

    // Sample stratified MULTISAMPLE_COUNT * MULTISAMPLE_COUNT
    // amount of samples over the region of the texel.
    // Filter these according to the filter
    static constexpr float FILTER_RADIUS = 2.0f;
    static constexpr uint32_t MULTISAMPLE_COUNT = 5;
    // Mitchell-Netravali Filter
    // TODO: Mitchell-Netravali filter causes hard ringing artifacts near the sun
    // (we currently only use mipmaps for environment boundary lights)
    // fix it later. Now use gaussian instead
    //const GPUMitchellFilterFunctor filterFunctor(FILTER_RADIUS, 0.3333f, 0.3333f);
    // Gaussian Filter
    const GPUGaussianFilterFunctor filterFunctor(FILTER_RADIUS, 0.5f);

    // Construct mips level by level
    upToMip = std::min(upToMip, newTexture.MipmapCount() - 1);
    for(uint32_t mipLevel = 1; mipLevel <= upToMip; mipLevel++)
    {
        CudaSurfaceRAII surfaceObject = newTexture.GetMipLevelSurface(mipLevel);
        // Find out the return type
        Vector2ui mipDim = Vector2ui::Max(texture.Dimensions() / (1 << mipLevel), Vector2ui(1));

        static constexpr uint32_t TPB = StaticThreadPerBlock1D;
        static constexpr uint32_t WARP_PER_BLOCK = TPB / WARP_SIZE;
        uint32_t totalThreadCount = mipDim.Multiply() * MULTISAMPLE_COUNT * MULTISAMPLE_COUNT;
        uint32_t totalWarpCount = (totalThreadCount + WARP_SIZE - 1) / WARP_SIZE;
        // TODO: make this utilize less blocks for multi kernel execution
        // Currently, it generates full amount of blocks
        uint32_t totalBlockCount = (totalWarpCount + WARP_PER_BLOCK - 1) / WARP_PER_BLOCK;

        textureGPU.ExactKC_X(0, (hipStream_t)0,
                             TPB, totalBlockCount,
                             //
                             //KCGenMipmap<StaticThreadPerBlock1D, Vector4f, GPUMitchellFilterFunctor>,
                             KCGenMipmap<StaticThreadPerBlock1D, Vector4f, GPUGaussianFilterFunctor>,
                             // Output
                             surfaceObject,
                             // Inputs
                             static_cast<hipTextureObject_t>(newTexture),
                             // Constants
                             mipDim,
                             texture.Dimensions(),
                             static_cast<uint32_t>(mipLevel),
                             MULTISAMPLE_COUNT,
                             FILTER_RADIUS,
                             filterFunctor);

        //Debug::DumpTextureMip(std::string("tex_") + std::to_string(mipLevel),
        //                      newTexture, mipLevel);

        // Defer destruction of the surface object until all kernels are finished
        surfaces.emplace_back(std::move(surfaceObject));
    }

    // Wait all events to finish before deleting surfaces (implicit)
    CUDA_CHECK(hipStreamSynchronize((hipStream_t)0));

    return newTexture;

}