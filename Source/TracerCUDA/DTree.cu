#include "hip/hip_runtime.h"
﻿#include "DTree.cuh"
#include "DTreeKC.cuh"
#include "ParallelReduction.cuh"
#include "CudaSystem.hpp"

#include "RayLib/Types.h"
#include "RayLib/MemoryAlignment.h"

#include "RayLib/Log.h"
#include "RayLib/CPUTimer.h"

static constexpr size_t AlignedOffsetDTreeGPU = Memory::AlignSize(sizeof(DTreeGPU));

void DTree::DTreeBuffer::FixPointers()
{
    Byte* nodeStart = static_cast<Byte*>(memory) + AlignedOffsetDTreeGPU;
    Byte* nodePtrLoc = static_cast<Byte*>(memory) + offsetof(DTreeGPU, gRoot);
    CUDA_CHECK(hipMemcpy(nodePtrLoc, &nodeStart, sizeof(DTreeNode*), 
                          hipMemcpyHostToDevice));
}

DTree::DTreeBuffer::DTreeBuffer()
    : dDTree(nullptr)
    , nodeCount(0)
{
    nodeCount = 1;
    DeviceMemory::EnlargeBuffer(memory, AlignedOffsetDTreeGPU + sizeof(DTreeNode));
    dDTree = static_cast<DTreeGPU*>(memory);
    DTreeNode* dDTreeNodes = reinterpret_cast<DTreeNode*>(static_cast<Byte*>(memory) + AlignedOffsetDTreeGPU);


    Vector4f irrads = Vector4f(1.0f, 0.01f, 
                               1.0f, 0.01f);

    // Init Tree
    DTreeGPU hDTree;
    hDTree.gRoot = dDTreeNodes;
    hDTree.nodeCount = 1;
    hDTree.irradiance = 0.0f;
    hDTree.irradiance = irrads.Sum();
    hDTree.totalSamples = 0;
    CUDA_CHECK(hipMemcpy(dDTree, &hDTree, sizeof(DTreeGPU),
                          hipMemcpyHostToDevice));

    // Init very first node
    DTreeNode hNode;
    //hNode.irradianceEstimates = Zero4;
    hNode.irradianceEstimates = irrads;

    hNode.childIndices = Vector4ui(std::numeric_limits<uint32_t>::max());
    hNode.parentIndex = std::numeric_limits<uint16_t>::max();
    CUDA_CHECK(hipMemcpy(dDTreeNodes, &hNode, sizeof(DTreeNode),
                          hipMemcpyHostToDevice));    
}

DTree::DTreeBuffer::DTreeBuffer(const DTreeBuffer& other)
    : memory(other.memory.Size())
    , nodeCount(other.nodeCount)
    , dDTree(static_cast<DTreeGPU*>(memory))
{
    CUDA_CHECK(hipMemcpy(memory, other.memory,
                          AlignedOffsetDTreeGPU + nodeCount * sizeof(DTreeNode),
                          hipMemcpyDeviceToDevice));
    FixPointers();
}

DTree::DTreeBuffer& DTree::DTreeBuffer::operator=(const DTreeBuffer& other)
{
    DeviceMemory::EnlargeBuffer(memory, other.memory.Size());
    nodeCount = other.nodeCount;
    dDTree = static_cast<DTreeGPU*>(memory);

    CUDA_CHECK(hipMemcpy(memory, other.memory,
                          AlignedOffsetDTreeGPU + nodeCount * sizeof(DTreeNode),
                          hipMemcpyDeviceToDevice));
    FixPointers();
    return *this;
}

void DTree::DTreeBuffer::ResetAndReserve(size_t newNodeCount,
                                         const CudaGPU& gpu,
                                         hipStream_t stream)
{   
    // Check capacity and if its not large enough
    // allocate larger memory
    size_t capacity = (memory.Size() - AlignedOffsetDTreeGPU) / sizeof(DTreeNode);
    if(capacity < newNodeCount)
    {
        size_t size = AlignedOffsetDTreeGPU + (newNodeCount * sizeof(DTreeNode));
        DeviceMemory::EnlargeBuffer(memory, size);
        dDTree = static_cast<DTreeGPU*>(memory);
        FixPointers();        
    }
    // Reset all node values
    gpu.GridStrideKC_X(0, stream, newNodeCount,
                        //
                       KCInitDTreeNodes,
                       //
                       dDTree,
                       static_cast<uint32_t>(newNodeCount));
    
    nodeCount = 0;
}

void DTree::DTreeBuffer::CopyGPUNodeCountToCPU()
{
    CUDA_CHECK(hipMemcpy(&nodeCount, reinterpret_cast<Byte*>(dDTree) + offsetof(DTreeGPU, nodeCount),
                          sizeof(uint32_t), hipMemcpyDeviceToHost));
}

void DTree::DTreeBuffer::DumpTree(DTreeGPU& treeCPU, std::vector<DTreeNode>& nodesCPU) const
{    
    CUDA_CHECK(hipMemcpy(&treeCPU, dDTree, sizeof(DTreeGPU),
                          hipMemcpyDeviceToHost));
    nodesCPU.resize(nodeCount);
    const DTreeNode* dDTreeNodes = treeCPU.gRoot;
    CUDA_CHECK(hipMemcpy(nodesCPU.data(), dDTreeNodes, nodeCount * sizeof(DTreeNode),
                          hipMemcpyDeviceToHost));
}

#include "TracerDebug.h"

void DTree::SwapTrees(float fluxRatio, uint32_t depthLimit,
                      const CudaGPU& gpu)
{
    // Get an arbitrary stream
    hipStream_t stream = gpu.DetermineStream();

    DTreeGPU treeGPU;
    std::vector<DTreeNode> nodes;
    writeTree.DumpTree(treeGPU, nodes);
    Debug::DumpMemToFile("WT_PC_N", nodes.data(), nodes.size());
    Debug::DumpMemToFile("WT_PC", &treeGPU, 1);

    // Currently build tree that has its only leafs
    // are valid. Write values to the all nodes
    uint32_t nodeCount = static_cast<uint32_t>(writeTree.NodeCount());
    gpu.GridStrideKC_X(0, stream, nodeCount,
                       //
                       KCCalculateParentIrradiance,
                       //
                       writeTree.TreeGPU(),
                       nodeCount);

    writeTree.DumpTree(treeGPU, nodes);
    Debug::DumpMemToFile("WT_AC_N", nodes.data(), nodes.size());
    Debug::DumpMemToFile("WT_AC", &treeGPU, 1);

    //Byte* dIrrad = reinterpret_cast<Byte*>(readTree.TreeGPU()) + offsetof(DTreeGPU, irradiance);
    //METU_LOG("TOTAL");
    //Debug::DumpMemToStdout(reinterpret_cast<float*>(dIrrad), 1);
    //METU_LOG("===================================================");
    // We have a valid tree now
    // New tree will be reconsturcted from this tree
    // Ask each node that how many child they will need
    DeviceMemory childCountBuffer(nodeCount * sizeof(uint32_t));
    uint32_t* dNodeChildCounts = static_cast<uint32_t*>(childCountBuffer);
    gpu.GridStrideKC_X(0, stream, nodeCount,
                       //
                       KCMarkChildRequest,
                       //
                       dNodeChildCounts,
                       writeTree.TreeGPU(),
                       fluxRatio,
                       nodeCount);

    // Sum all values on the GPU
    uint32_t newNodeCount;
    ReduceArrayGPU<uint32_t, ReduceAdd<uint32_t>,
                   hipMemcpyDeviceToHost>
    (
        newNodeCount,
        dNodeChildCounts, 
        nodeCount, 
        0u, 
        stream
    );
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Add root node (DTree will atleast have a root node)
    // And above kernel only checks if childs should be generated
    // Root does not have any parent so we need to manually include here
    newNodeCount++;

    METU_LOG("NEW_NODE_COUT %u", newNodeCount);

    // Reserve enough nodes on the other tree for construction
    readTree.ResetAndReserve(newNodeCount, gpu, stream);
    // Reconstruct a new read tree from the findings
    gpu.GridStrideKC_X(0, stream, nodeCount,
                       //
                       KCReconstructEmptyTree,
                       //
                       readTree.TreeGPU(),
                       //
                       writeTree.TreeGPU(),
                       fluxRatio,
                       depthLimit,
                       nodeCount);
    readTree.CopyGPUNodeCountToCPU();

    readTree.DumpTree(treeGPU, nodes);
    Debug::DumpMemToFile("RT_FINAL_N", nodes.data(), nodes.size());
    Debug::DumpMemToFile("RT_FINAL", &treeGPU, 1);

    // Finally swap the trees
    std::swap(readTree, writeTree);
}

void DTree::AddRadiancesFromPaths(const uint32_t* dNodeIndexArray,
                                  const PathGuidingNode* dPathNodes,
                                  const ArrayPortion<uint32_t>& portion,
                                  uint32_t maxPathNodePerRay,
                                  const CudaGPU& gpu)
{
    hipStream_t stream = gpu.DetermineStream();
    uint32_t nodeCount = static_cast<uint32_t>(portion.count);

    gpu.GridStrideKC_X(0, stream, portion.count,
                       //
                       KCAccumulateRadianceToLeaf,
                       //
                       writeTree.TreeGPU(),
                       dNodeIndexArray,
                       dPathNodes,
                       nodeCount,
                       maxPathNodePerRay);

    uint32_t totalSampleCount = static_cast<uint32_t>(portion.count);
    CUDA_CHECK(hipMemcpy(&writeTree.TreeGPU()->totalSamples, &totalSampleCount,
                          sizeof(uint32_t), hipMemcpyHostToDevice));               
}

void DTree::GetReadTreeToCPU(DTreeGPU& tree, std::vector<DTreeNode>& nodes) const
{
    readTree.DumpTree(tree, nodes);
}
void DTree::GetWriteTreeToCPU(DTreeGPU& tree, std::vector<DTreeNode>& nodes) const
{
    writeTree.DumpTree(tree, nodes);
}