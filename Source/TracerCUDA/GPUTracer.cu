#include "GPUTracer.h"

#include "RayLib/Log.h"
#include "RayLib/TracerError.h"
#include "RayLib/TracerCallbacksI.h"
#include "RayLib/BitManipulation.h"
#include "RayLib/GPUSceneI.h"
#include "RayLib/MemoryAlignment.h"
#include "RayLib/SceneStructs.h"

#include "CudaConstants.h"
#include "GPUAcceleratorI.h"
#include "GPUWorkI.h"
#include "GPUTransformI.h"
#include "GPUMediumI.h"
#include "GPUMaterialI.h"
#include "GPUTransformI.h"
#include "GPULightI.h"
#include "GPUCameraI.h"

#include "TracerDebug.h"

TracerError GPUTracer::LoadCameras(std::vector<const GPUCameraI*>& dGPUCameras)
{
    TracerError e = TracerError::OK;
    for(auto& camera : cameras)
    {
        CPUCameraGroupI& c = *(camera.second);
        if((e = c.ConstructCameras(cudaSystem, dTransforms)) != TracerError::OK)
            return e;
        const auto& dCList = c.GPUCameras();
        dGPUCameras.insert(dGPUCameras.end(), dCList.begin(), dCList.end());
    }
    cameraCount = static_cast<uint32_t>(dGPUCameras.size());
    return TracerError::OK;
}

TracerError GPUTracer::LoadLights(std::vector<const GPULightI*>& dGPULights)
{
    TracerError e = TracerError::OK;
    for(auto& light : lights)
    {
        CPULightGroupI& l = *(light.second);
        if((e = l.ConstructLights(cudaSystem, dTransforms)) != TracerError::OK)
            return e;
        const auto& dLList = l.GPULights();
        dGPULights.insert(dGPULights.end(), dLList.begin(), dLList.end());
    }
    lightCount = static_cast<uint32_t>(dGPULights.size());
    return TracerError::OK;
}

TracerError GPUTracer::LoadTransforms(std::vector<const GPUTransformI*>& dGPUTransforms)
{
    TracerError e = TracerError::OK;
    for(auto& transform : transforms)
    {
        CPUTransformGroupI& t = *(transform.second);
        if((e = t.ConstructTransforms(cudaSystem)) != TracerError::OK)
            return e;
        const auto& dTList = t.GPUTransforms();
        dGPUTransforms.insert(dGPUTransforms.end(), dTList.begin(), dTList.end());
    }
    transformCount = static_cast<uint32_t>(dGPUTransforms.size());
    return TracerError::OK;
}

TracerError GPUTracer::LoadMediums(std::vector<const GPUMediumI*>& dGPUMediums)
{
    TracerError e = TracerError::OK;
    uint32_t indexOffset = 0;
    for(auto& medium : mediums)
    {
        CPUMediumGroupI& m = *(medium.second);
        if((e = m.ConstructMediums(cudaSystem, indexOffset)) != TracerError::OK)
            return e;
        const auto& dMList = m.GPUMediums();
        dGPUMediums.insert(dGPUMediums.end(), dMList.begin(), dMList.end());
        indexOffset += m.MediumCount();
    }
    mediumCount = static_cast<uint32_t>(dGPUMediums.size());
    return TracerError::OK;
}

GPUTracer::GPUTracer(const CudaSystem& system, 
                     const GPUSceneI& scene,
                     const TracerParameters& p)
    : cudaSystem(system)
    , baseAccelerator(*scene.BaseAccelerator())
    , accelBatches(scene.AcceleratorBatchMappings())
    , materialGroups(scene.MaterialGroups())
    , transforms(scene.Transforms())
    , mediums(scene.Mediums())
    , cameras(scene.Cameras())
    , lights(scene.Lights())
    , baseMediumIndex(scene.BaseMediumIndex())
    , identityTransformIndex(scene.IdentityTransformIndex())
    , maxAccelBits(Vector2i(Utility::FindFirstSet32(scene.MaxAccelIds()[0]) + 1,
                            Utility::FindFirstSet32(scene.MaxAccelIds()[1]) + 1))
    , maxWorkBits(Vector2i(Utility::FindFirstSet32(scene.MaxMatIds()[0]) + 1,
                           Utility::FindFirstSet32(scene.MaxMatIds()[1]) + 1))
    , params(p)
    , maxHitSize(scene.HitStructUnionSize())
    , rayMemory(system.BestGPU())
    , callbacks(nullptr)
    , crashed(false)
    , currentRayCount(0)
{}

TracerError GPUTracer::Initialize()
{
    // Init RNGs for each block
    TracerError e = TracerError::OK;
    rngMemory = RNGMemory(params.seed, cudaSystem);

    std::vector<const GPUTransformI*> dGPUTransforms;
    std::vector<const GPUMediumI*> dGPUMediums;
    std::vector<const GPULightI*> dGPULights;
    std::vector<const GPUCameraI*> dGPUCameras;

    // Calculate Total Sizes
    size_t tCount = 0;
    size_t mCount = 0;
    size_t lCount = 0;
    size_t cCount = 0;
    std::for_each(transforms.cbegin(), transforms.cend(),
                  [&tCount](const auto& transform)
                  {
                      tCount += transform.second->TransformCount();
                  });
    std::for_each(mediums.cbegin(), mediums.cend(),
                  [&mCount](const auto& medium)
                  {
                      mCount += medium.second->MediumCount();
                  });
    std::for_each(lights.cbegin(), lights.cend(),
                  [&lCount](const auto& light)
                  {
                      lCount += light.second->LightCount();
                  });
    std::for_each(cameras.cbegin(), cameras.cend(),
                  [&cCount](const auto& camera)
                  {
                      cCount += camera.second->CameraCount();
                  });
    transformCount = static_cast<uint32_t>(tCount);
    mediumCount = static_cast<uint32_t>(mCount);
    lightCount = static_cast<uint32_t>(lCount);
    cameraCount = static_cast<uint32_t>(cCount);

    // Allocate
    size_t transformSize = transformCount * sizeof(GPUTransformI*);
    transformSize = Memory::AlignSize(transformSize, AlignByteCount);
    size_t mediumSize = mediumCount * sizeof(GPUMediumI*);
    mediumSize = Memory::AlignSize(mediumSize, AlignByteCount);
    size_t lightSize = lightCount * sizeof(GPULightI*);
    lightSize = Memory::AlignSize(lightSize, AlignByteCount);
    size_t cameraSize = cameraCount * sizeof(GPUCameraI*);
    cameraSize = Memory::AlignSize(cameraSize, AlignByteCount);

    size_t totalSize = (transformSize +
                        mediumSize +
                        lightSize +
                        cameraSize);

    DeviceMemory::EnlargeBuffer(commonTypeMemory, totalSize);

    // Determine pointers from allocation
    size_t offset = 0;
    Byte* memory = static_cast<Byte*>(commonTypeMemory);
    dTransforms = reinterpret_cast<const GPUTransformI**>(memory + offset);
    offset += transformSize;
    dMediums = reinterpret_cast<const GPUMediumI**>(memory + offset);
    offset += mediumSize;
    dCameras = reinterpret_cast<const GPUCameraI**>(memory + offset);
    offset += cameraSize;
    dLights = reinterpret_cast<const GPULightI**>(memory + offset);
    offset += lightSize;
    assert(offset == totalSize);

    // Transforms
    if((e = LoadTransforms(dGPUTransforms)) != TracerError::OK)
        return e;
    CUDA_CHECK(hipMemcpy(const_cast<GPUTransformI**>(dTransforms),
                          dGPUTransforms.data(),
                          dGPUTransforms.size() * sizeof(GPUTransformI*),
                          hipMemcpyHostToDevice));
    // Mediums
    if((e = LoadMediums(dGPUMediums)) != TracerError::OK)
        return e;
    CUDA_CHECK(hipMemcpy(const_cast<GPUMediumI**>(dMediums),
                          dGPUMediums.data(),
                          dGPUMediums.size() * sizeof(GPUMediumI*),
                          hipMemcpyHostToDevice));
    // Lights
    if((e = LoadLights(dGPULights)) != TracerError::OK)
        return e;
    CUDA_CHECK(hipMemcpy(const_cast<GPULightI**>(dLights),
                          dGPULights.data(),
                          dGPULights.size() * sizeof(GPULightI*),
                          hipMemcpyHostToDevice));
    // Cameras
    if((e = LoadCameras(dGPUCameras)) != TracerError::OK)
        return e;
    CUDA_CHECK(hipMemcpy(const_cast<GPUCameraI**>(dCameras),
                          dGPUCameras.data(),
                          dGPUCameras.size() * sizeof(GPUCameraI*),
                          hipMemcpyHostToDevice));

    // Attach Medium gpu pointer to Material Groups
    for(const auto& mg : materialGroups)
        mg.second->AttachGlobalMediumArray(dMediums, baseMediumIndex);
        
    // Attach Transform gpu pointer to the Accelerator Batches
    for(const auto& acc : accelBatches)
        acc.second->AttachGlobalTransformArray(dTransforms, identityTransformIndex);

    // Construct Accelerators
    SurfaceAABBList allSurfaceAABBs;
    for(const auto& accBatch : accelBatches)
    {
        GPUAcceleratorGroupI* acc = accBatch.second;
        if((e = acc->ConstructAccelerators(cudaSystem)) != TracerError::OK)
            return e;
        // Acquire surface aabb listings for base accelerator consrtuction  
        allSurfaceAABBs.insert(acc->AcceleratorAABBs().cbegin(), 
                               acc->AcceleratorAABBs().cend());

    }

    // Construct Base accelerator using aabb list
    if((e = baseAccelerator.Constrcut(cudaSystem, allSurfaceAABBs)) != TracerError::OK)
        return e;

    cudaSystem.SyncGPUAll();
    return TracerError::OK;
}

void GPUTracer::ResetHitMemory(uint32_t rayCount, HitKey baseBoundMatKey)
{
    currentRayCount = rayCount;
    rayMemory.ResizeRayOut(rayCount, baseBoundMatKey);
}

void GPUTracer::HitAndPartitionRays()
{   
    if(crashed) return;

    // Sort and Partition happens on the leader device
    CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice().DeviceId()));

    // Tracer Logic interface
    const Vector2i& accBitCounts = maxAccelBits;
    const AcceleratorBatchMap& subAccelerators = accelBatches;
    // Reset Hit Memory for hit loop
    rayMemory.ResetHitMemory(identityTransformIndex, currentRayCount, maxHitSize);
    // Make Base Accelerator to get ready for hitting
    baseAccelerator.GetReady(cudaSystem, currentRayCount);
    // Ray Memory Pointers
    RayGMem* dRays = rayMemory.Rays();
    HitKey* dWorkKeys = rayMemory.WorkKeys();
    TransformId* dTransfomIds = rayMemory.TransformIds();
    PrimitiveId* dPrimitiveIds = rayMemory.PrimitiveIds();
    HitStructPtr dHitStructs = rayMemory.HitStructs();
    // These are sorted etc.
    HitKey* dCurrentKeys = rayMemory.CurrentKeys();
    RayId*  dCurrentRayIds = rayMemory.CurrentIds();

    //CUDA_CHECK(hipMemset(dTransfomIds, 0xFF, currentRayCount * sizeof(TransformId)));
    //Debug::DumpMemToFile("dTransforms", dTransfomIds, currentRayCount);


    // Try to hit rays until no ray is left
    // (these rays will be assigned with a material)
    // outside rays are also assigned with a material (which is special)
    uint32_t rayCount = currentRayCount;
    // At start all rays are valid
    uint32_t validRayOffset = 0;
    while(rayCount > 0)
    {
        //Debug::DumpMemToFile("dAccKeys", dCurrentKeys, currentRayCount);
        //Debug::DumpMemToFile("dRayIds", dCurrentRayIds, currentRayCount);

        // Traverse accelerator
        // Base accelerator provides potential hits
        // Cannot provide an absolute hit (its not its job)
        baseAccelerator.Hit(cudaSystem,
                            dCurrentKeys + validRayOffset,
                            dRays,
                            dCurrentRayIds + validRayOffset,
                            rayCount);

        // Wait all GPUs to finish...
        cudaSystem.SyncGPUMainStreamAll();

        //Debug::DumpMemToFile("dAccKeys", dCurrentKeys, currentRayCount);
        //Debug::DumpMemToFile("dRayIds", dCurrentRayIds, currentRayCount);

        // Base accelerator traverses the data partially
        // Updates current key (which represents inner accelerator batch and id)

        // After that, system sorts rays according to the keys
        // and partitions the array according to batches

        // Sort and Partition happens on the leader device
        CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice().DeviceId()));

        // Sort initial results (in order to partition and launch kernels accordingly)
        // Sort is radix sort.
        // We sort inner indices in addition to batches results for better data locality
        // We only sort up-to a certain bit (radix sort) which is tied to
        // accelerator count
        // Move offset to skip null bathces
        rayMemory.SortKeys(dCurrentRayIds, dCurrentKeys, rayCount, accBitCounts);
        // Parition to sub accelerators
        //
        // There may be invalid rays sprinkled along the array.
        // Holes occur in the structure since in previous iteration,
        // a material may required to write N rays for its output (which is defined
        // by the material) but it wrote < N rays.
        //
        // One of the main examples for such behaviour can be transparent objects
        // where ray may be only reflected (instead of refrating and reflecting) because
        // of the total internal reflection phenomena.
        auto portions = rayMemory.Partition(rayCount);

        //Debug::DumpMemToFile("dAccKeys", dCurrentKeys, currentRayCount);
        //Debug::DumpMemToFile("dRayIds", dCurrentRayIds, currentRayCount);

        // Reorder partitions for efficient calls
        // (group partitions into gpus and order for better async access)
        // ....
        // TODO:
        const int totalGPU = static_cast<int>(cudaSystem.GPUList().size());
        const auto& gpus = cudaSystem.GPUList();
        auto currentGPU = gpus.begin();

        // For each partition
        for(const auto& p : portions)
        {
            // Find Accelerator
            // Since there is no batch for invalid keys
            // that partition will be automatically be skipped
            auto loc = subAccelerators.find(p.portionId);
            if(loc == subAccelerators.end()) continue;

            RayId* dRayIdStart = dCurrentRayIds + validRayOffset + p.offset;
            HitKey* dCurrentKeyStart = dCurrentKeys + validRayOffset + p.offset;

            // Run local hit kernels
            // Local hit kernels returns a material key
            // and primitive inner id.
            // Since materials are batched for both material and
            loc->second->Hit(*currentGPU,
                             // O
                             dWorkKeys,
                             dTransfomIds,
                             dPrimitiveIds,
                             dHitStructs,
                             // I-O
                             dRays,
                             // Input
                             dRayIdStart,
                             dCurrentKeyStart,
                             static_cast<uint32_t>(p.count));

            // Split to GPUs
            currentGPU++;
            if(currentGPU == gpus.end()) currentGPU = gpus.begin();

            // Hit function updates material key,
            // primitive id and struct if this hit is accepted
        }
        //printf("=====================================================\n");

        // Update new ray count
        // On partition array check first partition
        // it may contain invalid key meaning
        // those rays are totally processed
        // change the offset so that
        // we skip those rays
        auto nullPortion = portions.begin();
        if(nullPortion->portionId == HitKey::NullBatch)
            rayCount = static_cast<uint32_t>(nullPortion->offset);

        // Iteration is done
        // We cant continue loop untill these kernels are finished
        // on gpu(s)
        //
        // Tracer logic mostly utilizies mutiple GPUs so we need to
        // wait all GPUs to finish
        cudaSystem.SyncGPUAll();
    }

    // At the end of iteration all rays found a material, primitive
    // and interpolation weights (which should be on hitStruct)

    // Partition rays for work kernel calls
    // Copy materialKeys to currentKeys
    // to make it ready for sorting
    rayMemory.FillMatIdsForSort(currentRayCount);
    // Sort with respect to the materials keys
    rayMemory.SortKeys(dCurrentRayIds, dCurrentKeys, currentRayCount, maxWorkBits);

    //Debug::DumpMemToFile("MatKeysIn", dCurrentKeys, currentRayCount);
    //Debug::DumpMemToFile("workKeyIn", rayMemory.WorkKeys(), currentRayCount);

    // Parition w.r.t. material batch
    workPartition.clear();
    workPartition = rayMemory.Partition(currentRayCount);


    //Debug::DumpMemToFile("dTransforms", dTransfomIds, currentRayCount);
    //printf("HIT PORTION END\n");
}

void GPUTracer::WorkRays(const WorkBatchMap& workMap, 
                         const RayPartitions<uint32_t>& outPortions,
                         uint32_t totalRayOut,
                         HitKey baseBoundMatKey)
{
    // Sort and Partition happens on leader device
    CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice().DeviceId()));

    // Ray Memory Pointers
    const RayGMem* dRays = rayMemory.Rays();
    const HitStructPtr dHitStructs = rayMemory.HitStructs();
    const PrimitiveId* dPrimitiveIds = rayMemory.PrimitiveIds();
    const TransformId* dTransformIds = rayMemory.TransformIds();
    // These are sorted etc.
    HitKey* dCurrentKeys = rayMemory.CurrentKeys();
    RayId* dCurrentRayIds = rayMemory.CurrentIds();

    // Allocate output ray memory
    rayMemory.ResizeRayOut(totalRayOut, baseBoundMatKey);
    RayGMem* dRaysOut = rayMemory.RaysOut();
    HitKey* dBoundKeyOut = rayMemory.WorkKeys();

    // Reorder partitions for efficient calls
    // (sort by gpu and order for better async access)
    // ....
    // TODO:

    // For each partition
    //for(auto pIt = workPartition.crbegin();
    //    pIt != workPartition.crend(); pIt++)
    for(const auto& p : workPartition)
    {
        //const auto& p = (*pIt);

        // Skip if null batch or unfound material
        if(p.portionId == HitKey::NullBatch) continue;
        auto loc = workMap.find(p.portionId);
        if(loc == workMap.end()) continue;

        // TODO: change this loop to combine iterator instead of find
        //const auto& pIn = *(workPartition.find<uint32_t>(p.portionId));
        const auto& pOut = *(outPortions.find(ArrayPortion<uint32_t>{p.portionId}));

        // Relativize input & output pointers
        const RayId* dRayIdStart = dCurrentRayIds + p.offset;
        const HitKey* dKeyStart = dCurrentKeys + p.offset;
        // Output
        RayGMem* dRayOutStart = dRaysOut + pOut.offset;
        HitKey* dBoundKeyStart = dBoundKeyOut + pOut.offset;

        // Actual Shade Call
        loc->second->Work(dBoundKeyStart,
                          dRayOutStart,
                          //  Input
                          dRays,
                          dPrimitiveIds,
                          dTransformIds,
                          dHitStructs,
                          // Ids
                          dKeyStart,
                          dRayIdStart,
                          //
                          static_cast<uint32_t>(p.count),
                          rngMemory);

        //cudaSystem.SyncGPUAll();
        //METU_LOG("--------------------------");
    }
    currentRayCount = totalRayOut;

    // Again wait all of the GPU's since
    // CUDA functions will be on multiple-gpus
    cudaSystem.SyncGPUAll();

    //Debug::DumpMemToFile("workKeyOut", rayMemory.WorkKeys(), totalRayOut);

    // Shading complete
    // Now make "RayOut" to "RayIn"
    // and continue
    rayMemory.SwapRays();
}

void GPUTracer::SetParameters(const TracerParameters& p)
{
    if(params.seed != p.seed)
        rngMemory = std::move(RNGMemory(p.seed, cudaSystem));
    params = p;
}

void GPUTracer::SetImagePixelFormat(PixelFormat f)
{
    imgMemory.SetPixelFormat(f, cudaSystem);
}

void GPUTracer::ReportionImage(Vector2i start,
                                Vector2i end)
{
    imgMemory.Reportion(start, end, cudaSystem);
}

void GPUTracer::ResizeImage(Vector2i resolution)
{
    imgMemory.Resize(resolution);
}

void GPUTracer::ResetImage()
{
    imgMemory.Reset(cudaSystem);
}

template <class... Args>
inline void GPUTracer::SendLog(const char* format, Args... args)
{
    if(!params.verbose) return;

    size_t size = snprintf(nullptr, 0, format, args...);
    std::string s(size, '\0');
    snprintf(&s[0], size, format, args...);
    if(callbacks) callbacks->SendLog(s);
}

void GPUTracer::SendError(TracerError e, bool isFatal)
{
    if(callbacks) callbacks->SendError(e);
    crashed = isFatal;
}

RayPartitions<uint32_t> GPUTracer::PartitionOutputRays(uint32_t& totalOutRay,
                                                       const WorkBatchMap& workMap) const
{
    RayPartitions<uint32_t> outPartitions;

    // Find total ray out
    totalOutRay = 0;
    for(auto pIt = workPartition.crbegin();
        pIt != workPartition.crend(); pIt++)
    {
        const auto& p = (*pIt);

        // Skip if null batch or unfound material
        if(p.portionId == HitKey::NullBatch) continue;
        auto loc = workMap.find(p.portionId);
        if(loc == workMap.end()) continue;

        uint32_t count = (static_cast<uint32_t>(p.count) * 
                          loc->second->OutRayCount());

        outPartitions.emplace(ArrayPortion<uint32_t>
        {
            p.portionId,
                totalOutRay,
            count
        });
        totalOutRay += count;
    }
    return outPartitions;
}

void GPUTracer::Finalize()
{
    if(crashed) return;
    SendLog("Finalizing...");
   
    // Determine Size
    Vector2i pixelCount = imgMemory.SegmentSize();
    Vector2i start = imgMemory.SegmentOffset();
    Vector2i end = start + imgMemory.SegmentSize();
    size_t offset = (static_cast<size_t>(pixelCount[0])* pixelCount[1] *
                     imgMemory.PixelSize());

    // Flush Devices and Get the Image
    cudaSystem.SyncGPUAll();
    std::vector<Byte> imageData = imgMemory.GetImageToCPU(cudaSystem);

    size_t pixelCount1D = static_cast<size_t>(pixelCount[0]) * pixelCount[1];

    // Launch finished image
    if(callbacks) callbacks->SendImage(std::move(imageData),
                                       imgMemory.Format(),
                                       offset,
                                       start, end);
    SendLog("Image sent!");
}

void GPUTracer::AskParameters()
{
    if(callbacks) callbacks->SendCurrentParameters(params);
}
