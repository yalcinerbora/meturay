#include "hip/hip_runtime.h"
#include "AnisoSVO.cuh"

#include "CudaSystem.h"
#include "CudaSystem.hpp"
#include "PathNode.cuh"

#include "RayLib/ColorConversion.h"
#include "RayLib/HitStructs.h"
#include "RayLib/BitManipulation.h"
#include "RayLib/CPUTimer.h"

#include "GPUAcceleratorI.h"
#include "ParallelReduction.cuh"
#include "ParallelScan.cuh"
#include "ParallelMemset.cuh"
#include "ParallelSequence.cuh"
#include "BinarySearch.cuh"

#include <hipcub/hipcub.hpp>
#include <numeric>

#include "TracerDebug.h"

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCGetLightKeys(HitKey* gKeys,
                    const GPULightI** gLights,
                    uint32_t totalLightCount)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < totalLightCount;
        threadId += (blockDim.x * gridDim.x))
    {
        gKeys[threadId] = gLights[threadId]->WorkKey();
    }
}


__global__ CUDA_LAUNCH_BOUNDS_1D
void KCMarkMortonChanges(uint32_t* gMarks,
                         const uint64_t* gVoxels,
                         uint32_t voxelCount,
                         uint32_t level,
                         uint32_t maxLevel)
{
    static constexpr uint32_t DIMENSION = 3;
    const uint32_t voxelMSBStart = sizeof(uint64_t) * BYTE_BITS - (maxLevel * DIMENSION);

    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < voxelCount;
        threadId += (blockDim.x * gridDim.x))
    {
        uint64_t voxMorton = gVoxels[threadId];
        assert((voxMorton & (~((1ull << (maxLevel * DIMENSION)) - 1))) == 0);
        uint64_t voxMortonRev = __brevll(voxMorton) >> voxelMSBStart;
        voxMortonRev &= (1ull << level * DIMENSION) - 1;

        uint64_t voxMortonNext = gVoxels[threadId + 1];
        assert((voxMortonNext & (~((1ull << (maxLevel * DIMENSION)) - 1))) == 0);
        uint64_t voxMortonNextRev = __brevll(voxMortonNext) >> voxelMSBStart;
        voxMortonNextRev &= (1ull << level * DIMENSION) - 1;

        gMarks[threadId] = (voxMortonRev != voxMortonNextRev) ? 1 : 0;
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCMarkChild(// I-O
                 uint64_t* gNodes,
                 // Input
                 const uint64_t* gVoxels,
                 // Constants
                 uint32_t voxelCount,
                 uint32_t level,
                 uint32_t maxLevel)
{
    // Useful constants
    static constexpr uint32_t DIMENSION = 3;
    static constexpr uint32_t DIM_MASK = (1 << DIMENSION) - 1;

    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < voxelCount;
        threadId += (blockDim.x * gridDim.x))
    {
        uint64_t voxelMortonCode = gVoxels[threadId];
        uint32_t mortonLevelShift = (maxLevel - 1) * DIMENSION;
        // Start traversing (Root node is the very first node)
        uint32_t currentNodeIndex = 0;
        for(int i = 0; i < level; i++)
        {
            uint64_t currentNode = gNodes[currentNodeIndex];
            // Fetch the current bit triples of the level from the
            // morton code
            uint32_t childId = (voxelMortonCode >> mortonLevelShift) & DIM_MASK;
            uint32_t childOffset = AnisoSVOctreeGPU::FindChildOffset(currentNode, childId);
            uint32_t childrenIndex = AnisoSVOctreeGPU::ChildrenIndex(currentNode);
            // Go to next child
            currentNodeIndex = childrenIndex + childOffset;
            // Strip the processed bits
            mortonLevelShift -= DIMENSION;
        }
        // Now we are at the not that does not set its children ptr and mask is set
        // Atomically mark the required child
        uint32_t childId = (voxelMortonCode >> mortonLevelShift) & DIM_MASK;
        uint32_t childBit = (1 << childId);
        assert(childId < 8);
        assert(__popc(childBit) == 1);
        // Atomically set the child bit on the packed node
        AnisoSVOctreeGPU::AtomicSetChildMaskBit(gNodes + currentNodeIndex,
                                                childBit);
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCExtractChildrenCounts(uint32_t* gChildrenCounts,
                             const uint64_t* gLevelNodes,
                             uint32_t levelNodeCount)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < levelNodeCount;
        threadId += (blockDim.x * gridDim.x))
    {
        uint64_t node = gLevelNodes[threadId];
        uint32_t childrenCount = AnisoSVOctreeGPU::ChildrenCount(node);
        // Write the count
        gChildrenCounts[threadId] = childrenCount;
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCSetChildrenPtrs(uint64_t* gLevelNodes,
                       const uint32_t* gChildrenOffsets,
                       uint32_t nextLevelStartIndex,
                       uint32_t levelNodeCount,
                       bool markIsChildrenLeaf)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < levelNodeCount;
        threadId += (blockDim.x * gridDim.x))
    {
        uint64_t node = gLevelNodes[threadId];
        uint32_t offset = gChildrenOffsets[threadId];

        // Children offsets are relative to the level
        // we need to put global pointer (index)
        uint32_t globalOffset = nextLevelStartIndex + offset;
        AnisoSVOctreeGPU::SetChildrenIndex(node, globalOffset);
        // If this the last non-leaf level we need to mark the children
        if(markIsChildrenLeaf)
            AnisoSVOctreeGPU::SetIsChildrenLeaf(node, true);
        // Write back the modified node
        gLevelNodes[threadId] = node;
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCSetParentOfChildren(uint64_t* gNodes,
                           const uint64_t* gLevelNodes,
                           uint32_t levelNodeCount)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < levelNodeCount;
        threadId += (blockDim.x * gridDim.x))
    {
        uint64_t node = gLevelNodes[threadId];
        uint32_t childrenCount = AnisoSVOctreeGPU::ChildrenCount(node);
        uint32_t childrenIndex = AnisoSVOctreeGPU::ChildrenIndex(node);

        // Find the parent id using pointer arithmetic
        uint32_t currentNodeGlobalId  = (gLevelNodes + threadId) - gNodes;
        // Set ptrs for all children
        for(uint32_t i = 0; i < childrenCount; i++)
        {
            uint64_t* gChildNode = gNodes + childrenIndex + i;
            AnisoSVOctreeGPU::SetParentIndex(*gChildNode, currentNodeGlobalId);
        }
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCSetParentOfLeafChildren(uint32_t* gLeafParents,
                               const uint64_t* gNodes,
                               const uint64_t* gLevelNodes,
                               uint32_t levelNodeCount)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < levelNodeCount;
        threadId += (blockDim.x * gridDim.x))
    {
        uint64_t node = gLevelNodes[threadId];
        uint32_t childrenCount = AnisoSVOctreeGPU::ChildrenCount(node);
        uint32_t childrenIndex = AnisoSVOctreeGPU::ChildrenIndex(node);
        assert(AnisoSVOctreeGPU::IsChildrenLeaf(node));
        // Find the parent id using pointer arithmetic
        uint32_t currentNodeGlobalId = (gLevelNodes + threadId) - gNodes;
        // Set ptrs for all children
        for(uint32_t i = 0; i < childrenCount; i++)
        {
            uint32_t* gChildParent = gLeafParents + childrenIndex + i;
            *gChildParent = currentNodeGlobalId;
        }
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCReduceVoxelPayload(// I-O
                          AnisoSVOctreeGPU treeGPU,
                          // Input
                          const uint32_t* gVoxelIndexOffsets,
                          const uint64_t* gUniqueVoxels,
                          // non-unique voxel index array
                          const uint32_t* gSortedVoxelIndices,
                          // Voxel payload that will be reduced
                          const HitKey* gVoxelLightKeys,
                          const Vector2us* gVoxelNormals,
                          // Binary Search for light
                          const HitKey* gLightKeys,
                          const GPULightI** gLights,
                          uint32_t lightCount,
                          // Constants
                          uint32_t uniqueVoxCount,
                          uint32_t lightKeyCount,
                          const AABB3f svoAABB,
                          uint32_t resolutionXYZ)
{
    // At most this value of voxels will be processed to cluster normals
    static constexpr uint32_t MAX_CLUSTERED_VOXEL_COUNT = 8;

    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < uniqueVoxCount;
        threadId += (blockDim.x * gridDim.x))
    {
        // Voxel Key (Morton Code)
        uint64_t mortonCode = gUniqueVoxels[threadId];
        // And converted world position
        Vector3ui denseIndex = MortonCode::Decompose<uint64_t>(mortonCode);
        Vector3f worldPos = treeGPU.VoxelToWorld(denseIndex);


        Vector2ui reduceRange = Vector2ui(gVoxelIndexOffsets[threadId],
                                          gVoxelIndexOffsets[threadId + 1]);
        uint32_t dupVoxCount = reduceRange[1] - reduceRange[0];

        // First do a clustering of the normals
        // For each colliding voxel
        Vector3f normals[MAX_CLUSTERED_VOXEL_COUNT];
        #pragma unroll
        for(uint32_t i = 0; i < MAX_CLUSTERED_VOXEL_COUNT; i++)
        {
            if(i < dupVoxCount)
            {
                uint32_t index = gSortedVoxelIndices[reduceRange[0] + i];
                Vector2us normal = gVoxelNormals[index];

                Vector2f normalSphr = Vector2f(static_cast<float>(normal[0]) / 65535.0f,
                                               static_cast<float>(normal[1]) / 65535.0f);
                normalSphr[0] *= MathConstants::Pi * 2.0f - MathConstants::Pi;
                normalSphr[1] *= MathConstants::Pi;
                normals[i] = Utility::SphericalToCartesianUnit(normalSphr);

            }
            else normals[i] = Zero3f;
        }

        // Mean calculation variables
        Vector3f normalMeans[2][2] = {{normals[0], normals[1]},
                                      {Zero3f, Zero3f}};
        // Cluster id of each normal
        uint8_t clusterIds[MAX_CLUSTERED_VOXEL_COUNT];

        uint8_t availableNormalCount = min(MAX_CLUSTERED_VOXEL_COUNT, dupVoxCount);
        uint32_t meanBufferId = 0;
        // Do 2-means clustering
        static constexpr uint32_t K_MEANS_CLUSTER_ITER_COUNT = 4;
        #pragma unroll
        for(uint32_t l = 0; l < K_MEANS_CLUSTER_ITER_COUNT; l++)
        {
            for(uint32_t i = 0; i < availableNormalCount; i++)
            {
                // Calculate offsets
                float angularDist0 = normalMeans[meanBufferId][0].Dot(normals[i]);
                float angularDist1 = normalMeans[meanBufferId][1].Dot(normals[i]);

                uint32_t updateIndex = (angularDist0 < angularDist1) ? 0 : 1;
                normalMeans[(meanBufferId + 1) % 2][updateIndex] += normals[i];
                clusterIds[i] = updateIndex;
            }

            // One iteration done calculate actual mean by normalizing;
            // Reset for next iteration
            normalMeans[meanBufferId][0] = Zero3f;
            normalMeans[meanBufferId][1] = Zero3f;

            // Swap mean buffers
            meanBufferId = (meanBufferId + 1) % 2;

            // Calculate actual mean
            normalMeans[meanBufferId][0].NormalizeSelf();
            normalMeans[meanBufferId][1].NormalizeSelf();
        }

        // We estimated some normals using clustering.
        // Now find the variance and select the smaller cluster for the actual normal
        // TODO: Change this
        Vector3f combinedNormal = normalMeans[meanBufferId][0];

        // Then calculate the light luminance for each non unique voxel
        // For each colliding voxel
        Vector2f combinedLuminance = Zero2f;
        for(uint32_t i = 0; i < dupVoxCount; i++)
        {
            // Get Index
            uint32_t voxelIndex = gSortedVoxelIndices[reduceRange[0] + i];
            HitKey lightKey = gVoxelLightKeys[voxelIndex];

            // Skip if it is not a light
            if(lightKey == HitKey::InvalidKey) continue;

            // Binary search the light with key
            float lightIndexF;
            bool found = GPUFunctions::BinarySearchInBetween(lightIndexF, lightKey,
                                                             gLightKeys, lightCount);
            uint32_t lightIndex = static_cast<uint32_t>(lightIndexF);
            assert(found);
            if(!found)
            {
                KERNEL_DEBUG_LOG("Error: SVO light not found!\n");
                continue;
            }
            const GPULightI* gLight = gLights[lightIndex];

            // Query both sides of the surface
            // Towards normal
            Vector3f radiance = gLight->Emit(combinedNormal, worldPos, UVSurface{});
            combinedLuminance[0] += Utility::RGBToLuminance(radiance);

            // Query both sides of the surface
            radiance = gLight->Emit(-combinedNormal, worldPos, UVSurface{});
            combinedLuminance[1] += Utility::RGBToLuminance(radiance);
        }
        // Don't forget to average
        combinedLuminance /= static_cast<float>(dupVoxCount);

        // We are setting initial sample count to this voxel
        // there shouldn't be any updates to this voxel anyway but just to be sure
        uint32_t initialSampleCount = 10'000;
        // Set the combined values
        treeGPU.SetLeafRadiance(mortonCode, combinedLuminance, initialSampleCount);
        treeGPU.SetLeafNormal(mortonCode, combinedNormal);
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCAccumulateRadianceToLeaf(AnisoSVOctreeGPU svo,
                                // Input
                                const PathGuidingNode* gPathNodes,
                                uint32_t nodeCount,
                                uint32_t maxPathNodePerRay)
{
    bool unableToAccum = false;

    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < nodeCount;
        threadId += (blockDim.x * gridDim.x))
    {
        const uint32_t nodeIndex = threadId;
        const uint32_t pathStartIndex = nodeIndex / maxPathNodePerRay * maxPathNodePerRay;

        PathGuidingNode gPathNode = gPathNodes[nodeIndex];

        // Skip if this node cannot calculate wo
        if(!gPathNode.HasPrev()) continue;

        Vector3f wo = gPathNode.Wo<PathGuidingNode>(gPathNodes, pathStartIndex);
        float luminance = Utility::RGBToLuminance(gPathNode.totalRadiance);
        unableToAccum |= !svo.DepositRadiance(gPathNode.worldPosition, wo, luminance);
    }
    // Debug
    if(unableToAccum)
    {
        printf("Unable to accumulate some radiance values!\n");
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCCollapseRayCounts(// I-O
                         uint16_t* gBinInfo,
                         // Input
                         const uint64_t* gNodes,
                         // Constants
                         Vector2ui levelRange,
                         uint32_t level,
                         uint32_t minLevel,
                         uint32_t minRayCount)
{
    uint32_t nodeCount = levelRange[1] - levelRange[0];

    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < nodeCount;
        threadId += (blockDim.x * gridDim.x))
    {
        uint32_t nodeId = levelRange[0] + threadId;

        // We are at the user option limit,
        // Directly mark this node then leave
        if(level == minLevel)
        {
            AnisoSVOctreeGPU::SetBinAsMarked(gBinInfo[nodeId]);
            continue;
        }
        // Fetch Ray Count
        uint32_t rayCount = AnisoSVOctreeGPU::GetRayCount(gBinInfo[nodeId]);
        // If ray count is not enough on this voxel
        // collaborate with the other children
        if(rayCount < minRayCount)
        {
            uint32_t parent = AnisoSVOctreeGPU::ParentIndex(gNodes[nodeId]);
            AnisoSVOctreeGPU::AtomicAddUInt16(gBinInfo + parent, rayCount);
        }
        // We have enough rays in this node use it as is
        else
        {
            AnisoSVOctreeGPU::SetBinAsMarked(gBinInfo[nodeId]);
        }
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCCollapseRayCountsLeaf(// I-O
                             uint16_t* gLeafBinInfo,
                             uint16_t* gBinInfo,
                             // Input
                             const uint32_t* gLeafParents,
                             // Constants
                             uint32_t leafCount,
                             uint32_t level,
                             uint32_t minLevel,
                             uint32_t minRayCount)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < leafCount;
        threadId += (blockDim.x * gridDim.x))
    {
        // We are at the user option limit,
        // Directly mark this node then leave
        if(level == minLevel)
        {
            AnisoSVOctreeGPU::SetBinAsMarked(gLeafBinInfo[threadId]);
            continue;
        }
        // Fetch Ray Count
        uint16_t rayCount = AnisoSVOctreeGPU::GetRayCount(gLeafBinInfo[threadId]);
        if(rayCount == 0) continue;

        // If ray count is not enough on this voxel
        // collaborate with the other children
        if(rayCount < minRayCount)
        {
            uint32_t parent = gLeafParents[threadId];
            AnisoSVOctreeGPU::AtomicAddUInt16(gBinInfo + parent, rayCount);
        }
        // We have enough rays in this node use it as is
        else
        {
            AnisoSVOctreeGPU::SetBinAsMarked(gLeafBinInfo[threadId]);
        }
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCCCopyRadianceToHalfBufferLeaf(// I-O
                                     AnisoSVOctreeGPU::AnisoRadiance* dLeafRadianceRead,
                                     // Input
                                     const AnisoSVOctreeGPU::AnisoRadianceF* dLeafRadianceWrite,
                                     const AnisoSVOctreeGPU::AnisoCount* dLeafSampleCountWrite,
                                     // Constants
                                     uint32_t leafCount,
                                     float totalRadianceScene)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < leafCount;
        threadId += (blockDim.x * gridDim.x))
    {
        AnisoSVOctreeGPU::AnisoRadianceF anisoRad = dLeafRadianceWrite[threadId];
        AnisoSVOctreeGPU::AnisoCount anisoCount = dLeafSampleCountWrite[threadId];
        AnisoSVOctreeGPU::AnisoRadiance anisoOut;

        for(int i = 0; i < AnisoSVOctreeGPU::VOXEL_DIR_DATA_COUNT; i++)
        {
            uint32_t count = anisoCount.Read(i);
            float radClamped = 0.0f;
            if(count != 0)
            {
                float radiance = anisoRad.Read(i);
                float avgRadiance = radiance / count;
                // Normalize & Clamp the half range for now
                radClamped = fmin(MRAY_HALF_MAX, avgRadiance);
            }
            anisoOut.Write(i, radClamped);
        }
        dLeafRadianceRead[threadId] = anisoOut;
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCConvertToAnisoFloat(AnisoSVOctreeGPU::AnisoRadianceF* gAnisoOut,
                           const AnisoSVOctreeGPU::AnisoRadiance* gAnisoIn,
                           uint32_t anisoCount)
{
    using AnisoRadianceF = AnisoSVOctreeGPU::AnisoRadianceF;
    using AnisoRadiance = AnisoSVOctreeGPU::AnisoRadiance;

    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < anisoCount;
        threadId += (blockDim.x * gridDim.x))
    {
        AnisoRadianceF& gOut = gAnisoOut[threadId];
        AnisoRadiance in = gAnisoIn[threadId];

        gOut.data[0][0] = in.data[0][0];
        gOut.data[0][1] = in.data[0][1];
        gOut.data[0][2] = in.data[0][2];
        gOut.data[0][3] = in.data[0][3];

        gOut.data[1][0] = in.data[1][0];
        gOut.data[1][1] = in.data[1][1];
        gOut.data[1][2] = in.data[1][2];
        gOut.data[1][3] = in.data[1][3];
    }
}

TracerError AnisoSVOctreeCPU::Constrcut(const AABB3f& sceneAABB, uint32_t resolutionXYZ,
                                        const AcceleratorBatchMap& accels,
                                        const GPULightI** dSceneLights,
                                        uint32_t totalLightCount,
                                        HitKey boundaryLightKey,
                                        const CudaSystem& system)
{
    Utility::CPUTimer timer;
    timer.Start();

    // Find The SVO AABB
    Vector3f span = sceneAABB.Span();
    int maxDimIndex = span.Max();
    float worldSizeXYZ = span[maxDimIndex];
    float sizePadding = (worldSizeXYZ / static_cast<float>(resolutionXYZ));
    treeGPU.svoAABB = AABB3f(sceneAABB.Min() - Vector3f(sizePadding),
                             sceneAABB.Min() + Vector3f(sizePadding + worldSizeXYZ));
    treeGPU.leafDepth = Utility::FindLastSet(resolutionXYZ);
    treeGPU.leafVoxelSize = (worldSizeXYZ + 2.0f * sizePadding) / static_cast<float>(resolutionXYZ);
    treeGPU.voxelResolution = resolutionXYZ;

    // Find out the sort memory requirement of Light Keys
    size_t lightSortMemSize;
    HitKey* dLightKeys = nullptr;
    const GPULightI** dSortedLights = nullptr;
    HitKey* dSortedLightKeys = nullptr;
    CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(nullptr, lightSortMemSize,
                                               reinterpret_cast<HitKey::Type*>(dLightKeys),
                                               reinterpret_cast<HitKey::Type*>(dSortedLightKeys),
                                               dSceneLights, dSortedLights,
                                               totalLightCount));

    DeviceMemory lightMemory;
    GPUMemFuncs::AllocateMultiData(std::tie(dLightKeys, dSortedLights,
                                            dSortedLightKeys),
                                   lightMemory,
                                   {totalLightCount, totalLightCount,
                                    totalLightCount});

    // For each accelerator
    // First allocate per prim voxel count array
    std::vector<size_t> primOffsets;
    primOffsets.reserve(accels.size() + 1);
    primOffsets.push_back(0);
    for(const auto [_, accel] : accels)
    {
        primOffsets.push_back(accel->TotalPrimitiveCount());
    }

    std::inclusive_scan(primOffsets.cbegin(), primOffsets.cend(),
                        primOffsets.begin());

    // Allocate Voxel Count memory
    uint64_t* dVoxelCounts;
    uint64_t* dVoxelOffsets;
    uint64_t* dPrimOffsets;
    Byte* dLightSortTempMem;
    DeviceMemory voxOffsetMem;
    GPUMemFuncs::AllocateMultiData(std::tie(dVoxelCounts,
                                            dVoxelOffsets,
                                            dPrimOffsets,
                                            dLightSortTempMem),
                                   voxOffsetMem,
                                   {primOffsets.back(),
                                    primOffsets.back() + 1,
                                    accels.size() + 1,
                                    lightSortMemSize});

    // Copy prim offsets for segmented reduction
    CUDA_CHECK(hipMemcpy(dPrimOffsets, primOffsets.data(),
                          sizeof(uint64_t) * accels.size(),
                          hipMemcpyHostToDevice));

    // Ask each primitive for rasterized voxel count
    uint32_t i = 0;
    for(const auto [_, accel] : accels)
    {
        accel->EachPrimVoxelCount(dVoxelCounts + primOffsets[i],
                                  resolutionXYZ,
                                  treeGPU.svoAABB,
                                  system);
        i++;
    }
    // Find Global Voxel Offsets
    ExclusiveScanArrayGPU<uint64_t, ReduceAdd<uint64_t>>(dVoxelOffsets,
                                                         dVoxelCounts,
                                                         primOffsets.back() + 1,
                                                         0u);

    // Acquire total voxel count (last element of scan operation)
    uint64_t hTotalVoxCount;
    CUDA_CHECK(hipMemcpy(&hTotalVoxCount, dVoxelOffsets + primOffsets.back(),
                          sizeof(uint64_t), hipMemcpyDeviceToHost));

    // Allocate enough memory for temp voxels (these may overlap)
    Vector2us* dVoxelNormals;
    HitKey* dVoxelLightKeys;

    uint64_t* dVoxels;
    uint32_t* dVoxelIndices;
    DeviceMemory voxelMemory;
    GPUMemFuncs::AllocateMultiData(std::tie(dVoxels, dVoxelIndices),
                                   voxelMemory,
                                   {hTotalVoxCount, hTotalVoxCount + 1});
    DeviceMemory voxelPayloadMemory;
    GPUMemFuncs::AllocateMultiData(std::tie(dVoxelNormals,
                                            dVoxelLightKeys),
                                   voxelPayloadMemory,
                                   {hTotalVoxCount, hTotalVoxCount});

    // Generate Light / HitKey sorted array (for binary search)
    const CudaGPU& gpu = system.BestGPU();
    if(totalLightCount != 0)
    {
        gpu.GridStrideKC_X(0, (hipStream_t)0,
                           totalLightCount,
                           //
                           KCGetLightKeys,
                           //
                           dLightKeys,
                           dSceneLights,
                           totalLightCount);
        // Sort these for binary search
        CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(dLightSortTempMem, lightSortMemSize,
                                                   reinterpret_cast<HitKey::Type*>(dLightKeys),
                                                   reinterpret_cast<HitKey::Type*>(dSortedLightKeys),
                                                   dSceneLights, dSortedLights,
                                                   totalLightCount));

    }

    // Generate Iota for sorting
    IotaGPU(dVoxelIndices, 0u, hTotalVoxCount);

    // For each accelerator
    // Actually rasterize the primitives
    // and push to the memory (find the light key; if available, here)
    i = 0;
    for(const auto [_, accel] : accels)
    {
        accel->VoxelizeSurfaces(// Outputs
                                dVoxels,
                                dVoxelNormals,
                                dVoxelLightKeys,
                                // Inputs
                                dVoxelOffsets + primOffsets[i],
                                // Light Lookup Table (Binary Search)
                                dSortedLightKeys,
                                totalLightCount,
                                // Constants
                                resolutionXYZ,
                                treeGPU.svoAABB,
                                system);
        i++;
    }

    // Temporary Data Structures are not needed from now on
    // Deallocate
    voxOffsetMem = DeviceMemory();
    dVoxelCounts = nullptr;
    dVoxelOffsets = nullptr;
    dPrimOffsets = nullptr;
    dLightSortTempMem = nullptr;

    // Cub operation temporary buffers
    size_t rleTempMemSize;
    size_t sortTempMemSize;
    size_t scanTempMemSize;

    uint64_t* dSortedVoxels = nullptr;
    uint32_t* dSortedVoxelIndices = nullptr;
    // Duplicate counts
    uint32_t* dDuplicateCounts = nullptr;
    uint32_t* dUniqueVoxelCount = nullptr;
    Byte* dTempMemory = nullptr;

    // Acquire Temp Memory Requirements
    CUDA_CHECK(hipcub::DeviceScan::ExclusiveSum(nullptr, scanTempMemSize,
                                             dDuplicateCounts, dDuplicateCounts,
                                             static_cast<uint32_t>(hTotalVoxCount + 1)));
    CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(nullptr, sortTempMemSize,
                                               dVoxels, dSortedVoxels,
                                               dVoxelIndices, dSortedVoxelIndices,
                                               static_cast<uint32_t>(hTotalVoxCount),
                                               0, treeGPU.leafDepth * 3 + 1));
    CUDA_CHECK(hipcub::DeviceRunLengthEncode::Encode(nullptr,
                                                  rleTempMemSize,
                                                  dSortedVoxels, dVoxels,
                                                  dDuplicateCounts, dUniqueVoxelCount,
                                                  static_cast<uint32_t>(hTotalVoxCount)));
    size_t tempMemSize = std::max(rleTempMemSize, sortTempMemSize);
    tempMemSize = std::max(tempMemSize, scanTempMemSize);

    // Allocation
    DeviceMemory sortedVoxelMemory;
    GPUMemFuncs::AllocateMultiData(std::tie(dSortedVoxels, dSortedVoxelIndices,
                                            dDuplicateCounts, dTempMemory,
                                            dUniqueVoxelCount),
                                   sortedVoxelMemory,
                                   {hTotalVoxCount, hTotalVoxCount,
                                    hTotalVoxCount + 1, tempMemSize,
                                    1});

    // Sort and RLE
    CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(dTempMemory, sortTempMemSize,
                                               dVoxels, dSortedVoxels,
                                               dVoxelIndices, dSortedVoxelIndices,
                                               static_cast<uint32_t>(hTotalVoxCount),
                                               0, treeGPU.leafDepth * 3 + 1));
    CUDA_CHECK(hipcub::DeviceRunLengthEncode::Encode(dTempMemory,
                                                  rleTempMemSize,
                                                  dSortedVoxels, dVoxels,
                                                  dDuplicateCounts, dUniqueVoxelCount,
                                                  static_cast<uint32_t>(hTotalVoxCount)));

    // Load the found unique voxel count (non-duplicate) to host memory for kernel calls
    uint32_t hUniqueVoxelCount;
    CUDA_CHECK(hipMemcpy(&hUniqueVoxelCount, dUniqueVoxelCount, sizeof(uint32_t),
                          hipMemcpyDeviceToHost));
    assert(hUniqueVoxelCount <= hTotalVoxCount);

    // Temp reuse the voxel indices array for scan operation
    uint32_t* dVoxelIndexOffsets = reinterpret_cast<uint32_t*>(dVoxelIndices);
    CUDA_CHECK(hipcub::DeviceScan::ExclusiveSum(dTempMemory, scanTempMemSize,
                                             dDuplicateCounts, dVoxelIndexOffsets,
                                             hUniqueVoxelCount + 1));

    // Copy the scanned result back to the duplicate counts variable
    CUDA_CHECK(hipMemcpy(dDuplicateCounts, dVoxelIndexOffsets,
                          sizeof(uint32_t) * (hUniqueVoxelCount + 1),
                          hipMemcpyDeviceToDevice));
    // Rename the allocated buffer to the proper name
    uint32_t* dIndexOffsets = dDuplicateCounts;
    dDuplicateCounts = nullptr;

    // Voxel are sorted and RLE is run
    // Non-unique voxel array is not required copy the unique voxels
    // (which is in dVoxels) to dSortedVoxels array and rename
    CUDA_CHECK(hipMemcpy(dSortedVoxels, dVoxels,
                          sizeof(uint64_t) * hUniqueVoxelCount,
                          hipMemcpyDeviceToDevice));
    // Rename the dVoxels array to sorted unique voxels
    uint64_t* dSortedUniqueVoxels = dSortedVoxels;

    // Now we can deallocate the large non-unique voxel buffers
    voxelMemory = DeviceMemory();
    dVoxels = nullptr;
    dVoxelIndices = nullptr;

    // Now Allocate another temp memory for SVO Construction
    uint32_t* dDiffBitBuffer = nullptr;
    uint32_t* dChildOffsetBuffer = nullptr;
    Byte* dScanMemory = nullptr;
    DeviceMemory svoTempMemory;
    // Check Scan Memory for child reduction on SVO
    CUDA_CHECK(hipcub::DeviceScan::ExclusiveSum(nullptr, scanTempMemSize,
                                             dDiffBitBuffer, dChildOffsetBuffer,
                                             hUniqueVoxelCount + 1));

    // Allocate
    GPUMemFuncs::AllocateMultiData(std::tie(dDiffBitBuffer, dChildOffsetBuffer,
                                            dScanMemory),
                                   svoTempMemory,
                                   {hUniqueVoxelCount, hUniqueVoxelCount + 1,
                                   scanTempMemSize});

    // Top-down find the required voxel counts by looking at morton codes
    assert(Utility::BitCount(resolutionXYZ) == 1);
    uint32_t levelCount = treeGPU.leafDepth;
    std::vector<uint32_t> levelNodeCounts(levelCount + 1, 0);
    // Root node is always available
    levelNodeCounts[0] = 1;
    for(uint32_t i = 1; i <= levelCount; i++)
    {
        // Mark the differences between neighbors
        gpu.GridStrideKC_X(0, (hipStream_t)0, hUniqueVoxelCount - 1,
                           //
                           KCMarkMortonChanges,
                           //
                           dDiffBitBuffer,
                           dSortedUniqueVoxels,
                           hUniqueVoxelCount - 1,
                           i,
                           levelCount);

        // Reduce the marks to find level node count
        ReduceArrayGPU<uint32_t, ReduceAdd<uint32_t>, hipMemcpyDeviceToHost>
        (
            levelNodeCounts[i],
            dDiffBitBuffer,
            hUniqueVoxelCount - 1,
            0u
        );
        gpu.WaitMainStream();
        // n different slices means n+1 segments
        levelNodeCounts[i] += 1;
    }
    assert(levelNodeCounts.back() == hUniqueVoxelCount);

    // Populate node offset buffer
    levelNodeOffsets.resize(levelCount + 2, 0);
    std::inclusive_scan(levelNodeCounts.cbegin(), levelNodeCounts.cend(),
                        levelNodeOffsets.begin() + 1);
    levelNodeOffsets.front() = 0;

    uint32_t totalNodeCount = levelNodeOffsets[levelNodeOffsets.size() - 2];

    treeGPU.nodeCount = totalNodeCount;
    treeGPU.leafCount = hUniqueVoxelCount;
    // Allocate required memories now
    // since we found out the total node count
    GPUMemFuncs::AllocateMultiData(std::tie(// Node Related,
                                            treeGPU.dNodes,
                                            treeGPU.dRadianceRead,
                                            treeGPU.dBinInfo,
                                            // Leaf Related
                                            treeGPU.dLeafParents,
                                            treeGPU.dLeafRadianceRead,
                                            treeGPU.dLeafBinInfo,
                                            treeGPU.dLeafRadianceWrite,
                                            treeGPU.dLeafSampleCountWrite,
                                            // Node Offsets
                                            treeGPU.dLevelNodeOffsets),
                                   octreeMem,
                                   {totalNodeCount, totalNodeCount,
                                    totalNodeCount,
                                    hUniqueVoxelCount, hUniqueVoxelCount,
                                    hUniqueVoxelCount,
                                    hUniqueVoxelCount, hUniqueVoxelCount,
                                    levelNodeOffsets.size()});

    // Set Node and leaf parents to max to early catch errors
    // Rest is set to zero
    gpu.GridStrideKC_X(0, (hipStream_t)0, totalNodeCount,
                       //
                       KCMemset<uint64_t>,
                       //
                       treeGPU.dNodes,
                       AnisoSVOctreeGPU::INVALID_NODE,
                       totalNodeCount);
    CUDA_CHECK(hipMemset(treeGPU.dRadianceRead, 0x00, totalNodeCount * sizeof(AnisoSVOctreeGPU::AnisoRadiance)));
    CUDA_CHECK(hipMemset(treeGPU.dBinInfo, 0x00, totalNodeCount * sizeof(uint64_t)));

    CUDA_CHECK(hipMemset(treeGPU.dLeafParents, 0xFF, hUniqueVoxelCount * sizeof(uint32_t)));
    CUDA_CHECK(hipMemset(treeGPU.dLeafRadianceRead, 0x00, hUniqueVoxelCount * sizeof(AnisoSVOctreeGPU::AnisoRadiance)));
    CUDA_CHECK(hipMemset(treeGPU.dLeafBinInfo, 0x00, hUniqueVoxelCount * sizeof(uint32_t)));
    CUDA_CHECK(hipMemset(treeGPU.dLeafRadianceWrite, 0x00, hUniqueVoxelCount * sizeof(AnisoSVOctreeGPU::AnisoRadianceF)));
    CUDA_CHECK(hipMemset(treeGPU.dLeafSampleCountWrite, 0x00, hUniqueVoxelCount * sizeof(AnisoSVOctreeGPU::AnisoCount)));

    CUDA_CHECK(hipMemcpy(treeGPU.dLevelNodeOffsets, levelNodeOffsets.data(),
                          levelNodeOffsets.size() * sizeof(uint32_t),
                          hipMemcpyHostToDevice));
    // Top down-generate voxels
    // For each level save the node range for
    // efficient kernel calls later (level by level kernel calls)
    // Now start voxel generation level by level
    for(uint32_t i = 0; i < levelCount; i++)
    {
        gpu.GridStrideKC_X(0, (hipStream_t)0, hUniqueVoxelCount,
                           //
                           KCMarkChild,
                           // I-O
                           treeGPU.dNodes,
                           // Input
                           dSortedUniqueVoxels,
                           // Constants
                           hUniqueVoxelCount,
                           i,
                           levelCount);

        gpu.GridStrideKC_X(0, (hipStream_t)0, levelNodeCounts[i],
                           //
                           KCExtractChildrenCounts,
                           //
                           dDiffBitBuffer,
                           treeGPU.dNodes + levelNodeOffsets[i],
                           levelNodeCounts[i]);

        CUDA_CHECK(hipcub::DeviceScan::ExclusiveSum(dScanMemory, scanTempMemSize,
                                                 dDiffBitBuffer, dChildOffsetBuffer,
                                                 levelNodeCounts[i] + 1));

        // Check
        uint32_t hReducedSum;
        CUDA_CHECK(hipMemcpy(&hReducedSum, dChildOffsetBuffer + levelNodeCounts[i],
                              sizeof(uint32_t), hipMemcpyDeviceToHost));
        if(hReducedSum != levelNodeCounts[i + 1])
        {
            METU_ERROR_LOG("SVO children count allocation mismatch (Level {:d}.", i);
            return TracerError::TRACER_INTERNAL_ERROR;
        }

        bool lastNonLeafLevel = (i == (levelCount - 1));
        uint32_t nextLevelOffset = (lastNonLeafLevel) ? 0 : levelNodeOffsets[i + 1];
        gpu.GridStrideKC_X(0, (hipStream_t)0, levelNodeCounts[i],
                           //
                           KCSetChildrenPtrs,
                           //
                           treeGPU.dNodes + levelNodeOffsets[i],
                           dChildOffsetBuffer,
                           nextLevelOffset,
                           levelNodeCounts[i],
                           lastNonLeafLevel);

        if(!lastNonLeafLevel)
        {
            gpu.GridStrideKC_X(0, (hipStream_t)0, levelNodeCounts[i],
                               //
                               KCSetParentOfChildren,
                               //
                               treeGPU.dNodes,
                               treeGPU.dNodes + levelNodeOffsets[i],
                               levelNodeCounts[i]);
        }
        else
        {
            gpu.GridStrideKC_X(0, (hipStream_t)0, levelNodeCounts[i],
                               //
                               KCSetParentOfLeafChildren,
                               //
                               treeGPU.dLeafParents,
                               treeGPU.dNodes,
                               treeGPU.dNodes + levelNodeOffsets[i],
                               levelNodeCounts[i]);
        }
    }
    // Only Direct light information deposition is left
    // Call the kernel for it
    gpu.GridStrideKC_X(0, (hipStream_t)0, hTotalVoxCount,
                       //
                       KCReduceVoxelPayload,
                       // I-O
                       treeGPU,
                       // Input
                       dIndexOffsets,
                       dSortedUniqueVoxels,
                       dSortedVoxelIndices, // This is non-unique (we need to reduce it)
                       dVoxelLightKeys, // Voxel payload that will be reduced
                       dVoxelNormals,   // Voxel payload that will be reduced

                       // Binary Search for light
                       dSortedLightKeys,
                       dSortedLights,
                       totalLightCount,
                       // Constants
                       hUniqueVoxelCount,
                       static_cast<uint32_t>(hTotalVoxCount),
                       treeGPU.svoAABB,
                       resolutionXYZ);
    // Log some stuff
    timer.Stop();
    double svoMemSize = static_cast<double>(octreeMem.Size()) / 1024.0 / 1024.0;
    double radMemSize = static_cast<double>(totalNodeCount * sizeof(AnisoSVOctreeGPU::AnisoRadiance) +
                                            hUniqueVoxelCount * sizeof(AnisoSVOctreeGPU::AnisoRadiance) +
                                            hUniqueVoxelCount * sizeof(AnisoSVOctreeGPU::AnisoRadianceF) +
                                            hUniqueVoxelCount * sizeof(AnisoSVOctreeGPU::AnisoCount)) / 1024.0 / 1024.0;
    double irradMemSize = static_cast<double>(totalNodeCount * sizeof(half) +
                                              hUniqueVoxelCount * sizeof(uint32_t) +
                                              hUniqueVoxelCount * sizeof(float) +
                                              hUniqueVoxelCount * sizeof(half)) / 1024.0 / 1024.0;

    // New SVO size
    double newStyleSize = static_cast<double>(VoxelPayload::TotalSize(totalNodeCount, hUniqueVoxelCount) +
                                              // Pointer Hierarchy
                                              sizeof(uint64_t) * totalNodeCount +
                                              // Leaf parent pointers
                                              sizeof(uint32_t) * hUniqueVoxelCount +
                                              // Bin info
                                              sizeof(uint32_t) * totalNodeCount +
                                              sizeof(uint32_t) * hUniqueVoxelCount) / 1024.0 / 1024.0;

    METU_LOG("Scene Aniso-SVO [N: {:L}, L: {:L}] Generated in {:f} seconds. (Total {:.2f} MiB, Rad Cache {:.2f} MiB, If Irrad {:.2f} MiB), New {:.2f} MiB",
             treeGPU.nodeCount, treeGPU.leafCount,
             timer.Elapsed<CPUTimeSeconds>(),
             svoMemSize, radMemSize, irradMemSize,
             newStyleSize);

    // All Done!
    return TracerError::OK;
}

void AnisoSVOctreeCPU::NormalizeAndFilterRadiance(const CudaSystem& system)
{
    // From leaf (leaf-write) to root
    // Average the radiance
    // Down-sample the radiance for lowest n levels as well maybe? (n= 2 or 3)


    // TODO: Do some proper filtering
    // Just copy it to for now
    // Assume that the ray counts are set for leaves
    const CudaGPU& bestGPU = system.BestGPU();
    // Leaf has different memory layout do it separately
    bestGPU.GridStrideKC_X(0, (hipStream_t)0, treeGPU.leafCount,
                           //
                           KCCCopyRadianceToHalfBufferLeaf,
                           // I-O
                           treeGPU.dLeafRadianceRead,
                           // Input
                           treeGPU.dLeafRadianceWrite,
                           treeGPU.dLeafSampleCountWrite,
                           // Constants
                           treeGPU.leafCount,
                           1.0f);
}

void AnisoSVOctreeCPU::CollapseRayCounts(uint32_t minLevel, uint32_t minRayCount,
                                         const CudaSystem& system)
{
    // Assume that the ray counts are set for leaves
    const CudaGPU& bestGPU = system.BestGPU();

    // Leaf has different memory layout do it separately
    bestGPU.GridStrideKC_X(0, (hipStream_t)0, treeGPU.leafCount,
                           //
                           KCCollapseRayCountsLeaf,
                           // I-O
                           treeGPU.dLeafBinInfo,
                           treeGPU.dBinInfo,
                           // Input
                           treeGPU.dLeafParents,
                           // Constants
                           treeGPU.leafCount,
                           treeGPU.leafDepth,
                           minLevel,
                           minRayCount);

    //Debug::DumpMemToFile(std::to_string(treeGPU.leafDepth) + std::string("_binInfo"),
    //                     treeGPU.dLeafBinInfo,
    //                     treeGPU.leafCount, false, true);

    // Bottom-up process bins
    int32_t bottomNodeLevel = static_cast<int32_t>(treeGPU.leafDepth - 1);
    for(int32_t i = bottomNodeLevel; i >= static_cast<int32_t>(minLevel); i--)
    {
        Vector2ui range(levelNodeOffsets[i],
                        levelNodeOffsets[i + 1]);
        uint32_t nodeCount = range[1] - range[0];

        bestGPU.GridStrideKC_X(0, (hipStream_t)0, nodeCount,
                               //
                               KCCollapseRayCounts,
                               // I-O
                               treeGPU.dBinInfo,
                               // Input
                               treeGPU.dNodes,
                               // Constants
                               range,
                               i,
                               minLevel,
                               minRayCount);
    }
    // Leaf->Parent chain now there is at least a single mark
    // Rays will re-check and find their marked bin and set their id accordingly
}


void AnisoSVOctreeCPU::AccumulateRaidances(const PathGuidingNode* dPGNodes,
                                           uint32_t totalNodeCount,
                                           uint32_t maxPathNodePerRay,
                                           const CudaSystem& system)
{
    // Directly call the appropriate kernel
    const CudaGPU& bestGPU = system.BestGPU();
    bestGPU.GridStrideKC_X(0, (hipStream_t)0, totalNodeCount,
                           //
                           KCAccumulateRadianceToLeaf,
                           //
                           treeGPU,
                           dPGNodes,
                           totalNodeCount,
                           maxPathNodePerRay);
    bestGPU.WaitMainStream();
}

void AnisoSVOctreeCPU::ClearRayCounts(const CudaSystem&)
{
    CUDA_CHECK(hipMemset(treeGPU.dLeafBinInfo, 0x00, sizeof(uint32_t) * treeGPU.leafCount));
    CUDA_CHECK(hipMemset(treeGPU.dBinInfo, 0x00, sizeof(uint32_t) * treeGPU.nodeCount));
}

void AnisoSVOctreeCPU::DumpSVOAsBinary(std::vector<Byte>& data,
                                       const CudaSystem& system) const
{
    using AnisoRadianceF = AnisoSVOctreeGPU::AnisoRadianceF;
    using AnisoRadiance = AnisoSVOctreeGPU::AnisoRadiance;

    // Temp Float Buffer for Conversion
    assert(treeGPU.leafCount >= treeGPU.nodeCount);
    DeviceMemory halfConvertedMemory(treeGPU.leafCount * sizeof(AnisoRadianceF));

    // Conversion Function
    auto ConvertAnisoHalfToFloat = [&](const AnisoRadiance* dRadiance,
                                       uint32_t totalSize)
    {
        const CudaGPU& gpu = system.BestGPU();
        gpu.GridStrideKC_X(0, (hipStream_t)0, totalSize,
                           //
                           KCConvertToAnisoFloat,
                           //
                           static_cast<AnisoRadianceF*>(halfConvertedMemory),
                           dRadiance,
                           totalSize);
    };

    // Get Sizes
    std::array<size_t, 4> byteSizes;
    byteSizes[0]  = treeGPU.nodeCount * sizeof(uint64_t);       // dNodesSize
    byteSizes[1]  = treeGPU.nodeCount * sizeof(AnisoRadianceF); // dRadianceReadSize
    // Leaf Related
    byteSizes[2] = treeGPU.leafCount * sizeof(uint32_t);        // dLeafParentSize
    byteSizes[3] = treeGPU.leafCount * sizeof(AnisoRadianceF);  // dLeafRadianceReadSize
    // Calculate the offsets and total size
    size_t bufferTotalSize = std::reduce(byteSizes.cbegin(), byteSizes.cend(), 0ull);

    size_t totalSize = (bufferTotalSize + sizeof(AABB3f) +
                        5 * sizeof(uint32_t) +
                        sizeof(float));

    data.resize(totalSize);
    // Memcpy the data from the memory
    size_t offset = 0;
    std::memcpy(data.data() + offset, &treeGPU.svoAABB, sizeof(AABB3f));
    offset += sizeof(AABB3f);
    std::memcpy(data.data() + offset, &treeGPU.voxelResolution, sizeof(uint32_t));
    offset += sizeof(uint32_t);
    std::memcpy(data.data() + offset, &treeGPU.leafDepth, sizeof(uint32_t));
    offset += sizeof(uint32_t);
    std::memcpy(data.data() + offset, &treeGPU.nodeCount, sizeof(uint32_t));
    offset += sizeof(uint32_t);
    std::memcpy(data.data() + offset, &treeGPU.leafCount, sizeof(uint32_t));
    offset += sizeof(uint32_t);
    std::memcpy(data.data() + offset, &treeGPU.leafVoxelSize, sizeof(float));
    offset += sizeof(float);
    std::memcpy(data.data() + offset, &treeGPU.levelOffsetCount, sizeof(uint32_t));
    offset += sizeof(uint32_t);

    // Nodes
    CUDA_CHECK(hipMemcpy(data.data() + offset, treeGPU.dNodes,
                          byteSizes[0], hipMemcpyDeviceToHost));
    offset += byteSizes[0];
    // Radiance Cache Node
    ConvertAnisoHalfToFloat(treeGPU.dRadianceRead, treeGPU.nodeCount);
    CUDA_CHECK(hipMemcpy(data.data() + offset,
                          static_cast<void*>(halfConvertedMemory),
                          byteSizes[1], hipMemcpyDeviceToHost));
    offset += byteSizes[1];
    // Leaf Parents
    CUDA_CHECK(hipMemcpy(data.data() + offset, treeGPU.dLeafParents,
                          byteSizes[2], hipMemcpyDeviceToHost));
    offset += byteSizes[2];
    // Radiance Cache Leaf
    ConvertAnisoHalfToFloat(treeGPU.dLeafRadianceRead, treeGPU.leafCount);
    CUDA_CHECK(hipMemcpy(data.data() + offset,
                          static_cast<void*>(halfConvertedMemory),
                          byteSizes[3], hipMemcpyDeviceToHost));
    offset += byteSizes[3];
    assert(offset == data.size());

    // All Done!
}