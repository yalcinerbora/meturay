#include "hip/hip_runtime.h"
#include "AnisoSVO.cuh"
#include "AnisoSVOKC.cuh"

#include "CudaSystem.h"
#include "CudaSystem.hpp"
#include "PathNode.cuh"

#include "RayLib/ColorConversion.h"
#include "RayLib/HitStructs.h"
#include "RayLib/BitManipulation.h"
#include "RayLib/CPUTimer.h"

#include "GPUAcceleratorI.h"
#include "ParallelReduction.cuh"
#include "ParallelScan.cuh"
#include "ParallelMemset.cuh"
#include "ParallelSequence.cuh"
#include "BinarySearch.cuh"

#include <hipcub/hipcub.hpp>
#include <numeric>

#include "TracerDebug.h"

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCGetLightKeys(HitKey* gKeys,
                    const GPULightI** gLights,
                    uint32_t totalLightCount)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < totalLightCount;
        threadId += (blockDim.x * gridDim.x))
    {
        gKeys[threadId] = gLights[threadId]->WorkKey();
    }
}


__global__ CUDA_LAUNCH_BOUNDS_1D
void KCMarkMortonChanges(uint32_t* gMarks,
                         const uint64_t* gVoxels,
                         uint32_t voxelCount,
                         uint32_t level,
                         uint32_t maxLevel)
{
    static constexpr uint32_t DIMENSION = 3;
    const uint32_t voxelMSBStart = sizeof(uint64_t) * BYTE_BITS - (maxLevel * DIMENSION);

    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < voxelCount;
        threadId += (blockDim.x * gridDim.x))
    {
        uint64_t voxMorton = gVoxels[threadId];
        assert((voxMorton & (~((1ull << (maxLevel * DIMENSION)) - 1))) == 0);
        uint64_t voxMortonRev = __brevll(voxMorton) >> voxelMSBStart;
        voxMortonRev &= (1ull << level * DIMENSION) - 1;

        uint64_t voxMortonNext = gVoxels[threadId + 1];
        assert((voxMortonNext & (~((1ull << (maxLevel * DIMENSION)) - 1))) == 0);
        uint64_t voxMortonNextRev = __brevll(voxMortonNext) >> voxelMSBStart;
        voxMortonNextRev &= (1ull << level * DIMENSION) - 1;

        gMarks[threadId] = (voxMortonRev != voxMortonNextRev) ? 1 : 0;
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCMarkChild(// I-O
                 uint64_t* gNodes,
                 // Input
                 const uint64_t* gVoxels,
                 // Constants
                 uint32_t voxelCount,
                 uint32_t level,
                 uint32_t maxLevel)
{
    // Useful constants
    static constexpr uint32_t DIMENSION = 3;
    static constexpr uint32_t DIM_MASK = (1 << DIMENSION) - 1;

    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < voxelCount;
        threadId += (blockDim.x * gridDim.x))
    {
        uint64_t voxelMortonCode = gVoxels[threadId];
        uint32_t mortonLevelShift = (maxLevel - 1) * DIMENSION;
        // Start traversing (Root node is the very first node)
        uint32_t currentNodeIndex = 0;
        for(int i = 0; i < level; i++)
        {
            uint64_t currentNode = gNodes[currentNodeIndex];
            // Fetch the current bit triples of the level from the
            // morton code
            uint32_t childId = (voxelMortonCode >> mortonLevelShift) & DIM_MASK;
            uint32_t childOffset = AnisoSVOctreeGPU::FindChildOffset(currentNode, childId);
            uint32_t childrenIndex = AnisoSVOctreeGPU::ChildrenIndex(currentNode);
            // Go to next child
            currentNodeIndex = childrenIndex + childOffset;
            // Strip the processed bits
            mortonLevelShift -= DIMENSION;
        }
        // Now we are at the not that does not set its children ptr and mask is set
        // Atomically mark the required child
        uint32_t childId = (voxelMortonCode >> mortonLevelShift) & DIM_MASK;
        uint32_t childBit = (1 << childId);
        assert(childId < 8);
        assert(__popc(childBit) == 1);
        // Atomically set the child bit on the packed node
        AnisoSVOctreeGPU::AtomicSetChildMaskBit(gNodes + currentNodeIndex,
                                                childBit);
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCExtractChildrenCounts(uint32_t* gChildrenCounts,
                             const uint64_t* gLevelNodes,
                             uint32_t levelNodeCount)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < levelNodeCount;
        threadId += (blockDim.x * gridDim.x))
    {
        uint64_t node = gLevelNodes[threadId];
        uint32_t childrenCount = AnisoSVOctreeGPU::ChildrenCount(node);
        // Write the count
        gChildrenCounts[threadId] = childrenCount;
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCSetChildrenPtrs(uint64_t* gLevelNodes,
                       const uint32_t* gChildrenOffsets,
                       uint32_t nextLevelStartIndex,
                       uint32_t levelNodeCount,
                       bool markIsChildrenLeaf)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < levelNodeCount;
        threadId += (blockDim.x * gridDim.x))
    {
        uint64_t node = gLevelNodes[threadId];
        uint32_t offset = gChildrenOffsets[threadId];

        // Children offsets are relative to the level
        // we need to put global pointer (index)
        uint32_t globalOffset = nextLevelStartIndex + offset;
        AnisoSVOctreeGPU::SetChildrenIndex(node, globalOffset);
        // If this the last non-leaf level we need to mark the children
        if(markIsChildrenLeaf)
            AnisoSVOctreeGPU::SetIsChildrenLeaf(node, true);
        // Write back the modified node
        gLevelNodes[threadId] = node;
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCSetParentOfChildren(uint64_t* gNodes,
                           const uint64_t* gLevelNodes,
                           uint32_t levelNodeCount)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < levelNodeCount;
        threadId += (blockDim.x * gridDim.x))
    {
        uint64_t node = gLevelNodes[threadId];
        uint32_t childrenCount = AnisoSVOctreeGPU::ChildrenCount(node);
        uint32_t childrenIndex = AnisoSVOctreeGPU::ChildrenIndex(node);

        // Find the parent id using pointer arithmetic
        uint32_t currentNodeGlobalId  = (gLevelNodes + threadId) - gNodes;
        // Set ptrs for all children
        for(uint32_t i = 0; i < childrenCount; i++)
        {
            uint64_t* gChildNode = gNodes + childrenIndex + i;
            AnisoSVOctreeGPU::SetParentIndex(*gChildNode, currentNodeGlobalId);
        }
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCSetParentOfLeafChildren(uint32_t* gLeafParents,
                               const uint64_t* gNodes,
                               const uint64_t* gLevelNodes,
                               uint32_t levelNodeCount)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < levelNodeCount;
        threadId += (blockDim.x * gridDim.x))
    {
        uint64_t node = gLevelNodes[threadId];
        uint32_t childrenCount = AnisoSVOctreeGPU::ChildrenCount(node);
        uint32_t childrenIndex = AnisoSVOctreeGPU::ChildrenIndex(node);
        assert(AnisoSVOctreeGPU::IsChildrenLeaf(node));
        // Find the parent id using pointer arithmetic
        uint32_t currentNodeGlobalId = (gLevelNodes + threadId) - gNodes;
        // Set ptrs for all children
        for(uint32_t i = 0; i < childrenCount; i++)
        {
            uint32_t* gChildParent = gLeafParents + childrenIndex + i;
            *gChildParent = currentNodeGlobalId;
        }
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCAccumulateRadianceToLeaf(AnisoSVOctreeGPU svo,
                                // Input
                                const WFPGPathNode* gPathNodes,
                                uint32_t nodeCount,
                                uint32_t maxPathNodePerRay)
{
    bool unableToAccum = false;

    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < nodeCount;
        threadId += (blockDim.x * gridDim.x))
    {
        const uint32_t nodeIndex = threadId;
        const uint32_t pathStartIndex = nodeIndex / maxPathNodePerRay * maxPathNodePerRay;

        WFPGPathNode gPathNode = gPathNodes[nodeIndex];

        // Skip if this node cannot calculate wo
        if(!gPathNode.HasPrev()) continue;

        Vector3f wo = gPathNode.Wo<WFPGPathNode>(gPathNodes, pathStartIndex);
        Vector3f wi = gPathNode.Wi<WFPGPathNode>(gPathNodes, pathStartIndex);

        float luminance = Utility::RGBToLuminance(gPathNode.totalRadiance);
        unableToAccum |= !svo.DepositRadiance(gPathNode.worldPosition,
                                              gPathNode.Normal(),
                                              wi, wo, luminance);
    }
    // Debug
    if(unableToAccum)
    {
        printf("Unable to accumulate some radiance values!\n");
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCCollapseRayCounts(// I-O
                         uint16_t* gBinInfo,
                         // Input
                         const uint64_t* gNodes,
                         // Constants
                         Vector2ui levelRange,
                         uint32_t level,
                         uint32_t minLevel,
                         uint32_t minRayCount)
{
    uint32_t nodeCount = levelRange[1] - levelRange[0];

    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < nodeCount;
        threadId += (blockDim.x * gridDim.x))
    {
        uint32_t nodeId = levelRange[0] + threadId;

        // We are at the user option limit,
        // Directly mark this node then leave
        if(level == minLevel)
        {
            AnisoSVOctreeGPU::SetBinAsMarked(gBinInfo[nodeId]);
            continue;
        }
        // Fetch Ray Count
        uint32_t rayCount = AnisoSVOctreeGPU::GetRayCount(gBinInfo[nodeId]);
        // If ray count is not enough on this voxel
        // collaborate with the other children
        if(rayCount < minRayCount)
        {
            uint32_t parent = AnisoSVOctreeGPU::ParentIndex(gNodes[nodeId]);
            AnisoSVOctreeGPU::AtomicAddUInt16(gBinInfo + parent, rayCount);
        }
        // We have enough rays in this node use it as is
        else
        {
            AnisoSVOctreeGPU::SetBinAsMarked(gBinInfo[nodeId]);
        }
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCCollapseRayCountsLeaf(// I-O
                             uint16_t* gLeafBinInfo,
                             uint16_t* gBinInfo,
                             // Input
                             const uint32_t* gLeafParents,
                             // Constants
                             uint32_t leafCount,
                             uint32_t level,
                             uint32_t minLevel,
                             uint32_t minRayCount)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < leafCount;
        threadId += (blockDim.x * gridDim.x))
    {
        // We are at the user option limit,
        // Directly mark this node then leave
        if(level == minLevel)
        {
            AnisoSVOctreeGPU::SetBinAsMarked(gLeafBinInfo[threadId]);
            continue;
        }
        // Fetch Ray Count
        uint16_t rayCount = AnisoSVOctreeGPU::GetRayCount(gLeafBinInfo[threadId]);
        if(rayCount == 0) continue;

        // If ray count is not enough on this voxel
        // collaborate with the other children
        if(rayCount < minRayCount)
        {
            uint32_t parent = gLeafParents[threadId];
            AnisoSVOctreeGPU::AtomicAddUInt16(gBinInfo + parent, rayCount);
        }
        // We have enough rays in this node use it as is
        else
        {
            AnisoSVOctreeGPU::SetBinAsMarked(gLeafBinInfo[threadId]);
        }
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCCCopyRadianceToHalfBufferLeaf(// I-O
                                     Vector2h* dLeafRadianceRead,
                                     // Input
                                     const Vector2f* dLeafRadianceWrite,
                                     const Vector2ui* dLeafSampleCountWrite,
                                     // Constants
                                     uint32_t leafCount,
                                     float totalRadianceScene)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < leafCount;
        threadId += (blockDim.x * gridDim.x))
    {
        // Read from the value;
        Vector2ui count = dLeafSampleCountWrite[threadId];
        Vector2f irradiance = dLeafRadianceWrite[threadId];
        Vector2f avgRadiance = Vector2f(irradiance[0] / static_cast<float>(count[0]),
                                        irradiance[1] / static_cast<float>(count[1]));
        // Avoid NaN if not accumulation occured
        avgRadiance[0] = (count[0] == 0) ? 0.0f : avgRadiance[0];
        avgRadiance[1] = (count[1] == 0) ? 0.0f : avgRadiance[1];
        // Normalize & Clamp the half range for now
        Vector2f irradClampled = Vector2f::Min(avgRadiance, Vector2f(MRAY_HALF_MAX));

        Vector2h irradHalf = Vector2h(irradClampled);
        dLeafRadianceRead[threadId] = irradHalf;
    }
}

__global__ CUDA_LAUNCH_BOUNDS_1D
void KCConvertToAnisoFloat(Vector2f* gAnisoOut,
                           const Vector2h* gAnisoIn,
                           uint32_t anisoCount)
{
    for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
        threadId < anisoCount;
        threadId += (blockDim.x * gridDim.x))
    {
        Vector2f& gOut = gAnisoOut[threadId];
        Vector2h in = gAnisoIn[threadId];


        gOut[0] = in[0];
        gOut[1] = in[1];
    }
}

TracerError AnisoSVOctreeCPU::Constrcut(const AABB3f& sceneAABB, uint32_t resolutionXYZ,
                                        const AcceleratorBatchMap& accels,
                                        const GPULightI** dSceneLights,
                                        uint32_t totalLightCount,
                                        HitKey boundaryLightKey,
                                        const CudaSystem& system)
{
    Utility::CPUTimer timer;
    timer.Start();

    // Find The SVO AABB
    Vector3f span = sceneAABB.Span();
    int maxDimIndex = span.Max();
    float worldSizeXYZ = span[maxDimIndex];
    float sizePadding = (worldSizeXYZ / static_cast<float>(resolutionXYZ));
    treeGPU.svoAABB = AABB3f(sceneAABB.Min() - Vector3f(sizePadding),
                             sceneAABB.Min() + Vector3f(sizePadding + worldSizeXYZ));
    treeGPU.leafDepth = Utility::FindLastSet(resolutionXYZ);
    treeGPU.leafVoxelSize = (worldSizeXYZ + 2.0f * sizePadding) / static_cast<float>(resolutionXYZ);
    treeGPU.voxelResolution = resolutionXYZ;

    // Find out the sort memory requirement of Light Keys
    size_t lightSortMemSize;
    HitKey* dLightKeys = nullptr;
    const GPULightI** dSortedLights = nullptr;
    HitKey* dSortedLightKeys = nullptr;
    CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(nullptr, lightSortMemSize,
                                               reinterpret_cast<HitKey::Type*>(dLightKeys),
                                               reinterpret_cast<HitKey::Type*>(dSortedLightKeys),
                                               dSceneLights, dSortedLights,
                                               totalLightCount));

    DeviceMemory lightMemory;
    GPUMemFuncs::AllocateMultiData(std::tie(dLightKeys, dSortedLights,
                                            dSortedLightKeys),
                                   lightMemory,
                                   {totalLightCount, totalLightCount,
                                    totalLightCount});

    // For each accelerator
    // First allocate per prim voxel count array
    std::vector<size_t> primOffsets;
    primOffsets.reserve(accels.size() + 1);
    primOffsets.push_back(0);
    for(const auto [_, accel] : accels)
    {
        primOffsets.push_back(accel->TotalPrimitiveCount());
    }

    std::inclusive_scan(primOffsets.cbegin(), primOffsets.cend(),
                        primOffsets.begin());

    // Allocate Voxel Count memory
    uint64_t* dVoxelCounts;
    uint64_t* dVoxelOffsets;
    uint64_t* dPrimOffsets;
    Byte* dLightSortTempMem;
    DeviceMemory voxOffsetMem;
    GPUMemFuncs::AllocateMultiData(std::tie(dVoxelCounts,
                                            dVoxelOffsets,
                                            dPrimOffsets,
                                            dLightSortTempMem),
                                   voxOffsetMem,
                                   {primOffsets.back(),
                                    primOffsets.back() + 1,
                                    accels.size() + 1,
                                    lightSortMemSize});

    // Copy prim offsets for segmented reduction
    CUDA_CHECK(hipMemcpy(dPrimOffsets, primOffsets.data(),
                          sizeof(uint64_t) * accels.size(),
                          hipMemcpyHostToDevice));

    // Ask each primitive for rasterized voxel count
    uint32_t i = 0;
    for(const auto [_, accel] : accels)
    {
        accel->EachPrimVoxelCount(dVoxelCounts + primOffsets[i],
                                  resolutionXYZ,
                                  treeGPU.svoAABB,
                                  system);
        i++;
    }
    // Find Global Voxel Offsets
    ExclusiveScanArrayGPU<uint64_t, ReduceAdd<uint64_t>>(dVoxelOffsets,
                                                         dVoxelCounts,
                                                         primOffsets.back() + 1,
                                                         0u);

    // Acquire total voxel count (last element of scan operation)
    uint64_t hTotalVoxCount;
    CUDA_CHECK(hipMemcpy(&hTotalVoxCount, dVoxelOffsets + primOffsets.back(),
                          sizeof(uint64_t), hipMemcpyDeviceToHost));

    // Allocate enough memory for temp voxels (these may overlap)
    Vector2us* dVoxelNormals;
    HitKey* dVoxelLightKeys;

    uint64_t* dVoxels;
    uint32_t* dVoxelIndices;
    DeviceMemory voxelMemory;
    GPUMemFuncs::AllocateMultiData(std::tie(dVoxels, dVoxelIndices),
                                   voxelMemory,
                                   {hTotalVoxCount, hTotalVoxCount + 1});
    DeviceMemory voxelPayloadMemory;
    GPUMemFuncs::AllocateMultiData(std::tie(dVoxelNormals,
                                            dVoxelLightKeys),
                                   voxelPayloadMemory,
                                   {hTotalVoxCount, hTotalVoxCount});

    // Generate Light / HitKey sorted array (for binary search)
    const CudaGPU& gpu = system.BestGPU();
    if(totalLightCount != 0)
    {
        gpu.GridStrideKC_X(0, (hipStream_t)0,
                           totalLightCount,
                           //
                           KCGetLightKeys,
                           //
                           dLightKeys,
                           dSceneLights,
                           totalLightCount);
        // Sort these for binary search
        CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(dLightSortTempMem, lightSortMemSize,
                                                   reinterpret_cast<HitKey::Type*>(dLightKeys),
                                                   reinterpret_cast<HitKey::Type*>(dSortedLightKeys),
                                                   dSceneLights, dSortedLights,
                                                   totalLightCount));

    }

    // Generate Iota for sorting
    IotaGPU(dVoxelIndices, 0u, hTotalVoxCount);

    // For each accelerator
    // Actually rasterize the primitives
    // and push to the memory (find the light key; if available, here)
    i = 0;
    for(const auto [_, accel] : accels)
    {
        accel->VoxelizeSurfaces(// Outputs
                                dVoxels,
                                dVoxelNormals,
                                dVoxelLightKeys,
                                // Inputs
                                dVoxelOffsets + primOffsets[i],
                                // Light Lookup Table (Binary Search)
                                dSortedLightKeys,
                                totalLightCount,
                                // Constants
                                resolutionXYZ,
                                treeGPU.svoAABB,
                                system);
        i++;
    }

    // Temporary Data Structures are not needed from now on
    // Deallocate
    voxOffsetMem = DeviceMemory();
    dVoxelCounts = nullptr;
    dVoxelOffsets = nullptr;
    dPrimOffsets = nullptr;
    dLightSortTempMem = nullptr;

    // Cub operation temporary buffers
    size_t rleTempMemSize;
    size_t sortTempMemSize;
    size_t scanTempMemSize;

    uint64_t* dSortedVoxels = nullptr;
    uint32_t* dSortedVoxelIndices = nullptr;
    // Duplicate counts
    uint32_t* dDuplicateCounts = nullptr;
    uint32_t* dUniqueVoxelCount = nullptr;
    Byte* dTempMemory = nullptr;

    // Acquire Temp Memory Requirements
    CUDA_CHECK(hipcub::DeviceScan::ExclusiveSum(nullptr, scanTempMemSize,
                                             dDuplicateCounts, dDuplicateCounts,
                                             static_cast<uint32_t>(hTotalVoxCount + 1)));
    CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(nullptr, sortTempMemSize,
                                               dVoxels, dSortedVoxels,
                                               dVoxelIndices, dSortedVoxelIndices,
                                               static_cast<uint32_t>(hTotalVoxCount),
                                               0, treeGPU.leafDepth * 3 + 1));
    CUDA_CHECK(hipcub::DeviceRunLengthEncode::Encode(nullptr,
                                                  rleTempMemSize,
                                                  dSortedVoxels, dVoxels,
                                                  dDuplicateCounts, dUniqueVoxelCount,
                                                  static_cast<uint32_t>(hTotalVoxCount)));
    size_t tempMemSize = std::max(rleTempMemSize, sortTempMemSize);
    tempMemSize = std::max(tempMemSize, scanTempMemSize);

    // Allocation
    DeviceMemory sortedVoxelMemory;
    GPUMemFuncs::AllocateMultiData(std::tie(dSortedVoxels, dSortedVoxelIndices,
                                            dDuplicateCounts, dTempMemory,
                                            dUniqueVoxelCount),
                                   sortedVoxelMemory,
                                   {hTotalVoxCount, hTotalVoxCount,
                                    hTotalVoxCount + 1, tempMemSize,
                                    1});

    // Sort and RLE
    CUDA_CHECK(hipcub::DeviceRadixSort::SortPairs(dTempMemory, sortTempMemSize,
                                               dVoxels, dSortedVoxels,
                                               dVoxelIndices, dSortedVoxelIndices,
                                               static_cast<uint32_t>(hTotalVoxCount),
                                               0, treeGPU.leafDepth * 3 + 1));
    CUDA_CHECK(hipcub::DeviceRunLengthEncode::Encode(dTempMemory,
                                                  rleTempMemSize,
                                                  dSortedVoxels, dVoxels,
                                                  dDuplicateCounts, dUniqueVoxelCount,
                                                  static_cast<uint32_t>(hTotalVoxCount)));

    // Load the found unique voxel count (non-duplicate) to host memory for kernel calls
    uint32_t hUniqueVoxelCount;
    CUDA_CHECK(hipMemcpy(&hUniqueVoxelCount, dUniqueVoxelCount, sizeof(uint32_t),
                          hipMemcpyDeviceToHost));
    assert(hUniqueVoxelCount <= hTotalVoxCount);

    // Temp reuse the voxel indices array for scan operation
    uint32_t* dVoxelIndexOffsets = reinterpret_cast<uint32_t*>(dVoxelIndices);
    CUDA_CHECK(hipcub::DeviceScan::ExclusiveSum(dTempMemory, scanTempMemSize,
                                             dDuplicateCounts, dVoxelIndexOffsets,
                                             hUniqueVoxelCount + 1));

    // Copy the scanned result back to the duplicate counts variable
    CUDA_CHECK(hipMemcpy(dDuplicateCounts, dVoxelIndexOffsets,
                          sizeof(uint32_t) * (hUniqueVoxelCount + 1),
                          hipMemcpyDeviceToDevice));
    // Rename the allocated buffer to the proper name
    uint32_t* dIndexOffsets = dDuplicateCounts;
    dDuplicateCounts = nullptr;

    // Voxel are sorted and RLE is run
    // Non-unique voxel array is not required copy the unique voxels
    // (which is in dVoxels) to dSortedVoxels array and rename
    CUDA_CHECK(hipMemcpy(dSortedVoxels, dVoxels,
                          sizeof(uint64_t) * hUniqueVoxelCount,
                          hipMemcpyDeviceToDevice));
    // Rename the dVoxels array to sorted unique voxels
    uint64_t* dSortedUniqueVoxels = dSortedVoxels;

    // Now we can deallocate the large non-unique voxel buffers
    voxelMemory = DeviceMemory();
    dVoxels = nullptr;
    dVoxelIndices = nullptr;

    // Now Allocate another temp memory for SVO Construction
    uint32_t* dDiffBitBuffer = nullptr;
    uint32_t* dChildOffsetBuffer = nullptr;
    Byte* dScanMemory = nullptr;
    DeviceMemory svoTempMemory;
    // Check Scan Memory for child reduction on SVO
    CUDA_CHECK(hipcub::DeviceScan::ExclusiveSum(nullptr, scanTempMemSize,
                                             dDiffBitBuffer, dChildOffsetBuffer,
                                             hUniqueVoxelCount + 1));

    // Allocate
    GPUMemFuncs::AllocateMultiData(std::tie(dDiffBitBuffer, dChildOffsetBuffer,
                                            dScanMemory),
                                   svoTempMemory,
                                   {hUniqueVoxelCount, hUniqueVoxelCount + 1,
                                   scanTempMemSize});

    // Top-down find the required voxel counts by looking at morton codes
    assert(Utility::BitCount(resolutionXYZ) == 1);
    uint32_t levelCount = treeGPU.leafDepth;
    std::vector<uint32_t> levelNodeCounts(levelCount + 1, 0);
    // Root node is always available
    levelNodeCounts[0] = 1;
    for(uint32_t i = 1; i <= levelCount; i++)
    {
        // Mark the differences between neighbors
        gpu.GridStrideKC_X(0, (hipStream_t)0, hUniqueVoxelCount - 1,
                           //
                           KCMarkMortonChanges,
                           //
                           dDiffBitBuffer,
                           dSortedUniqueVoxels,
                           hUniqueVoxelCount - 1,
                           i,
                           levelCount);

        // Reduce the marks to find level node count
        ReduceArrayGPU<uint32_t, ReduceAdd<uint32_t>, hipMemcpyDeviceToHost>
        (
            levelNodeCounts[i],
            dDiffBitBuffer,
            hUniqueVoxelCount - 1,
            0u
        );
        gpu.WaitMainStream();
        // n different slices means n+1 segments
        levelNodeCounts[i] += 1;
    }
    assert(levelNodeCounts.back() == hUniqueVoxelCount);

    // Populate node offset buffer
    levelNodeOffsets.resize(levelCount + 2, 0);
    std::inclusive_scan(levelNodeCounts.cbegin(), levelNodeCounts.cend(),
                        levelNodeOffsets.begin() + 1);
    levelNodeOffsets.front() = 0;

    uint32_t totalNodeCount = levelNodeOffsets[levelNodeOffsets.size() - 2];

    treeGPU.nodeCount = totalNodeCount;
    treeGPU.leafCount = hUniqueVoxelCount;
    // Allocate required memories now
    // since we found out the total node count
    GPUMemFuncs::AllocateMultiData(std::tie(// Node Related,
                                            treeGPU.dNodes,
                                            treeGPU.dBinInfo,
                                            // Leaf Related
                                            treeGPU.dLeafParents,
                                            treeGPU.dLeafBinInfo,
                                            // Payload Node
                                            treeGPU.payload.dAvgIrradianceNode,
                                            treeGPU.payload.dNormalAndSpecNode,
                                            treeGPU.payload.dGuidingFactorNode,
                                            treeGPU.payload.dMicroQuadTreeNode,
                                            // Payload Leaf
                                            treeGPU.payload.dTotalIrradianceLeaf,
                                            treeGPU.payload.dSampleCountLeaf,
                                            treeGPU.payload.dAvgIrradianceLeaf,
                                            treeGPU.payload.dNormalAndSpecLeaf,
                                            treeGPU.payload.dGuidingFactorLeaf,
                                            treeGPU.payload.dMicroQuadTreeLeaf,
                                            // Node Offsets
                                            treeGPU.dLevelNodeOffsets),
                                   octreeMem,
                                   {
                                        // Node Related
                                        totalNodeCount, totalNodeCount,
                                        // Leaf Related
                                        hUniqueVoxelCount, hUniqueVoxelCount,
                                        // Payload Node
                                        totalNodeCount, totalNodeCount,
                                        totalNodeCount, totalNodeCount,
                                        // Payload Leaf
                                        hUniqueVoxelCount, hUniqueVoxelCount, hUniqueVoxelCount,
                                        hUniqueVoxelCount, hUniqueVoxelCount, hUniqueVoxelCount,
                                        // Offsets
                                        levelNodeOffsets.size()
                                   });

    // Set Node and leaf parents to max to early catch errors
    gpu.GridStrideKC_X(0, (hipStream_t)0, totalNodeCount,
                       //
                       KCMemset<uint64_t>,
                       //
                       treeGPU.dNodes,
                       AnisoSVOctreeGPU::INVALID_NODE,
                       totalNodeCount);
    CUDA_CHECK(hipMemset(treeGPU.dLeafParents, 0xFF, hUniqueVoxelCount * sizeof(uint32_t)));
    // Bin info initially should be zero (every bounce we will set it to again zero as well)
    CUDA_CHECK(hipMemset(treeGPU.dBinInfo, 0x00, totalNodeCount * sizeof(uint16_t)));
    CUDA_CHECK(hipMemset(treeGPU.dLeafBinInfo, 0x00, hUniqueVoxelCount * sizeof(uint16_t)));
    // Set accumulators to zero
    CUDA_CHECK(hipMemset(treeGPU.payload.dTotalIrradianceLeaf, 0x00, hUniqueVoxelCount * sizeof(Vector2f)));
    CUDA_CHECK(hipMemset(treeGPU.payload.dSampleCountLeaf, 0x00, hUniqueVoxelCount * sizeof(Vector2ui)));
    // Copy the generated offsets
    CUDA_CHECK(hipMemcpy(treeGPU.dLevelNodeOffsets, levelNodeOffsets.data(),
                          levelNodeOffsets.size() * sizeof(uint32_t),
                          hipMemcpyHostToDevice));
    // Top down-generate voxels
    // For each level save the node range for
    // efficient kernel calls later (level by level kernel calls)
    // Now start voxel generation level by level
    for(uint32_t i = 0; i < levelCount; i++)
    {
        gpu.GridStrideKC_X(0, (hipStream_t)0, hUniqueVoxelCount,
                           //
                           KCMarkChild,
                           // I-O
                           treeGPU.dNodes,
                           // Input
                           dSortedUniqueVoxels,
                           // Constants
                           hUniqueVoxelCount,
                           i,
                           levelCount);

        gpu.GridStrideKC_X(0, (hipStream_t)0, levelNodeCounts[i],
                           //
                           KCExtractChildrenCounts,
                           //
                           dDiffBitBuffer,
                           treeGPU.dNodes + levelNodeOffsets[i],
                           levelNodeCounts[i]);

        CUDA_CHECK(hipcub::DeviceScan::ExclusiveSum(dScanMemory, scanTempMemSize,
                                                 dDiffBitBuffer, dChildOffsetBuffer,
                                                 levelNodeCounts[i] + 1));

        // Check
        uint32_t hReducedSum;
        CUDA_CHECK(hipMemcpy(&hReducedSum, dChildOffsetBuffer + levelNodeCounts[i],
                              sizeof(uint32_t), hipMemcpyDeviceToHost));
        if(hReducedSum != levelNodeCounts[i + 1])
        {
            METU_ERROR_LOG("SVO children count allocation mismatch (Level {:d}.", i);
            return TracerError::TRACER_INTERNAL_ERROR;
        }

        bool lastNonLeafLevel = (i == (levelCount - 1));
        uint32_t nextLevelOffset = (lastNonLeafLevel) ? 0 : levelNodeOffsets[i + 1];
        gpu.GridStrideKC_X(0, (hipStream_t)0, levelNodeCounts[i],
                           //
                           KCSetChildrenPtrs,
                           //
                           treeGPU.dNodes + levelNodeOffsets[i],
                           dChildOffsetBuffer,
                           nextLevelOffset,
                           levelNodeCounts[i],
                           lastNonLeafLevel);

        if(!lastNonLeafLevel)
        {
            gpu.GridStrideKC_X(0, (hipStream_t)0, levelNodeCounts[i],
                               //
                               KCSetParentOfChildren,
                               //
                               treeGPU.dNodes,
                               treeGPU.dNodes + levelNodeOffsets[i],
                               levelNodeCounts[i]);
        }
        else
        {
            gpu.GridStrideKC_X(0, (hipStream_t)0, levelNodeCounts[i],
                               //
                               KCSetParentOfLeafChildren,
                               //
                               treeGPU.dLeafParents,
                               treeGPU.dNodes,
                               treeGPU.dNodes + levelNodeOffsets[i],
                               levelNodeCounts[i]);
        }
    }
    // Only Direct light information deposition is left
    // Call the kernel for it

    // DEBUG
    //Debug::DumpMemToFile("voxelDuplicates", dIndexOffsets,
    //                     hUniqueVoxelCount + 1);

    //std::vector<uint32_t> countsss(hUniqueVoxelCount + 1);
    //std::adjacent_difference(dIndexOffsets, dIndexOffsets + hUniqueVoxelCount + 1,
    //                         countsss.begin());
    //Debug::DumpMemToFile("voxelDuplicateCounts", countsss.data(),
    //                     hUniqueVoxelCount + 1);

    gpu.GridStrideKC_X(0, (hipStream_t)0, hUniqueVoxelCount * WARP_SIZE,
                       //
                       KCReduceVoxelPayload<StaticThreadPerBlock1D>,
                       // I-O
                       treeGPU,
                       // Input
                       dIndexOffsets,
                       dSortedUniqueVoxels,
                       dSortedVoxelIndices, // This is non-unique (we need to reduce it)
                       dVoxelLightKeys, // Voxel payload that will be reduced
                       dVoxelNormals,   // Voxel payload that will be reduced

                       // Binary Search for light
                       dSortedLightKeys,
                       dSortedLights,
                       totalLightCount,
                       // Constants
                       hUniqueVoxelCount,
                       static_cast<uint32_t>(hTotalVoxCount),
                       treeGPU.svoAABB,
                       resolutionXYZ);

    // Log some stuff
    timer.Stop();
    double svoMemSize = static_cast<double>(octreeMem.Size()) / 1024.0 / 1024.0;

    METU_LOG("Scene Aniso-SVO [N: {:L}, L: {:L}] Generated in {:f} seconds. (Total {:.2f} MiB)",
             treeGPU.nodeCount, treeGPU.leafCount, timer.Elapsed<CPUTimeSeconds>(), svoMemSize);

    // All Done!
    return TracerError::OK;
}

void AnisoSVOctreeCPU::NormalizeAndFilterRadiance(const CudaSystem& system)
{
    // From leaf (leaf-write) to root
    // Average the radiance
    // Down-sample the radiance for lowest n levels as well maybe? (n= 2 or 3)


    // TODO: Do some proper filtering
    // Just copy it to for now
    // Assume that the ray counts are set for leaves
    const CudaGPU& bestGPU = system.BestGPU();
    // Leaf has different memory layout do it separately
    bestGPU.GridStrideKC_X(0, (hipStream_t)0, treeGPU.leafCount,
                           //
                           KCCCopyRadianceToHalfBufferLeaf,
                           // I-O
                           treeGPU.payload.dAvgIrradianceLeaf,
                           // Input
                           treeGPU.payload.dTotalIrradianceLeaf,
                           treeGPU.payload.dSampleCountLeaf,
                           // Constants
                           treeGPU.leafCount,
                           1.0f);
}

void AnisoSVOctreeCPU::CollapseRayCounts(uint32_t minLevel, uint32_t minRayCount,
                                         const CudaSystem& system)
{
    // Assume that the ray counts are set for leaves
    const CudaGPU& bestGPU = system.BestGPU();

    // Leaf has different memory layout do it separately
    bestGPU.GridStrideKC_X(0, (hipStream_t)0, treeGPU.leafCount,
                           //
                           KCCollapseRayCountsLeaf,
                           // I-O
                           treeGPU.dLeafBinInfo,
                           treeGPU.dBinInfo,
                           // Input
                           treeGPU.dLeafParents,
                           // Constants
                           treeGPU.leafCount,
                           treeGPU.leafDepth,
                           minLevel,
                           minRayCount);

    //Debug::DumpMemToFile(std::to_string(treeGPU.leafDepth) + std::string("_binInfo"),
    //                     treeGPU.dLeafBinInfo,
    //                     treeGPU.leafCount, false, true);

    // Bottom-up process bins
    int32_t bottomNodeLevel = static_cast<int32_t>(treeGPU.leafDepth - 1);
    for(int32_t i = bottomNodeLevel; i >= static_cast<int32_t>(minLevel); i--)
    {
        Vector2ui range(levelNodeOffsets[i],
                        levelNodeOffsets[i + 1]);
        uint32_t nodeCount = range[1] - range[0];

        bestGPU.GridStrideKC_X(0, (hipStream_t)0, nodeCount,
                               //
                               KCCollapseRayCounts,
                               // I-O
                               treeGPU.dBinInfo,
                               // Input
                               treeGPU.dNodes,
                               // Constants
                               range,
                               i,
                               minLevel,
                               minRayCount);
    }
    // Leaf->Parent chain now there is at least a single mark
    // Rays will re-check and find their marked bin and set their id accordingly
}


void AnisoSVOctreeCPU::AccumulateRaidances(const WFPGPathNode* dPGNodes,
                                           uint32_t totalNodeCount,
                                           uint32_t maxPathNodePerRay,
                                           const CudaSystem& system)
{
    // Directly call the appropriate kernel
    const CudaGPU& bestGPU = system.BestGPU();
    bestGPU.GridStrideKC_X(0, (hipStream_t)0, totalNodeCount,
                           //
                           KCAccumulateRadianceToLeaf,
                           //
                           treeGPU,
                           dPGNodes,
                           totalNodeCount,
                           maxPathNodePerRay);
    bestGPU.WaitMainStream();
}

void AnisoSVOctreeCPU::ClearRayCounts(const CudaSystem&)
{
    CUDA_CHECK(hipMemset(treeGPU.dLeafBinInfo, 0x00, sizeof(uint32_t) * treeGPU.leafCount));
    CUDA_CHECK(hipMemset(treeGPU.dBinInfo, 0x00, sizeof(uint32_t) * treeGPU.nodeCount));
}

void AnisoSVOctreeCPU::DumpSVOAsBinary(std::vector<Byte>& data,
                                       const CudaSystem& system) const
{
    // Temp Float Buffer for Conversion
    assert(treeGPU.leafCount >= treeGPU.nodeCount);
    DeviceMemory halfConvertedMemory(treeGPU.leafCount * sizeof(Vector2f));

    // Conversion Function
    auto ConvertAnisoHalfToFloat = [&](const Vector2h* dRadiance,
                                       uint32_t totalSize)
    {
        const CudaGPU& gpu = system.BestGPU();
        gpu.GridStrideKC_X(0, (hipStream_t)0, totalSize,
                           //
                           KCConvertToAnisoFloat,
                           //
                           static_cast<Vector2f*>(halfConvertedMemory),
                           dRadiance,
                           totalSize);
    };

    // Get Sizes
    std::array<size_t, 4> byteSizes;
    byteSizes[0]  = treeGPU.nodeCount * sizeof(uint64_t);   // "dNodes" Size
    byteSizes[1]  = treeGPU.nodeCount * sizeof(Vector2f);   // "dAvgIrradianceNode" Size
    // Leaf Related
    byteSizes[2] = treeGPU.leafCount * sizeof(uint32_t);    // "dLeafParent" Size
    byteSizes[3] = treeGPU.leafCount * sizeof(Vector2f);    // "dAvgIrradianceLeaf" Size
    // Calculate the offsets and total size
    size_t bufferTotalSize = std::reduce(byteSizes.cbegin(), byteSizes.cend(), 0ull);

    size_t totalSize = (bufferTotalSize + sizeof(AABB3f) +
                        5 * sizeof(uint32_t) +
                        sizeof(float));

    data.resize(totalSize);
    // Memcpy the data from the memory
    size_t offset = 0;
    std::memcpy(data.data() + offset, &treeGPU.svoAABB, sizeof(AABB3f));
    offset += sizeof(AABB3f);
    std::memcpy(data.data() + offset, &treeGPU.voxelResolution, sizeof(uint32_t));
    offset += sizeof(uint32_t);
    std::memcpy(data.data() + offset, &treeGPU.leafDepth, sizeof(uint32_t));
    offset += sizeof(uint32_t);
    std::memcpy(data.data() + offset, &treeGPU.nodeCount, sizeof(uint32_t));
    offset += sizeof(uint32_t);
    std::memcpy(data.data() + offset, &treeGPU.leafCount, sizeof(uint32_t));
    offset += sizeof(uint32_t);
    std::memcpy(data.data() + offset, &treeGPU.leafVoxelSize, sizeof(float));
    offset += sizeof(float);
    std::memcpy(data.data() + offset, &treeGPU.levelOffsetCount, sizeof(uint32_t));
    offset += sizeof(uint32_t);

    // Nodes
    CUDA_CHECK(hipMemcpy(data.data() + offset, treeGPU.dNodes,
                          byteSizes[0], hipMemcpyDeviceToHost));
    offset += byteSizes[0];
    // Radiance Cache Node
    ConvertAnisoHalfToFloat(treeGPU.payload.dAvgIrradianceNode, treeGPU.nodeCount);
    CUDA_CHECK(hipMemcpy(data.data() + offset,
                          static_cast<void*>(halfConvertedMemory),
                          byteSizes[1], hipMemcpyDeviceToHost));
    offset += byteSizes[1];
    // Leaf Parents
    CUDA_CHECK(hipMemcpy(data.data() + offset, treeGPU.dLeafParents,
                          byteSizes[2], hipMemcpyDeviceToHost));
    offset += byteSizes[2];
    // Radiance Cache Leaf
    ConvertAnisoHalfToFloat(treeGPU.payload.dAvgIrradianceLeaf, treeGPU.leafCount);
    CUDA_CHECK(hipMemcpy(data.data() + offset,
                          static_cast<void*>(halfConvertedMemory),
                          byteSizes[3], hipMemcpyDeviceToHost));
    offset += byteSizes[3];
    assert(offset == data.size());

    // All Done!
}