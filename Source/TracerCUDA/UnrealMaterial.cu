#include "UnrealMaterial.cuh"

SceneError UnrealMat::InitializeGroup(const NodeListing& materialNodes,
                                      const std::map<uint32_t, uint32_t>& mediumIdIndexPairs,
                                      double time, const std::string& scenePath)
{    

    std::vector<Vector3> albedoCPU;
    std::vector<Vector3> rougnessMetallicSheenCPU;
    uint32_t i = 0;
    for(const auto& sceneNode : materialNodes)
    {        
        const auto& ids = sceneNode->Ids();
        for(IdPair id : ids)
        {
            innerIds.emplace(std::make_pair(id.first, i));
            i++;
        }
    }

    // Generate Id List
    SceneError e = SceneError::OK;
    if((e = GenerateInnerIds(materialNodes)) != SceneError::OK)
        return e;

    // Alloc etc
    //size_t dIrradianceSize = irradianceCPU.size() * sizeof(Vector3);
    //memory = std::move(DeviceMemory(dIrradianceSize));
    //Vector3f* dIrradiance = static_cast<Vector3f*>(memory);
    //CUDA_CHECK(hipMemcpy(dIrradiance, irradianceCPU.data(), dIrradianceSize,
    //                      hipMemcpyHostToDevice));

    //dData = EmissiveMatData{dIrradiance};
    return SceneError::OK;

    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}

SceneError UnrealMat::ChangeTime(const NodeListing& materialNodes, double time,
                                 const std::string& scenePath)
{
    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}