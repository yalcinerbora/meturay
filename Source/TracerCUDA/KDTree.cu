#include "hip/hip_runtime.h"
#include "KDTree.cuh"
#include <queue>


KDTreeCPU::KDTreeCPU()
{
    treeGPU.gLeafs = nullptr;
    treeGPU.gSplits = nullptr;
    treeGPU.gPackedData = nullptr;
    treeGPU.rootNodeId = UINT32_MAX;
}

TracerError KDTreeCPU::Construct(const Vector3f* dPositionList,
                                 uint32_t leafCount,
                                 const CudaSystem& system)
{
    static constexpr uint32_t MAX_BASE_DEPTH = 64;
    // Partition Function
    auto GenKdTreeNode = [](// Output
                            uint64_t& packedInfo,
                            float& splitPlane,
                            size_t& splitLoc,
                            bool& isLeaf,
                            // I-O
                            std::vector<Vector3f>& positions,
                            // Args
                            uint32_t childIndex,
                            uint32_t parentIndex,
                            size_t start, size_t end)
    {
        // Base Case
        if(end - start == 1)
        {
            splitLoc = std::numeric_limits<size_t>::max();
            KDTreeGPU::PackInfo(parentIndex,
                                start,
                                true,
                                KDTreeGPU::AXIS_END);
            isLeaf = true;
        }
        else
        {
            Vector3f maxPoint = Vector3f(-FLT_MAX);
            Vector3f minPoint = Vector3f(FLT_MAX);
            Vector3f center = Zero3f;
            for(size_t j = start; j < end; j++)
            {
                maxPoint = Vector3f::Max(maxPoint, positions[j]);
                minPoint = Vector3f::Min(minPoint, positions[j]);
                center += positions[j];
            }
            center /= (start - end);

            // Determine the split
            int maxIndex = (maxPoint - minPoint).Max();
            KDTreeGPU::AxisType axis = static_cast<KDTreeGPU::AxisType>(maxIndex);

            // Partition
            splitLoc = 0;
            int testAxis = maxIndex;
            // Partition wrt. avg center
            int64_t splitStart = static_cast<int64_t>(start - 1);
            int64_t splitEnd = static_cast<int64_t>(end);
            while(splitStart < splitEnd)
            {
                // Hoare Like Partition
                float leftAxisCenter;
                do
                {
                    if(splitStart >= static_cast<int64_t>(end - 1)) break;
                    splitStart++;
                    leftAxisCenter = positions[splitStart][testAxis];
                }
                while(leftAxisCenter >= center[testAxis]);
                float rightAxisCenter;
                do
                {
                    if(splitEnd <= static_cast<int64_t>(start + 1)) break;
                    splitEnd--;
                    rightAxisCenter = positions[splitEnd][testAxis];
                }
                while(rightAxisCenter <= center[testAxis]);

                if(splitStart < splitEnd)
                    std::swap(positions[splitEnd], positions[splitStart]);
            }
            // If cant find any proper split
            // Just cut in half
            if(splitLoc == 0) splitLoc = (end - start) / 2;

            // Sanity Check
            assert(splitLoc != start);
            assert(splitLoc != end);

            // Return
            splitPlane = center[static_cast<int>(axis)];
            KDTreeGPU::PackInfo(parentIndex,
                                childIndex,
                                false,
                                axis);
            isLeaf = false;
        }
    };

    // Load Leafs to Memory
    std::vector<Vector3f> hPositions(leafCount);
    CUDA_CHECK(hipMemcpy(hPositions.data(), dPositionList,
                          sizeof(Vector3f) * leafCount,
                          hipMemcpyDeviceToHost));
    // CPU Memory
    std::vector<uint64_t> hPackInfo;
    std::vector<float> hSplitPlanes;
    //
    struct SplitWork
    {
        bool isLeft;
        size_t start;
        size_t end;
        uint32_t parentId;
        uint32_t depth;
    };

    // Start Partitioning
    std::queue<SplitWork> partitionQueue;
    partitionQueue.emplace(SplitWork
                           {
                               false,
                               0, leafCount,
                               std::numeric_limits<uint32_t>::max(),
                               0
                           });

    // Breath first tree generation (top-down)
    uint8_t maxDepth = 0;
    while(!partitionQueue.empty())
    {
        SplitWork current = partitionQueue.front();
        partitionQueue.pop();

        size_t splitLoc;
        uint64_t packedInfo;
        float splitPlane;
        bool isLeaf;
        // Do Generation
        GenKdTreeNode(packedInfo,
                      splitPlane,
                      splitLoc,
                      isLeaf,
                      // I-O
                      hPositions,
                       // Args
                      static_cast<uint32_t>(hPackInfo.size() + 1),
                      current.parentId,
                      current.start, current.end);

        // Save
        hPackInfo.push_back(packedInfo);
        hSplitPlanes.push_back(splitPlane);

        // Next parent id
        uint32_t nextParentId = static_cast<uint32_t>(hPackInfo.size() - 1);

        // Update parent
        // Since nodes are adjacent only left can update the parent
        if(current.isLeft)
        {
            // Update the packed child Id of the parent
            KDTreeGPU::UpdateChildIndex(hPackInfo[current.parentId],
                                        nextParentId);
        }
        // Check if not base case and add more generation
        if(splitLoc != std::numeric_limits<size_t>::max())
        {
            partitionQueue.emplace(SplitWork{true, current.start, splitLoc, nextParentId, current.depth + 1});
            partitionQueue.emplace(SplitWork{false, splitLoc, current.end, nextParentId, current.depth + 1});
            maxDepth = static_cast<uint8_t>(current.depth + 1);

            if((current.depth + 1) > MAX_BASE_DEPTH)
                return TracerError::TRACER_INTERNAL_ERROR;
        }
    }
    // BVH cannot hold this surface return error
    if(maxDepth > MAX_BASE_DEPTH)
        return TracerError::TRACER_INTERNAL_ERROR;

    assert(hPackInfo.size() == hSplitPlanes.size());
    // Finally Allocate the entire node array
    uint64_t* dPackInfo;
    float* dSplitPlanes;
    Vector3f* dPositions;
    GPUMemFuncs::AllocateMultiData(std::tie(dPackInfo, dSplitPlanes, dPositions),
                                   memory,
                                   {hPackInfo.size(), hSplitPlanes.size(),
                                   hPositions.size()});
    CUDA_CHECK(hipMemcpy(dPackInfo, hPackInfo.data(),
                          sizeof(uint64_t) * hPackInfo.size(),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dSplitPlanes, hSplitPlanes.data(),
                          sizeof(float) * hSplitPlanes.size(),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dPositions, dPositionList,
                          sizeof(Vector3f) * hPositions.size(),
                          hipMemcpyDeviceToDevice));

    treeGPU.gPackedData = dPackInfo;
    treeGPU.gLeafs = dPositions;
    treeGPU.gSplits = dSplitPlanes;
    treeGPU.rootNodeId = 0;
    return TracerError::OK;
}


const KDTreeGPU& KDTreeCPU::TreeGPU() const
{
    return treeGPU;
}