#include "hip/hip_runtime.h"
#include "KDTree.cuh"
#include <queue>
#include <ostream>

KDTreeCPU::KDTreeCPU()
{
    treeGPU.gLeafs = nullptr;
    treeGPU.gSplits = nullptr;
    treeGPU.gPackedData = nullptr;
    treeGPU.rootNodeId = UINT32_MAX;
    treeGPU.voronoiCenterSize = 0;
}

TracerError KDTreeCPU::Construct(const Vector3f* dPositionList,
                                 uint32_t leafCount,
                                 const CudaSystem& system)
{
    static constexpr uint32_t MAX_BASE_DEPTH = 64;
    // Partition Function
    Vector3f pointMin;
    Vector3f pointMax;

    auto GenKdTreeNode = [&pointMin, &pointMax, leafCount]
    (
        // Output
        uint64_t& packedInfo,
        float& splitPlane,
        size_t& splitLoc,
        bool& isLeaf,
        // I-O
        std::vector<Vector3f>& positions,
        // Args
        uint32_t childIndex,
        uint32_t parentIndex,
        size_t start, size_t end
    )
    {
        // Base Case
        if(end - start == 1)
        {
            splitLoc = std::numeric_limits<size_t>::max();
            packedInfo = KDTreeGPU::PackInfo(parentIndex,
                                             static_cast<uint32_t>(start),
                                             true,
                                             KDTreeGPU::AXIS_END);
            isLeaf = true;
        }
        else
        {
            Vector3f maxPoint = Vector3f(-FLT_MAX);
            Vector3f minPoint = Vector3f(FLT_MAX);
            for(size_t j = start; j < end; j++)
            {
                maxPoint = Vector3f::Max(maxPoint, positions[j]);
                minPoint = Vector3f::Min(minPoint, positions[j]);
            }
            Vector3f center = (minPoint + (maxPoint - minPoint) * 0.5f);

            // Save Min Max Point for debug visualization
            if(start == 0 && end == leafCount)
            {
                pointMax = maxPoint;
                pointMin = minPoint;
            }

            // Determine the split
            int maxIndex = (maxPoint - minPoint).Max();
            KDTreeGPU::AxisType axis = static_cast<KDTreeGPU::AxisType>(maxIndex);

            // Partition
            splitLoc = 0;
            int testAxis = maxIndex;
            // Partition wrt. avg center
            int64_t splitStart = static_cast<int64_t>(start - 1);
            int64_t splitEnd = static_cast<int64_t>(end);
            while(splitStart < splitEnd)
            {
                // Hoare Like Partition
                float leftAxisCenter;
                do
                {
                    if(splitStart >= static_cast<int64_t>(end - 1)) break;
                    splitStart++;
                    leftAxisCenter = positions[splitStart][testAxis];
                }
                while(leftAxisCenter >= center[testAxis]);
                float rightAxisCenter;
                do
                {
                    if(splitEnd <= static_cast<int64_t>(start + 1)) break;
                    splitEnd--;
                    rightAxisCenter = positions[splitEnd][testAxis];
                }
                while(rightAxisCenter <= center[testAxis]);

                if(splitStart < splitEnd)
                    std::swap(positions[splitEnd], positions[splitStart]);
            }
            // If cant find any proper split
            // Just cut in half
            if(splitStart != static_cast<int64_t>(start) ||
               splitStart != static_cast<int64_t>(end))
                splitLoc = splitStart;
            else
                splitLoc = (end - start) / 2;

            // Sanity Check
            assert(splitLoc != start);
            assert(splitLoc != end);

            // Return
            splitPlane = center[static_cast<int>(axis)];
            packedInfo = KDTreeGPU::PackInfo(parentIndex & 0x3FFFFFFF,
                                             childIndex,
                                             false,
                                             axis);
            isLeaf = false;
        }
    };

    // Load Leafs to Memory
    std::vector<Vector3f> hPositions(leafCount);
    CUDA_CHECK(hipMemcpy(hPositions.data(), dPositionList,
                          sizeof(Vector3f) * leafCount,
                          hipMemcpyDeviceToHost));
    // CPU Memory
    std::vector<uint64_t> hPackInfo;
    std::vector<float> hSplitPlanes;
    //
    struct SplitWork
    {
        bool isLeft;
        size_t start;
        size_t end;
        uint32_t parentId;
        uint32_t depth;
    };

    // Start Partitioning
    std::queue<SplitWork> partitionQueue;
    partitionQueue.emplace(SplitWork
                           {
                               false,
                               0, leafCount,
                               std::numeric_limits<uint32_t>::max(),
                               0
                           });

    // Breath first tree generation (top-down)
    uint8_t maxDepth = 0;
    while(!partitionQueue.empty())
    {
        SplitWork current = partitionQueue.front();
        partitionQueue.pop();

        size_t splitLoc;
        uint64_t packedInfo;
        float splitPlane;
        bool isLeaf;
        // Do Generation
        GenKdTreeNode(packedInfo,
                      splitPlane,
                      splitLoc,
                      isLeaf,
                      // I-O
                      hPositions,
                       // Args
                      static_cast<uint32_t>(hPackInfo.size() + 1),
                      current.parentId,
                      current.start, current.end);

        // Save
        hPackInfo.push_back(packedInfo);
        hSplitPlanes.push_back(splitPlane);

        // Next parent id
        uint32_t nextParentId = static_cast<uint32_t>(hPackInfo.size() - 1);

        // Update parent
        // Since nodes are adjacent only left can update the parent
        if(current.isLeft)
        {
            // Update the packed child Id of the parent
            KDTreeGPU::UpdateChildIndex(hPackInfo[current.parentId],
                                        nextParentId);
        }
        // Check if not base case and add more generation
        if(splitLoc != std::numeric_limits<size_t>::max())
        {
            partitionQueue.emplace(SplitWork{true, current.start, splitLoc, nextParentId, current.depth + 1});
            partitionQueue.emplace(SplitWork{false, splitLoc, current.end, nextParentId, current.depth + 1});
            maxDepth = static_cast<uint8_t>(current.depth + 1);

            if((current.depth + 1) > MAX_BASE_DEPTH)
                return TracerError::TRACER_INTERNAL_ERROR;
        }
    }
    // BVH cannot hold this surface return error
    if(maxDepth > MAX_BASE_DEPTH)
        return TracerError::TRACER_INTERNAL_ERROR;

    assert(hPackInfo.size() == hSplitPlanes.size());
    // Finally Allocate the entire node array
    uint64_t* dPackInfo;
    float* dSplitPlanes;
    Vector3f* dPositions;
    GPUMemFuncs::AllocateMultiData(std::tie(dPackInfo, dSplitPlanes, dPositions),
                                   memory,
                                   {hPackInfo.size(), hSplitPlanes.size(),
                                   hPositions.size()});
    CUDA_CHECK(hipMemcpy(dPackInfo, hPackInfo.data(),
                          sizeof(uint64_t) * hPackInfo.size(),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dSplitPlanes, hSplitPlanes.data(),
                          sizeof(float) * hSplitPlanes.size(),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dPositions, hPositions.data(),
                          sizeof(Vector3f) * hPositions.size(),
                          hipMemcpyHostToDevice));

    this->leafCount = leafCount;
    nodeCount = static_cast<uint32_t>(hPackInfo.size());

    treeGPU.voronoiCenterSize = CalculateVoronoiCenterSize(AABB3f(pointMin, pointMax));
    treeGPU.gPackedData = dPackInfo;
    treeGPU.gLeafs = dPositions;
    treeGPU.gSplits = dSplitPlanes;
    treeGPU.rootNodeId = 0;

    std::ofstream file("surfaceTree");
    DumpTreeToStream(file);
    file.close();
    return TracerError::OK;
}


const KDTreeGPU& KDTreeCPU::TreeGPU() const
{
    return treeGPU;
}

size_t KDTreeCPU::UsedGPUMemory() const
{
    return memory.Size();
}

size_t KDTreeCPU::UsedCPUMemory() const
{
    return sizeof(KDTreeCPU);
}

float KDTreeCPU::CalculateVoronoiCenterSize(const AABB3f& sceneAABB)
{
    Vector3f span = sceneAABB.Span();
    float sceneSize = span.Length();
    static constexpr float VORONOI_RATIO = 1.0f / 1'300.0f;
    return sceneSize * VORONOI_RATIO;
}

void KDTreeCPU::DumpTreeToStream(std::ostream& s) const
{
    std::vector<uint64_t> hPackedData(nodeCount);
    std::vector<float> hSplitPlanes(nodeCount);

    CUDA_CHECK(hipMemcpy(hPackedData.data(), treeGPU.gPackedData,
                          sizeof(uint64_t) * nodeCount,
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hSplitPlanes.data(), treeGPU.gSplits,
                          sizeof(float) * nodeCount,
                          hipMemcpyDeviceToHost));

    static constexpr uint32_t UINT30_MAX = (UINT32_MAX & 0x3FFFFFFF);

    for(uint32_t i = 0; i < nodeCount; i++)
    {
        uint32_t parent;
        uint32_t child;
        bool isLeaf;
        KDTreeGPU::AxisType axis;
        KDTreeGPU::UnPackInfo(parent, child,
                              isLeaf, axis,
                              hPackedData[i]);

        s << std::string("P[");
        if(parent == UINT30_MAX) s << "-";
        else s << parent;
        s << "] ";

        if(isLeaf)
        {
            s << "L[";
            if(child == UINT30_MAX) s << "-";
            else s << child;
            s << "]";
        }
        else
        {
            s << "C[";
            if(child == UINT30_MAX) s << "-";
            else s << child;
            s << ", ";
            if(child == UINT30_MAX) s << "-";
            else s << (child + 1);
            s << "] ";
            s << "Split [" << hSplitPlanes[i] << "]";
        }
        s << "\n";
    }
}

void KDTreeCPU::DumpTreeAsBinary(std::vector<Byte>& data) const
{

}