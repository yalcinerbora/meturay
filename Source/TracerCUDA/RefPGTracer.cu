#include "hip/hip_runtime.h"
﻿#include "RefPGTracer.h"
#include "RayTracer.hpp"

#include "RayLib/GPUSceneI.h"
#include "RayLib/TracerCallbacksI.h"
#include "RayLib/BitManipulation.h"
#include "RayLib/FileUtility.h"

#include "PPGTracerWork.cuh"
#include "GPULightSamplerUniform.cuh"
#include "GenerationKernels.cuh"
#include "GPUWork.cuh"
#include "GPUAcceleratorI.h"

#include "TracerDebug.h"

#include "GPUCameraSpherical.cuh"

__global__
void KCConstructSingleGPUCameraSpherical(GPUCameraSpherical* gCameraLocations,
                                         bool deletePrevious,
                                         //
                                         float pixelRatio,
                                         Vector3 position,
                                         Vector3 direction,
                                         Vector3 up,
                                         Vector2 nearFar,
                                         //
                                         const TransformId gTransformId,
                                         const uint16_t gMediumIndex,
                                         const HitKey gCameraMaterialId,
                                         //
                                         const GPUTransformI& gTransform)
{
    if(deletePrevious) delete gCameraLocations;
    new (gCameraLocations) GPUCameraSpherical(pixelRatio,
                                              position,
                                              direction,
                                              up,
                                              nearFar,
                                              gTransform,
                                              //
                                              gMediumIndex,
                                              gCameraMaterialId);
}

RefPGTracer::RefPGTracer(const CudaSystem& s,
                         const GPUSceneI& scene,
                         const TracerParameters& p)    
    : currentPixel(0)
    , currentDepth(0)    
    , currentSample(0)
    , pathTracer(s, scene, p)
    , directTracer(s, scene, p)    
    , cudaSystem(s)
{}

TracerError RefPGTracer::Initialize()
{
    // Generate Tracers
    TracerError err = TracerError::OK;
    if((err = pathTracer.Initialize()) != TracerError::OK)
        return err;

    if((err = directTracer.Initialize()) != TracerError::OK)
        return err;

    // Allocate a SphericalCamera Memory (construct when needed)
    memory = DeviceMemory(sizeof(GPUCameraSpherical));

    return TracerError::OK;
}

TracerError RefPGTracer::SetOptions(const TracerOptionsI& opts)
{
    TracerError err = TracerError::OK;
    if((err = opts.GetUInt(options.maximumDepth, MAX_DEPTH_NAME)) != TracerError::OK)
        return err;
    if((err = opts.GetInt(options.sampleCount, SAMPLE_NAME)) != TracerError::OK)
        return err;
    if((err = opts.GetUInt(options.rrStart, RR_START_NAME)) != TracerError::OK)
        return err;
    if((err = opts.GetString(options.lightSamplerType, LIGHT_SAMPLER_TYPE_NAME)) != TracerError::OK)
        return err;

    if((err = opts.GetBool(options.nextEventEstimation, NEE_NAME)) != TracerError::OK)
        return err;
    if((err = opts.GetBool(options.directLightMIS, DIRECT_LIGHT_MIS_NAME)) != TracerError::OK)
        return err;

    if((err = opts.GetUInt(options.maxSampleCount, MAX_SAMPLE_NAME)) != TracerError::OK)
        return err;    

    ...
    return TracerError::OK;
}

bool RefPGTracer::Render()
{
    ...
    return true;
}

void RefPGTracer::Finalize()
{
    ...;

    //
    cudaSystem.BestGPU().AsyncGridStrideKC_X(0, 1,
                                             // Function
                                             KCConstructSingleGPUCameraSpherical,
                                             // Args
                                             memory,
                                             ...);

}

void RefPGTracer::GenerateWork(int cameraId)
{
    ...
}

void RefPGTracer::GenerateWork(const VisorCamera& cam)
{
    ...
}

void RefPGTracer::SetParameters(const TracerParameters& p)
{
    directTracer.SetParameters(p);
    pathTracer.SetParameters(p);
}

void RefPGTracer::AskParameters()
{
    if(callbacks) callbacks->SendCurrentParameters(params);
}

void RefPGTracer::SetImagePixelFormat(PixelFormat f)
{
    directTracer.SetImagePixelFormat(f);
}

void RefPGTracer::ReportionImage(Vector2i start, Vector2i end)
{
    directTracer.ReportionImage(start, end);
    portionStart = start;
    portionEnd = end;
}

void RefPGTracer::ResizeImage(Vector2i resolution)
{
    directTracer.ResizeImage(resolution);
}

void RefPGTracer::ResetImage()
{
    directTracer.ResetImage();
    if(callbacks)
    {
        Vector2i start = portionStart;
        Vector2i end = portionEnd;
        callbacks->SendImageSectionReset(start, end);
    }
}