#include <gtest/gtest.h>
#include <gmock/gmock.h>

#include "RayLib/CudaConstants.h"
#include "RayLib/DeviceMemory.h"
#include "RayLib/Log.h"
#include "RayLib/CameraKernels.cuh"
#include "RayLib/TracerError.h"

TEST(CameraRayGPU, Test)
{
	TracerError e = CudaSystem::Initialize();

	static constexpr size_t ThreadCount = 32;
	static constexpr size_t StateSize = 32 * sizeof(uint32_t);

	static constexpr size_t NumberPerThread = 2;
	static constexpr size_t NumberCount = NumberPerThread * ThreadCount;
	static constexpr size_t NumberSize = NumberCount * sizeof(uint32_t);

	DeviceMemory randomState(StateSize);
	DeviceMemory numbers(NumberSize);

	//// Set State
	//std::mt19937 engine;
	//uint32_t* seeds = static_cast<uint32_t*>(randomState);
	//for(size_t i = 0; i < ThreadCount; i++)
	//{
	//	seeds[i] = engine();
	//}

	// Kernel Call
	uint32_t* h_data = static_cast<uint32_t*>(numbers);

	//KCGenerateCameraRays();


	CUDA_KERNEL_CHECK();
	CUDA_CHECK(hipDeviceSynchronize());

	//for(int i = 0; i < NumberCount; i++)
	//{
	//	METU_LOG("%u", h_data[i]);
	//}
}