#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <gmock/gmock.h>

#include "RayLib/Vector.h"

#include <hip/hip_runtime.h>

using ::testing::ElementsAre;
using ::testing::FloatEq;

//static constexpr unsigned int THREAD_COUNT = 50'000'000;

__global__ void KGlobalLoadStore(const Vector2* v2Input,
								 const Vector3* v3Input,
								 const Vector4* v4Input,
								 const float* fInput,

								 Vector2* v2Output,
								 Vector3* v3Output,
								 Vector4* v4Output,
								 float* fV2Output,
								 float* fV3Output,
								 float* fV4Output)
{
	unsigned int gId = threadIdx.x + blockIdx.x * blockDim.x;

	// Testing Direct Load and Store
	Vector2 v0 = v2Input[gId];
	Vector3 v1 = v3Input[gId];
	Vector4 v2 = v4Input[gId];

	Vector2 v3 = fInput[gId * 4];
	Vector3 v4 = fInput[gId * 4];
	Vector4 v5 = fInput[gId * 4];
	//
	v2Output[gId] = v0;
	v3Output[gId] = v1;
	v4Output[gId] = v2;

	reinterpret_cast<Vector2*>(fV2Output)[gId] = v3;
	reinterpret_cast<Vector3*>(fV3Output)[gId] = v4;
	reinterpret_cast<Vector4*>(fV4Output)[gId] = v5;
}

__global__ void KConstruction(Vector4* resuults)
{
	const float dataArray[] = {1.0f, 2.0f, 3.0f, 4.0f};
	const float dataArrayLarge[] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};

	Vector2 vecAssign0(1.0f, 2.0f);
	Vector3 vecAssign1(1.0f, 2.0f, 3.0f);
	Vector4 vecAssign2(1.0f, 2.0f, 3.0f, 4.0f);

	//
	Vector4 vec0;
	Vector4 vec1(1.0f);
	Vector4 vec2(1.0f, 1u, 2.0f, 3.0f);
	Vector4 vec3(dataArray);

	Vector4 vec4(dataArrayLarge);

	//
	Vector4 vec5(vecAssign0);
	Vector4 vec6(vecAssign1);

	// Copy Constructor (default)
	Vector4 vec7(vecAssign2);

	// Partial Constructor
	Vector4 vec9(vecAssign0, 3.0f, 4.0f);
	Vector4 vec10(vecAssign1, 4.0f);
}

__global__ void KOperators(Vector4* results)
{
	Vector4 a(2.0f, 2.0f, 2.0f, 2.0f);
	Vector4 b(1.0f, 1.0f, 1.0f, 1.0f);
	Vector4 c(2.0f, 4.0f, 6.0f, 8.0f);

	// Artihmetic
	Vector4 v0 = a + b;
	Vector4 v1 = a - b;
	Vector4 v2 = a * c;
	Vector4 v3 = a / c;
	Vector4 v4a = a * 2.0f;
	Vector4 v4b = 2.0f * a;
	Vector4 v5 = a / 2.0f;

	// Assignment with arithmetic
	Vector4 v6 = Vector4(1.0f, 1.0f, 1.0f, 1.0f);
	v6 += a;

	Vector4 v7 = Vector4(1.0f, 1.0f, 1.0f, 1.0f);
	v7 -= a;

	Vector4 v8 = Vector4(2.0f, 4.0f, 6.0f, 8.0f);
	v8 *= a;

	Vector4 v9 = Vector4(1.0f, 1.0f, 1.0f, 1.0f);
	v9 *= 2.0f;

	Vector4 v10 = Vector4(2.0f, 2.0f, 2.0f, 2.0f);
	v10 /= c;

	Vector4 v11 = Vector4(1.0f, 1.0f, 1.0f, 1.0f);
	v11 /= 2.0f;
}

__global__ void KFunctions1(float* floatResults,
							Vector4* results)
{
	Vector4 a(2.0f, 2.0f, 2.0f, 2.0f);
	Vector4 b(1.0f, 1.0f, 1.0f, 1.0f);
	Vector4 c(2.0f, 4.0f, 6.0f, 8.0f);
	Vector2 d(3.0f, 4.0f);
	Vector3 e(1.0f, 2.0f, 3.0f);
	Vector3 f(3.0f, 5.0f, 7.0f);

	// Non-Selfs
	float v0 = a.Dot(b);
	float v1 = d.Length();
	float v2 = d.LengthSqr();
	Vector4 v3 = c.Normalize();
	Vector4 v4 = c.Clamp(Vector4(3.0f), Vector4(4.0f));
	Vector4 v5 = c.Clamp(3.0f, 4.0f);

	// Selfs
	Vector4 v6 = c;
	v6.NormalizeSelf();

	Vector4 v7 = c;
	v7.ClampSelf(Vector4(3.0f), Vector4(4.0f));

	Vector4 v8 = c;
	v8.ClampSelf(3.0f, 4.0f);

	// Cross Product (3D Vector Special)
	Vector3 v9 = Cross(e, f);
}

__global__ void KFunctions2(Vector4* results)
{
	Vector4 b(-2.12f, -2.5f, 2.60f, 2.3f);
	Vector4 c(-1.0f, 0.0f, -0.0f, 1.0f);

	Vector4 v0 = c.Abs();
	Vector4 v1 = b.Round();
	Vector4 v2 = b.Floor();
	Vector4 v3 = b.Ceil();

	// Self Equavilents
	Vector4 v4 = c;
	v4.AbsSelf();
	Vector4 v5 = b;
	v5.RoundSelf();
	Vector4 v6 = b;
	v6.FloorSelf();
	Vector4 v7 = b;
	v7.CeilSelf();
}

__global__ void KFunctions3(Vector4* results)
{
	Vector4 a(2.12f, 2.5f, -2.60f, -2.3f);
	Vector4 b(-2.12f, -2.5f, 2.60f, 2.3f);
	Vector4 c(0.0f);
	Vector4 d(1.0f);

	Vector4 v0 = Vector4::Max(a, b);
	Vector4 v1 = Vector4::Max(a, 999.0f);

	Vector4 v2 = Vector4::Min(a, b);
	Vector4 v3 = Vector4::Min(a, -999.0f);

	Vector4 v4 = Vector4::Lerp(c, d, 0.5f);
}

TEST(VectorGPU, GlobalLoadStore)
{

}

TEST(VectorGPU, Construction)
{
	Vector4 h_data[11];
	Vector4* d_data;


	//	
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[0]),
								   static_cast<const float*>(h_data[0]) + 4),
				ElementsAre(FloatEq(0.0f), FloatEq(0.0f), FloatEq(0.0f), FloatEq(0.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[1]),
								   static_cast<const float*>(h_data[1]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(1.0f), FloatEq(1.0f), FloatEq(1.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[2]),
								   static_cast<const float*>(h_data[2]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(1.0f), FloatEq(2.0f), FloatEq(3.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[3]),
								   static_cast<const float*>(h_data[3]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(2.0f), FloatEq(3.0f), FloatEq(4.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[4]),
								   static_cast<const float*>(h_data[4]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(2.0f), FloatEq(3.0f), FloatEq(4.0f)));


	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[5]),
								   static_cast<const float*>(h_data[5]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(2.0f), FloatEq(0.0f), FloatEq(0.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[6]),
								   static_cast<const float*>(h_data[6]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(2.0f), FloatEq(3.0f), FloatEq(0.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[7]),
								   static_cast<const float*>(h_data[7]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(2.0f), FloatEq(3.0f), FloatEq(4.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[8]),
								   static_cast<const float*>(h_data[8]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(2.0f), FloatEq(3.0f), FloatEq(4.0f)));

	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[9]),
								   static_cast<const float*>(h_data[9]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(2.0f), FloatEq(3.0f), FloatEq(4.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[10]),
								   static_cast<const float*>(h_data[10]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(2.0f), FloatEq(3.0f), FloatEq(4.0f)));
}

TEST(VectorGPU, Operators)
{
	Vector4 h_data[13];
	Vector4* d_data;

	//
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[0]),
								   static_cast<const float*>(h_data[0]) + 4),
				ElementsAre(FloatEq(3.0f), FloatEq(3.0f), FloatEq(3.0f), FloatEq(3.0f)));

	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[1]),
								   static_cast<const float*>(h_data[1]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(1.0f), FloatEq(1.0f), FloatEq(1.0f)));

	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[2]),
								   static_cast<const float*>(h_data[2]) + 4),
				ElementsAre(FloatEq(4.0f), FloatEq(8.0f), FloatEq(12.0f), FloatEq(16.0f)));

	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[3]),
								   static_cast<const float*>(h_data[3]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(0.5f), FloatEq(0.3333333f), FloatEq(0.25f)));

	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[4]),
								   static_cast<const float*>(h_data[4]) + 4),
				ElementsAre(FloatEq(4.0f), FloatEq(4.0f), FloatEq(4.0f), FloatEq(4.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[5]),
								   static_cast<const float*>(h_data[5]) + 4),
				ElementsAre(FloatEq(4.0f), FloatEq(4.0f), FloatEq(4.0f), FloatEq(4.0f)));

	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[6]),
								   static_cast<const float*>(h_data[6]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(1.0f), FloatEq(1.0f), FloatEq(1.0f)));
	//
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[7]),
								   static_cast<const float*>(h_data[7]) + 4),
				ElementsAre(FloatEq(3.0f), FloatEq(3.0f), FloatEq(3.0f), FloatEq(3.0f)));

	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[8]),
								   static_cast<const float*>(h_data[8]) + 4),
				ElementsAre(FloatEq(-1.0f), FloatEq(-1.0f), FloatEq(-1.0f), FloatEq(-1.0f)));

	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[9]),
								   static_cast<const float*>(h_data[9]) + 4),
				ElementsAre(FloatEq(4.0f), FloatEq(8.0f), FloatEq(12.0f), FloatEq(16.0f)));

	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[10]),
								   static_cast<const float*>(h_data[10]) + 4),
				ElementsAre(FloatEq(2.0f), FloatEq(2.0f), FloatEq(2.0f), FloatEq(2.0f)));

	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[11]),
								   static_cast<const float*>(h_data[11]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(0.5f), FloatEq(0.3333333f), FloatEq(0.25f)));

	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[12]),
								   static_cast<const float*>(h_data[12]) + 4),
				ElementsAre(FloatEq(0.5f), FloatEq(0.5f), FloatEq(0.5f), FloatEq(0.5f)));
}

TEST(VectorGPU, Functions1)
{
	float h_data_f[3];
	float* d_data_f;

	Vector4 h_data_v4[6];
	Vector4* d_data_v4;

	Vector3 h_data_v3;
	Vector3* d_data;


	//
	EXPECT_FLOAT_EQ(8.0f, h_data_f[0]);
	EXPECT_FLOAT_EQ(5.0f, h_data_f[1]);
	EXPECT_FLOAT_EQ(25.0f, h_data_f[2]);
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data_v4[0]),
								   static_cast<const float*>(h_data_v4[0]) + 4),
				ElementsAre(FloatEq(0.18257418f), FloatEq(0.36514837f), FloatEq(0.54772255f), FloatEq(0.73029674f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data_v4[1]),
								   static_cast<const float*>(h_data_v4[1]) + 4),
				ElementsAre(FloatEq(3.0f), FloatEq(4.0f), FloatEq(4.0f), FloatEq(4.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data_v4[2]),
								   static_cast<const float*>(h_data_v4[2]) + 4),
				ElementsAre(FloatEq(3.0f), FloatEq(4.0f), FloatEq(4.0f), FloatEq(4.0f)));
	//
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data_v4[3]),
								   static_cast<const float*>(h_data_v4[3]) + 4),
				ElementsAre(FloatEq(0.18257418f), FloatEq(0.36514837f), FloatEq(0.54772255f), FloatEq(0.73029674f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data_v4[4]),
								   static_cast<const float*>(h_data_v4[4]) + 4),
				ElementsAre(FloatEq(3.0f), FloatEq(4.0f), FloatEq(4.0f), FloatEq(4.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data_v4[5]),
								   static_cast<const float*>(h_data_v4[5]) + 4),
				ElementsAre(FloatEq(3.0f), FloatEq(4.0f), FloatEq(4.0f), FloatEq(4.0f)));
	//
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data_v3),
								   static_cast<const float*>(h_data_v3) + 3),
				ElementsAre(FloatEq(-1.0f), FloatEq(2.0f), FloatEq(-1.0f)));

}

TEST(VectorGPU, Functions2)
{
	Vector4 h_data[8];
	Vector4* d_data;


	//
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[0]),
								   static_cast<const float*>(h_data[0]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(0.0f), FloatEq(-0.0f), FloatEq(1.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[1]),
								   static_cast<const float*>(h_data[1]) + 4),
				ElementsAre(FloatEq(-2.0f), FloatEq(-3.0f), FloatEq(3.0f), FloatEq(2.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[2]),
								   static_cast<const float*>(h_data[2]) + 4),
				ElementsAre(FloatEq(-3.0f), FloatEq(-3.0f), FloatEq(2.0f), FloatEq(2.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[3]),
								   static_cast<const float*>(h_data[3]) + 4),
				ElementsAre(FloatEq(-2.0f), FloatEq(-2.0f), FloatEq(3.0f), FloatEq(3.0f)));
	//
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[4]),
								   static_cast<const float*>(h_data[4]) + 4),
				ElementsAre(FloatEq(1.0f), FloatEq(0.0f), FloatEq(-0.0f), FloatEq(1.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[5]),
								   static_cast<const float*>(h_data[5]) + 4),
				ElementsAre(FloatEq(-2.0f), FloatEq(-3.0f), FloatEq(3.0f), FloatEq(2.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[6]),
								   static_cast<const float*>(h_data[6]) + 4),
				ElementsAre(FloatEq(-3.0f), FloatEq(-3.0f), FloatEq(2.0f), FloatEq(2.0f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[7]),
								   static_cast<const float*>(h_data[7]) + 4),
				ElementsAre(FloatEq(-2.0f), FloatEq(-2.0f), FloatEq(3.0f), FloatEq(3.0f)));
}

TEST(VectorGPU, Functions3)
{
	Vector4 h_data[5];
	Vector4* d_data;


	//
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[0]),
								   static_cast<const float*>(h_data[0]) + 4),
				ElementsAre(FloatEq(2.12f), FloatEq(2.5f), FloatEq(2.6f), FloatEq(2.3f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[1]),
								   static_cast<const float*>(h_data[1]) + 4),
				ElementsAre(FloatEq(999.0f), FloatEq(999.0f), FloatEq(999.0f), FloatEq(999.0f)));

	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[2]),
								   static_cast<const float*>(h_data[2]) + 4),
				ElementsAre(FloatEq(-2.12f), FloatEq(-2.5f), FloatEq(-2.6f), FloatEq(-2.3f)));
	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[3]),
								   static_cast<const float*>(h_data[3]) + 4),
				ElementsAre(FloatEq(-999.0f), FloatEq(-999.0f), FloatEq(-999.0f), FloatEq(-999.0f)));

	EXPECT_THAT(std::vector<float>(static_cast<const float*>(h_data[4]),
								   static_cast<const float*>(h_data[4]) + 4),
				ElementsAre(FloatEq(0.5f), FloatEq(0.5f), FloatEq(0.5f), FloatEq(0.5f)));
}