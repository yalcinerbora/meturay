#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <gmock/gmock.h>
#include <random>
#include <chrono>
using namespace std::chrono_literals;

#include "RayLib/Log.h"
#include "RayLib/TracerError.h"

#include "TracerLib/TracerBase.h"
#include "TracerLib/TracerLogicI.h"
#include "TracerLib/GPUAcceleratorI.h"
#include "TracerLib/GPUMaterialI.h"

#include "TracerLib/Random.cuh"
#include "TracerLib/RNGMemory.h"
#include "TracerLib/RayMemory.h"

#include "TracerLib/CameraKernels.cuh"

#include "TracerLib/TracerLoader.h"

struct RayAuxGMem {};
struct RayAuxBaseData{};

//template <class RayAuxGMem, class RayAuxBaseData>
__device__ void AuxInitEmpty(const RayAuxGMem,
							 const uint32_t writeLoc,
							 // Input
							 const RayAuxBaseData,
							 // Index
							 const Vector2ui& globalPixelId,
							 const Vector2ui& localSampleId,
							 const uint32_t samplePerPixel)
{}

class MockTracerLogic : public TracerBaseLogicI
{
	public:
		class BaseAcceleratorMock : public GPUBaseAcceleratorI
		{
			private:
				MockTracerLogic& 	mockLogic;

			public:
				// Constructors & Destructor
									BaseAcceleratorMock(MockTracerLogic& r) : mockLogic(r) {}
									~BaseAcceleratorMock() = default;
				
				// Type(as string) of the accelerator group
				const char*			Type() const override { return "MockBaseAccel"; }
				// KC
				void				Hit(// Output
										TransformId* dTransformIds,
										HitKey* dAcceleratorKeys,
										// Inputs
										const RayGMem* dRays,
										const RayId* dRayIds,
										const uint32_t rayCount) const override;

				void			Constrcut(// List of allocator hitkeys of surfaces
										  const std::map<uint32_t, HitKey>&,
										  // List of all Surface/Transform pairs
										  // that will be constructed
										  const std::map<uint32_t, uint32_t>&) override {};
				void			Reconstruct(// List of allocator hitkeys of surfaces
											const std::map<uint32_t, HitKey>&,
											// List of changed Surface/Transform pairs
											const std::map<uint32_t, uint32_t>&) override {};
		};

		class AcceleratorMock : public GPUAcceleratorBatchI
		{
			private:
				MockTracerLogic& 				mockLogic;
				uint32_t						myKey;
				const void*						groupNullPtr = nullptr;

			public:
				// Constructors & Destructor
												AcceleratorMock(MockTracerLogic& r, uint32_t myKey) 
													: mockLogic(r), myKey(myKey)  {}
												~AcceleratorMock() = default;

				// Type(as string) of the accelerator group
				const char*						Type() const override { return "MockAccelBatch"; }
				// KC
				void							Hit(// O
													HitKey* dMaterialKeys,
													PrimitiveId* dPrimitiveIds,
													HitStructPtr dHitStructs,
													// I-O													
													RayGMem* dRays,
													// Input
													const TransformId* dTransformIds,
													const RayId* dRayIds,
													const HitKey* dAcceleratorKeys,
													const uint32_t rayCount) const override;

				const GPUPrimitiveGroupI&		PrimitiveGroup() const override;
				const GPUAcceleratorGroupI&		AcceleratorGroup() const override;
		};

		class MaterialMock : public GPUMaterialBatchI
		{
			private:
				MockTracerLogic& 	mockLogic;
				bool				isMissMaterial;
				const void*			groupNullPtr = nullptr;

			public:
				// Constructors & Destructor
											MaterialMock(MockTracerLogic& r, bool missMat) 
												: mockLogic(r)
												, isMissMaterial(missMat) {}
											~MaterialMock() = default;

				// Type(as string) of the accelerator group
				const char*					Type() const override { return "MockMatBatch"; }
				// KC
				void						ShadeRays(// Output
													  RayGMem* dRayOut,
													  void* dRayAuxOut,
													  //  Input
													  const RayGMem* dRayIn,
													  const void* dRayAuxIn,
													  const PrimitiveId* dPrimitiveIds,
													  const HitStructPtr dHitStructs,
													  //
													  const HitKey* dMatIds,
													  const RayId* dRayIds,

													  const uint32_t rayCount,
													  RNGMemory& rngMem) const override;

				// Every MaterialBatch is available for a specific primitive / material data
				const GPUPrimitiveGroupI&	PrimitiveGroup() const override;
				const GPUMaterialGroupI&	MaterialGroup() const override;

				uint8_t						OutRayCount() const override { return isMissMaterial ? 0 : 1; }
		};

	private:
		std::mt19937								rng;	
		std::uniform_int_distribution<>				matIndexGenerator;

		uint32_t									seed;

		HitOpts										optsHit;
		ShadeOpts									optsShade;

		static constexpr Vector2i					MaterialRange = Vector2i(0, 24);
		static constexpr Vector2i					AcceleratorRange = Vector2i(24, 32);

		static const std::string					HitName;
		static const std::string					ShadeName;

		// Mock Implementations
		std::unique_ptr<BaseAcceleratorMock>		baseAccelerator;
		std::vector<AcceleratorMock>				mockAccelerators;
		std::vector<MaterialMock>					mockMaterials;

		AcceleratorBatchMappings					accelerators;
		MaterialBatchMappings						materials;
		
		// Convenience
		std::vector<HitKey>							materialKeys;
		std::vector<HitKey>							acceleratorKeys;
			   
		static constexpr int						AcceleratorCount = 2;
		static constexpr int						MaterialCount = 4;

	protected:
	public:
		// Constructors & Destructor
													MockTracerLogic(uint32_t seed);
		virtual										~MockTracerLogic() = default;

		// Init & Load
		TracerError									Initialize() override;
	
		// Generate Camera Rays
		void										GenerateCameraRays(RayMemory&, RNGMemory&,
																	   const CameraPerspective& camera,
																	   const uint32_t samplePerPixel,
																	   const Vector2ui& resolution,
																	   const Vector2ui& pixelStart,
																	   const Vector2ui& pixelCount) override;
		void										GenerateRays(RayMemory&, RNGMemory&,
																 const uint32_t rayCount) override;

		

		// Interface fetching for logic
		const GPUBaseAcceleratorI&					BaseAcelerator() override { return *baseAccelerator; }
		const AcceleratorBatchMappings&				AcceleratorBatches() override { return accelerators; }
		const MaterialBatchMappings&				MaterialBatches() override { return materials; }

		// Returns max bits of keys (for batch and id respectively)
		const Vector2i								SceneMaterialMaxBits() const override { return MaterialRange; }
		const Vector2i								SceneAcceleratorMaxBits() const override { return AcceleratorRange; }

		// Options of the Hitman & Shademan
		const HitOpts&								HitOptions() const override { return optsHit; }
		const ShadeOpts&							ShadeOptions() const override { return optsShade; }

		// Misc
		// Retuns "sizeof(RayAux)"
		size_t										PerRayAuxDataSize() const override { return 0; }
		// Return mimimum size of an arbitrary struct which holds all hit results
		size_t										HitStructSize() const override { return sizeof(uint32_t); };
};

const GPUPrimitiveGroupI& MockTracerLogic::AcceleratorMock::PrimitiveGroup() const 
{ 
	return *static_cast<const GPUPrimitiveGroupI*>(groupNullPtr);
}

const GPUAcceleratorGroupI& MockTracerLogic::AcceleratorMock::AcceleratorGroup() const
{
	return *static_cast<const GPUAcceleratorGroupI*>(groupNullPtr);
}

const GPUPrimitiveGroupI& MockTracerLogic::MaterialMock::PrimitiveGroup() const
{
	return *static_cast<const GPUPrimitiveGroupI*>(groupNullPtr);
}

const GPUMaterialGroupI& MockTracerLogic::MaterialMock::MaterialGroup() const
{
	return *static_cast<const GPUMaterialGroupI*>(groupNullPtr);
}

const std::string MockTracerLogic::HitName = "";
const std::string MockTracerLogic::ShadeName = "";

void MockTracerLogic::BaseAcceleratorMock::Hit(// Output
											   TransformId* dTransformIds,
											   HitKey* dAcceleratorKeys,
											   // Inputs
											   const RayGMem* dRays,
											   const RayId* dRayIds,
											   const uint32_t rayCount) const
{
	// Go To CPU
	CUDA_CHECK(hipDeviceSynchronize());

	METU_LOG("-----------------------------");

	// Delegate Stuff Interleaved
	for(uint32_t i = 0; i < rayCount; i++)
	{
		// Key index is used to acces RayGMem (this program does not care about actual ray)
		//uint32_t keyIndex = dRayIds[i];

		// Each Iteration some of the rays are missed (only first ray in this case)
		uint32_t index = i % (AcceleratorCount * MaterialCount + 1);
		if(index == 0)
			dAcceleratorKeys[i] = HitKey::OutsideMatKey;
		else
			dAcceleratorKeys[i] = mockLogic.acceleratorKeys[index - 1];
	}
}

void MockTracerLogic::AcceleratorMock::Hit(// O
										   HitKey* dMaterialKeys,
										   PrimitiveId* dPrimitiveIds,
										   HitStructPtr dHitStructs,
										   // I-O													
										   RayGMem* dRays,
										   // Input
										   const TransformId* dTransformIds,
										   const RayId* dRayIds,
										   const HitKey* dAcceleratorKeys,
										   const uint32_t rayCount) const
{
	// Go To CPU
	CUDA_CHECK(hipDeviceSynchronize());

	// Each Individual Hit segment writes the actual hit result
	METU_LOG("Stub Accelerator Work %u", rayCount);
	std::stringstream s;	
	for(uint32_t i = 0; i < rayCount; i++)
	{
		RayId rayId = dRayIds[i];
		printf("%d, ", rayId);


		double random01 = static_cast<double>(mockLogic.rng()) /
						  static_cast<double>(mockLogic.rng.max());

		// %50 Make it hit
		if(random01 <= 0.5)
		{
			// We found a hit,
			// Randomly select a material for hit			
			HitKey materialId = mockLogic.materialKeys[mockLogic.matIndexGenerator(mockLogic.rng)];
			dMaterialKeys[rayId] = materialId;
			// Put primitive id
			dPrimitiveIds[rayId] = 0;
			// Put a struct
			struct Test
			{
				int i;
			};
			//Test a;
			//dHitStructs[static_cast<int>(i)] = a;
			
		}
	}
	printf("\n");
}

void MockTracerLogic::MaterialMock::ShadeRays(// Output
											  RayGMem* dRayOut,
											  void* dRayAuxOut,
											  //  Input
											  const RayGMem* dRayIn,
											  const void* dRayAuxIn,
											  const PrimitiveId* dPrimitiveIds,
											  const HitStructPtr dHitStructs,
											  //
											  const HitKey* dMatIds,
											  const RayId* dRayIds,

											  const uint32_t rayCount,
											  RNGMemory& rngMem) const
{
	// Go To CPU
	CUDA_CHECK(hipDeviceSynchronize());

	METU_LOG("Stub Material Work %u", rayCount);
	for(uint32_t i = 0; i < rayCount; i++)
	{
		RayId rayId = dRayIds[i];
		printf("%d, ", rayId);
	}
	printf("\n");
}

TracerError MockTracerLogic::Initialize()
{
	// Initialize Single Here Also
	TracerError e(TracerError::END);
	if((e = CudaSystem::Initialize()) != TracerError::OK)
	{
		return e;
	}

	rng.seed(seed);
	baseAccelerator = std::make_unique<BaseAcceleratorMock>(*this);

	// Generate Accelerators and Id Mappings
	// Be careful pointers will be invalidated 
	// if vector reallocates. Thus pre-allocate for all
	// the data.
	mockAccelerators.reserve(AcceleratorCount);
	mockMaterials.reserve((MaterialCount * AcceleratorCount) + 1);
	for(int i = 0; i < AcceleratorCount; i++)
	{
		mockAccelerators.emplace_back(*this, static_cast<uint32_t>(i));
		accelerators.emplace(std::make_pair(static_cast<uint32_t>(i),
											&mockAccelerators.back()));

		for(int j = 0; j < MaterialCount; j++)
		{
			int combinedIndex = i << MaterialRange[1];
			combinedIndex |= j;

			mockMaterials.emplace_back(*this, false);
			materials.emplace(std::make_pair(static_cast<uint32_t>(combinedIndex),
											 &mockMaterials.back()));
			materialKeys.emplace_back(static_cast<uint32_t>(combinedIndex));
		}
	}

	// Create miss material
	mockMaterials.emplace_back(*this, true);
	materials.emplace(std::make_pair(static_cast<uint32_t>(HitKey::OutsideMatKey),
									 &mockMaterials.back()));

	// We have total of 8 material seperated by 2 accelerators
	return TracerError::OK;
}


void MockTracerLogic::GenerateCameraRays(RayMemory& rMem,
										 RNGMemory& rngMemory,
										 const CameraPerspective& camera,
										 const uint32_t samplePerPixel,
										 const Vector2ui& resolution,
										 const Vector2ui& pixelStart,
										 const Vector2ui& pixelCount)
{
	RayAuxGMem rAux;
	RayAuxBaseData rAuxBase;

	// Camera Ray Generation Kernel Check
	constexpr int GPUId = 0;
	CudaSystem::GPUCallX(GPUId, rngMemory.SharedMemorySize(GPUId), 0,
						 KCGenerateCameraRays<RayAuxGMem, RayAuxBaseData, AuxInitEmpty>,
						 rMem.RaysOut(),
						 rAux,
						 // Input
						 rngMemory.RNGData(GPUId),
						 camera,
						 samplePerPixel,
						 resolution,
						 pixelStart,
						 pixelCount,
						 //
						 rAuxBase);
	// We do not use this actual data but w/e
}

void MockTracerLogic::GenerateRays(RayMemory&, RNGMemory&,
								   const uint32_t rayCount)
{}

MockTracerLogic::MockTracerLogic(uint32_t seed)
	: seed(seed)
	, matIndexGenerator(0, MaterialCount - 1)
{}

TEST(MockTracerTest, Test)
{
	constexpr Vector2ui resolution = Vector2ui(3, 3);
	constexpr uint32_t seed = 0;

	// Create our mock
	MockTracerLogic mockLogic(seed);

	// Load Tracer DLL
	TracerBase tracer;
	TracerI* tracerI = &tracer;
	tracerI->Initialize(seed, mockLogic);
	tracerI->ResizeImage(resolution);
	tracerI->ReportionImage();

	// Generate Camera Rays
	// Mock Tracer only cares about pixel count and sample size
	tracerI->GenerateCameraRays(CameraPerspective{}, 1);
	// Loop until all rays are processed
	while(tracerI->Continue())
	{
		tracerI->Render();
	}
	tracerI->FinishSamples();

	// All Done!!
}