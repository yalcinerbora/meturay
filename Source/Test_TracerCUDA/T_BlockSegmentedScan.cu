#include "hip/hip_runtime.h"
#include "TracerCUDA/BlockSegmentedScan.cuh"
#include "TracerCUDA/CudaSystem.h"
#include "TracerCUDA/CudaSystem.hpp"
#include "TracerCUDA/DeviceMemory.h"

#include "RayLib/Constants.h"

#include <gtest/gtest.h>
#include <random>
#include <numeric>

template <class T, uint32_t TPB, uint32_t SEGMENT_SIZE>
__global__ __launch_bounds__(TPB)
void KCSegmentedInclusiveScanTest(T* gOut,
                                  T* gSegmentAggregates,
                                  const T* gData)
{
    static_assert(T() == static_cast<T>(0));

    // Block Segmented Scan Operating Class
    using BSegScan = BlockSegmentedScan<T, TPB, SEGMENT_SIZE>;

    // Shared Memory that is required by the scan
    __shared__ typename BSegScan::TempStorage sMem;

    const uint32_t globalId = threadIdx.x + blockDim.x * blockIdx.x;
    T myData = gData[globalId];

    T scanResult, segmentAggregates;
    BSegScan(sMem).InclusiveSum(scanResult, segmentAggregates, myData, T());

    gSegmentAggregates[globalId] = segmentAggregates;
    gOut[globalId] = scanResult;
}

template <class T, uint32_t TPB, uint32_t SEGMENT_SIZE>
__global__ __launch_bounds__(TPB)
void KCSegmentedExclusiveScanTest(T* gOut,
                                  T* gSegmentAggregates,
                                  const T* gData)
{
    static_assert(T() == static_cast<T>(0));

    // Block Segmented Scan Operating Class
    using BSegScan = BlockSegmentedScan<T, TPB, SEGMENT_SIZE>;

    // Shared Memory that is required by the scan
    __shared__ typename BSegScan::TempStorage sMem;

    const uint32_t globalId = threadIdx.x + blockDim.x * blockIdx.x;
    T myData = gData[globalId];

    T scanResult, segmentAggregates;
    BSegScan(sMem).ExclusiveSum(scanResult, segmentAggregates, myData, T());

    gSegmentAggregates[globalId] = segmentAggregates;
    gOut[globalId] = scanResult;
}

template <uint32_t TPB_VAL, uint32_t SEGMENT_SIZE_VAL>
struct BlockScanTestParams
{
    static constexpr auto TPB           = TPB_VAL;
    static constexpr auto SEGMENT_SIZE  = SEGMENT_SIZE_VAL;
    static constexpr auto SEGMENT_COUNT = TPB / SEGMENT_SIZE;
};

template <class T>
class BlockSegScanTest : public testing::Test
{};

using Implementations = ::testing::Types<BlockScanTestParams<512, 64>,
                                         BlockScanTestParams<64, 32>,
                                         BlockScanTestParams<128, 64>,
                                         BlockScanTestParams<64, 16>,
                                         BlockScanTestParams<64, 8>>;

TYPED_TEST_SUITE(BlockSegScanTest, Implementations);

TYPED_TEST(BlockSegScanTest, FloatInclusiveSumBasic)
{
    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());

    constexpr auto TPB = TypeParam::TPB;
    constexpr auto SEGMENT_SIZE = TypeParam::SEGMENT_SIZE;

    // Copy all ones to GPU
    std::vector<float> data(TypeParam::TPB, 1.0f);

    // GPU Allocations
    float* dData;
    float* dScanOutputs;
    float* dSegmentAggregates;
    DeviceMemory mem;
    GPUMemFuncs::AllocateMultiData(std::tie(dData, dScanOutputs, dSegmentAggregates),
                                   mem, {TPB, TPB, TPB});

    CUDA_CHECK(hipMemcpy(dData, data.data(), sizeof(float) * TPB,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(dScanOutputs, 0xFF, sizeof(float) * TPB));
    CUDA_CHECK(hipMemset(dSegmentAggregates, 0xFF, sizeof(float) * TPB));

    // Scan Call
    const CudaGPU& bestGPU = system.BestGPU();
    bestGPU.ExactKC_X(0, (hipStream_t)0, TPB, 1,
                      //
                      KCSegmentedInclusiveScanTest<float, TPB, SEGMENT_SIZE>,
                      //
                      dScanOutputs,
                      dSegmentAggregates,
                      dData);

    // Copy to Host to check
    std::vector<float> hSegmentAggregates(TPB);
    std::vector<float> hScanResults(TPB);
    CUDA_CHECK(hipMemcpy(hSegmentAggregates.data(),
                          dSegmentAggregates,
                          sizeof(float) * TPB,
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hScanResults.data(),
                          dScanOutputs,
                          sizeof(float) * TPB,
                          hipMemcpyDeviceToHost));

    // Checks
    for(float r : hSegmentAggregates)
    {
        EXPECT_FLOAT_EQ(r, static_cast<float>(SEGMENT_SIZE));
    }

    for(uint32_t i = 0; i < static_cast<uint32_t>(hScanResults.size()); i++)
    {
        float r = hScanResults[i];
        uint32_t segmentLocalId = i % SEGMENT_SIZE;
        EXPECT_FLOAT_EQ(r, static_cast<float>(segmentLocalId + 1));
    }
}

TYPED_TEST(BlockSegScanTest, FloatInclusiveSumStress)
{
    static constexpr uint32_t ITERATION_COUNT = 100;

    static constexpr auto TPB = TypeParam::TPB;
    static constexpr auto SEGMENT_SIZE = TypeParam::SEGMENT_SIZE;
    static constexpr auto SEGMENT_COUNT = TypeParam::SEGMENT_COUNT;

    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());
    // CPU Allocations
    std::vector<float> hData(TPB, 1.0f);
    std::vector<float> hSegmentAggregates(TPB);
    std::vector<float> hScanResults(TPB);
    std::vector<float> hSegAggregatesExpected(TPB);
    std::vector<float> hScanResultsExpected(TPB);
    // GPU Allocations
    float* dData;
    float* dScanOutputs;
    float* dSegmentAggregates;
    DeviceMemory mem;
    GPUMemFuncs::AllocateMultiData(std::tie(dData, dScanOutputs, dSegmentAggregates),
                                   mem, {TPB, TPB, TPB});


    std::mt19937 rng;
    rng.seed(0);
    std::uniform_real_distribution<float> uniformDist(0.0f, 10.0f);

    for(uint32_t ik = 0; ik < ITERATION_COUNT; ik++)
    {
        // Generate new batch of random numbers
        for(float& d : hData)
        {
            d = uniformDist(rng);
        }

        CUDA_CHECK(hipMemcpy(dData, hData.data(), sizeof(float) * TPB,
                              hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemset(dScanOutputs, 0xFF, sizeof(float) * TPB));
        CUDA_CHECK(hipMemset(dSegmentAggregates, 0xFF, sizeof(float) * TPB));

        // Scan Call
        const CudaGPU& bestGPU = system.BestGPU();
        bestGPU.ExactKC_X(0, (hipStream_t)0, TPB, 1,
                          //
                          KCSegmentedInclusiveScanTest<float, TPB, SEGMENT_SIZE>,
                          //
                          dScanOutputs,
                          dSegmentAggregates,
                          dData);

        // Copy to Host to check
        CUDA_CHECK(hipMemcpy(hSegmentAggregates.data(),
                              dSegmentAggregates,
                              sizeof(float) * TPB,
                              hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(hScanResults.data(),
                              dScanOutputs,
                              sizeof(float) * TPB,
                              hipMemcpyDeviceToHost));

        // Generate results by hand to check
        for(uint32_t i = 0; i < SEGMENT_COUNT; i++)
        {
            uint32_t segmentStart = i * SEGMENT_SIZE;
            uint32_t nextSegmentStart = (i + 1) * SEGMENT_SIZE;

            std::inclusive_scan(hData.begin() + segmentStart,
                                hData.begin() + nextSegmentStart,
                                hScanResultsExpected.begin() + segmentStart);
        }
        hSegAggregatesExpected.clear();
        for(uint32_t i = 1; i <= SEGMENT_COUNT; i++)
        {
            uint32_t segmentAggregateIndex = (i * SEGMENT_SIZE) - 1;
            hSegAggregatesExpected.insert(hSegAggregatesExpected.end(),
                                          SEGMENT_SIZE,
                                          hScanResultsExpected[segmentAggregateIndex]);
        }
        // Checks
        for(uint32_t i = 0; i < TPB; i++)
        {
            float result = hScanResults[i];
            float expected = hScanResultsExpected[i];
            EXPECT_NEAR(result, expected, MathConstants::VeryLargeEpsilon);
        }
        for(uint32_t i = 0; i < TPB; i++)
        {
            float result = hSegmentAggregates[i];
            float expected = hSegAggregatesExpected[i];
            EXPECT_NEAR(result, expected, MathConstants::VeryLargeEpsilon);
        }
    }
}

TYPED_TEST(BlockSegScanTest, IntInclusiveSumStress)
{
    static constexpr uint32_t ITERATION_COUNT = 100;

    static constexpr auto TPB = TypeParam::TPB;
    static constexpr auto SEGMENT_SIZE = TypeParam::SEGMENT_SIZE;
    static constexpr auto SEGMENT_COUNT = TypeParam::SEGMENT_COUNT;

    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());
    // CPU Allocations
    std::vector<uint32_t> hData(TPB, 1u);
    std::vector<uint32_t> hSegmentAggregates(TPB);
    std::vector<uint32_t> hScanResults(TPB);
    std::vector<uint32_t> hSegAggregatesExpected(TPB);
    std::vector<uint32_t> hScanResultsExpected(TPB);
    // GPU Allocations
    uint32_t* dData;
    uint32_t* dScanOutputs;
    uint32_t* dSegmentAggregates;
    DeviceMemory mem;
    GPUMemFuncs::AllocateMultiData(std::tie(dData, dScanOutputs, dSegmentAggregates),
                                   mem, {TPB, TPB, TPB});


    std::mt19937 rng;
    rng.seed(0);
    std::uniform_int_distribution<uint32_t> uniformDist(0u, 10u);

    for(uint32_t ik = 0; ik < ITERATION_COUNT; ik++)
    {
        // Generate new batch of random numbers
        for(uint32_t& d : hData)
        {
            d = uniformDist(rng);
        }

        CUDA_CHECK(hipMemcpy(dData, hData.data(), sizeof(uint32_t) * TPB,
                              hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemset(dScanOutputs, 0xFF, sizeof(uint32_t) * TPB));
        CUDA_CHECK(hipMemset(dSegmentAggregates, 0xFF, sizeof(uint32_t) * TPB));

        // Scan Call
        const CudaGPU& bestGPU = system.BestGPU();
        bestGPU.ExactKC_X(0, (hipStream_t)0, TPB, 1,
                          //
                          KCSegmentedInclusiveScanTest<uint32_t, TPB, SEGMENT_SIZE>,
                          //
                          dScanOutputs,
                          dSegmentAggregates,
                          dData);

        // Copy to Host to check
        CUDA_CHECK(hipMemcpy(hSegmentAggregates.data(),
                              dSegmentAggregates,
                              sizeof(uint32_t) * TPB,
                              hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(hScanResults.data(),
                              dScanOutputs,
                              sizeof(uint32_t) * TPB,
                              hipMemcpyDeviceToHost));

        // Generate results by hand to check
        for(uint32_t i = 0; i < SEGMENT_COUNT; i++)
        {
            uint32_t segmentStart = i * SEGMENT_SIZE;
            uint32_t nextSegmentStart = (i + 1) * SEGMENT_SIZE;

            std::inclusive_scan(hData.begin() + segmentStart,
                                hData.begin() + nextSegmentStart,
                                hScanResultsExpected.begin() + segmentStart);
        }
        hSegAggregatesExpected.clear();
        for(uint32_t i = 1; i <= SEGMENT_COUNT; i++)
        {
            uint32_t segmentAggregateIndex = (i * SEGMENT_SIZE) - 1;
            hSegAggregatesExpected.insert(hSegAggregatesExpected.end(),
                                          SEGMENT_SIZE,
                                          hScanResultsExpected[segmentAggregateIndex]);
        }
        // Checks
        for(uint32_t i = 0; i < TPB; i++)
        {
            uint32_t result = hScanResults[i];
            uint32_t expected = hScanResultsExpected[i];
            EXPECT_EQ(result, expected);
        }
        for(uint32_t i = 0; i < TPB; i++)
        {
            uint32_t result = hSegmentAggregates[i];
            uint32_t expected = hSegAggregatesExpected[i];
            EXPECT_EQ(result, expected);
        }
    }
}

TYPED_TEST(BlockSegScanTest, FloatExclusiveSumBasic)
{
    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());

    static constexpr auto TPB = TypeParam::TPB;
    static constexpr auto SEGMENT_SIZE = TypeParam::SEGMENT_SIZE;

    // Copy all ones to GPU
    std::vector<float> data(TPB, 1.0f);

    // GPU Allocations
    float* dData;
    float* dScanOutputs;
    float* dSegmentAggregates;
    DeviceMemory mem;
    GPUMemFuncs::AllocateMultiData(std::tie(dData, dScanOutputs, dSegmentAggregates),
                                   mem, {TPB, TPB, TPB});

    CUDA_CHECK(hipMemcpy(dData, data.data(), sizeof(float) * TPB,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(dScanOutputs, 0xFF, sizeof(float) * TPB));
    CUDA_CHECK(hipMemset(dSegmentAggregates, 0xFF, sizeof(float) * TPB));

    // Scan Call
    const CudaGPU& bestGPU = system.BestGPU();
    bestGPU.ExactKC_X(0, (hipStream_t)0, TPB, 1,
                      //
                      KCSegmentedExclusiveScanTest<float, TPB, SEGMENT_SIZE>,
                      //
                      dScanOutputs,
                      dSegmentAggregates,
                      dData);

    // Copy to Host to check
    std::vector<float> hSegmentAggregates(TPB);
    std::vector<float> hScanResults(TPB);
    CUDA_CHECK(hipMemcpy(hSegmentAggregates.data(),
                          dSegmentAggregates,
                          sizeof(float) * TPB,
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hScanResults.data(),
                          dScanOutputs,
                          sizeof(float) * TPB,
                          hipMemcpyDeviceToHost));

    // Checks
    for(float r : hSegmentAggregates)
    {
        EXPECT_FLOAT_EQ(r, static_cast<float>(SEGMENT_SIZE - 1));
    }

    for(uint32_t i = 0; i < static_cast<uint32_t>(hScanResults.size()); i++)
    {
        float r = hScanResults[i];
        uint32_t segmentLocalId = i % SEGMENT_SIZE;
        EXPECT_FLOAT_EQ(r, static_cast<float>(segmentLocalId));
    }
}

TYPED_TEST(BlockSegScanTest, FloatExclusiveSumStress)
{
    static constexpr uint32_t ITERATION_COUNT = 100;

    static constexpr auto TPB = TypeParam::TPB;
    static constexpr auto SEGMENT_SIZE = TypeParam::SEGMENT_SIZE;
    static constexpr auto SEGMENT_COUNT = TypeParam::SEGMENT_COUNT;

    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());
    // CPU Allocations
    std::vector<float> hData(TPB, 1.0f);
    std::vector<float> hSegmentAggregates(TPB);
    std::vector<float> hScanResults(TPB);
    std::vector<float> hSegAggregatesExpected(TPB);
    std::vector<float> hScanResultsExpected(TPB);
    // GPU Allocations
    float* dData;
    float* dScanOutputs;
    float* dSegmentAggregates;
    DeviceMemory mem;
    GPUMemFuncs::AllocateMultiData(std::tie(dData, dScanOutputs, dSegmentAggregates),
                                   mem, {TPB, TPB, TPB});


    std::mt19937 rng;
    rng.seed(0);
    std::uniform_real_distribution<float> uniformDist(0.0f, 10.0f);

    for(uint32_t ik = 0; ik < ITERATION_COUNT; ik++)
    {
        // Generate new batch of random numbers
        for(float& d : hData)
        {
            d = uniformDist(rng);
        }

        CUDA_CHECK(hipMemcpy(dData, hData.data(), sizeof(float) * TPB,
                              hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemset(dScanOutputs, 0xFF, sizeof(float) * TPB));
        CUDA_CHECK(hipMemset(dSegmentAggregates, 0xFF, sizeof(float) * TPB));

        // Scan Call
        const CudaGPU& bestGPU = system.BestGPU();
        bestGPU.ExactKC_X(0, (hipStream_t)0, TPB, 1,
                          //
                          KCSegmentedExclusiveScanTest<float, TPB, SEGMENT_SIZE>,
                          //
                          dScanOutputs,
                          dSegmentAggregates,
                          dData);

        // Copy to Host to check
        CUDA_CHECK(hipMemcpy(hSegmentAggregates.data(),
                              dSegmentAggregates,
                              sizeof(float) * TPB,
                              hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(hScanResults.data(),
                              dScanOutputs,
                              sizeof(float) * TPB,
                              hipMemcpyDeviceToHost));

        // Generate results by hand to check
        for(uint32_t i = 0; i < SEGMENT_COUNT; i++)
        {
            uint32_t segmentStart = i * SEGMENT_SIZE;
            uint32_t nextSegmentStart = (i + 1) * SEGMENT_SIZE;

            std::exclusive_scan(hData.begin() + segmentStart,
                                hData.begin() + nextSegmentStart,
                                hScanResultsExpected.begin() + segmentStart,
                                0.0f);
        }
        hSegAggregatesExpected.clear();
        for(uint32_t i = 1; i <= SEGMENT_COUNT; i++)
        {
            uint32_t segmentAggregateIndex = (i * SEGMENT_SIZE) - 1;
            hSegAggregatesExpected.insert(hSegAggregatesExpected.end(),
                                          SEGMENT_SIZE,
                                          hScanResultsExpected[segmentAggregateIndex]);
        }
        // Checks
        for(uint32_t i = 0; i < TPB; i++)
        {
            float result = hScanResults[i];
            float expected = hScanResultsExpected[i];
            EXPECT_NEAR(result, expected, MathConstants::VeryLargeEpsilon);
        }
        for(uint32_t i = 0; i < TPB; i++)
        {
            float result = hSegmentAggregates[i];
            float expected = hSegAggregatesExpected[i];
            EXPECT_NEAR(result, expected, MathConstants::VeryLargeEpsilon);
        }
    }
}

TYPED_TEST(BlockSegScanTest, IntExclusiveSumStress)
{
    static constexpr uint32_t ITERATION_COUNT = 100;
    static constexpr auto TPB = TypeParam::TPB;
    static constexpr auto SEGMENT_SIZE = TypeParam::SEGMENT_SIZE;
    static constexpr auto SEGMENT_COUNT = TypeParam::SEGMENT_COUNT;

    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());
    // CPU Allocations
    std::vector<uint32_t> hData(TPB, 1u);
    std::vector<uint32_t> hSegmentAggregates(TPB);
    std::vector<uint32_t> hScanResults(TPB);
    std::vector<uint32_t> hSegAggregatesExpected(TPB);
    std::vector<uint32_t> hScanResultsExpected(TPB);
    // GPU Allocations
    uint32_t* dData;
    uint32_t* dScanOutputs;
    uint32_t* dSegmentAggregates;
    DeviceMemory mem;
    GPUMemFuncs::AllocateMultiData(std::tie(dData, dScanOutputs, dSegmentAggregates),
                                   mem, {TPB, TPB, TPB});


    std::mt19937 rng;
    rng.seed(0);
    std::uniform_int_distribution<uint32_t> uniformDist(0u, 10u);

    for(uint32_t ik = 0; ik < ITERATION_COUNT; ik++)
    {
        // Generate new batch of random numbers
        for(uint32_t& d : hData)
        {
            d = uniformDist(rng);
        }

        CUDA_CHECK(hipMemcpy(dData, hData.data(), sizeof(uint32_t) * TPB,
                              hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemset(dScanOutputs, 0xFF, sizeof(uint32_t) * TPB));
        CUDA_CHECK(hipMemset(dSegmentAggregates, 0xFF, sizeof(uint32_t) * TPB));

        // Scan Call
        const CudaGPU& bestGPU = system.BestGPU();
        bestGPU.ExactKC_X(0, (hipStream_t)0, TPB, 1,
                          //
                          KCSegmentedExclusiveScanTest<uint32_t, TPB, SEGMENT_SIZE>,
                          //
                          dScanOutputs,
                          dSegmentAggregates,
                          dData);

        // Copy to Host to check
        CUDA_CHECK(hipMemcpy(hSegmentAggregates.data(),
                              dSegmentAggregates,
                              sizeof(uint32_t) * TPB,
                              hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(hScanResults.data(),
                              dScanOutputs,
                              sizeof(uint32_t) * TPB,
                              hipMemcpyDeviceToHost));

        // Generate results by hand to check
        for(uint32_t i = 0; i < SEGMENT_COUNT; i++)
        {
            uint32_t segmentStart = i * SEGMENT_SIZE;
            uint32_t nextSegmentStart = (i + 1) * SEGMENT_SIZE;

            std::exclusive_scan(hData.begin() + segmentStart,
                                hData.begin() + nextSegmentStart,
                                hScanResultsExpected.begin() + segmentStart,
                                0u);
        }
        hSegAggregatesExpected.clear();
        for(uint32_t i = 1; i <= SEGMENT_COUNT; i++)
        {
            uint32_t segmentAggregateIndex = (i * SEGMENT_SIZE) - 1;
            hSegAggregatesExpected.insert(hSegAggregatesExpected.end(),
                                          SEGMENT_SIZE,
                                          hScanResultsExpected[segmentAggregateIndex]);
        }
        // Checks
        for(uint32_t i = 0; i < TPB; i++)
        {
            uint32_t result = hScanResults[i];
            uint32_t expected = hScanResultsExpected[i];
            EXPECT_EQ(result, expected);
        }
        for(uint32_t i = 0; i < TPB; i++)
        {
            uint32_t result = hSegmentAggregates[i];
            uint32_t expected = hSegAggregatesExpected[i];
            EXPECT_EQ(result, expected);
        }
    }
}