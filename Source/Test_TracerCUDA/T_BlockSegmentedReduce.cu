#include "hip/hip_runtime.h"
#include "TracerCUDA/BlockSegmentedReduce.cuh"
#include "TracerCUDA/CudaSystem.h"
#include "TracerCUDA/CudaSystem.hpp"
#include "TracerCUDA/DeviceMemory.h"

#include "RayLib/Constants.h"

#include <gtest/gtest.h>
#include <random>
#include <numeric>

template <class T, uint32_t TPB, uint32_t SEGMENT_SIZE>
__global__ __launch_bounds__(TPB)
void KCSegmentedReduceTest(T* gOut,
                           const T* gData)
{
    static_assert(T() == static_cast<T>(0));
    const uint32_t globalId = threadIdx.x + blockDim.x * blockIdx.x;
    auto IsSegmentLeader = [&]() -> uint32_t
    {
        return globalId % SEGMENT_SIZE == 0;
    };
    auto SegmentId = [&]() -> uint32_t
    {
        return globalId / SEGMENT_SIZE;
    };

    // Block Segmented Scan Operating Class
    using BSegReduce = BlockSegmentedReduce<T, TPB, SEGMENT_SIZE>;

    // Shared Memory that is required by the scan
    __shared__ typename BSegReduce::TempStorage sMem;

    T myData = gData[globalId];
    T reduceResult = BSegReduce(sMem).Sum(myData, T());

    if(IsSegmentLeader())
        gOut[SegmentId()] = reduceResult;
}

template <uint32_t TPB_VAL, uint32_t SEGMENT_SIZE_VAL>
struct BlockReduceTestParams
{
    static constexpr auto TPB = TPB_VAL;
    static constexpr auto SEGMENT_SIZE = SEGMENT_SIZE_VAL;
    static constexpr auto SEGMENT_COUNT = TPB / SEGMENT_SIZE;
};

template <class T>
class BlockSegReduceTest : public testing::Test
{};

using Implementations = ::testing::Types<BlockReduceTestParams<64, 32>,
                                         BlockReduceTestParams<128, 64>,
                                         BlockReduceTestParams<64, 16>,
                                         BlockReduceTestParams<64, 8>>;

TYPED_TEST_SUITE(BlockSegReduceTest, Implementations);

TYPED_TEST(BlockSegReduceTest, FloatSumBasic)
{
    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());

    constexpr auto TPB = TypeParam::TPB;
    constexpr auto SEGMENT_SIZE = TypeParam::SEGMENT_SIZE;
    constexpr auto SEGMENT_COUNT = TypeParam::SEGMENT_COUNT;

    // Copy all ones to GPU
    std::vector<float> data(TPB, 1.0f);

    // GPU Allocations
    float* dData;
    float* dReduceOutputs;
    DeviceMemory mem;
    GPUMemFuncs::AllocateMultiData(std::tie(dData, dReduceOutputs),
                                   mem, {TPB, SEGMENT_COUNT});
    CUDA_CHECK(hipMemcpy(dData, data.data(), sizeof(float) * TPB,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(dReduceOutputs, 0xFF, sizeof(float) * SEGMENT_COUNT));

    // Scan Call
    const CudaGPU& bestGPU = system.BestGPU();
    bestGPU.ExactKC_X(0, (hipStream_t)0, TPB, 1,
                      //
                      KCSegmentedReduceTest<float, TPB, SEGMENT_SIZE>,
                      //
                      dReduceOutputs,
                      dData);

    // Copy to Host to check
    std::vector<float> hReduceResults(SEGMENT_COUNT);
    CUDA_CHECK(hipMemcpy(hReduceResults.data(),
                          dReduceOutputs,
                          sizeof(float) * SEGMENT_COUNT,
                          hipMemcpyDeviceToHost));

    // Checks
    for(float r : hReduceResults)
    {
        EXPECT_FLOAT_EQ(r, static_cast<float>(SEGMENT_SIZE));
    }
}

TYPED_TEST(BlockSegReduceTest, FloatSumStress)
{
    static constexpr uint32_t ITERATION_COUNT = 100;
    constexpr auto TPB = TypeParam::TPB;
    constexpr auto SEGMENT_SIZE = TypeParam::SEGMENT_SIZE;
    constexpr auto SEGMENT_COUNT = TypeParam::SEGMENT_COUNT;

    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());
    // CPU Allocations
    std::vector<float> hData(TPB, 1.0f);
    std::vector<float> hReduceResults(SEGMENT_COUNT);
    std::vector<float> hReduceResultsExpected(SEGMENT_COUNT);
    // GPU Allocations
    float* dData;
    float* dReduceOutputs;
    DeviceMemory mem;
    GPUMemFuncs::AllocateMultiData(std::tie(dData, dReduceOutputs),
                                   mem, {TPB, SEGMENT_COUNT});


    std::mt19937 rng;
    rng.seed(0);
    std::uniform_real_distribution<float> uniformDist(0.0f, 10.0f);

    for(uint32_t ik = 0; ik < ITERATION_COUNT; ik++)
    {
        // Generate new batch of random numbers
        for(float& d : hData)
        {
            d = uniformDist(rng);
        }

        CUDA_CHECK(hipMemcpy(dData, hData.data(), sizeof(float) * TPB,
                              hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemset(dReduceOutputs, 0xFF, sizeof(float) * SEGMENT_COUNT));

        // Scan Call
        const CudaGPU& bestGPU = system.BestGPU();
        bestGPU.ExactKC_X(0, (hipStream_t)0, TPB, 1,
                          //
                          KCSegmentedReduceTest<float, TPB, SEGMENT_SIZE>,
                          //
                          dReduceOutputs,
                          dData);

        // Copy to Host to check
        CUDA_CHECK(hipMemcpy(hReduceResults.data(),
                              dReduceOutputs,
                              sizeof(float) * SEGMENT_COUNT,
                              hipMemcpyDeviceToHost));

        // Generate results by hand to check
        for(uint32_t i = 0; i < SEGMENT_COUNT; i++)
        {
            uint32_t segmentStart = i * SEGMENT_SIZE;
            uint32_t nextSegmentStart = (i + 1) * SEGMENT_SIZE;

            hReduceResultsExpected[i] = std::reduce(hData.begin() + segmentStart,
                                                    hData.begin() + nextSegmentStart);
        }
        // Checks
        for(uint32_t i = 0; i < SEGMENT_COUNT; i++)
        {
            float result = hReduceResults[i];
            float expected = hReduceResultsExpected[i];
            EXPECT_NEAR(result, expected, MathConstants::VeryLargeEpsilon);
        }
    }
}

TYPED_TEST(BlockSegReduceTest, IntSumStress)
{
     static constexpr uint32_t ITERATION_COUNT = 100;
     constexpr auto TPB = TypeParam::TPB;
     constexpr auto SEGMENT_SIZE = TypeParam::SEGMENT_SIZE;
     constexpr auto SEGMENT_COUNT = TypeParam::SEGMENT_COUNT;

    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());
    // CPU Allocations
    std::vector<uint32_t> hData(TPB, 1u);
    std::vector<uint32_t> hReduceResults(SEGMENT_COUNT);
    std::vector<uint32_t> hReduceResultsExpected(SEGMENT_COUNT);
    // GPU Allocations
    uint32_t* dData;
    uint32_t* dReduceOutputs;
    DeviceMemory mem;
    GPUMemFuncs::AllocateMultiData(std::tie(dData, dReduceOutputs),
                                   mem, {TPB, SEGMENT_COUNT});


    std::mt19937 rng;
    rng.seed(0);
    std::uniform_int_distribution<uint32_t> uniformDist(0, 10);

    for(uint32_t i = 0; i < ITERATION_COUNT; i++)
    {
        // Generate new batch of random numbers
        for(uint32_t& d : hData)
        {
            d = uniformDist(rng);
        }

        CUDA_CHECK(hipMemcpy(dData, hData.data(), sizeof(uint32_t) * TPB,
                              hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemset(dReduceOutputs, 0xFF, sizeof(uint32_t) * SEGMENT_COUNT));

        // Scan Call
        const CudaGPU& bestGPU = system.BestGPU();
        bestGPU.ExactKC_X(0, (hipStream_t)0, TPB, 1,
                          //
                          KCSegmentedReduceTest<uint32_t, TPB, SEGMENT_SIZE>,
                          //
                          dReduceOutputs,
                          dData);

        // Copy to Host to check
        CUDA_CHECK(hipMemcpy(hReduceResults.data(),
                              dReduceOutputs,
                              sizeof(uint32_t) * SEGMENT_COUNT,
                              hipMemcpyDeviceToHost));

        // Generate results by hand to check
        for(uint32_t j = 0; j < SEGMENT_COUNT; j++)
        {
            uint32_t segmentStart = j * SEGMENT_SIZE;
            uint32_t nextSegmentStart = (j + 1) * SEGMENT_SIZE;

            hReduceResultsExpected[j] = std::reduce(hData.begin() + segmentStart,
                                                    hData.begin() + nextSegmentStart);
        }
        // Checks
        for(uint32_t j = 0; j < SEGMENT_COUNT; j++)
        {
            uint32_t result = hReduceResults[j];
            uint32_t expected = hReduceResultsExpected[j];
            EXPECT_EQ(result, expected);
        }
    }
}
