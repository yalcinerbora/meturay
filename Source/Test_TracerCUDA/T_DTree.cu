#include "hip/hip_runtime.h"
﻿#include <gtest/gtest.h>
#include <gmock/gmock.h>

#include <numeric>
#include <random>

#include "TracerCUDA/DTree.cuh"
#include "TracerCUDA/CudaSystem.h"
#include "TracerCUDA/DTreeKC.cuh"

#include "TracerCUDA/TracerDebug.h"

using ::testing::FloatEq;

static std::ostream& operator<<(std::ostream& s, const PathGuidingNode& n)
{
    s << "{"   << std::endl
      << "   " << n.worldPosition[0] << ", " 
               << n.worldPosition[1] << ", "
               << n.worldPosition[2] << std::endl
      << "   " << static_cast<uint32_t>(n.prevNext[0]) << ", "
               << static_cast<uint32_t>(n.prevNext[1]) << std::endl
      << "}";
    return s;
}

static std::ostream& operator<<(std::ostream& s, const DTreeNode& n)
{
    constexpr uint32_t UINT32_T_MAX = std::numeric_limits<uint32_t>::max();
    constexpr uint16_t UINT16_T_MAX = std::numeric_limits<uint16_t>::max();

    s << "P{"; 
    if(n.parentIndex == UINT16_T_MAX) s << "-";
    else s << n.parentIndex;
    s << "} ";
    s << "C{";
    if(n.childIndices[0] == UINT32_T_MAX) s << "-";
    else s << n.childIndices[0];
    s << ", ";
    if(n.childIndices[1] == UINT32_T_MAX) s << "-";
    else s << n.childIndices[1];
    s << ", ";
    if(n.childIndices[2] == UINT32_T_MAX) s << "-";
    else s << n.childIndices[2];
    s << ", ";
    if(n.childIndices[3] == UINT32_T_MAX) s << "-";
    else s << n.childIndices[3];
    s << "} ";
    s << "I{"
      << n.irradianceEstimates[0] << ", "
      << n.irradianceEstimates[1] << ", "
      << n.irradianceEstimates[2] << ", "
      << n.irradianceEstimates[3] << "}";
    return s;
}

static std::ostream& operator<<(std::ostream& s, const DTreeGPU& n)
{
    s << "Irradiane  : " << n.irradiance << std::endl;
    s << "NodeCount  : " << n.nodeCount << std::endl;
    s << "SampleCount: " << n.totalSamples << std::endl;        
    return s;
}

TEST(PPG_DTree, Empty)
{
    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());

    // Constants
    // If a node has %10 or more total energy, split
    static constexpr float FLUX_RATIO = 0.1f;
    // Maximum allowed depth of the tree
    static constexpr uint32_t DEPTH_LIMIT = 10;

    std::vector<DTreeNode> nodes;
    DTreeGPU tree;

    // Initialize Check
    DTree testTree;
    testTree.GetReadTreeToCPU(tree, nodes);
    EXPECT_EQ(0.0f, tree.irradiance);
    EXPECT_EQ(0, tree.totalSamples);
    EXPECT_EQ(1, tree.nodeCount);
    EXPECT_EQ(1, nodes.size());
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[0]);
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[1]);
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[2]);
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[3]);
    EXPECT_EQ(std::numeric_limits<uint16_t>::max(), nodes.front().parentIndex);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[0]);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[1]);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[2]);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[3]);
    testTree.GetWriteTreeToCPU(tree, nodes);
    EXPECT_EQ(0.0f, tree.irradiance);
    EXPECT_EQ(0, tree.totalSamples);
    EXPECT_EQ(1, tree.nodeCount);
    EXPECT_EQ(1, nodes.size());
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[0]);
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[1]);
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[2]);
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[3]);
    EXPECT_EQ(std::numeric_limits<uint16_t>::max(), nodes.front().parentIndex);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[0]);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[1]);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[2]);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[3]);

    // After Swap Check
    testTree.SwapTrees(FLUX_RATIO, DEPTH_LIMIT, system.BestGPU());
    system.SyncAllGPUs();
    testTree.GetReadTreeToCPU(tree, nodes);
    EXPECT_EQ(0.0f, tree.irradiance);
    EXPECT_EQ(0, tree.totalSamples);
    EXPECT_EQ(1, tree.nodeCount);
    EXPECT_EQ(1, nodes.size());
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[0]);
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[1]);
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[2]);
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[3]);
    EXPECT_EQ(std::numeric_limits<uint16_t>::max(), nodes.front().parentIndex);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[0]);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[1]);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[2]);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[3]);
    testTree.GetWriteTreeToCPU(tree, nodes);
    EXPECT_EQ(0.0f, tree.irradiance);
    EXPECT_EQ(0, tree.totalSamples);
    EXPECT_EQ(1, tree.nodeCount);
    EXPECT_EQ(1, nodes.size());
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[0]);
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[1]);
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[2]);
    EXPECT_EQ(std::numeric_limits<uint32_t>::max(), nodes.front().childIndices[3]);
    EXPECT_EQ(std::numeric_limits<uint16_t>::max(), nodes.front().parentIndex);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[0]);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[1]);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[2]);
    EXPECT_EQ(0.0f, nodes.front().irradianceEstimates[3]);
}

TEST(PPG_DTree, AddThenSwap)
{
    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());

    // Constants
    // If a node has %10 or more total energy, split
    static constexpr float FLUX_RATIO = 0.001f;
    // Maximum allowed depth of the tree
    static constexpr uint32_t DEPTH_LIMIT = 10;

    // Check buffers
    std::vector<DTreeNode> nodes;
    DTreeGPU treeGPU;

    PathGuidingNode camNode;
    camNode.worldPosition = Vector3f{0.0f, 0.0f, 0.0f};
    camNode.prevNext = Vector<2, PathNode::IndexType>(PathNode::InvalidIndex, 1);
    camNode.radFactor = Zero3;
    camNode.nearestDTreeIndex = 0;
    camNode.totalRadiance = Vector3f{10.0f, 10.0f, 10.0f};
    PathGuidingNode midNode0;
    midNode0.worldPosition = Vector3f{1.0f, 1.0f, 1.0f};
    midNode0.prevNext = Vector<2, PathNode::IndexType>(0, 2);
    midNode0.radFactor = Zero3;
    midNode0.nearestDTreeIndex = 0;
    midNode0.totalRadiance = Vector3f{30.0f, 30.0f, 30.0f};
    PathGuidingNode midNode1;
    midNode1.worldPosition = Vector3f{0.0f, 0.0f, 0.0f};
    midNode1.prevNext = Vector<2, PathNode::IndexType>(1, 3);
    midNode1.radFactor = Zero3;
    midNode1.nearestDTreeIndex = 0;
    midNode1.totalRadiance = Vector3f{40.0f, 40.0f, 40.0f};
    PathGuidingNode midNode2;
    midNode2.worldPosition = Vector3f{1.0f, 1.0f, -1.0f};
    midNode2.prevNext = Vector<2, PathNode::IndexType>(2, 4);
    midNode2.radFactor = Zero3;
    midNode2.nearestDTreeIndex = 0;
    midNode2.totalRadiance = Vector3f{50.0f, 50.0f, 50.0f};
    PathGuidingNode endNode;
    endNode.worldPosition = Vector3f{0.0f, 0.0f, 0.0f};
    endNode.prevNext = Vector<2, PathNode::IndexType>(3, PathNode::InvalidIndex);
    endNode.radFactor = Zero3;
    endNode.nearestDTreeIndex = 0;
    endNode.totalRadiance = Vector3f{0.0f, 0.0f, 0.0f};

    std::vector<PathGuidingNode> pathNodes =
    {
        camNode,
        midNode0,
        midNode1,
        midNode2,
        endNode
    };

    std::vector<Vector3f> directions;
    for(const PathGuidingNode& p : pathNodes)
    {
        if(p.prevNext[1] != PathNode::InvalidIndex)
        {
            directions.push_back(pathNodes[p.prevNext[1]].worldPosition - p.worldPosition);
            directions.back().NormalizeSelf();
        }            
    }

    // Create Tree
    DTree testTree;
    // Copy Vertices to the GPU
    DeviceMemory pathNodeMemory(pathNodes.size() * sizeof(PathGuidingNode));
    PathGuidingNode* dPathNodes = static_cast<PathGuidingNode*>(pathNodeMemory);
    CUDA_CHECK(hipMemcpy(dPathNodes, pathNodes.data(),
                          pathNodes.size() * sizeof(PathGuidingNode),
                          hipMemcpyHostToDevice));

    DeviceMemory indexMemory(pathNodes.size() * sizeof(uint32_t));
    uint32_t* dIndices = static_cast<uint32_t*>(indexMemory);
    std::vector<uint32_t> hIndices(pathNodes.size());
    std::iota(hIndices.begin(), hIndices.end(), 0);
    CUDA_CHECK(hipMemcpy(dIndices, hIndices.data(),
                          pathNodes.size() * sizeof(uint32_t),
                          hipMemcpyHostToDevice));

    // Push these values to the Tree
    const uint32_t PathNodePerRay = static_cast<uint32_t>(pathNodes.size());
    testTree.AddRadiancesFromPaths(dIndices, dPathNodes,
                                   ArrayPortion<uint32_t>{0, 0, pathNodes.size()},
                                   PathNodePerRay, system.BestGPU());
    system.SyncAllGPUs();

    // Check Tree
    testTree.GetWriteTreeToCPU(treeGPU, nodes);
    for(size_t i = 0; i < nodes.size(); i++)
    {
        const DTreeNode& node = nodes[i];
        if(node.parentIndex == std::numeric_limits<uint16_t>::max())
        {
            // This is root
            // Root should be the very first element
            EXPECT_EQ(0, i);
            EXPECT_EQ(treeGPU.totalSamples, pathNodes.size());
            continue;
        }

        // Only leafs should have value
        if(node.childIndices[0] != std::numeric_limits<uint32_t>::max())
            EXPECT_EQ(0.0f, node.irradianceEstimates[0]);
        if(node.childIndices[1] != std::numeric_limits<uint32_t>::max())
            EXPECT_EQ(0.0f, node.irradianceEstimates[1]);
        if(node.childIndices[2] != std::numeric_limits<uint32_t>::max())
            EXPECT_EQ(0.0f, node.irradianceEstimates[2]);
        if(node.childIndices[3] != std::numeric_limits<uint32_t>::max())
            EXPECT_EQ(0.0f, node.irradianceEstimates[3]);
    }

    // Do the swap
    testTree.SwapTrees(FLUX_RATIO, DEPTH_LIMIT, system.BestGPU());
    system.SyncAllGPUs();

    // Check again
    testTree.GetReadTreeToCPU(treeGPU, nodes);
    for(size_t i = 0; i < nodes.size(); i++)
    {
        const DTreeNode& node = nodes[i];
        float total = node.irradianceEstimates.Sum();
        if(node.parentIndex == std::numeric_limits<uint16_t>::max())
        {
            // This is root
            // Root should be the very first element
            EXPECT_EQ(0, i);
            EXPECT_EQ(treeGPU.totalSamples, pathNodes.size());
            EXPECT_FLOAT_EQ(treeGPU.irradiance, total);
            continue;
        }

        const DTreeNode& parent = nodes[node.parentIndex];
        uint32_t childId = UINT32_MAX;
        childId = (parent.childIndices[0] == i) ? 0 : childId;
        childId = (parent.childIndices[1] == i) ? 1 : childId;
        childId = (parent.childIndices[2] == i) ? 2 : childId;
        childId = (parent.childIndices[3] == i) ? 3 : childId;
        EXPECT_FLOAT_EQ(total, parent.irradianceEstimates[childId]);
    }    
    testTree.GetWriteTreeToCPU(treeGPU, nodes);
    for(size_t i = 0; i < nodes.size(); i++)
    {
        const DTreeNode& node = nodes[i];
        float total = node.irradianceEstimates.Sum();
        if(node.parentIndex == std::numeric_limits<uint16_t>::max())
        {
            // This is root
            // Root should be the very first element
            EXPECT_EQ(0, i);
            EXPECT_EQ(treeGPU.totalSamples, 0);
            EXPECT_FLOAT_EQ(treeGPU.irradiance, 0);
            EXPECT_EQ(treeGPU.nodeCount, 5);
        }
        EXPECT_FLOAT_EQ(0.0f, node.irradianceEstimates[0]);
        EXPECT_FLOAT_EQ(0.0f, node.irradianceEstimates[1]);
        EXPECT_FLOAT_EQ(0.0f, node.irradianceEstimates[2]);
        EXPECT_FLOAT_EQ(0.0f, node.irradianceEstimates[3]);
    }  
}

TEST(PPG_DTree, Stress)
{
    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());

    constexpr int ITERATION_COUNT = 500;
    constexpr int PATH_PER_ITERATION = 5000;
    constexpr int RAY_COUNT = 500;
    constexpr int PATH_PER_RAY = PATH_PER_ITERATION / RAY_COUNT;

    constexpr int DTREE_ID = 0;
    constexpr Vector3f MAX_TOTAL_RADIANCE = Vector3f(1, 1, 1);
    constexpr Vector3f MIN_WORLD_BOUND = Vector3f(-10, -10, -10);
    constexpr Vector3f MAX_WORLD_BOUND = Vector3f(10, 10, 10);
    // Change depth on each iteration
    // just sto stress
    constexpr uint32_t DEPTH_MIN = 0;
    constexpr uint32_t DEPTH_MAX = 16;    
    // Also change the flux
    constexpr float FLUX_MIN = 0.001f;
    constexpr float FLUX_MAX = 0.1f;

    const Vector3f worldBound = MAX_WORLD_BOUND - MIN_WORLD_BOUND;
    std::uniform_real_distribution<float> uniformDist(0.0f, 1.0f);

    std::mt19937 rng;
    rng.seed(0);

    // GPU Buffers
    DeviceMemory pathNodeMemory(PATH_PER_ITERATION * sizeof(PathGuidingNode));
    PathGuidingNode* dPathNodes = static_cast<PathGuidingNode*>(pathNodeMemory);
    DeviceMemory indexMemory(PATH_PER_ITERATION * sizeof(uint32_t));
    uint32_t* dIndices = static_cast<uint32_t*>(indexMemory);

    // Copy redundant incrementing buffer to GPU (since we are not sorting stuff)
    std::vector<uint32_t> hIndices(PATH_PER_ITERATION);
    std::iota(hIndices.begin(), hIndices.end(), 0);
    CUDA_CHECK(hipMemcpy(dIndices, hIndices.data(),
                          PATH_PER_ITERATION * sizeof(uint32_t),
                          hipMemcpyHostToDevice));
    hIndices.clear();
    
    // Check buffer
    DTreeGPU treeGPU;
    std::vector<DTreeNode> nodes;

    // Stress the Tree by randomly adding data multiple times
    DTree testTree;
    std::vector<PathGuidingNode> paths(PATH_PER_ITERATION);
    for(int iCount = 0; iCount < ITERATION_COUNT; iCount++)
    {
        // Constants for this itertion
        // If a node has %X or more total energy, split
        const float fluxRatio = FLUX_MIN + uniformDist(rng) * (FLUX_MAX - FLUX_MIN);
        // Maximum allowed depth of the tree
        uint32_t depthLimit = DEPTH_MIN + static_cast<uint32_t>(uniformDist(rng) * (DEPTH_MAX - DEPTH_MIN));

        //METU_LOG("Depth %u, Flux %f", depthLimit, fluxRatio);

        for(size_t i = 0; i < PATH_PER_ITERATION; i++)
        {
            uint32_t localIndex = i % PATH_PER_RAY;
            uint32_t prev = (localIndex == 0) ? PathGuidingNode::InvalidIndex : localIndex - 1;
            uint32_t next = (localIndex == (PATH_PER_RAY - 1)) ? PathGuidingNode::InvalidIndex : localIndex + 1;
           
            Vector3f worldUniform(uniformDist(rng), uniformDist(rng), uniformDist(rng));
            Vector3f radianceUniform(uniformDist(rng), uniformDist(rng), uniformDist(rng));

            PathGuidingNode p;
            p.worldPosition = MIN_WORLD_BOUND + worldUniform * worldBound;
            p.prevNext = Vector<2, PathGuidingNode::IndexType>(prev, next);
            p.totalRadiance = radianceUniform * MAX_TOTAL_RADIANCE;
            // Unnecessary Data for this operation
            p.nearestDTreeIndex = DTREE_ID;
            p.radFactor = Zero3;
            paths[i] = p;
        }


        // Copy Vertices to the GPU
        CUDA_CHECK(hipMemcpy(dPathNodes, paths.data(),
                              PATH_PER_ITERATION * sizeof(PathGuidingNode),
                              hipMemcpyHostToDevice));
        // Do add radiance kernel
        testTree.AddRadiancesFromPaths(dIndices, dPathNodes,
                                       ArrayPortion<uint32_t>{DTREE_ID, 0, PATH_PER_ITERATION},
                                       PATH_PER_RAY, system.BestGPU());
        system.SyncAllGPUs();

        // Check if radiance is properly added
        testTree.GetWriteTreeToCPU(treeGPU, nodes);
        for(size_t i = 0; i < nodes.size(); i++)
        {
            const DTreeNode& node = nodes[i];
            if(node.parentIndex == std::numeric_limits<uint16_t>::max())
            {
                // This is root
                // Root should be the very first element
                EXPECT_EQ(0, i);
                EXPECT_EQ(treeGPU.totalSamples, PATH_PER_ITERATION);
                continue;
            }

            // Only leafs should have value
            if(node.childIndices[0] != std::numeric_limits<uint32_t>::max())
                EXPECT_EQ(0.0f, node.irradianceEstimates[0]);
            if(node.childIndices[1] != std::numeric_limits<uint32_t>::max())
                EXPECT_EQ(0.0f, node.irradianceEstimates[1]);
            if(node.childIndices[2] != std::numeric_limits<uint32_t>::max())
                EXPECT_EQ(0.0f, node.irradianceEstimates[2]);
            if(node.childIndices[3] != std::numeric_limits<uint32_t>::max())
                EXPECT_EQ(0.0f, node.irradianceEstimates[3]);            
        }

        testTree.SwapTrees(fluxRatio, depthLimit, system.BestGPU());
        system.SyncAllGPUs();

        // Check integrity of the new write tree
        testTree.GetWriteTreeToCPU(treeGPU, nodes);
        for(size_t i = 0; i < nodes.size(); i++)
        {
            const DTreeNode& node = nodes[i];
            EXPECT_FLOAT_EQ(0.0f, node.irradianceEstimates[0]);
            EXPECT_FLOAT_EQ(0.0f, node.irradianceEstimates[1]);
            EXPECT_FLOAT_EQ(0.0f, node.irradianceEstimates[2]);
            EXPECT_FLOAT_EQ(0.0f, node.irradianceEstimates[3]);

            if(node.parentIndex == std::numeric_limits<uint16_t>::max())
            {
                // This is root
                // Root should be the very first element
                EXPECT_EQ(0, i);
                EXPECT_EQ(treeGPU.totalSamples, 0);
                EXPECT_EQ(0.0f, treeGPU.irradiance);
                continue;
            }

            // Try to go to the parent
            const DTreeNode* n = &node;
            while(n->parentIndex != std::numeric_limits<uint16_t>::max())
            {
                n = &nodes[n->parentIndex];
            }
            // After back propogation
            // check if we actually reached to the parent
            ptrdiff_t index = n - nodes.data();
            EXPECT_EQ(0, index);
        }
        //// DEBUG
        //Debug::DumpMemToFile("WT", &treeGPU, 1);
        //Debug::DumpMemToFile("WTN", nodes.data(), nodes.size());

        // Check integrity of the new read tree
        testTree.GetReadTreeToCPU(treeGPU, nodes);
        for(size_t i = 0; i < nodes.size(); i++)
        {
            const DTreeNode& node = nodes[i];
            float total = node.irradianceEstimates.Sum();
            if(node.parentIndex == std::numeric_limits<uint16_t>::max())
            {
                // This is root
                // Root should be the very first element
                EXPECT_EQ(0, i);
                EXPECT_EQ(treeGPU.totalSamples, PATH_PER_ITERATION);
                EXPECT_FLOAT_EQ(treeGPU.irradiance, total);
                continue;
            }

            const DTreeNode& parent = nodes[node.parentIndex];
            uint32_t childId = UINT32_MAX;
            childId = (parent.childIndices[0] == i) ? 0 : childId;
            childId = (parent.childIndices[1] == i) ? 1 : childId;
            childId = (parent.childIndices[2] == i) ? 2 : childId;
            childId = (parent.childIndices[3] == i) ? 3 : childId;
            EXPECT_FLOAT_EQ(total, parent.irradianceEstimates[childId]);
        }
        //// DEBUG
        //Debug::DumpMemToFile("RT", &treeGPU, 1);
        //Debug::DumpMemToFile("RTN", nodes.data(), nodes.size());    
    }
 }

TEST(PPG_DTree, ZeroAdd)
{
        CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());

    constexpr int ITERATION_COUNT = 500;
    constexpr int PATH_PER_ITERATION = 5000;
    constexpr int RAY_COUNT = 500;
    constexpr int PATH_PER_RAY = PATH_PER_ITERATION / RAY_COUNT;

    constexpr int DTREE_ID = 0;
    constexpr Vector3f MIN_WORLD_BOUND = Vector3f(-10, -10, -10);
    constexpr Vector3f MAX_WORLD_BOUND = Vector3f(10, 10, 10);
    // Change depth on each iteration
    // just sto stress
    constexpr uint32_t DEPTH_MIN = 0;
    constexpr uint32_t DEPTH_MAX = 16;    
    // Also change the flux
    constexpr float FLUX_MIN = 0.001f;
    constexpr float FLUX_MAX = 0.1f;

    const Vector3f worldBound = MAX_WORLD_BOUND - MIN_WORLD_BOUND;
    std::uniform_real_distribution<float> uniformDist(0.0f, 1.0f);

    std::mt19937 rng;
    rng.seed(0);

    // GPU Buffers
    DeviceMemory pathNodeMemory(PATH_PER_ITERATION * sizeof(PathGuidingNode));
    PathGuidingNode* dPathNodes = static_cast<PathGuidingNode*>(pathNodeMemory);
    DeviceMemory indexMemory(PATH_PER_ITERATION * sizeof(uint32_t));
    uint32_t* dIndices = static_cast<uint32_t*>(indexMemory);

    // Copy redundant incrementing buffer to GPU (since we are not sorting stuff)
    std::vector<uint32_t> hIndices(PATH_PER_ITERATION);
    std::iota(hIndices.begin(), hIndices.end(), 0);
    CUDA_CHECK(hipMemcpy(dIndices, hIndices.data(),
                          PATH_PER_ITERATION * sizeof(uint32_t),
                          hipMemcpyHostToDevice));
    hIndices.clear();
    
    // Check buffer
    DTreeGPU treeGPU;
    std::vector<DTreeNode> nodes;

    // Stress the Tree by randomly adding data multiple times
    DTree testTree;
    std::vector<PathGuidingNode> paths(PATH_PER_ITERATION);
    for(int iCount = 0; iCount < ITERATION_COUNT; iCount++)
    {
        // Constants for this itertion
        // If a node has %X or more total energy, split
        const float fluxRatio = FLUX_MIN + uniformDist(rng) * (FLUX_MAX - FLUX_MIN);
        // Maximum allowed depth of the tree
        uint32_t depthLimit = DEPTH_MIN + static_cast<uint32_t>(uniformDist(rng) * (DEPTH_MAX - DEPTH_MIN));

        //METU_LOG("Depth %u, Flux %f", depthLimit, fluxRatio);

        for(size_t i = 0; i < PATH_PER_ITERATION; i++)
        {
            uint32_t localIndex = i % PATH_PER_RAY;
            uint32_t prev = (localIndex == 0) ? PathGuidingNode::InvalidIndex : localIndex - 1;
            uint32_t next = (localIndex == (PATH_PER_RAY - 1)) ? PathGuidingNode::InvalidIndex : localIndex + 1;
           
            Vector3f worldUniform(uniformDist(rng), uniformDist(rng), uniformDist(rng));
            Vector3f radianceUniform(uniformDist(rng), uniformDist(rng), uniformDist(rng));

            PathGuidingNode p;
            p.worldPosition = MIN_WORLD_BOUND + worldUniform * worldBound;
            p.prevNext = Vector<2, PathGuidingNode::IndexType>(prev, next);
            p.totalRadiance = Zero3;
            // Unnecessary Data for this operation
            p.nearestDTreeIndex = DTREE_ID;
            p.radFactor = Zero3;
            paths[i] = p;
        }


        // Copy Vertices to the GPU
        CUDA_CHECK(hipMemcpy(dPathNodes, paths.data(),
                              PATH_PER_ITERATION * sizeof(PathGuidingNode),
                              hipMemcpyHostToDevice));
        // Do add radiance kernel
        testTree.AddRadiancesFromPaths(dIndices, dPathNodes,
                                       ArrayPortion<uint32_t>{DTREE_ID, 0, PATH_PER_ITERATION},
                                       PATH_PER_RAY, system.BestGPU());
        system.SyncAllGPUs();

        // Check if radiance is properly added
        testTree.GetWriteTreeToCPU(treeGPU, nodes);
        for(size_t i = 0; i < nodes.size(); i++)
        {
            const DTreeNode& node = nodes[i];
            if(node.parentIndex == std::numeric_limits<uint16_t>::max())
            {
                // This is root
                // Root should be the very first element
                EXPECT_EQ(0, i);
                EXPECT_EQ(treeGPU.totalSamples, PATH_PER_ITERATION);
                continue;
            }

            // Only leafs should have value
            if(node.childIndices[0] != std::numeric_limits<uint32_t>::max())
                EXPECT_EQ(0.0f, node.irradianceEstimates[0]);
            if(node.childIndices[1] != std::numeric_limits<uint32_t>::max())
                EXPECT_EQ(0.0f, node.irradianceEstimates[1]);
            if(node.childIndices[2] != std::numeric_limits<uint32_t>::max())
                EXPECT_EQ(0.0f, node.irradianceEstimates[2]);
            if(node.childIndices[3] != std::numeric_limits<uint32_t>::max())
                EXPECT_EQ(0.0f, node.irradianceEstimates[3]);            
        }

        testTree.SwapTrees(fluxRatio, depthLimit, system.BestGPU());
        system.SyncAllGPUs();

        // Check integrity of the new write tree
        testTree.GetWriteTreeToCPU(treeGPU, nodes);
        for(size_t i = 0; i < nodes.size(); i++)
        {
            const DTreeNode& node = nodes[i];
            EXPECT_FLOAT_EQ(0.0f, node.irradianceEstimates[0]);
            EXPECT_FLOAT_EQ(0.0f, node.irradianceEstimates[1]);
            EXPECT_FLOAT_EQ(0.0f, node.irradianceEstimates[2]);
            EXPECT_FLOAT_EQ(0.0f, node.irradianceEstimates[3]);

            if(node.parentIndex == std::numeric_limits<uint16_t>::max())
            {
                // This is root
                // Root should be the very first element
                EXPECT_EQ(0, i);
                EXPECT_EQ(treeGPU.totalSamples, 0);
                EXPECT_EQ(0.0f, treeGPU.irradiance);
                continue;
            }

            // Try to go to the parent
            const DTreeNode* n = &node;
            while(n->parentIndex != std::numeric_limits<uint16_t>::max())
            {
                n = &nodes[n->parentIndex];
            }
            // After back propogation
            // check if we actually reached to the parent
            ptrdiff_t index = n - nodes.data();
            EXPECT_EQ(0, index);
        }
        //// DEBUG
        //Debug::DumpMemToFile("WT", &treeGPU, 1);
        //Debug::DumpMemToFile("WTN", nodes.data(), nodes.size());

        // Check integrity of the new read tree
        testTree.GetReadTreeToCPU(treeGPU, nodes);
        for(size_t i = 0; i < nodes.size(); i++)
        {
            const DTreeNode& node = nodes[i];
            float total = node.irradianceEstimates.Sum();
            if(node.parentIndex == std::numeric_limits<uint16_t>::max())
            {
                // This is root
                // Root should be the very first element
                EXPECT_EQ(0, i);
                EXPECT_EQ(treeGPU.totalSamples, PATH_PER_ITERATION);
                EXPECT_FLOAT_EQ(treeGPU.irradiance, total);
                continue;
            }

            const DTreeNode& parent = nodes[node.parentIndex];
            uint32_t childId = UINT32_MAX;
            childId = (parent.childIndices[0] == i) ? 0 : childId;
            childId = (parent.childIndices[1] == i) ? 1 : childId;
            childId = (parent.childIndices[2] == i) ? 2 : childId;
            childId = (parent.childIndices[3] == i) ? 3 : childId;
            EXPECT_FLOAT_EQ(total, parent.irradianceEstimates[childId]);
        }
        //// DEBUG
        //Debug::DumpMemToFile("RT", &treeGPU, 1);
        //Debug::DumpMemToFile("RTN", nodes.data(), nodes.size());    
    }
}