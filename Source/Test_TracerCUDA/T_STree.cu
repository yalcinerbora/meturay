#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <gmock/gmock.h>

#include <numeric>
#include <random>

#include "TracerCUDA/STree.cuh"
#include "TracerCUDA/STreeKC.cuh"
#include "TracerCUDA/CudaSystem.h"
#include "TracerCUDA/CudaSystem.hpp"
#include "TracerCUDA/DTreeKC.cuh"

#include "RayLib/CPUTimer.h"

#include "TracerCUDA/TracerDebug.h"

__global__
static void KCFindNearestDTree(uint32_t* dTreeIndices,
                               const Vector3f* gWorldPositions,
                               const STreeGPU& gSTree,
                               uint32_t totalSampleCount)
{
    // Grid Stride Loop
    for(uint32_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
        globalId < totalSampleCount;
        globalId += blockDim.x * gridDim.x)
    {
        uint32_t dTreeId;
        gSTree.AcquireNearestDTree(dTreeId, gWorldPositions[globalId]);
        dTreeIndices[globalId] = dTreeId;
    }
}

TEST(PPG_STree, Empty)
{
    static constexpr Vector3f WORLD_MIN = Vector3f(10.0f, 10.0f, 10.0f);
    static constexpr Vector3f WORLD_MAX = Vector3f(-10.0f, -10.0f, -10.0f);
    static const AABB3f WorldAABB = AABB3f(WORLD_MIN, WORLD_MAX);
    static constexpr uint32_t SAMPLE_COUNT = 100;
    // RNG
    std::mt19937 rng;
    rng.seed(0);
    // Cuda System
    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());
    // Default Constructed STree
    STree tree(WorldAABB);
    
    DeviceMemory outIndices(sizeof(uint32_t) * SAMPLE_COUNT);
    DeviceMemory inWorldPositions(sizeof(Vector3f) * SAMPLE_COUNT);

    // Do some random accesses to find a tree
    std::vector<Vector3f> worldSamples(SAMPLE_COUNT);
    for(Vector3f& worldPos : worldSamples)
    {
        worldPos = WorldAABB.Min() + WorldAABB.Span() * Vector3f(rng(), rng(), rng());           
    }
    CUDA_CHECK(hipMemcpy(static_cast<Vector3f*>(inWorldPositions),
                          worldSamples.data(),
                          sizeof(Vector3f) * SAMPLE_COUNT,
                          hipMemcpyHostToDevice));


    const CudaGPU& bestGPU = system.BestGPU();

    const STreeGPU* dSTreeGPU;
    const DTreeGPU** dReadDTrees;
    DTreeGPU** dWriteDTrees;
    tree.TreeGPU(dSTreeGPU, dReadDTrees, dWriteDTrees);
    bestGPU.GridStrideKC_X(0, 0, SAMPLE_COUNT,
                           //
                           KCFindNearestDTree,
                           //
                           static_cast<uint32_t*>(outIndices),
                           static_cast<Vector3f*>(inWorldPositions),
                           *dSTreeGPU,
                           SAMPLE_COUNT);

    std::vector<uint32_t> indicesCPU(SAMPLE_COUNT);
    CUDA_CHECK(hipMemcpy(indicesCPU.data(),
                          static_cast<uint32_t*>(outIndices),
                          sizeof(uint32_t) * SAMPLE_COUNT,
                          hipMemcpyDeviceToHost));

    for(uint32_t index : indicesCPU)
    {
        EXPECT_EQ(0, index);
    }
}

TEST(PPG_STree, Split)
{
    static constexpr Vector3f WORLD_MIN = Vector3f(10.0f, 10.0f, 10.0f);
    static constexpr Vector3f WORLD_MAX = Vector3f(-10.0f, -10.0f, -10.0f);
    static const AABB3f WorldAABB = AABB3f(WORLD_MIN, WORLD_MAX);
    // Maximum of 5 depth for each DTree
    static constexpr uint32_t D_MAX_DEPT = 50;
    // Split a DTree when it reaches more than %10 of total energy
    static constexpr float D_FLUX_SPLIT = 0.001f;
    // Split a STree leaf when it reaches 100 samples
    static constexpr uint32_t S_SPLIT = 100;
    // 
    static constexpr uint32_t ITERATION_COUNT = 15;
    static constexpr uint32_t PATH_PER_ITERATION = 100'000;
    static constexpr uint32_t RAY_COUNT = 10'000;
    static constexpr uint32_t PATH_PER_RAY = PATH_PER_ITERATION / RAY_COUNT;
    //
    static constexpr Vector3f MAX_TOTAL_RADIANCE = Vector3f(1, 1, 1);
    // Cuda System
    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());
    // Rng
    std::mt19937 rng;
    rng.seed(0);
    std::uniform_real_distribution<float> uniformDist(0.0f, 1.0f);

    // GPU Buffers
    DeviceMemory pathNodeMemory(PATH_PER_ITERATION * sizeof(PathGuidingNode));
    PathGuidingNode* dPathNodes = static_cast<PathGuidingNode*>(pathNodeMemory);

    STree testTree(WorldAABB);
    std::vector<PathGuidingNode> paths(PATH_PER_ITERATION);
    for(uint32_t iCount = 0; iCount < ITERATION_COUNT; iCount++)
    {
        std::uniform_int_distribution<uint32_t> treeCountDist(0, testTree.TotalTreeCount() - 1);
        // Generate Random Paths
        for(size_t i = 0; i < PATH_PER_ITERATION; i++)
        {
            uint32_t localIndex = i % PATH_PER_RAY;
            uint32_t prev = (localIndex == 0) ? PathGuidingNode::InvalidIndex : localIndex - 1;
            uint32_t next = (localIndex == (PATH_PER_RAY - 1)) ? PathGuidingNode::InvalidIndex : localIndex + 1;

            Vector3f worldUniform(uniformDist(rng), uniformDist(rng), uniformDist(rng));
            Vector3f radianceUniform(uniformDist(rng), uniformDist(rng), uniformDist(rng));

            PathGuidingNode p;
            p.worldPosition = WORLD_MIN + worldUniform * WorldAABB.Span();
            p.prevNext = Vector<2, PathGuidingNode::IndexType>(prev, next);
            p.totalRadiance = radianceUniform * MAX_TOTAL_RADIANCE;
            // Unnecessary Data for this operation
            p.nearestDTreeIndex = treeCountDist(rng);
            p.radFactor = Zero3;
            paths[i] = p;
        }

        // Add Paths to GPU
        // Copy Vertices to the GPU
        CUDA_CHECK(hipMemcpy(dPathNodes, paths.data(),
                              PATH_PER_ITERATION * sizeof(PathGuidingNode),
                              hipMemcpyHostToDevice));

        //// DEBUGGING
        //Utility::CPUTimer t;
        //STreeGPU treeGPU;
        //std::vector<STreeNode> nodes;
        //testTree.GetTreeToCPU(treeGPU, nodes);
        //Debug::DumpMemToFile("BS-Nodes", nodes.data(), nodes.size());
        // Accumulate Radiances
        //t.Start();
        testTree.AccumulateRaidances(dPathNodes, PATH_PER_ITERATION, PATH_PER_RAY, system);
        system.SyncAllGPUs();
        //t.Lap();
        //METU_LOG("Accum-Rad {:f}", t.Elapsed<CPUTimeSeconds>());

        // Split and Swap trees
        testTree.SplitAndSwapTrees(S_SPLIT, D_FLUX_SPLIT, D_MAX_DEPT, system);
        system.SyncAllGPUs();

        //// DEBUGGING
        //t.Lap();
        //METU_LOG("Split&Swap {:f}", t.Elapsed<CPUTimeSeconds>());
        //testTree.GetTreeToCPU(treeGPU, nodes);
        //Debug::DumpMemToFile("AS-Nodes", nodes.data(), nodes.size());
        //METU_LOG("iter {:d}", iCount);
        //METU_LOG("-----------------------------------------------------");
    }

    //// DEBUGGING
    //STreeGPU treeGPU;
    //std::vector<STreeNode> nodes;
    //testTree.GetTreeToCPU(treeGPU, nodes);
    //Debug::DumpMemToFile("Final Nodes", nodes.data(), nodes.size());
}