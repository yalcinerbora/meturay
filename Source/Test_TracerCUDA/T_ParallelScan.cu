#include <gtest/gtest.h>
#include <gmock/gmock.h>

#include "RayLib/Vector.h"

#include "TracerCUDA/DeviceMemory.h"
#include "TracerCUDA/ParallelScan.cuh"

using ::testing::FloatEq;

TEST(ParallelScan, GenericExclusive)
{
    static constexpr int ElementCount = 50'000;

    DeviceMemory iDataIn(ElementCount * sizeof(int));
    DeviceMemory fDataIn(ElementCount * sizeof(float));
    DeviceMemory iDataOut(ElementCount * sizeof(int));
    DeviceMemory fDataOut(ElementCount * sizeof(float));
    std::fill_n(static_cast<int*>(iDataIn), ElementCount, 1);
    std::fill_n(static_cast<float*>(fDataIn), ElementCount, 1.0f);

    // Integer Kernel
    ExclusiveScanArrayGPU<int, ReduceAdd<int>>
    (
      static_cast<int*>(iDataOut),
      static_cast<int*>(iDataIn),
      ElementCount, 0
    );
    // Float Kernel
    ExclusiveScanArrayGPU<float, ReduceAdd<float>>
    (
      static_cast<float*>(fDataOut),
      static_cast<float*>(fDataIn),
      ElementCount, 0.0f
    );
    //EXPECT_TRUE(false);

    // Wait Kernel Finish and Check
    CUDA_CHECK(hipDeviceSynchronize());

    for(int i = 0; i < ElementCount; i++)
    {
        EXPECT_EQ(static_cast<int*>(iDataOut)[i], i);
        EXPECT_FLOAT_EQ(static_cast<float*>(fDataOut)[i],
                        static_cast<float>(i));
    }
}

TEST(ParallelScan, GenericInclusive)
{
    static constexpr int ElementCount = 50'000;
    //static constexpr int ElementCount = 50;

    DeviceMemory iDataIn(ElementCount * sizeof(int));
    DeviceMemory fDataIn(ElementCount * sizeof(float));
    DeviceMemory iDataOut(ElementCount * sizeof(int));
    DeviceMemory fDataOut(ElementCount * sizeof(float));
    std::fill_n(static_cast<int*>(iDataIn), ElementCount, 1);
    std::fill_n(static_cast<float*>(fDataIn), ElementCount, 1.0f);

    // Integer Kernel
    InclusiveScanArrayGPU<int, ReduceAdd<int>>
    (
      static_cast<int*>(iDataOut),
      static_cast<int*>(iDataIn),
      ElementCount
    );
    // Float Kernel
    InclusiveScanArrayGPU<float, ReduceAdd<float>>
    (
      static_cast<float*>(fDataOut),
      static_cast<float*>(fDataIn),
      ElementCount
    );

    // Wait Kernel Finish and Check
    CUDA_CHECK(hipDeviceSynchronize());

    for(int i = 0; i < ElementCount; i++)
    {
        EXPECT_EQ(static_cast<int*>(iDataOut)[i], i + 1);
        EXPECT_FLOAT_EQ(static_cast<float*>(fDataOut)[i],
                        static_cast<float>(i + 1));
    }
}