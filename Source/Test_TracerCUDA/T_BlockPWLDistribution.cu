#include "hip/hip_runtime.h"
#include "TracerCUDA/GPUBlockPWLDistribution.cuh"
#include "TracerCUDA/CudaSystem.h"
#include "TracerCUDA/CudaSystem.hpp"
#include "TracerCUDA/DeviceMemory.h"

#include "RayLib/Constants.h"

#include <gtest/gtest.h>
#include <random>
#include <numeric>

template <uint32_t TPB, uint32_t X, uint32_t Y>
__global__ __launch_bounds__(TPB)
void KCPiecewiseLinearDistInitCheck(float* gPDFXOut,
                                    float* gCDFXOut,
                                    float* gPDFYOut,
                                    float* gCDFYOut,
                                    const float* gData)
{
    using BlockPWL2D = BlockPWLDistribution2D<TPB, X, Y>;
    static constexpr auto DATA_PER_THREAD = BlockPWL2D::DATA_PER_THREAD;
    static constexpr auto PIXEL_COUNT = BlockPWL2D::PIX_COUNT;

    // Allocate shared memory for Block Operations
    __shared__ typename BlockPWL2D::TempStorage sPWLMem;

    const uint32_t threadId = threadIdx.x;

    float data[DATA_PER_THREAD];
    for(uint32_t i = 0; i < DATA_PER_THREAD; i++)
    {
        data[i] = (threadId < PIXEL_COUNT) ? gData[i * TPB + threadId] : 0.0f;
    }

    // Init the class
    BlockPWL2D dist2D(sPWLMem, data);
    // Directly dump the calculated PDF / CDF
    dist2D.DumpSharedMem(gPDFXOut,
                         gCDFXOut,
                         gPDFYOut,
                         gCDFYOut);

}

template <uint32_t TPB_VAL, uint32_t X_VAL, uint32_t Y_VAL>
struct BlockPWL2DTestParams
{
    static constexpr uint32_t TPB = TPB_VAL;
    static constexpr uint32_t X = X_VAL;
    static constexpr uint32_t Y = Y_VAL;
    static constexpr uint32_t PIX_COUNT = X * Y;
    static constexpr uint32_t DATA_PER_BLOCK = PIX_COUNT / TPB;
    static constexpr uint32_t X_CDF_COUNT = X * Y;
    static constexpr uint32_t Y_CDF_COUNT = Y;

};

template <class T>
class BlockPWL2DTest : public testing::Test
{};

using Implementations = ::testing::Types<BlockPWL2DTestParams<512, 64, 64>,
                                         BlockPWL2DTestParams<512, 64, 32>,
                                         BlockPWL2DTestParams<512, 32, 32>,
                                         BlockPWL2DTestParams<256, 32, 16>,
                                         BlockPWL2DTestParams<256, 16, 16>,
                                         BlockPWL2DTestParams<128, 16, 8>>;

TYPED_TEST_SUITE(BlockPWL2DTest, Implementations);

TYPED_TEST(BlockPWL2DTest, BasicInit)
{
    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());

    constexpr uint32_t TPB = TypeParam::TPB;
    constexpr uint32_t X = TypeParam::X;
    constexpr uint32_t Y = TypeParam::Y;
    constexpr uint32_t PIX_COUNT = TypeParam::PIX_COUNT;
    constexpr uint32_t X_CDF_COUNT = TypeParam::X_CDF_COUNT;
    constexpr uint32_t Y_CDF_COUNT = TypeParam::Y_CDF_COUNT;

    // Copy all ones to GPU
    std::vector<float> data(PIX_COUNT, 10.0f);
    // GPU Allocations
    float* dData;
    float* dPDFX;
    float* dCDFX;
    float* dPDFY;
    float* dCDFY;
    DeviceMemory mem;
    GPUMemFuncs::AllocateMultiData(std::tie(dData,
                                            dPDFX, dCDFX,
                                            dPDFY, dCDFY),
                                   mem,
                                   {PIX_COUNT,
                                    PIX_COUNT, X_CDF_COUNT,
                                    Y, Y_CDF_COUNT});
    CUDA_CHECK(hipMemcpy(dData, data.data(), sizeof(float) * PIX_COUNT,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(dPDFX, 0xFF, sizeof(float) * PIX_COUNT));
    CUDA_CHECK(hipMemset(dCDFX, 0xFF, sizeof(float) * X_CDF_COUNT));
    CUDA_CHECK(hipMemset(dPDFY, 0xFF, sizeof(float) * Y));
    CUDA_CHECK(hipMemset(dCDFY, 0xFF, sizeof(float) * Y_CDF_COUNT));

    // PWL Initialization and Dump to Global Memory Call
    const CudaGPU& bestGPU = system.BestGPU();
    bestGPU.ExactKC_X(0, (hipStream_t)0, TPB, 1,
                      //
                      KCPiecewiseLinearDistInitCheck<TPB, X, Y>,
                      //
                      dPDFX,
                      dCDFX,
                      dPDFY,
                      dCDFY,
                      dData);

    // Copy to Host to check
    std::vector<float> hPDFX(PIX_COUNT);
    std::vector<float> hCDFX(X_CDF_COUNT);
    std::vector<float> hPDFY(Y);
    std::vector<float> hCDFY(Y_CDF_COUNT);
    CUDA_CHECK(hipMemcpy(hPDFX.data(), dPDFX, sizeof(float) * PIX_COUNT,
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hCDFX.data(), dCDFX, sizeof(float) * X_CDF_COUNT,
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hPDFY.data(), dPDFY, sizeof(float) * Y,
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hCDFY.data(), dCDFY, sizeof(float) * Y_CDF_COUNT,
                          hipMemcpyDeviceToHost));
    // Checks
    for(float pdf : hPDFX)
    {
        EXPECT_FLOAT_EQ(pdf, 1.0f);
    }

    for(uint32_t j = 0; j < Y; j++)
    for(uint32_t i = 0; i < X; i++)
    {
        static constexpr float DELTA_X = 1.0f / (static_cast<float>(X) - 1.0f);

        // Technically we should check accumulated trapezoid area
        // but in this basic case it is a rectangle (entire portion)
        uint32_t index = j * X + i;
        float value = static_cast<float>(i) * DELTA_X;
        EXPECT_FLOAT_EQ(value, hCDFX[index]);
    }
    // Marginal
    for(float pdf : hPDFY)
    {
        EXPECT_FLOAT_EQ(pdf, 1.0f);
    }
    for(uint32_t i = 0; i < Y_CDF_COUNT; i++)
    {
        // Technically we should check accumulated trapezoid area
        // but in this basic case it is a rectangle (entire portion)
        static constexpr float DELTA_Y = 1.0f / (static_cast<float>(Y) - 1.0f);
        float value = static_cast<float>(i) * DELTA_Y;
        EXPECT_FLOAT_EQ(value, hCDFY[i]);
    }
}

TYPED_TEST(BlockPWL2DTest, Stress)
{
    static constexpr uint32_t ITERATION_COUNT = 100;
    constexpr uint32_t TPB = TypeParam::TPB;
    constexpr uint32_t X = TypeParam::X;
    constexpr uint32_t Y = TypeParam::Y;
    constexpr uint32_t PIX_COUNT = TypeParam::PIX_COUNT;
    constexpr uint32_t X_CDF_COUNT = TypeParam::X_CDF_COUNT;
    constexpr uint32_t Y_CDF_COUNT = TypeParam::Y_CDF_COUNT;

    CudaSystem system;
    ASSERT_EQ(CudaError::OK, system.Initialize());

    auto TrapezoidArea = [](float a, float b, float h)
    {
        return (a + b) * h * 0.5f;
    };

    // CPU Allocations
    // Random PDFtion
    std::vector<float> hData(PIX_COUNT);
    // Results returned by the PWL
    std::vector<float> hPDFX(PIX_COUNT);
    std::vector<float> hCDFX(X_CDF_COUNT);
    std::vector<float> hPDFY(Y);
    std::vector<float> hCDFY(Y_CDF_COUNT);
    // Calculated Values
    std::vector<float> hPDFYExpected(Y);
    std::vector<float> hPDFXExpected(PIX_COUNT);
    std::vector<float> hCDFXExpected(X_CDF_COUNT);
    std::vector<float> hCDFYExpected(Y_CDF_COUNT);
    // Temp
    std::vector<float> hScratchpad;
    hScratchpad.reserve(std::max(X, Y));

    // GPU Allocations
    float* dData;
    float* dPDFX;
    float* dCDFX;
    float* dPDFY;
    float* dCDFY;
    DeviceMemory mem;
    GPUMemFuncs::AllocateMultiData(std::tie(dData,
                                            dPDFX, dCDFX,
                                            dPDFY, dCDFY),
                                   mem,
                                   {PIX_COUNT,
                                    PIX_COUNT, X_CDF_COUNT,
                                    Y, Y_CDF_COUNT});

    std::mt19937 rng;
    rng.seed(0);
    std::uniform_real_distribution<float> uniformDist(0.0f, 10.0f);
    for(uint32_t i = 0; i < ITERATION_COUNT; i++)
    {
        // Generate new batch of random numbers
        for(float& d : hData)
        {
            d = uniformDist(rng);
        }

        // Copy to GPU
        CUDA_CHECK(hipMemcpy(dData, hData.data(), sizeof(float) * PIX_COUNT,
                              hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemset(dPDFX, 0xFF, sizeof(float) * PIX_COUNT));
        CUDA_CHECK(hipMemset(dCDFX, 0xFF, sizeof(float) * X_CDF_COUNT));
        CUDA_CHECK(hipMemset(dPDFY, 0xFF, sizeof(float) * Y));
        CUDA_CHECK(hipMemset(dCDFY, 0xFF, sizeof(float) * Y_CDF_COUNT));

        // PWC Initialization and Dump to Global Memory Call
        const CudaGPU& bestGPU = system.BestGPU();
        bestGPU.ExactKC_X(0, (hipStream_t)0, TPB, 1,
                          //
                          KCPiecewiseLinearDistInitCheck<TPB, X, Y>,
                          //
                          dPDFX,
                          dCDFX,
                          dPDFY,
                          dCDFY,
                          dData);

        // Copy to Host to check
        CUDA_CHECK(hipMemcpy(hPDFX.data(), dPDFX, sizeof(float) * PIX_COUNT,
                              hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(hCDFX.data(), dCDFX, sizeof(float) * X_CDF_COUNT,
                              hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(hPDFY.data(), dPDFY, sizeof(float) * Y,
                              hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(hCDFY.data(), dCDFY, sizeof(float) * Y_CDF_COUNT,
                              hipMemcpyDeviceToHost));

        // Generate Marginal and Conditional Data here
        hScratchpad.resize(X);
        for(uint32_t j = 0; j < Y; j++)
        {
            static constexpr float DELTA_X = 1.0f / static_cast<float>(X - 1);
            const uint32_t dataRowStart = j * X;
            const uint32_t dataRowEnd = (j + 1) * X;

            hScratchpad[0] = 0.0f;
            for(uint32_t i = 1; i < X; i++)
            {
                hScratchpad[i] = TrapezoidArea(hData[dataRowStart + i - 1],
                                               hData[dataRowStart + i],
                                               DELTA_X);
            }
            std::copy(hData.cbegin() + dataRowStart,
                      hData.cbegin() + dataRowEnd,
                      hPDFXExpected.begin() + dataRowStart);

            uint32_t cdfRowStart = j * X;
            uint32_t cdfRowEnd = (j + 1) * X;
            std::inclusive_scan(hScratchpad.cbegin(), hScratchpad.cbegin() + X,
                                hCDFXExpected.begin() + cdfRowStart);
            float totalSum = hCDFXExpected[cdfRowEnd - 1];
            hPDFYExpected[j] = totalSum;

            if(totalSum != 0.0f)
            {
                std::transform(hPDFXExpected.cbegin() + dataRowStart,
                               hPDFXExpected.cbegin() + dataRowEnd,
                               hPDFXExpected.begin() + dataRowStart,
                               [&](float f)
                               {
                                   return f / totalSum;
                               });
                std::transform(hCDFXExpected.cbegin() + cdfRowStart,
                               hCDFXExpected.cbegin() + cdfRowEnd,
                               hCDFXExpected.begin() + cdfRowStart,
                               [&](float f)
                               {
                                   return f / totalSum;
                               });
            }
        }

        static constexpr float DELTA_Y = 1.0f / static_cast<float>(Y - 1);
        hScratchpad.resize(Y);
        hScratchpad[0] = 0.0f;
        for(uint32_t i = 1; i < Y; i++)
        {
            hScratchpad[i] = TrapezoidArea(hPDFYExpected[i - 1], hPDFYExpected[i], DELTA_Y);
        }
        std::inclusive_scan(hScratchpad.cbegin(), hScratchpad.cbegin() + Y,
                            hCDFYExpected.begin());
        float totalSum = hCDFYExpected.back();

        if(totalSum != 0.0f)
        {
            std::transform(hPDFYExpected.cbegin(), hPDFYExpected.cend(),
                           hPDFYExpected.begin(),
                           [&](float f)
                           {
                               return f / totalSum;
                           });
            std::transform(hCDFYExpected.cbegin(), hCDFYExpected.cend(),
                           hCDFYExpected.begin(),
                           [&](float f)
                           {
                               return f / totalSum;
                           });
        }

        // Checks
        for(uint32_t i = 0; i < PIX_COUNT; i++)
        {
            EXPECT_NEAR(hPDFXExpected[i], hPDFX[i], MathConstants::LargeEpsilon);
        }
        for(uint32_t i = 0; i < X_CDF_COUNT; i++)
        {
            EXPECT_NEAR(hCDFXExpected[i], hCDFX[i], MathConstants::LargeEpsilon);
        }
        for(uint32_t i = 0; i < Y; i++)
        {
            EXPECT_NEAR(hPDFYExpected[i], hPDFY[i], MathConstants::LargeEpsilon);
        }
        for(uint32_t i = 0; i < Y_CDF_COUNT; i++)
        {
            EXPECT_NEAR(hCDFYExpected[i], hCDFY[i], MathConstants::LargeEpsilon);
        }
    }
}