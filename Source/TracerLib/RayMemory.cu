#include "hip/hip_runtime.h"
#include "RayMemory.h"

#include <hipcub/hipcub.hpp>
#include <type_traits>

#include "RayLib/Log.h"
#include "RayLib/MemoryAlignment.h"

#include "CudaConstants.h"
#include "TracerDebug.h"

static constexpr uint32_t INVALID_LOCATION = std::numeric_limits<uint32_t>::max();

struct ValidSplit
{
    __device__ __host__
    __forceinline__ bool operator()(const uint32_t &ids) const
    {
        return (ids != INVALID_LOCATION);
    }
};

__global__ void FillMatIdsForSortKC(HitKey* gKeys, RayId* gIds,
                                    const HitKey* gWorkKeys,
                                    uint32_t rayCount)
{
    // Grid Stride Loop
    for(uint32_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
        globalId < rayCount;
        globalId += blockDim.x * gridDim.x)
    {
        gKeys[globalId] = gWorkKeys[globalId];
        gIds[globalId] = globalId;
    }
}

__global__ void ResetHitKeysKC(HitKey* gKeys,
                               HitKey key, uint32_t rayCount)
{
    // Grid Stride Loop
    for(uint32_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
        globalId < rayCount;
        globalId += blockDim.x * gridDim.x)
    {
        gKeys[globalId] = key;
    }
}

__global__ void ResetHitIdsKC(HitKey* gAcceleratorKeys, RayId* gIds,
                              const RayGMem* gRays, uint32_t rayCount)
{
    // Grid Stride Loop
    for(uint32_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
        globalId < rayCount;
        globalId += blockDim.x * gridDim.x)
    {
        gIds[globalId] = globalId;
        gAcceleratorKeys[globalId] = HitKey::InvalidKey;
    }
}

__global__ void FindSplitsSparseKC(uint32_t* gPartLoc,
                                   const HitKey* gKeys,
                                   const uint32_t locCount)
{
    for(uint32_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
        globalId < locCount;
        globalId += blockDim.x * gridDim.x)
    {
        HitKey key = gKeys[globalId];
        HitKey keyN = gKeys[globalId + 1];

        uint16_t keyBatch = HitKey::FetchBatchPortion(key);
        uint16_t keyNBatch = HitKey::FetchBatchPortion(keyN);

        // Write location if split is found
        if(keyBatch != keyNBatch) gPartLoc[globalId + 1] = globalId + 1;
        else gPartLoc[globalId + 1] = INVALID_LOCATION;
    }

    // Init first location also
    if((blockIdx.x * blockDim.x + threadIdx.x) == 0)
        gPartLoc[0] = 0;
}

__global__ void FindSplitBatchesKC(uint16_t* gBatches,
                                   const uint32_t* gDenseIds,
                                   const HitKey* gSparseKeys,
                                   const uint32_t locCount)
{
    for(uint32_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
        globalId < locCount;
        globalId += blockDim.x * gridDim.x)
    {
        uint32_t index = gDenseIds[globalId];
        HitKey key = gSparseKeys[index];
        gBatches[globalId] = HitKey::FetchBatchPortion(key);
    }
}

void RayMemory::ResizeRayOut(uint32_t rayCount, HitKey baseBoundMatKey)
{
    // Align to proper memory strides
    size_t sizeOfWorkKeys = sizeof(HitKey) * rayCount;
    sizeOfWorkKeys = Memory::AlignSize(sizeOfWorkKeys);
    size_t sizeOfRays = rayCount * sizeof(RayGMem);
    sizeOfRays =  Memory::AlignSize(sizeOfRays);
    //size_t sizeOfAuxiliary = rayCount * perRayAuxSize;
    //sizeOfAuxiliary = Memory::AlignSize(sizeOfAuxiliary);

    size_t requiredSize = sizeOfRays + sizeOfWorkKeys;
    if(memOut.Size() < requiredSize)
    {
        memOut = DeviceMemory();
        memOut = std::move(DeviceMemory(requiredSize));
    }

    size_t offset = 0;
    std::uint8_t* dRay = static_cast<uint8_t*>(memOut);
    dRayOut = reinterpret_cast<RayGMem*>(dRay + offset);
    offset += sizeOfRays;
    //dRayAuxOut = reinterpret_cast<void*>(dRay + offset);
    //offset += sizeOfAuxiliary;
    dWorkKeys = reinterpret_cast<HitKey*>(dRay + offset);
    offset += sizeOfWorkKeys;
    assert(requiredSize == offset);

    // Initialize memory
    if(rayCount != 0)
        leaderDevice.GridStrideKC_X(0, 0, rayCount,
                                    ResetHitKeysKC,
                                    dWorkKeys, baseBoundMatKey,
                                    rayCount);
}

RayMemory::RayMemory(const CudaGPU& g)
    : leaderDevice(g)
{}

void RayMemory::SwapRays()
{
    std::swap(memIn, memOut);
    std::swap(dRayIn, dRayOut);
}

void RayMemory::ResetHitMemory(uint32_t rayCount, size_t hitStructSize)
{
    size_t sizeOfTransformIds = sizeof(TransformId) * rayCount;
    sizeOfTransformIds = Memory::AlignSize(sizeOfTransformIds);

    size_t sizeOfPrimitiveIds = sizeof(PrimitiveId) * rayCount;
    sizeOfPrimitiveIds = Memory::AlignSize(sizeOfPrimitiveIds);
    
    size_t sizeOfHitStructs = hitStructSize * rayCount;
    sizeOfHitStructs = Memory::AlignSize(sizeOfHitStructs);
    
    size_t sizeOfIds = sizeof(RayId) * rayCount;
    sizeOfIds = Memory::AlignSize(sizeOfIds);

    size_t sizeOfAcceleratorKeys = sizeof(HitKey) * rayCount;
    sizeOfAcceleratorKeys = Memory::AlignSize(sizeOfAcceleratorKeys);

    // Find out sort auxiliary storage
    hipcub::DoubleBuffer<HitKey::Type> dbKeys(nullptr, nullptr);
    hipcub::DoubleBuffer<RayId> dbIds(nullptr, nullptr);
    CUDA_CHECK(hipcub::DeviceRadixSort::SortPairsDescending(nullptr, cubSortMemSize,
                                                         dbKeys, dbIds,
                                                         static_cast<int>(rayCount)));

    // Check if while partitioning  double buffer data is
    // enough for using (Unique and Scan) algos
    uint32_t* in = nullptr;
    uint32_t* out = nullptr;
    uint32_t* count = nullptr;
    CUDA_CHECK(hipcub::DeviceSelect::If(nullptr, cubIfMemSize,
                                     in, out, count,
                                     static_cast<int>(rayCount),
                                     ValidSplit()));

    // Select algo reads from split locations and writes to backbuffer Ids (half is used)
    // uses backbuffer ids other half as auxiliary buffer
    // This code tries to increase it accordingly
    // Output Count of If also should be considered (add sizeof uint32_t)
    size_t sizeOfTempMemory = std::max(cubSortMemSize, cubIfMemSize + sizeof(uint32_t));
    sizeOfTempMemory = Memory::AlignSize(sizeOfTempMemory);

    // Finally allocate
    size_t requiredSize = ((sizeOfIds + sizeOfAcceleratorKeys) * 2 +
                           sizeOfTransformIds +
                           sizeOfPrimitiveIds +
                           sizeOfHitStructs +
                           sizeOfTempMemory);

    // Reallocate if memory is not enough
    if(memHit.Size() < requiredSize)
    {
        memHit = DeviceMemory();
        memHit = std::move(DeviceMemory(requiredSize));
    }
 
    // Populate pointers
    size_t offset = 0;
    std::uint8_t* dBasePtr = static_cast<uint8_t*>(memHit);
    dTransformIds = reinterpret_cast<TransformId*>(dBasePtr + offset);
    offset += sizeOfTransformIds;
    dPrimitiveIds = reinterpret_cast<PrimitiveId*>(dBasePtr + offset);
    offset += sizeOfPrimitiveIds;
    dHitStructs = HitStructPtr(reinterpret_cast<void*>(dBasePtr + offset), static_cast<int>(hitStructSize));
    offset += sizeOfHitStructs;
    dIds0 = reinterpret_cast<RayId*>(dBasePtr + offset);
    offset += sizeOfIds;
    dKeys0 = reinterpret_cast<HitKey*>(dBasePtr + offset);
    offset += sizeOfAcceleratorKeys;
    dIds1 = reinterpret_cast<RayId*>(dBasePtr + offset);
    offset += sizeOfIds;
    dKeys1 = reinterpret_cast<HitKey*>(dBasePtr + offset);
    offset += sizeOfAcceleratorKeys;
    dTempMemory = reinterpret_cast<void*>(dBasePtr + offset);
    offset += sizeOfTempMemory;
    assert(requiredSize == offset);

    dCurrentIds = dIds0;
    dCurrentKeys = dKeys0;

    // Make nullptr if no hitstruct is needed
    if(sizeOfHitStructs == 0)
        dHitStructs = HitStructPtr(nullptr, static_cast<int>(hitStructSize));

    // Initialize memory
    leaderDevice.GridStrideKC_X(0, 0, rayCount,
                                ResetHitIdsKC,
                                dCurrentKeys, dCurrentIds, dRayIn,
                                static_cast<uint32_t>(rayCount));
}

void RayMemory::SortKeys(RayId*& ids, HitKey*& keys,
                         uint32_t count,
                         const Vector2i& bitMaxValues)
{
    // Sort Call over buffers
    HitKey* keysOther = (dCurrentKeys == dKeys0) ? dKeys1 : dKeys0;
    RayId* idsOther = (dCurrentIds == dIds0) ? dIds1 : dIds0;
    hipcub::DoubleBuffer<HitKey::Type> dbKeys(reinterpret_cast<HitKey::Type*>(dCurrentKeys),
                                           reinterpret_cast<HitKey::Type*>(keysOther));
    hipcub::DoubleBuffer<RayId> dbIds(dCurrentIds,
                                   idsOther);
    int bitStart = 0;
    int bitEnd = bitMaxValues[1];

    // First sort internals
    if(bitStart != bitEnd)
    {
        CUDA_CHECK(hipcub::DeviceRadixSort::SortPairsDescending(dTempMemory, cubSortMemSize,
                                                             dbKeys, dbIds,
                                                             static_cast<int>(count),
                                                             bitStart, bitEnd,
                                                             (hipStream_t)0,
                                                             false));
    }

    // Then sort batches
    bitStart = HitKey::IdBits;
    bitEnd = HitKey::IdBits + bitMaxValues[0];
    if(bitStart != bitEnd)
    {
        CUDA_CHECK(hipcub::DeviceRadixSort::SortPairsDescending(dTempMemory, cubSortMemSize,
                                                             dbKeys, dbIds,
                                                             static_cast<int>(count),
                                                             bitStart, bitEnd,
                                                             (hipStream_t)0,
                                                             false));
    }

    ids = dbIds.Current();
    keys = reinterpret_cast<HitKey*>(dbKeys.Current());
    dCurrentIds = ids;
    dCurrentKeys = keys;
}

RayPartitions<uint32_t> RayMemory::Partition(uint32_t rayCount)
{
    // Use double buffers for partition auxilary data
    RayId* dEmptyIds = (dCurrentIds == dIds0) ? dIds1 : dIds0;
    HitKey* dEmptyKeys = (dCurrentKeys == dKeys0) ? dKeys1 : dKeys0;

    // Generate Names that make sense for the operation
    // We have total of three buffers
    // Temp Memory will be used for temp memory
    // (it holds enough space for both sort and select)
    //
    // dSparseSplitIndices (a.k.a. dEmptyKeys)
    // will be used as intermediate buffer
    uint32_t* dSparseSplitIndices = reinterpret_cast<uint32_t*>(dEmptyKeys);
    uint32_t* dDenseSplitIndices = reinterpret_cast<uint32_t*>(dEmptyIds);
    uint32_t* dSelectCount = static_cast<uint32_t*>(dTempMemory);
    void* dSelectTempMemory = dSelectCount + 1;

    // Find Split Locations
    // Read from dKeys -> dEmptyKeys
    uint32_t locCount = rayCount - 1;
    leaderDevice.GridStrideKC_X(0, 0, rayCount,
                                FindSplitsSparseKC,
                                dSparseSplitIndices, dCurrentKeys, locCount);

    // Make Splits Dense
    // From dEmptyKeys -> dEmptyIds
    CUDA_CHECK(hipcub::DeviceSelect::If(dSelectTempMemory, cubIfMemSize,
                                     dSparseSplitIndices, dDenseSplitIndices, dSelectCount,
                                     static_cast<int>(rayCount),
                                     ValidSplit(),
                                     (hipStream_t)0,
                                     false));

    // Copy Reduced Count
    uint32_t hSelectCount;
    CUDA_CHECK(hipMemcpy(&hSelectCount, dSelectCount,
                          sizeof(uint32_t), hipMemcpyDeviceToHost));

    // Find The Hit Keys for each split
    // From dEmptyIds, dKeys -> dEmptyKeys
    uint16_t* dBatches = reinterpret_cast<uint16_t*>(dSparseSplitIndices);
    leaderDevice.GridStrideKC_X(0, 0, rayCount,
                                FindSplitBatchesKC,
                                dBatches,
                                dDenseSplitIndices,
                                dCurrentKeys,
                                hSelectCount);

    // We need to get dDenseIndices & dDenseKeys
    // Memcopy to vectors
    std::vector<uint16_t> hDenseKeys(hSelectCount);
    std::vector<uint32_t> hDenseIndices(hSelectCount);
    CUDA_CHECK(hipMemcpy(hDenseKeys.data(), dBatches,
                          sizeof(uint16_t) * hSelectCount,
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hDenseIndices.data(), dDenseSplitIndices,
                          sizeof(uint32_t) * hSelectCount,
                          hipMemcpyDeviceToHost));

    // Construct The Set
    // Add extra index to end as rayCount for cleaner code
    hDenseIndices.push_back(rayCount);
    RayPartitions<uint32_t> partitions;
    for(uint32_t i = 0; i < hSelectCount; i++)
    {
        uint32_t id = hDenseKeys[i];
        uint32_t offset = hDenseIndices[i];
        size_t count = hDenseIndices[i + 1] - hDenseIndices[i];
        partitions.emplace(ArrayPortion<uint32_t>{id, offset, count});
    }
    // Done!
    return std::move(partitions);
}

void RayMemory::FillMatIdsForSort(uint32_t rayCount)
{
    leaderDevice.GridStrideKC_X(0, 0, rayCount,
                                FillMatIdsForSortKC,
                                dCurrentKeys, dCurrentIds, dWorkKeys,
                                rayCount);
}

size_t RayMemory::TotalMemorySize()
{
    return memIn.Size() + memOut.Size() + memHit.Size();
}