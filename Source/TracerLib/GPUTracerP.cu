#include "GPUTracerP.h"

#include "RayLib/Camera.h"
#include "RayLib/Log.h"
#include "RayLib/TracerError.h"
#include "RayLib/TracerCallbacksI.h"
#include "RayLib/GPUSceneI.h"

#include "TracerDebug.h"
#include "GPUAcceleratorI.h"
#include "GPUWorkI.h"


//struct RayAuxBasic
//{
//    Vector3f        totalRadiance;
//    uint32_t        pixelId;
//    uint32_t        pixelSampleId;
//};
//
//std::ostream& operator<<(std::ostream& stream, const RayAuxBasic& v)
//{
//    stream << std::setw(0)
//            << v.pixelId << ", "
//            << v.pixelSampleId << ", "
//            << "{" << v.totalRadiance[0]
//            << "," << v.totalRadiance[0]
//            << "," << v.totalRadiance[0] << "}";
//    return stream;
//}

//GPUTracerP::GPUTracerP(CudaSystem& s,
//                       // Accelerators that are required
//                       // for hit loop
//                       GPUBaseAcceleratorI& ba,
//                       AcceleratorBatchMap& am,
//                       // Bits for sorting
//                       const Vector2i maxAccelBits,
//                       const Vector2i maxWorkBits,
//                       // Hit size for union allocation
//                       const uint32_t maxHitSize,
//                       // Initialization Param of tracer
//                       const TracerParameters& p)
//{
//    : cudaSystem(s)
//    , maxAccelBits(maxAccelBits)
//    , maxWorkBits(maxWorkBits)
//    , baseAccelerator(ba)
//    , accelBatches(am)
//    , params(p),
//    , maxHitSize(maxHitSize)
//    , rayMemory(*(s.GPUList().begin()))
//    , callbacks(nullptr)
//    , crashed(false)
//{}

TracerError GPUTracerP::Initialize()
{
    rngMemory = RNGMemory(params.seed, cudaSystem);
    return TracerError::OK;
}

void GPUTracerP::ResetHitMemory(uint32_t rayCount, HitKey baseBoundMatKey)
{
    currentRayCount = rayCount;
    rayMemory.ResizeRayOut(rayCount, baseBoundMatKey);
}

void GPUTracerP::HitRays()
{   
    if(crashed) return;

    // Sort and Partition happens on the leader device
    CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice().DeviceId()));

    // Tracer Logic interface
    const Vector2i& accBitCounts = maxAccelBits;
    const AcceleratorBatchMap& subAccelerators = accelBatches;
    // Reset Hit Memory for hit loop
    rayMemory.ResetHitMemory(currentRayCount, maxHitSize);
    // Make Base Accelerator to get ready for hitting
    baseAccelerator.GetReady(cudaSystem, currentRayCount);
    // Ray Memory Pointers
    RayGMem* dRays = rayMemory.Rays();
    HitKey* dWorkKeys = rayMemory.WorkKeys();
    TransformId* dTransfomIds = rayMemory.TransformIds();
    PrimitiveId* dPrimitiveIds = rayMemory.PrimitiveIds();
    HitStructPtr dHitStructs = rayMemory.HitStructs();
    // These are sorted etc.
    HitKey* dCurrentKeys = rayMemory.CurrentKeys();
    RayId*  dCurrentRayIds = rayMemory.CurrentIds();

    // Try to hit rays until no ray is left
    // (these rays will be assigned with a material)
    // outside rays are also assigned with a material (which is special)
    uint32_t rayCount = currentRayCount;
    // At start all rays are valid
    uint32_t validRayOffset = 0;
    while(rayCount > 0)
    {
        //Debug::DumpMemToFile("dAccKeys", dCurrentKeys, currentRayCount);
        //Debug::DumpMemToFile("dRayIds", dCurrentRayIds, currentRayCount);

        // Traverse accelerator
        // Base accelerator provides potential hits
        // Cannot provide an absolute hit (its not its job)
        baseAccelerator.Hit(cudaSystem,
                            dTransfomIds, 
                            dCurrentKeys + validRayOffset,
                            dRays,
                            dCurrentRayIds + validRayOffset,
                            rayCount);

        // Wait all GPUs to finish...
        cudaSystem.SyncGPUMainStreamAll();

        //Debug::DumpMemToFile("dAccKeys", dCurrentKeys, currentRayCount);
        //Debug::DumpMemToFile("dRayIds", dCurrentRayIds, currentRayCount);

        // Base accelerator traverses the data partially
        // Updates current key (which represents inner accelerator batch and id)

        // After that, system sorts rays according to the keys
        // and partitions the array according to batches

        // Sort and Partition happens on the leader device
        CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice().DeviceId()));

        // Sort initial results (in order to partition and launch kernels accordingly)
        // Sort is radix sort.
        // We sort inner indices in addition to batches results for better data locality
        // We only sort up-to a certain bit (radix sort) which is tied to
        // accelerator count
        // Move offset to skip null bathces
        rayMemory.SortKeys(dCurrentRayIds, dCurrentKeys, rayCount, accBitCounts);
        // Parition to sub accelerators
        //
        // There may be invalid rays sprinkled along the array.
        // Holes occur in the structure since in previous iteration,
        // a material may required to write N rays for its output (which is defined
        // by the material) but it wrote < N rays.
        //
        // One of the main examples for such behaviour can be transparent objects
        // where ray may be only reflected (instead of refrating and reflecting) because
        // of the total internal reflection phenomena.
        auto portions = rayMemory.Partition(rayCount);

        //Debug::DumpMemToFile("dAccKeys", dCurrentKeys, currentRayCount);
        //Debug::DumpMemToFile("dRayIds", dCurrentRayIds, currentRayCount);

        // Reorder partitions for efficient calls
        // (group partitions into gpus and order for better async access)
        // ....
        // TODO:
        const int totalGPU = static_cast<int>(cudaSystem.GPUList().size());
        const auto& gpus = cudaSystem.GPUList();
        auto currentGPU = gpus.begin();

        // For each partition
        for(const auto& p : portions)
        {
            // Find Accelerator
            // Since there is no batch for invalid keys
            // that partition will be automatically be skipped
            auto loc = subAccelerators.find(p.portionId);
            if(loc == subAccelerators.end()) continue;

            RayId* dRayIdStart = dCurrentRayIds + validRayOffset + p.offset;
            HitKey* dCurrentKeyStart = dCurrentKeys + validRayOffset + p.offset;

            // Run local hit kernels
            // Local hit kernels returns a material key
            // and primitive inner id.
            // Since materials are batched for both material and
            loc->second->Hit(*currentGPU,
                             // O
                             dWorkKeys,
                             dPrimitiveIds,
                             dHitStructs,
                             // I-O
                             dRays,
                             // Input
                             dTransfomIds,
                             dRayIdStart,
                             dCurrentKeyStart,
                             static_cast<uint32_t>(p.count));

            // Split to GPUs
            currentGPU++;
            if(currentGPU == gpus.end()) currentGPU = gpus.begin();

            // Hit function updates material key,
            // primitive id and struct if this hit is accepted
        }
        //printf("=====================================================\n");

        // Update new ray count
        // On partition array check first partition
        // it may contain invalid key meaning
        // those rays are totally processed
        // change the offset so that
        // we skip those rays
        auto nullPortion = portions.begin();
        if(nullPortion->portionId == HitKey::NullBatch)
            rayCount = static_cast<uint32_t>(nullPortion->offset);

        // Iteration is done
        // We cant continue loop untill these kernels are finished
        // on gpu(s)
        //
        // Tracer logic mostly utilizies mutiple GPUs so we need to
        // wait all GPUs to finish
        cudaSystem.SyncGPUAll();
    }

    // At the end of iteration all rays found a material, primitive
    // and interpolation weights (which should be on hitStruct)
    //printf("FRAME END\n");
}

void GPUTracerP::WorkRays(const WorkBatchMap& workMap, HitKey baseBoundMatKey)
{
    // Sort and Partition happens on leader device
    CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice().DeviceId()));

    // Ray Memory Pointers
    const RayGMem* dRays = rayMemory.Rays();
    const void* dRayAux = rayMemory.RayAux<void>();
    const HitStructPtr dHitStructs = rayMemory.HitStructs();
    const PrimitiveId* dPrimitiveIds = rayMemory.PrimitiveIds();
    // These are sorted etc.
    HitKey* dCurrentKeys = rayMemory.CurrentKeys();
    RayId* dCurrentRayIds = rayMemory.CurrentIds();

    // Material Interfaces    
    uint32_t rayCount = currentRayCount;

    // Copy materialKeys to currentKeys
    // to make it ready for sorting
    rayMemory.FillMatIdsForSort(rayCount);

    // Sort with respect to the materials keys
    rayMemory.SortKeys(dCurrentRayIds, dCurrentKeys, rayCount, maxWorkBits);

    // Parition w.r.t. material batch
    auto portions = rayMemory.Partition(rayCount);
    // Use partition lis to find out
    // total potential output ray count
    uint32_t totalOutRayCount = 0;
    for(const auto& p : portions)
    {
        // Skip if null batch or unfound material
        if(p.portionId == HitKey::NullBatch) continue;
        auto loc = workMap.find(p.portionId);
        if(loc == workMap.end()) continue;

        totalOutRayCount += static_cast<uint32_t>(p.count) *
                            loc->second->OutRayCount();
    }

    // Allocate output ray memory
    rayMemory.ResizeRayOut(totalOutRayCount, baseBoundMatKey);
    unsigned char* dAuxOut = rayMemory.RayAuxOut<unsigned char>();
    RayGMem* dRaysOut = rayMemory.RaysOut();
    HitKey* dBoundKeyOut = rayMemory.WorkKeys();

    // Reorder partitions for efficient calls
    // (sort by gpu and order for better async access)
    // ....
    // TODO:

    // For each partition
    uint32_t outOffset = 0;
    for(auto pIt = portions.crbegin();
        pIt != portions.crend(); pIt++)
    {
        const auto& p = (*pIt);

        // Skip if null batch or unfound material
        if(p.portionId == HitKey::NullBatch) continue;
        auto loc = workMap.find(p.portionId);
        if(loc == workMap.end()) continue;

        // Relativize input & output pointers
        const RayId* dRayIdStart = dCurrentRayIds + p.offset;
        const HitKey* dKeyStart = dCurrentKeys + p.offset;
        // Output
        RayGMem* dRayOutStart = dRaysOut + outOffset;        
        HitKey* dBoundKeyStart = dBoundKeyOut + outOffset;

        // Actual Shade Call
        loc->second->Work(dBoundKeyStart,
                          dRayOutStart,
                          //  Input
                          dRays,
                          dPrimitiveIds,
                          dHitStructs,
                          // Ids
                          dKeyStart,
                          dRayIdStart,
                          //
                          outOffset,
                          static_cast<uint32_t>(p.count),
                          rngMemory);

        // Since output is dynamic (each material may write multiple rays)
        // add offsets to find proper count
        outOffset += static_cast<uint32_t>(p.count * loc->second->OutRayCount());
    }
    assert(totalOutRayCount == outOffset);
    currentRayCount = totalOutRayCount;

    // Again wait all of the GPU's since
    // CUDA functions will be on multiple-gpus
    cudaSystem.SyncGPUAll();

    // Shading complete
    // Now make "RayOut" to "RayIn"
    // and continue
    rayMemory.SwapRays();
}

void GPUTracerP::SetImagePixelFormat(PixelFormat f)
{
    imgMemory.SetPixelFormat(f, cudaSystem);
}

void GPUTracerP::ReportionImage(Vector2i start,
                                Vector2i end)
{
    imgMemory.Reportion(start, end, cudaSystem);
}

void GPUTracerP::ResizeImage(Vector2i resolution)
{
    imgMemory.Resize(resolution);
}

void GPUTracerP::ResetImage()
{
    imgMemory.Reset(cudaSystem);
}

template <class... Args>
inline void GPUTracerP::SendLog(const char* format, Args... args)
{
    if(!options.verbose) return;

    size_t size = snprintf(nullptr, 0, format, args...);
    std::string s(size, '\0');
    snprintf(&s[0], size, format, args...);
    if(callbacks) callbacks->SendLog(s);
}

void GPUTracerP::SendError(TracerError e, bool isFatal)
{
    if(callbacks) callbacks->SendError(e);
    crashed = isFatal;
}

void GPUTracerP::SetCommonOptions(const TracerCommonOpts& opts)
{
    options = opts;
}

void GPUTracerP::Finalize()
{
    if(crashed) return;
    SendLog("Finalizing...");
   
    // Determine Size
    Vector2i pixelCount = imgMemory.SegmentSize();
    Vector2i start = imgMemory.SegmentOffset();
    Vector2i end = start + imgMemory.SegmentSize();
    size_t offset = (static_cast<size_t>(pixelCount[0])* pixelCount[1] *
                     imgMemory.PixelSize());

    // Flush Devices and Get the Image
    cudaSystem.SyncGPUAll();
    std::vector<Byte> imageData = imgMemory.GetImageToCPU(cudaSystem);

    size_t pixelCount1D = static_cast<size_t>(pixelCount[0]) * pixelCount[1];

    // Launch finished image
    if(callbacks) callbacks->SendImage(std::move(imageData),
                                       imgMemory.Format(),
                                       offset,
                                       start, end);
    SendLog("Image sent!");
}


