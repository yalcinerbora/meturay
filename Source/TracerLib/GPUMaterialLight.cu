#include "GPUMaterialLight.cuh"
#include "RayLib/ColorConversion.h"

SceneError LightMatConstant::InitializeGroup(const NodeListing& materialNodes, double time,
                                             const std::string& scenePath)
{
    constexpr const char* RADIANCE = "radiance";

    std::vector<Vector3> radianceCPU;
    uint32_t i = 0;
    for(const auto& sceneNode : materialNodes)
    {
        std::vector<Vector3> radiances = sceneNode->AccessVector3(RADIANCE);
        radianceCPU.insert(radianceCPU.end(), radiances.begin(), radiances.end());
    
        // Calculate Distributions
        for(const Vector3& r : radiances)
        {
            float lum = Utility::RGBToLuminance(r);
            lightRadianceDistributions.emplace_back(std::vector<float>(1, lum));
        }
        // Generate Id pairs
        const auto& ids = sceneNode->Ids();
        for(IdPair id : ids)
        {
            innerIds.emplace(std::make_pair(id.first, i));
            i++;
        }
    }

    // Alloc etc
    size_t dRadianceSize = radianceCPU.size() * sizeof(Vector3);
    memory = std::move(DeviceMemory(dRadianceSize));
    Vector3f* dRadiances = static_cast<Vector3f*>(memory);
    CUDA_CHECK(hipMemcpy(dRadiances, radianceCPU.data(), dRadianceSize,
                          hipMemcpyHostToDevice));

    dData = LightMatData{dRadiances};
    return SceneError::OK;
}

SceneError LightMatConstant::ChangeTime(const NodeListing& materialNodes, double time,
                                        const std::string& scenePath)
{
    // TODO: Implement
    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}

const GPUDistribution1D& LightMatConstant::LightDistribution(uint32_t materialId) const
{
    return lightRadianceDistributions[innerIds.at(materialId)].DistributionGPU();
}

SceneError LightMatTextured::InitializeGroup(const NodeListing& materialNodes, double time,
                                             const std::string& scenePath)
{
    // TODO: Implement
    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}

SceneError LightMatTextured::ChangeTime(const NodeListing& materialNodes, double time,
                                        const std::string& scenePath)
{
    // TODO: Implement
    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}

const GPUDistribution2D& LightMatTextured::LightDistribution(uint32_t materialId) const
{
    return lightRadianceDistributions[innerIds.at(materialId)].DistributionGPU();
}

SceneError LightMatCube::InitializeGroup(const NodeListing& materialNodes, double time,
                                         const std::string& scenePath)
{
    // TODO: Implement
    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}

SceneError LightMatCube::ChangeTime(const NodeListing& materialNodes, double time,
                                   const std::string& scenePath)
{
    // TODO: Implement
    return SceneError::MATERIAL_TYPE_INTERNAL_ERROR;
}

const GPUDistribution2D& LightMatCube::LightDistribution(uint32_t materialId) const
{
    return lightRadianceDistributions[innerIds.at(materialId)].DistributionGPU();
}