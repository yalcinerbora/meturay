#include "TracerBase.h"

#include "RayLib/Camera.h"
#include "RayLib/Log.h"
#include "RayLib/TracerError.h"
#include "RayLib/TracerCallbacksI.h"

#include "TracerDebug.h"
#include "GPUAcceleratorI.h"
#include "GPUMaterialI.h"
#include "TracerLogicI.h"


void TracerBase::SendError(TracerError e, bool isFatal)
{
    if(callbacks) callbacks->SendError(e);
    healthy = isFatal;
}

void TracerBase::HitRays()
{
    // Tracer Logic interface
    const Vector2i& accBitCounts = currentLogic->SceneAcceleratorMaxBits();
    GPUBaseAcceleratorI& baseAccelerator = currentLogic->BaseAcelerator();
    const AcceleratorBatchMappings& subAccelerators = currentLogic->AcceleratorBatches();

    // Reset Hit Memory for hit loop
    rayMemory.ResetHitMemory(currentRayCount, currentLogic->HitStructSize());

    // Make Base Accelerator to get ready for hitting
    baseAccelerator.GetReady(currentRayCount);

    // Ray Memory Pointers
    RayGMem* dRays = rayMemory.Rays();
    HitKey* dMaterialKeys = rayMemory.MaterialKeys();
    TransformId* dTransfomIds = rayMemory.TransformIds();
    PrimitiveId* dPrimitiveIds = rayMemory.PrimitiveIds();
    HitStructPtr dHitStructs = rayMemory.HitStructs();
    // These are sorted etc.
    HitKey* dCurrentKeys = rayMemory.CurrentKeys();
    RayId*  dCurrentRayIds = rayMemory.CurrentIds();

    // Try to hit rays until no ray is left
    // (these rays will be assigned with a material)
    // outside rays are also assigned with a material (which is special)
    uint32_t rayCount = currentRayCount;
    while(rayCount > 0)
    {
        // Traverse accelerator
        // Base accelerator provides potential hits
        // Cannot provide an absolute hit (its not its job)
        baseAccelerator.Hit(dTransfomIds, dCurrentKeys, dRays, dCurrentRayIds,
                             rayCount);

        // Base accelerator traverses the data partially
        // Updates current key (which represents innter accelerator batch and id)

        // After that, system sorts rays according to the keys
        // and partitions the array according to batches

        // Sort and Partition happens on the leader device
        CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice()));

        // Sort initial results (in order to partition and launch kernels accordingly)
        // Sort is radix sort.
        // We sort inner indices in addition to batches results for better data locality
        // We only sort up-to a certain bit (radix sort) which is tied to
        // accelerator count
        rayMemory.SortKeys(dCurrentRayIds, dCurrentKeys, rayCount, accBitCounts);
        // Parition to sub accelerators
        //
        // There may be invalid rays sprinkled along the array.
        // Holes occur in the structure since in previous iteration,
        // a material may required to write N rays for its output (which is defined
        // by the material) but it wrote < N rays.
        //
        // One of the main examples for such behaviour can be transparent objects
        // where ray may be only reflected (instead of refrating and reflecting) because
        // of the total internal reflection phenomena.
        auto portions = rayMemory.Partition(rayCount);

        // Reorder partitions for efficient calls
        // (group partitions into gpus and order for better async access)
        // ....
        // TODO:
        const int totalGPU = static_cast<int>(CudaSystem::GPUList().size());
        int currentGPU = 0;

        // For each partition
        for(const auto& p : portions)
        {
            // Find Accelerator
            // Since there is no batch for invalid keys
            // that partition will be automatically be skipped
            auto loc = subAccelerators.find(p.portionId);
            if(loc == subAccelerators.end()) continue;

            RayId* dRayIdStart = dCurrentRayIds + p.offset;
            HitKey* dCurrentKeyStart = dCurrentKeys + p.offset;

            // Run local hit kernels
            // Local hit kernels returns a material key
            // and primitive inner id.
            // Since materials are batched for both material and
            loc->second->Hit(currentGPU,
                             // O
                             dMaterialKeys,
                             dPrimitiveIds,
                             dHitStructs,
                             // I-O
                             dRays,
                             // Input
                             dTransfomIds,
                             dRayIdStart,
                             dCurrentKeyStart,
                             static_cast<uint32_t>(p.count));

            // Split to GPUs
            currentGPU = (currentGPU + 1) % totalGPU;

            // Hit function updates material key,
            // primitive id and struct if this hit is accepted
        }

        // Update new ray count
        // On partition array check last partition
        // it may contain invalid key meaning
        // those rays are totally processed
        auto iterator = portions.begin();
        if(iterator->portionId == HitKey::NullBatch)
        {
            rayCount = static_cast<uint32_t>(iterator->offset);
        }

        // Iteration is done
        // We cant continue loop untill these kernels are finished
        // on gpu(s)
        //
        // Tracer logic mostly utilizies mutiple GPUs so we need to
        // wait all GPUs to finish
        CudaSystem::SyncAllGPUs();
    }
    // At the end of iteration all rays found a material, primitive
    // and interpolation weights (which should be on hitStruct)
}

void TracerBase::ShadeRays()
{
    const Vector2i matMaxBits = currentLogic->SceneMaterialMaxBits();
    // Image Memory Pointers
    Vector4f* dImageMem = outputImage.GMem<Vector4f>();

    // Ray Memory Pointers
    const RayGMem* dRays = rayMemory.Rays();
    const void* dRayAux = rayMemory.RayAux<void>();
    const HitStructPtr dHitStructs = rayMemory.HitStructs();
    const PrimitiveId* dPrimitiveIds = rayMemory.PrimitiveIds();
    // These are sorted etc.
    HitKey* dCurrentKeys = rayMemory.CurrentKeys();
    RayId* dCurrentRayIds = rayMemory.CurrentIds();

    // Material Interfaces
    const MaterialBatchMappings& materials = currentLogic->MaterialBatches();
    uint32_t rayCount = currentRayCount;

    // Sort and Partition happens on leader device
    CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice()));

    // Copy materialKeys to currentKeys
    // to make it ready for sorting
    rayMemory.FillRayIdsForSort(rayCount);

    // Sort with respect to the materials keys
    rayMemory.SortKeys(dCurrentRayIds, dCurrentKeys, rayCount, matMaxBits);

    // Parition w.r.t. material batch
    auto portions = rayMemory.Partition(rayCount);

    // Use partition lis to find out
    // total potential output ray count
    size_t totalOutRayCount = 0;
    for(const auto& p : portions)
    {
        // Skip if null batch or unfound material
        if(p.portionId == HitKey::NullBatch) continue;
        auto loc = materials.find(p.portionId);
        if(loc == materials.end()) continue;

        totalOutRayCount += p.count * loc->second->OutRayCount();
    }

    // Allocate output ray memory
    rayMemory.ResizeRayOut(totalOutRayCount, currentLogic->PerRayAuxDataSize());
    unsigned char* dAuxOut = rayMemory.RayAuxOut<unsigned char>();
    RayGMem* dRaysOut = rayMemory.RaysOut();

    // Reorder partitions for efficient calls
    // (sort by gpu and order for better async access)
    // ....
    // TODO:

    // For each partition
    size_t outOffset = 0;
    for(const auto& p : portions)
    {
        // Skip if null batch or unfound material
        if(p.portionId == HitKey::NullBatch) continue;
        auto loc = materials.find(p.portionId);
        if(loc == materials.end()) continue;

        // Since output is dynamic (each material may write multiple rays)
        // add offsets to find proper count
        outOffset += p.count * loc->second->OutRayCount();

        // Relativize input & output pointers
        const RayId* dRayIdStart = dCurrentRayIds + p.offset;
        const HitKey* dKeyStart = dCurrentKeys + p.offset;
        RayGMem* dRayOutStart = dRaysOut + outOffset;
        void* dAuxOutStart = dAuxOut + (outOffset * currentLogic->PerRayAuxDataSize());

        // Actual Shade Call
        loc->second->ShadeRays(// Output
                               dImageMem,
                               //
                               dRayOutStart,
                               dAuxOutStart,
                               //  Input
                               dRays,
                               dRayAux,
                               dPrimitiveIds,
                               dHitStructs,
                               //
                               dKeyStart,
                               dRayIdStart,

                               static_cast<uint32_t>(p.count),
                               rngMemory);

    }
    assert(totalOutRayCount == outOffset);
    currentRayCount = static_cast<uint32_t>(totalOutRayCount);

    // Again wait all of the GPU's since
    // CUDA functions will be on multiple-gpus
    CudaSystem::SyncAllGPUs();

    // Shading complete
    // Now make "RayOut" to "RayIn"
    // and continue
    rayMemory.SwapRays();
}

TracerBase::TracerBase()
    : callbacks(nullptr)
    , currentRayCount(0)
    , currentLogic(nullptr)
    , healthy(false)
{}

TracerError TracerBase::Initialize(int leaderGPUId)
{
    // No logic set for initalization
    if(currentLogic == nullptr) return TracerError::NO_LOGIC_SET;

    // Device initalization
    rayMemory.SetLeaderDevice(leaderGPUId);

    // Construct Accelerators
    GPUBaseAcceleratorI& baseAccelerator = currentLogic->BaseAcelerator();
    const AcceleratorGroupList& acceleratorGroups = currentLogic->AcceleratorGroups();

    baseAccelerator.Constrcut();
    for(const auto& accel : acceleratorGroups)
    {
        accel->ConstructAccelerators();
    }

    CUDA_CHECK(hipSetDevice(leaderGPUId));

    // All seems fine mark tracer as healthy
    healthy = true;
    return TracerError::OK;
}

void TracerBase::SetOptions(const TracerOptions& opts)
{
    options = opts;
}

void TracerBase::RequestBaseAccelerator()
{}

void TracerBase::RequestAccelerator(HitKey key)
{}

void TracerBase::AttachLogic(TracerBaseLogicI& logic)
{
    // Init and set Tracer System
    TracerError e = TracerError::OK;
    if((e = logic.Initialize()) != TracerError::OK)
    {
        if(callbacks) callbacks->SendError(e);
    }
    currentLogic = &logic;

    // Initialize RNG Memory
    rngMemory = RNGMemory(logic.Seed());
}

void TracerBase::GenerateInitialRays(const GPUScene& scene,
                                     int cameraId,
                                     int samplePerLocation)
{
    if(!healthy) return;

    // Delegate camera ray generation to tracer system
    currentRayCount = static_cast<uint32_t>(currentLogic->GenerateRays(rayMemory, rngMemory,
                                                                       scene, cameraId, samplePerLocation,
                                                                       outputImage.Resolution(),
                                                                       outputImage.SegmentOffset(),
                                                                       outputImage.SegmentSize()));

    // You can only write to out buffer of the ray memory
    // Make that memory in rays for hit/shade system
    rayMemory.SwapRays();
    sampleCountPerRay = samplePerLocation * samplePerLocation;
}

bool TracerBase::Continue()
{
    return (currentRayCount > 0) && healthy;
}

void TracerBase::Render()
{
    if(!healthy) return;
    if(currentRayCount == 0) return;

    HitRays();
    METU_LOG("-----------------------------");
    ShadeRays();
    METU_LOG("-----------------------------");
    METU_LOG("-----------------------------");
}

void TracerBase::FinishSamples()
{
    // Normally if ray reaches to boundary material
    // its result is written to the ray data
    // but a ray not always reach to boundary material
    // if a pre determined

    // Determine Size
    Vector2i start = outputImage.SegmentOffset();
    Vector2i end = start + outputImage.SegmentSize();
    size_t size = (outputImage.SegmentSize()[0] *
                   outputImage.SegmentSize()[1] *
                   outputImage.PixelSize());

    // Data
    std::vector<Byte> data(size);
    CUDA_CHECK(hipMemcpy(data.data(), outputImage.GMem<Byte>(),
                          size, hipMemcpyDeviceToHost));

    // Launch finished image
    if(callbacks)callbacks->SendImage(std::move(data),
                                      outputImage.Format,
                                      sampleCountPerRay,
                                      start, end);

    // TODO: Do this
    if(!healthy) return;
}

void TracerBase::SetImagePixelFormat(PixelFormat f)
{
    outputImage.SetPixelFormat(f);
}

void TracerBase::ReportionImage(Vector2i start,
                                Vector2i end)
{
    outputImage.Reportion(start, end);
}

void TracerBase::ResizeImage(Vector2i resolution)
{
    outputImage.Resize(resolution);
}

void TracerBase::ResetImage()
{
    outputImage.Reset();
}
