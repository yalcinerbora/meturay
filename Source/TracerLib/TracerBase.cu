#include "TracerBase.h"

#include "RayLib/Camera.h"
#include "RayLib/Log.h"
#include "RayLib/TracerError.h"

#include "TracerDebug.h"
#include "GPUAcceleratorI.h"
#include "GPUMaterialI.h"
#include "TracerLogicI.h"

void TracerBase::SendError(TracerError e, bool isFatal)
{
	if(errorFunc) errorFunc(e);
	healthy = isFatal;
}

void TracerBase::HitRays()
{
	// Tracer Logic interface
	const Vector2i accBitRange = tracerSystem->AcceleratorBitRange();
	const GPUBaseAcceleratorI* baseAccelerator = tracerSystem->BaseAcelerator();
	const AcceleratorGroupMappings& subAccelerators = tracerSystem->AcceleratorGroups();

	// Reset Hit Memory for hit loop
	rayMemory.ResetHitMemory(currentRayCount, tracerSystem->HitStructMaxSize());

	// Ray Memory Pointers
	RayGMem* dRays = rayMemory.Rays();	
	HitKey* dCurrentHits = rayMemory.CurrentHits();
	void* dHitStructs = rayMemory.HitStructs<unsigned char>();
	// These are sorted etc.
	HitKey* dPotentialHits = rayMemory.PotentialHits();	
	RayId*	dRayIds = rayMemory.RayIds();	
	
	// Try to hit rays until no ray is left 
	// (these rays will be assigned with a material)
	// outside rays are also assigned with a material (which is special)
	uint32_t rayCount = currentRayCount;
	while(rayCount > 0)
	{
		// Traverse accelerator
		// Base accelerator provides potential hits
		// Cannot provide an absolute hit (its not its job)
		baseAccelerator->Hit(dPotentialHits, dRays, dRayIds,
							 rayCount);

		// Base accelerator traverses the data partially
		// It delegates the rays to smaller accelerators
		// by writing their Id's to its portion in the key.

		// After that systems sorts ray hit list and key
		// and partitions the array this partitioning scheme 

		// Sort and Partition happens on the leader device
		CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice()));

		// Sort initial results (to partition and launch kernels accordingly)				
		rayMemory.SortKeys(dRayIds, dPotentialHits, rayCount, accBitRange);
		// Parition to sub accelerators
		// Remove the rays that are invalid.
		//
		// Partition code does not return invalid rays.
		// Invalid rays include empty rays (which are holes in the array)
		// or missed rays (that does not hit anything).
		// If accelerator bit segment is used for partitioning,
		// portions structure omits both of these type of rays.
		//
		// Holes occur in the structure since in previous iteration,
		// a material may required to write N rays for its output (which is defined
		// by the material) but it wrote < N rays.
		// 
		// One of the main examples for such behaviour can be transparent objects
		// where ray may be only reflected (instead of refrating and reflecting) because
		// of the total internal reflection phenomena.
		auto portions = rayMemory.Partition(rayCount, accBitRange);

		// For each partition
		for(const auto& p : portions)
		{
			auto loc = subAccelerators.find(p.portionId);
			if(loc == subAccelerators.end()) continue;

			// Run local hit kernels
			// These hit kernels can only modify actual hits
			// Potential HitKeys are used to fetch inner data
			RayId* dRayIdStart = dRayIds + p.offset;
			HitKey* dPotentialHitStart = dPotentialHits + p.offset;

			loc->second->Hit(dRays, dHitStructs, dCurrentHits,
							 dRayIdStart, dPotentialHitStart,
							 static_cast<uint32_t>(p.count));

			// Hit function updates the hitIds structure with its appropirate data,
			// internally, also it changes HitId structure if new hit is found

		}
		// Iteration is done
		// We cant continue loop untill these kernels are finished 
		// on gpu(s)
		CUDA_CHECK(hipDeviceSynchronize());
	}
	// At the end of iteration each accelerator holds its custom struct array
	// And hit ids holds a index for that struct
}

void TracerBase::SendAndRecieveRays()
{
	// Here also generate RayOut and use that
	// Also pre allocate sort buffers
	// TODO:
}

void TracerBase::ShadeRays()
{
	const Vector2i matBitRange = tracerSystem->MaterialBitRange();

	// Ray Memory Pointers	
	const RayGMem* dRays = rayMemory.Rays();	
	HitKey* dPotentialHits = rayMemory.PotentialHits();
	const void* dHitStructs = rayMemory.HitStructs<void>();
	RayId*	dRayIds = rayMemory.RayIds();
	const void* dAux = rayMemory.RayAux<void>();
	
	// Material Interfaces
	const MaterialGroupMappings& materials = tracerSystem->MaterialGroups();
		
	// Now here conside incoming rays from different tracers
	// Consume ray array
	uint32_t rayCount = currentRayCount;

	// Sort and Partition happens on leader device
	CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice()));

	// Copy Keys (which are stored in HitGMem) to HitKeys
	// Make ready for sorting
	rayMemory.FillRayIdsForSort(rayCount);

	// Sort with respect to the hits that are returned
	rayMemory.SortKeys(dRayIds, dPotentialHits, rayCount, matBitRange);

	// Parition w.r.t. material (full range sort is required here)
	// Each same material on accelerator is actually considered a unique material
	// this is why we sort full range.
	// This is required since same material may fetch is data differently 
	// from different objects
	auto portions = rayMemory.Partition(rayCount, matBitRange);

	// Use partition lis to find out
	// total potential output ray count
	size_t totalOutRayCount = 0;
	for(const auto& p : portions)
	{
		auto loc = materials.find(p.portionId);
		if(loc == materials.end()) continue;

		totalOutRayCount += p.count * loc->second->MaxOutRayPerRay();
	}

	// Allocate
	rayMemory.ResizeRayOut(totalOutRayCount, tracerSystem->PerRayAuxDataSize());
	unsigned char* dAuxOut = rayMemory.RayAuxOut<unsigned char>();
	RayGMem* dRaysOut = rayMemory.RaysOut();

	// For each partition
	size_t outOffset = 0;
	for(const auto& p : portions)
	{
		auto loc = materials.find(p.portionId);
		if(loc == materials.end()) continue;

		// Since output is dynamic (each material may write multiple rays)
		// add offsets to find proper count
		outOffset += p.count * loc->second->MaxOutRayPerRay();
		
		// Run local hit kernels
		RayId* dRayIdStart = dRayIds + p.offset;
		RayGMem* dRayOutStart = dRaysOut + outOffset;
		void* dAuxOutStart = dAuxOut + (outOffset * tracerSystem->PerRayAuxDataSize());
	
		// Actual Shade Call
		// TODO: Defer this call if p.count is too low
		// Problem: What if it is always low ?
		// Probably it is better to launch it
		//
		// Another TODO: Implement multi-gpu load balancing
		// More TODO: Implement single-gpu SM load balacing
		loc->second->ShadeRays(dRayOutStart, dAuxOutStart,
							   dRays, dHitStructs, dAux,
							   dRayIdStart,
							   static_cast<uint32_t>(p.count),
							   rngMemory);
		
	}
	assert(totalOutRayCount == outOffset);	
	currentRayCount = static_cast<uint32_t>(totalOutRayCount);

	// Shading complete
	// Now make "RayOut" to "RayIn"
	rayMemory.SwapRays();
}

TracerBase::TracerBase()
	: rayDelegateFunc(nullptr)
	, errorFunc(nullptr)
	, analyticFunc(nullptr)
	, imageFunc(nullptr)
	, baseSendFunc(nullptr)
	, accSendFunc(nullptr)
	, currentRayCount(0)
	, tracerSystem(nullptr)
	, healthy(false)	
{}

void TracerBase::Initialize(uint32_t seed, TracerLogicI& logic)
{
	// Device initalization
	TracerError e(TracerError::END);
	if((e = CudaSystem::Initialize()) != TracerError::OK)
	{
		if(errorFunc) errorFunc(e);
	}

	// Init and set Tracer System
	if((e = logic.Initialize()) != TracerError::OK)
	{
		if(errorFunc) errorFunc(e);
	}
	tracerSystem = &logic;

	// Select a leader device that is responsible
	// for sorting and partitioning works
	// for different materials / accelerators
	// TODO: Determine a leader Device
	rayMemory.SetLeaderDevice(0);
	CUDA_CHECK(hipSetDevice(0));

	// Initialize RNG Memory
	rngMemory = RNGMemory(seed);

	// All seems fine mark tracer as healthy
	healthy = true;
}

void TracerBase::SetTime(double seconds)
{}

void TracerBase::SetParams(const TracerParameters&)
{}

void TracerBase::SetScene(const std::string& sceneFileName)
{}

void TracerBase::RequestBaseAccelerator()
{}

void TracerBase::RequestAccelerator(HitKey key)
{}

void TracerBase::AssignAllMaterials()
{}

void TracerBase::AssignMaterial(uint32_t matId)
{}

void TracerBase::UnassignAllMaterials()
{}

void TracerBase::UnassignMaterial(uint32_t matId)
{}

void TracerBase::GenerateCameraRays(const CameraPerspective& camera,
									const uint32_t samplePerPixel)
{
	if(!healthy) return;

	// Initial ray count
	currentRayCount = outputImage.SegmentSize()[0] *
					  outputImage.SegmentSize()[1] *
					  samplePerPixel * samplePerPixel;

	// Allocate enough space for ray
	rayMemory.ResizeRayOut(currentRayCount, tracerSystem->PerRayAuxDataSize());

	// Delegate camera ray generation to tracer system
	tracerSystem->GenerateCameraRays(rayMemory, rngMemory,
									 camera, samplePerPixel,
									 outputImage.Resolution(),
									 outputImage.SegmentOffset(),
									 outputImage.SegmentSize());

	

	// You can only write to out buffer of the ray memory
	// Make that memory in rays for hit/shade system
	rayMemory.SwapRays();
}

bool TracerBase::Continue()
{
	return (currentRayCount > 0) && healthy;
}

void TracerBase::Render()
{
	if(!healthy) return;
	if(currentRayCount == 0) return;
	
	HitRays();
	ShadeRays();

	METU_LOG("-----------------------------");
	METU_LOG("-----------------------------");
}

void TracerBase::FinishSamples()
{
	if(!healthy) return;
}

bool TracerBase::IsCrashed()
{
	return (!healthy);
}

void TracerBase::AddMaterialRays(const RayCPU&, const HitCPU&,
								 uint32_t rayCount, uint32_t matId)
{}

void TracerBase::SetImagePixelFormat(PixelFormat f)
{
	outputImage.SetPixelFormat(f);
}

void TracerBase::ReportionImage(const Vector2ui& offset,
								const Vector2ui& size)
{
	outputImage.Reportion(offset, size);
}

void TracerBase::ResizeImage(const Vector2ui& resolution)
{
	outputImage.Resize(resolution);
}

void TracerBase::ResetImage()
{
	outputImage.Reset();
}
