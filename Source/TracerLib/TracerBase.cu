#include "hip/hip_runtime.h"
#include "TracerBase.h"

#include "RayLib/Camera.h"
#include "RayLib/Log.h"
#include "RayLib/TracerError.h"
#include "RayLib/TracerCallbacksI.h"

#include "TracerDebug.h"
#include "GPUAcceleratorI.h"
#include "GPUMaterialI.h"
#include "TracerLogicI.h"


void TracerBase::SendError(TracerError e, bool isFatal)
{
	if(callbacks) callbacks->SendError(e);
	healthy = isFatal;
}

void TracerBase::HitRays()
{
	// Tracer Logic interface
	const Vector2i& accBitCounts = currentLogic->SceneAcceleratorMaxBits();
	GPUBaseAcceleratorI& baseAccelerator = currentLogic->BaseAcelerator();
	const AcceleratorBatchMappings& subAccelerators = currentLogic->AcceleratorBatches();

	// Reset Hit Memory for hit loop
	rayMemory.ResetHitMemory(currentRayCount, currentLogic->HitStructSize());

	// Make Base Accelerator to get ready for hitting
	baseAccelerator.GetReady(currentRayCount);

	// Ray Memory Pointers
	RayGMem* dRays = rayMemory.Rays();
	HitKey* dMaterialKeys = rayMemory.CurrentKeys();
	TransformId* dTransfomIds = rayMemory.TransformIds();
	PrimitiveId* dPrimitiveIds = rayMemory.PrimitiveIds();
	HitStructPtr dHitStructs = rayMemory.HitStructs();
	// These are sorted etc.
	HitKey* dCurrentKeys = rayMemory.CurrentKeys();	
	RayId*	dCurrentRayIds = rayMemory.CurrentIds();	
	
	// Try to hit rays until no ray is left 
	// (these rays will be assigned with a material)
	// outside rays are also assigned with a material (which is special)
	uint32_t rayCount = currentRayCount;
	while(rayCount > 0)
	{
		// Traverse accelerator
		// Base accelerator provides potential hits
		// Cannot provide an absolute hit (its not its job)
		baseAccelerator.Hit(dTransfomIds, dCurrentKeys, dRays, dCurrentRayIds,
							 rayCount);

		// Base accelerator traverses the data partially
		// Updates current key (which represents innter accelerator batch and id)
		
		// After that, system sorts rays according to the keys
		// and partitions the array according to batches

		// Sort and Partition happens on the leader device
		CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice()));

		// Sort initial results (in order to partition and launch kernels accordingly)
		// Sort is radix sort.
		// We sort inner indices in addition to batches results for better data locality
		// We only sort up-to a certain bit (radix sort) which is tied to 
		// accelerator count
		rayMemory.SortKeys(dCurrentRayIds, dCurrentKeys, rayCount, accBitCounts);
		// Parition to sub accelerators		
		//
		// There may be invalid rays sprinkled along the array.
		// Holes occur in the structure since in previous iteration,
		// a material may required to write N rays for its output (which is defined
		// by the material) but it wrote < N rays.
		// 
		// One of the main examples for such behaviour can be transparent objects
		// where ray may be only reflected (instead of refrating and reflecting) because
		// of the total internal reflection phenomena.
		auto portions = rayMemory.Partition(rayCount);

		// Reorder partitions for efficient calls
		// (sort by gpu and order for better async access)
		// ....
		// TODO:

		// For each partition
		for(const auto& p : portions)
		{
			// Find Accelerator
			// Since there is no batch for invalid keys
			// that partition will be automatically be skipped
			auto loc = subAccelerators.find(p.portionId);
			if(loc == subAccelerators.end()) continue;

			RayId* dRayIdStart = dCurrentRayIds + p.offset;
			HitKey* dCurrentKeyStart = dCurrentKeys + p.offset;

			// Run local hit kernels
			// Local hit kernels returns a material key 
			// and primitive inner id.
			// Since materials are batched for both material and
			loc->second->Hit(// O
							 dMaterialKeys,
							 dPrimitiveIds,
							 dHitStructs,
							 // I-O
							 dRays,
							 // Input
							 dTransfomIds,
							 dRayIdStart,
							 dCurrentKeyStart,
							 static_cast<uint32_t>(p.count));

			// Hit function updates material key,
			// primitive id and struct if this hit is accepted
		}

		// Update new ray count
		// On partition array check last two partitions
		// Those partitions may contain outside/invalid batches
		// Reduce ray count accordingly
		int iterationCount = std::min(static_cast<int>(portions.size()), 2);
		auto iterator = portions.rbegin();
		for(int i = 0; i < iterationCount; ++i)
		{
			const auto& portion = *iterator;
			if(portion.portionId == HitKey::NullBatch ||
			   portion.portionId == HitKey::BoundaryBatch)
			{
				rayCount = static_cast<uint32_t>(portion.offset);
			}			
			iterator++;
		}
		
		// Iteration is done
		// We cant continue loop untill these kernels are finished 
		// on gpu(s)
		//
		// Tracer logic mostly utilizies mutiple GPUs so we need to
		// wait all GPUs to finish
		CudaSystem::SyncAllGPUs();
	}
	// At the end of iteration all rays found a material, primitive
	// and interpolation weights (which should be on hitStruct)
}

void TracerBase::ShadeRays()
{
	const Vector2i matMaxBits = currentLogic->SceneMaterialMaxBits();

	// Ray Memory Pointers	
	const RayGMem* dRays = rayMemory.Rays();	
	const void* dRayAux = rayMemory.RayAux<void>();
	const HitStructPtr dHitStructs = rayMemory.HitStructs();
	const PrimitiveId* dPrimitiveIds = rayMemory.PrimitiveIds();	
	// These are sorted etc.
	HitKey* dCurrentKeys = rayMemory.CurrentKeys();
	RayId* dCurrentRayIds = rayMemory.CurrentIds();
		
	// Material Interfaces
	const MaterialBatchMappings& materials = currentLogic->MaterialBatches();
	uint32_t rayCount = currentRayCount;

	// Sort and Partition happens on leader device
	CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice()));

	// Copy materialKeys to currentKeys
	// to make it ready for sorting
	rayMemory.FillRayIdsForSort(rayCount);

	// Sort with respect to the materials keys
	rayMemory.SortKeys(dCurrentRayIds, dCurrentKeys, rayCount, matMaxBits);

	// Parition w.r.t. material batch
	auto portions = rayMemory.Partition(rayCount);

	// Update new ray count
	// Last partition may be invalid partition
	// Skip those partition and adjust ray count accordingly
	if(!portions.empty() &&
	   portions.rbegin()->portionId == HitKey::NullBatch)
	{
		rayCount = static_cast<uint32_t>(portions.rbegin()->offset);
	}

	// Use partition lis to find out
	// total potential output ray count
	size_t totalOutRayCount = 0;
	for(const auto& p : portions)
	{
		auto loc = materials.find(p.portionId);
		if(loc == materials.end()) continue;

		totalOutRayCount += p.count * loc->second->OutRayCount();
	}

	// Allocate output ray memory
	rayMemory.ResizeRayOut(totalOutRayCount, currentLogic->PerRayAuxDataSize());
	unsigned char* dAuxOut = rayMemory.RayAuxOut<unsigned char>();
	RayGMem* dRaysOut = rayMemory.RaysOut();

	// Reorder partitions for efficient calls
	// (sort by gpu and order for better async access)
	// ....
	// TODO:
	
	// For each partition
	size_t outOffset = 0;
	for(const auto& p : portions)
	{
		auto loc = materials.find(p.portionId);
		if(loc == materials.end()) continue;

		// Since output is dynamic (each material may write multiple rays)
		// add offsets to find proper count
		outOffset += p.count * loc->second->OutRayCount();
		
		// Relativize input & output pointers
		const RayId* dRayIdStart = dCurrentRayIds + p.offset;
		const HitKey* dKeyStart = dCurrentKeys + p.offset;
		RayGMem* dRayOutStart = dRaysOut + outOffset;
		void* dAuxOutStart = dAuxOut + (outOffset * currentLogic->PerRayAuxDataSize());
	
		// Actual Shade Call
		loc->second->ShadeRays(// Output
							   outputImage.GMem<Vector4f>(),
							   //
							   dRayOutStart,
							   dAuxOutStart,
							   //  Input
							   dRays,
							   dRayAux,
							   dPrimitiveIds,
							   dHitStructs,
							   //
							   dKeyStart,
							   dRayIdStart,

							   static_cast<uint32_t>(p.count),
							   rngMemory);
		
	}
	assert(totalOutRayCount == outOffset);	
	currentRayCount = static_cast<uint32_t>(totalOutRayCount);

	// Again wait all of the GPU's since
	// CUDA functions will be on multiple-gpus
	CudaSystem::SyncAllGPUs();
	
	// Shading complete
	// Now make "RayOut" to "RayIn"
	// and continue
	rayMemory.SwapRays();
}

TracerBase::TracerBase()
	: callbacks(nullptr)	
	, currentRayCount(0)
	, currentLogic(nullptr)
	, healthy(false)	
{}

void TracerBase::Initialize(int leaderGPUId)
{
	// Device initalization
	TracerError e(TracerError::END);
	if((e = CudaSystem::Initialize()) != TracerError::OK)
	{
		if(callbacks) callbacks->SendError(e);
	}
	rayMemory.SetLeaderDevice(leaderGPUId);
	CUDA_CHECK(hipSetDevice(leaderGPUId));

	// All seems fine mark tracer as healthy
	healthy = true;
}

void TracerBase::SetOptions(const TracerOptions& opts)
{
	options = opts;
}

void TracerBase::RequestBaseAccelerator()
{}

void TracerBase::RequestAccelerator(HitKey key)
{}

void TracerBase::AttachLogic(TracerBaseLogicI& logic)
{
	// Init and set Tracer System
	TracerError e = TracerError::OK;
	if((e = logic.Initialize()) != TracerError::OK)
	{
		if(callbacks) callbacks->SendError(e);
	}
	currentLogic = &logic;

	// Initialize RNG Memory
	rngMemory = RNGMemory(logic.Seed());
}

void TracerBase::GenerateInitialRays(const GPUScene& scene,
									 int cameraId,
									 int samplePerLocation)
{
	if(!healthy) return;

	// Delegate camera ray generation to tracer system
	currentRayCount = static_cast<uint32_t>(currentLogic->GenerateRays(rayMemory, rngMemory,
																	   scene, cameraId, samplePerLocation,
																	   outputImage.Resolution(),
																	   outputImage.SegmentOffset(),
																	   outputImage.SegmentSize()));

	// You can only write to out buffer of the ray memory
	// Make that memory in rays for hit/shade system
	rayMemory.SwapRays();
}

bool TracerBase::Continue()
{
	return (currentRayCount > 0) && healthy;
}

void TracerBase::Render()
{
	if(!healthy) return;
	if(currentRayCount == 0) return;
	
	HitRays();
	ShadeRays();

	METU_LOG("-----------------------------");
	METU_LOG("-----------------------------");
}

void TracerBase::FinishSamples()
{
	if(!healthy) return;
}

void TracerBase::SetImagePixelFormat(PixelFormat f)
{
	outputImage.SetPixelFormat(f);
}

void TracerBase::ReportionImage(Vector2i start,
								Vector2i end)
{
	outputImage.Reportion(start, end);
}

void TracerBase::ResizeImage(Vector2i resolution)
{
	outputImage.Resize(resolution);
}

void TracerBase::ResetImage()
{
	outputImage.Reset();
}
