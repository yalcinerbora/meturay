#include "GPUTracer.h"

#include "RayLib/Log.h"
#include "RayLib/TracerError.h"
#include "RayLib/TracerCallbacksI.h"
#include "RayLib/BitManipulation.h"
#include "RayLib/GPUSceneI.h"

#include "CudaConstants.h"
#include "GPUAcceleratorI.h"
#include "GPUWorkI.h"

#include "TracerDebug.h"

GPUTracer::GPUTracer(const CudaSystem& system, 
                     const GPUSceneI& scene,
                     const TracerParameters& p)
    : cudaSystem(system)
    , baseAccelerator(*scene.BaseAccelerator())
    , accelBatches(scene.AcceleratorBatchMappings())
    , maxAccelBits(Vector2i(Utility::FindFirstSet32(scene.MaxAccelIds()[0]) + 1,
                            Utility::FindFirstSet32(scene.MaxAccelIds()[1]) + 1))
    , maxWorkBits(Vector2i(Utility::FindFirstSet32(scene.MaxMatIds()[0]) + 1,
                           Utility::FindFirstSet32(scene.MaxMatIds()[1]) + 1))
    , params(p)
    , maxHitSize(scene.HitStructUnionSize())
    , rayMemory(system.BestGPU())
    , callbacks(nullptr)
    , crashed(false)
    , currentRayCount(0)
{}

TracerError GPUTracer::Initialize()
{
    // Init RNG
    rngMemory = RNGMemory(params.seed, cudaSystem);

    // Construct Tracers
    TracerError e = TracerError::OK;
    if((e = baseAccelerator.Constrcut(cudaSystem)) != TracerError::OK)
        return e;

    for(const auto& accBatch : accelBatches)
    {
        GPUAcceleratorGroupI* acc = accBatch.second;
        if((e = acc->ConstructAccelerators(cudaSystem)) != TracerError::OK)
            return e;
    }    
    cudaSystem.SyncGPUAll();
    return TracerError::OK;
}

void GPUTracer::ResetHitMemory(uint32_t rayCount, HitKey baseBoundMatKey)
{
    currentRayCount = rayCount;
    rayMemory.ResizeRayOut(rayCount, baseBoundMatKey);
}

void GPUTracer::HitAndPartitionRays()
{   
    if(crashed) return;

    // Sort and Partition happens on the leader device
    CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice().DeviceId()));

    // Tracer Logic interface
    const Vector2i& accBitCounts = maxAccelBits;
    const AcceleratorBatchMap& subAccelerators = accelBatches;
    // Reset Hit Memory for hit loop
    rayMemory.ResetHitMemory(currentRayCount, maxHitSize);
    // Make Base Accelerator to get ready for hitting
    baseAccelerator.GetReady(cudaSystem, currentRayCount);
    // Ray Memory Pointers
    RayGMem* dRays = rayMemory.Rays();
    HitKey* dWorkKeys = rayMemory.WorkKeys();
    TransformId* dTransfomIds = rayMemory.TransformIds();
    PrimitiveId* dPrimitiveIds = rayMemory.PrimitiveIds();
    HitStructPtr dHitStructs = rayMemory.HitStructs();
    // These are sorted etc.
    HitKey* dCurrentKeys = rayMemory.CurrentKeys();
    RayId*  dCurrentRayIds = rayMemory.CurrentIds();

    // Try to hit rays until no ray is left
    // (these rays will be assigned with a material)
    // outside rays are also assigned with a material (which is special)
    uint32_t rayCount = currentRayCount;
    // At start all rays are valid
    uint32_t validRayOffset = 0;
    while(rayCount > 0)
    {
        //Debug::DumpMemToFile("dAccKeys", dCurrentKeys, currentRayCount);
        //Debug::DumpMemToFile("dRayIds", dCurrentRayIds, currentRayCount);

        // Traverse accelerator
        // Base accelerator provides potential hits
        // Cannot provide an absolute hit (its not its job)
        baseAccelerator.Hit(cudaSystem,
                            dTransfomIds, 
                            dCurrentKeys + validRayOffset,
                            dRays,
                            dCurrentRayIds + validRayOffset,
                            rayCount);

        // Wait all GPUs to finish...
        cudaSystem.SyncGPUMainStreamAll();

        //Debug::DumpMemToFile("dAccKeys", dCurrentKeys, currentRayCount);
        //Debug::DumpMemToFile("dRayIds", dCurrentRayIds, currentRayCount);

        // Base accelerator traverses the data partially
        // Updates current key (which represents inner accelerator batch and id)

        // After that, system sorts rays according to the keys
        // and partitions the array according to batches

        // Sort and Partition happens on the leader device
        CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice().DeviceId()));

        // Sort initial results (in order to partition and launch kernels accordingly)
        // Sort is radix sort.
        // We sort inner indices in addition to batches results for better data locality
        // We only sort up-to a certain bit (radix sort) which is tied to
        // accelerator count
        // Move offset to skip null bathces
        rayMemory.SortKeys(dCurrentRayIds, dCurrentKeys, rayCount, accBitCounts);
        // Parition to sub accelerators
        //
        // There may be invalid rays sprinkled along the array.
        // Holes occur in the structure since in previous iteration,
        // a material may required to write N rays for its output (which is defined
        // by the material) but it wrote < N rays.
        //
        // One of the main examples for such behaviour can be transparent objects
        // where ray may be only reflected (instead of refrating and reflecting) because
        // of the total internal reflection phenomena.
        auto portions = rayMemory.Partition(rayCount);

        //Debug::DumpMemToFile("dAccKeys", dCurrentKeys, currentRayCount);
        //Debug::DumpMemToFile("dRayIds", dCurrentRayIds, currentRayCount);

        // Reorder partitions for efficient calls
        // (group partitions into gpus and order for better async access)
        // ....
        // TODO:
        const int totalGPU = static_cast<int>(cudaSystem.GPUList().size());
        const auto& gpus = cudaSystem.GPUList();
        auto currentGPU = gpus.begin();

        // For each partition
        for(const auto& p : portions)
        {
            // Find Accelerator
            // Since there is no batch for invalid keys
            // that partition will be automatically be skipped
            auto loc = subAccelerators.find(p.portionId);
            if(loc == subAccelerators.end()) continue;

            RayId* dRayIdStart = dCurrentRayIds + validRayOffset + p.offset;
            HitKey* dCurrentKeyStart = dCurrentKeys + validRayOffset + p.offset;

            // Run local hit kernels
            // Local hit kernels returns a material key
            // and primitive inner id.
            // Since materials are batched for both material and
            loc->second->Hit(*currentGPU,
                             // O
                             dWorkKeys,
                             dPrimitiveIds,
                             dHitStructs,
                             // I-O
                             dRays,
                             // Input
                             dTransfomIds,
                             dRayIdStart,
                             dCurrentKeyStart,
                             static_cast<uint32_t>(p.count));

            // Split to GPUs
            currentGPU++;
            if(currentGPU == gpus.end()) currentGPU = gpus.begin();

            // Hit function updates material key,
            // primitive id and struct if this hit is accepted
        }
        //printf("=====================================================\n");

        // Update new ray count
        // On partition array check first partition
        // it may contain invalid key meaning
        // those rays are totally processed
        // change the offset so that
        // we skip those rays
        auto nullPortion = portions.begin();
        if(nullPortion->portionId == HitKey::NullBatch)
            rayCount = static_cast<uint32_t>(nullPortion->offset);

        // Iteration is done
        // We cant continue loop untill these kernels are finished
        // on gpu(s)
        //
        // Tracer logic mostly utilizies mutiple GPUs so we need to
        // wait all GPUs to finish
        cudaSystem.SyncGPUAll();
    }

    // At the end of iteration all rays found a material, primitive
    // and interpolation weights (which should be on hitStruct)

    // Partition rays for work kernel calls
    // Copy materialKeys to currentKeys
    // to make it ready for sorting
    rayMemory.FillMatIdsForSort(currentRayCount);
    // Sort with respect to the materials keys
    rayMemory.SortKeys(dCurrentRayIds, dCurrentKeys, currentRayCount, maxWorkBits);

    // Parition w.r.t. material batch
    workPartition.clear();
    workPartition = rayMemory.Partition(currentRayCount);

    //printf("FRAME END\n");
}

void GPUTracer::WorkRays(const WorkBatchMap& workMap, HitKey baseBoundMatKey)
{
    // Sort and Partition happens on leader device
    CUDA_CHECK(hipSetDevice(rayMemory.LeaderDevice().DeviceId()));

    // Ray Memory Pointers
    const RayGMem* dRays = rayMemory.Rays();
    const HitStructPtr dHitStructs = rayMemory.HitStructs();
    const PrimitiveId* dPrimitiveIds = rayMemory.PrimitiveIds();
    // These are sorted etc.
    HitKey* dCurrentKeys = rayMemory.CurrentKeys();
    RayId* dCurrentRayIds = rayMemory.CurrentIds();

    // Use partition lis to find out
    // total potential output ray count
    uint32_t totalOutRayCount = 0;
    for(const auto& p : workPartition)
    {
        // Skip if null batch or unfound material
        if(p.portionId == HitKey::NullBatch) continue;
        auto loc = workMap.find(p.portionId);
        if(loc == workMap.end()) continue;

        totalOutRayCount += (static_cast<uint32_t>(p.count)*
                             loc->second->OutRayCount());
    }

    // Allocate output ray memory
    rayMemory.ResizeRayOut(totalOutRayCount, baseBoundMatKey);
    RayGMem* dRaysOut = rayMemory.RaysOut();
    HitKey* dBoundKeyOut = rayMemory.WorkKeys();

    // Reorder partitions for efficient calls
    // (sort by gpu and order for better async access)
    // ....
    // TODO:

    // For each partition
    uint32_t outOffset = 0;
    for(auto pIt = workPartition.crbegin();
        pIt != workPartition.crend(); pIt++)
    {
        const auto& p = (*pIt);

        // Skip if null batch or unfound material
        if(p.portionId == HitKey::NullBatch) continue;
        auto loc = workMap.find(p.portionId);
        if(loc == workMap.end()) continue;

        // Relativize input & output pointers
        const RayId* dRayIdStart = dCurrentRayIds + p.offset;
        const HitKey* dKeyStart = dCurrentKeys + p.offset;
        // Output
        RayGMem* dRayOutStart = dRaysOut + outOffset;        
        HitKey* dBoundKeyStart = dBoundKeyOut + outOffset;

        // Actual Shade Call
        loc->second->Work(dBoundKeyStart,
                          dRayOutStart,
                          //  Input
                          dRays,
                          dPrimitiveIds,
                          dHitStructs,
                          // Ids
                          dKeyStart,
                          dRayIdStart,
                          //
                          outOffset,
                          static_cast<uint32_t>(p.count),
                          rngMemory);

        //cudaSystem.SyncGPUAll();
        //METU_LOG("--------------------------");

        // Since output is dynamic (each material may write multiple rays)
        // add offsets to find proper count
        outOffset += static_cast<uint32_t>(p.count * loc->second->OutRayCount());
    }
    assert(totalOutRayCount == outOffset);
    currentRayCount = totalOutRayCount;

    // Again wait all of the GPU's since
    // CUDA functions will be on multiple-gpus
    cudaSystem.SyncGPUAll();

    // Shading complete
    // Now make "RayOut" to "RayIn"
    // and continue
    rayMemory.SwapRays();
}

void GPUTracer::SetParameters(const TracerParameters& p)
{
    if(params.seed != p.seed)
        rngMemory = std::move(RNGMemory(p.seed, cudaSystem));
    params = p;
}

void GPUTracer::SetImagePixelFormat(PixelFormat f)
{
    imgMemory.SetPixelFormat(f, cudaSystem);
}

void GPUTracer::ReportionImage(Vector2i start,
                                Vector2i end)
{
    imgMemory.Reportion(start, end, cudaSystem);
}

void GPUTracer::ResizeImage(Vector2i resolution)
{
    imgMemory.Resize(resolution);
}

void GPUTracer::ResetImage()
{
    imgMemory.Reset(cudaSystem);
}

template <class... Args>
inline void GPUTracer::SendLog(const char* format, Args... args)
{
    if(!params.verbose) return;

    size_t size = snprintf(nullptr, 0, format, args...);
    std::string s(size, '\0');
    snprintf(&s[0], size, format, args...);
    if(callbacks) callbacks->SendLog(s);
}

void GPUTracer::SendError(TracerError e, bool isFatal)
{
    if(callbacks) callbacks->SendError(e);
    crashed = isFatal;
}

void GPUTracer::Finalize()
{
    if(crashed) return;
    SendLog("Finalizing...");
   
    // Determine Size
    Vector2i pixelCount = imgMemory.SegmentSize();
    Vector2i start = imgMemory.SegmentOffset();
    Vector2i end = start + imgMemory.SegmentSize();
    size_t offset = (static_cast<size_t>(pixelCount[0])* pixelCount[1] *
                     imgMemory.PixelSize());

    // Flush Devices and Get the Image
    cudaSystem.SyncGPUAll();
    std::vector<Byte> imageData = imgMemory.GetImageToCPU(cudaSystem);

    size_t pixelCount1D = static_cast<size_t>(pixelCount[0]) * pixelCount[1];

    // Launch finished image
    if(callbacks) callbacks->SendImage(std::move(imageData),
                                       imgMemory.Format(),
                                       offset,
                                       start, end);
    SendLog("Image sent!");
}

void GPUTracer::AskParameters()
{
    if(callbacks) callbacks->SendCurrentParameters(params);
}
